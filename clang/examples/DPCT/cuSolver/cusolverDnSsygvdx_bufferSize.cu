
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, const float *a, int lda, const float *b,
          int ldb, float vl, float vu, int il, int iu, int *h_meig,
          const float *w) {
  // Start
  int buffer_size;
  hipsolverDnSsygvdx_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, range /*cusolverEigRange_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const float **/, lda /*int*/,
      b /*const float **/, ldb /*int*/, vl /*float*/, vu /*float*/, il /*int*/,
      iu /*int*/, h_meig /*int **/, w /*const float **/,
      &buffer_size /*int **/);
  // End
}
