
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipComplex *const *a, int lda, int *ipiv,
          int *info, int group_count) {
  // Start
  hipblasCgetrfBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*cuComplex *const **/, lda /*int*/, ipiv /*int **/,
                      info /*int **/, group_count /*int*/);
  // End
}
