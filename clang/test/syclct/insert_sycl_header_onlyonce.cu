// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/insert_sycl_header_onlyonce.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK: #include <stdio.h>
// CHECK-NOT:#include <CL/sycl.hpp>
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(){
  return 0;
}

