// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cublasTsyrkx.sycl.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

//CHECK: #define macro_a (mkl::transpose)1
#define macro_a (hipblasOperation_t)1

//CHECK: #define macro_b (mkl::uplo)1
#define macro_b (hipblasFillMode_t)1

hipblasFillMode_t foo(){
  return HIPBLAS_FILL_MODE_LOWER;
}

hipblasOperation_t bar(){
  return HIPBLAS_OP_T;
}

int main() {
  int n = 275;
  int k = 275;
  int lda = 1;
  int ldb = 1;
  int ldc = 1;

  float alpha_s = 1;
  float beta_s = 1;

  double alpha_d = 1;
  double beta_d = 1;

  hipblasHandle_t handle;
  hipblasStatus_t status;

  float* A_s=0;
  float* B_s=0;
  float* C_s=0;

  double* A_d=0;
  double* B_d=0;
  double* C_d=0;

  int trans0 = 0;
  int trans1 = 1;
  int fill0 = 0;
  int fill1 = 1;

  //CHECK: /*
  //CHECK-NEXT: SYCLCT1003:0: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  //CHECK-NEXT: */
  //CHECK-NEXT: {
  //CHECK-NEXT: auto transpose_ct_2 = trans0;
  //CHECK-NEXT: auto A_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_s);
  //CHECK-NEXT: cl::sycl::buffer<float,1> A_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(A_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  //CHECK-NEXT: auto B_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_s);
  //CHECK-NEXT: cl::sycl::buffer<float,1> B_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(B_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  //CHECK-NEXT: auto C_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_s);
  //CHECK-NEXT: cl::sycl::buffer<float,1> C_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(C_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  //CHECK-NEXT: status = (mkl::sgemmt(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct_2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct_2)), ((((int)transpose_ct_2)==0)?(mkl::transpose::trans):(mkl::transpose::nontrans)), n, k, *(&alpha_s), A_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_s), C_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc), 0);
  //CHECK-NEXT: }
  //CHECK-NEXT: {
  //CHECK-NEXT: auto transpose_ct_2 = trans1;
  //CHECK-NEXT: auto A_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_s);
  //CHECK-NEXT: cl::sycl::buffer<float,1> A_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(A_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  //CHECK-NEXT: auto B_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_s);
  //CHECK-NEXT: cl::sycl::buffer<float,1> B_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(B_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  //CHECK-NEXT: auto C_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_s);
  //CHECK-NEXT: cl::sycl::buffer<float,1> C_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(C_s_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  //CHECK-NEXT: mkl::sgemmt(handle, (((int)fill1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct_2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct_2)), ((((int)transpose_ct_2)==0)?(mkl::transpose::trans):(mkl::transpose::nontrans)), n, k, *(&alpha_s), A_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_s), C_s_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc);
  //CHECK-NEXT: }
  status = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);
  hipblasSsyrkx(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);

  //CHECK: /*
  //CHECK-NEXT: SYCLCT1003:1: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  //CHECK-NEXT: */
  //CHECK-NEXT: {
  //CHECK-NEXT: auto transpose_ct_2 = 0;
  //CHECK-NEXT: auto A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: status = (mkl::dgemmt(handle, (((int)0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct_2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct_2)), ((((int)transpose_ct_2)==0)?(mkl::transpose::trans):(mkl::transpose::nontrans)), n, k, *(&alpha_d), A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_d), C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc), 0);
  //CHECK-NEXT: }
  //CHECK-NEXT: {
  //CHECK-NEXT: auto transpose_ct_2 = 1;
  //CHECK-NEXT: auto A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: mkl::dgemmt(handle, (((int)1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct_2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct_2)), ((((int)transpose_ct_2)==0)?(mkl::transpose::trans):(mkl::transpose::nontrans)), n, k, *(&alpha_d), A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_d), C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc);
  //CHECK-NEXT: }
  status = hipblasDsyrkx(handle, (hipblasFillMode_t)0, (hipblasOperation_t)0, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, (hipblasFillMode_t)1, (hipblasOperation_t)1, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  //CHECK: {
  //CHECK-NEXT: auto transpose_ct_2 = macro_a;
  //CHECK-NEXT: auto A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: mkl::dgemmt(handle, foo(), (((int)transpose_ct_2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct_2)), ((((int)transpose_ct_2)==0)?(mkl::transpose::trans):(mkl::transpose::nontrans)), n, k, *(&alpha_d), A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_d), C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc);
  //CHECK-NEXT: }


  hipblasDsyrkx(handle, foo(), macro_a, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  //CHECK: {
  //CHECK-NEXT: auto transpose_ct_2 = bar();
  //CHECK-NEXT: auto A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: mkl::dgemmt(handle, (((int)macro_b)==0?(mkl::uplo::lower):(mkl::uplo::upper)), transpose_ct_2, ((transpose_ct_2)==(mkl::transpose::nontrans))?(mkl::transpose::trans):(mkl::transpose::nontrans), n, k, *(&alpha_d), A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_d), C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc);
  //CHECK-NEXT: }
  hipblasDsyrkx(handle, macro_b, bar(), n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  //CHECK: {
  //CHECK-NEXT: auto transpose_ct_2 = mkl::transpose::trans;
  //CHECK-NEXT: auto A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(A_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(A_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(B_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(B_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: auto C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(C_d);
  //CHECK-NEXT: cl::sycl::buffer<double,1> C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(C_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  //CHECK-NEXT: mkl::dgemmt(handle, mkl::uplo::lower, transpose_ct_2, ((transpose_ct_2)==(mkl::transpose::nontrans))?(mkl::transpose::trans):(mkl::transpose::nontrans), n, k, *(&alpha_d), A_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, B_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldb, *(&beta_d), C_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, ldc);
  //CHECK-NEXT: }
  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  return 0;
}
