// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/device002.sycl.cpp

void checkError(hipError_t err) {

}

int main(int argc, char **argv)
{
int deviceCount = 0;
// CHECK: deviceCount = syclct::get_device_manager().device_count();
hipGetDeviceCount(&deviceCount);

int dev_id;
// CHECK: dev_id = syclct::get_device_manager().current_device_id();
hipGetDevice(&dev_id);

hipDeviceProp_t deviceProp;
// CHECK: deviceProp = syclct::get_device_manager().get_device( 0).get_device_info();
hipGetDeviceProperties(&deviceProp, 0);

int atomicSupported;
// CHECK: atomicSupported = syclct::get_device_manager().get_device(  dev_id).is_native_atomic_supported();
hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: P2P Access is not supported in Sycl
// CHECK-NEXT:*/
// CHECK-NEXT: accessSupported = 0;
hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2);

// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: P2P Access is not supported in Sycl
// CHECK-NEXT:*/
// CHECK-NEXT: perfRank = 0;
hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2);

// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: P2P Access is not supported in Sycl
// CHECK-NEXT:*/
// CHECK-NEXT: atomicSupported = 0;
hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2);

// CHECK: syclct::get_device_manager().current_device().reset();
hipDeviceReset();
// CHECK: syclct::get_device_manager().select_device(device2);
hipSetDevice(device2);

// CHECK:syclct::get_device_manager().current_device().queues_wait_and_throw();
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:int err = (syclct::get_device_manager().current_device().queues_wait_and_throw(), 0);
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:checkError((syclct::get_device_manager().current_device().queues_wait_and_throw(), 0));
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:return (syclct::get_device_manager().current_device().queues_wait_and_throw(), 0);
hipDeviceSynchronize();
hipError_t err = hipDeviceSynchronize();
checkError(hipDeviceSynchronize());
return hipDeviceSynchronize();
// CHECK: int e = 0;
int e = hipGetLastError();
// CHECK:syclct::get_device_manager().current_device().queues_wait_and_throw();
hipDeviceSynchronize();
return 0;
}
