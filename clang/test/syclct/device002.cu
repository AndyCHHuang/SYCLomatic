// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/device002.sycl.cpp

#include <stdio.h>

void checkError(hipError_t err) {

}

int main(int argc, char **argv)
{
int devID = atoi(argv[1]);
hipDeviceProp_t cdp;
// CHECK:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT: int error_code = (syclct::get_device_manager().get_device(devID).get_device_info(cdp), 0);
hipError_t error_code = hipGetDeviceProperties(&cdp, devID);

if (error_code == hipSuccess) {
// CHECK: /*
// CHECK-NEXT:  SYCLCT1005:{{[0-9]+}}: The device version is different. You may want to rewrite this code
// CHECK-NEXT: */
// CHECK-NEXT: /*
// CHECK-NEXT:  SYCLCT1006:{{[0-9]+}}: SYCL doesn't provide standard API to differentiate between integrated/discrete GPU devices. Consider to re-implement the code which depends on this field
// CHECK-NEXT: */
// CHECK-NEXT:if (cdp.get_major_version() < 3 && cdp.get_integrated() != 1) {
    if (cdp.major < 3 && cdp.integrated != 1) {
            printf("do_complex_compute requires compute capability 3.0 or later and not integrated\n");
    }
}

int deviceCount = 0;
// CHECK: deviceCount = syclct::get_device_manager().device_count();
hipGetDeviceCount(&deviceCount);

int dev_id;
// CHECK: dev_id = syclct::get_device_manager().current_device_id();
hipGetDevice(&dev_id);

hipDeviceProp_t deviceProp;
// CHECK: syclct::get_device_manager().get_device(0).get_device_info(deviceProp);
hipGetDeviceProperties(&deviceProp, 0);

int atomicSupported;
// CHECK: atomicSupported = syclct::get_device_manager().get_device(dev_id).is_native_atomic_supported();
hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: P2P Access is not supported in DPC++
// CHECK-NEXT:*/
// CHECK-NEXT: accessSupported = 0;
hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2);

// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: P2P Access is not supported in DPC++
// CHECK-NEXT:*/
// CHECK-NEXT: perfRank = 0;
hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2);

// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: P2P Access is not supported in DPC++
// CHECK-NEXT:*/
// CHECK-NEXT: atomicSupported = 0;
hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2);


char pciBusId[80];
// CHECK:/*
// CHECK-NEXT:SYCLCT1004:{{[0-9]+}}: Get PCI BusId is not supported in DPC++
// CHECK-NEXT:*/
hipDeviceGetPCIBusId(pciBusId, 80, 0);


// CHECK: syclct::get_device_manager().current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (syclct::get_device_manager().current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK: syclct::get_device_manager().current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (syclct::get_device_manager().current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (syclct::get_device_manager().select_device(device2), 0);
error_code = hipSetDevice(device2);
// CHECK: syclct::get_device_manager().select_device(device2);
hipSetDevice(device2);

// CHECK:syclct::get_device_manager().current_device().queues_wait_and_throw();
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:int err = (syclct::get_device_manager().current_device().queues_wait_and_throw(), 0);
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:checkError((syclct::get_device_manager().current_device().queues_wait_and_throw(), 0));
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:return (syclct::get_device_manager().current_device().queues_wait_and_throw(), 0);
hipDeviceSynchronize();
hipError_t err = hipDeviceSynchronize();
checkError(hipDeviceSynchronize());
return hipDeviceSynchronize();
// CHECK:/*
// CHECK-NEXT:SYCLCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, hipGetLastError was replaced with 0. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e = 0;
int e = hipGetLastError();
// CHECK:/*
// CHECK-NEXT:SYCLCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, hipPeekAtLastError was replaced with 0. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e1 = 0;
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, hipPeekAtLastError was replaced with 0. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: 0;
int e1 = hipPeekAtLastError();
hipPeekAtLastError();
// CHECK:syclct::get_device_manager().current_device().queues_wait_and_throw();
hipDeviceSynchronize();
return 0;
}
