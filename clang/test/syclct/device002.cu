
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device002.sycl.cpp

int main(int argc, char **argv)
{
int deviceCount = 0;
// CHECK: deviceCount = syclct::get_device_manager().device_count();
hipGetDeviceCount(&deviceCount);

int dev_id;
// CHECK: dev_id = syclct::get_device_manager().current_device_id();
hipGetDevice(&dev_id);

hipDeviceProp_t deviceProp;
// CHECK: deviceProp = syclct::get_device_manager().get_device( 0).get_device_info();
hipGetDeviceProperties(&deviceProp, 0);

int atomicSupported;
// CHECK: atomicSupported = syclct::get_device_manager().get_device(  dev_id).is_native_atomic_supported();
hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;
// CHECK: accessSupported = 0;
hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2);
// CHECK: perfRank = 0;
hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2);
// CHECK: atomicSupported = 0;
hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2);

// CHECK: syclct::get_device_manager().current_device().reset();
hipDeviceReset();
// CHECK: syclct::get_device_manager().select_device(device2);
hipSetDevice(device2);

// CHECK:  syclct::get_device_manager().current_device().queues_wait_and_throw();
hipDeviceSynchronize();

// CHECK: int e = 0;
int e = hipGetLastError();

return 0;
}
