// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/cublasReturnType.sycl.cpp --match-full-lines %s
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: int foo(int m, int n) try {
hipblasStatus_t foo(int m, int n) {
  // CHECK: return 0;
  return HIPBLAS_STATUS_SUCCESS;
}

// CHECK: cl::sycl::queue foo1(int m) try {
hipblasHandle_t foo1(int m) {
  return 0;
}

// CHECK: std::complex<float> foo2(std::complex<float> m) try {
hipComplex foo2(hipComplex m) {
  // CHECK: return std::complex<float>(1, 0);
  return make_hipComplex(1, 0);
}

// CHECK: std::complex<double> foo3(std::complex<double> m) try {
hipDoubleComplex foo3(hipDoubleComplex m) {
  // CHECK: return std::complex<double>(1, 0);
  return make_hipDoubleComplex(1, 0);
}

// CHECK: mkl::transpose foo4(mkl::transpose m) try {
hipblasOperation_t foo4(hipblasOperation_t m) {
  // CHECK: return mkl::transpose::conjtrans;
  return HIPBLAS_OP_C;
}

// CHECK: mkl::uplo foo5(mkl::uplo m) try {
hipblasFillMode_t foo5(hipblasFillMode_t m) {
  // CHECK: return mkl::uplo::lower;
  return HIPBLAS_FILL_MODE_LOWER;
}

// CHECK: mkl::side foo6(mkl::side m) try {
hipblasSideMode_t foo6(hipblasSideMode_t m) {
  // CHECK: return mkl::side::right;
  return HIPBLAS_SIDE_RIGHT;
}

// CHECK: mkl::diag foo7(mkl::diag m) try {
hipblasDiagType_t foo7(hipblasDiagType_t m) {
  // CHECK: return mkl::diag::nonunit;
  return HIPBLAS_DIAG_NON_UNIT;
}
