// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/cublasReturnType.sycl.cpp --match-full-lines %s
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: int foo(int m, int n) {
hipblasStatus_t foo(int m, int n) {
  // CHECK: return 0;
  return HIPBLAS_STATUS_SUCCESS;
}

// CHECK: cl::sycl::queue foo1(int m) {
hipblasHandle_t foo1(int m) {
  return 0;
}

// CHECK: std::complex<float> foo2(std::complex<float> m) {
hipComplex foo2(hipComplex m) {
  // CHECK: return std::complex<float>(1, 0);
  return make_hipComplex(1, 0);
}

// CHECK: std::complex<double> foo3(std::complex<double> m) {
hipDoubleComplex foo3(hipDoubleComplex m) {
  // CHECK: return std::complex<double>(1, 0);
  return make_hipDoubleComplex(1, 0);
}

// CHECK: mkl::transpose foo4(mkl::transpose m) {
hipblasOperation_t foo4(hipblasOperation_t m) {
  // CHECK: return mkl::transpose::conjtrans;
  return HIPBLAS_OP_C;
}

// CHECK: mkl::uplo foo5(mkl::uplo m) {
hipblasFillMode_t foo5(hipblasFillMode_t m) {
  // CHECK: return mkl::uplo::lower;
  return HIPBLAS_FILL_MODE_LOWER;
}

// CHECK: mkl::side foo6(mkl::side m) {
hipblasSideMode_t foo6(hipblasSideMode_t m) {
  // CHECK: return mkl::side::right;
  return HIPBLAS_SIDE_RIGHT;
}

// CHECK: mkl::diag foo7(mkl::diag m) {
hipblasDiagType_t foo7(hipblasDiagType_t m) {
  // CHECK: return mkl::diag::nonunit;
  return HIPBLAS_DIAG_NON_UNIT;
}
