// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cusolverDnLn.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    // CHECK: status = 0;
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: status = 0;
    status = hipsolverDnSpotrf_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotrf_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotrf_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotrf_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    // CHECK: status = 0;
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: status = 0;
    status = hipsolverDnSgetrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnDgetrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnCgetrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnZgetrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::spotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::spotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dpotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dpotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cpotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cpotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zpotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zpotrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::spotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::spotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);
    hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dpotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dpotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);
    hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cpotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cpotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);
    hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zpotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&C_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zpotrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);
    hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);
    hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);
    hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);
    hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgetrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);
    hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);
    hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);
    hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgetrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);
    hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_tau(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgeqrf_get_lwork(*cusolverH, m, n, buffer_ct3, lda, buffer_ct_mkl_tau, result_temp_buffer5);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgeqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    status = hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sormqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sormqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    status = hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dormqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dormqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    status = hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cunmqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cunmqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cunmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cunmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    status = hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zunmqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&C_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zunmqr_get_lwork(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldc, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zunmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zunmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    status = hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sorgqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sorgqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sorgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sorgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dorgqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dorgqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dorgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dorgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cungqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cungqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zungqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zungqr_get_lwork(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT:*/
    // CHECK-NEXT:{
    // CHECK-NEXT:auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT:cl::sycl::buffer<float,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT:cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT:mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT:status = (mkl::ssytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4), 0);
    // CHECK-NEXT:buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    // CHECK-NEXT:{
    // CHECK-NEXT:auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT:cl::sycl::buffer<float,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT:cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT:mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT:mkl::ssytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4);
    // CHECK-NEXT:buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    // CHECK-NEXT:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT:*/
    // CHECK-NEXT:{
    // CHECK-NEXT:auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT:cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT:cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT:auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT:cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT:cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT:status = (mkl::ssytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT:buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    // CHECK-NEXT:{
    // CHECK-NEXT:auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT:cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT:cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT:auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT:cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT:cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT:mkl::ssytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT:buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    status = hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    status = hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dsytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4), 0);
    // CHECK-NEXT: buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dsytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4);
    // CHECK-NEXT: buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dsytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dsytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    status = hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::csytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4), 0);
    // CHECK-NEXT: buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::csytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4);
    // CHECK-NEXT: buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::csytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::csytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    status = hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);
    hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zsytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4), 0);
    // CHECK-NEXT: buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct2 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct2 = allocation_ct2.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct2.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct4 = dpct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct4 = allocation_ct4.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct4.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer4(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> buffer_ct_mkl_ipiv(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zsytrf_get_lwork(*cusolverH, uplo_ct_mkl_upper_lower, n, buffer_ct2, lda, buffer_ct_mkl_ipiv, result_temp_buffer4);
    // CHECK-NEXT: buffer_ct4.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer4.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zsytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::memory_manager::get_instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zsytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    status = hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
    hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
}
