
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -passes "IterationSpaceBuiltinRule" -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/pass-filter.sycl.cpp

// Test that only IterationSpaceBuiltinRule is being run
// CHECK:__global__ void test_00() {
__global__ void test_00() {
  // CHECK: size_t tix = item_{{[a-f0-9]+}}.get_local_id(0);
  // CHECK: size_t tiy = item_{{[a-f0-9]+}}.get_local_id(1);
  // CHECK: size_t tiz = item_{{[a-f0-9]+}}.get_local_id(2);
  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;
}
