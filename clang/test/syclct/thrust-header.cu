// UNSUPPORTED: cuda-8.0
// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/thrust-header.sycl.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK-NEXT: #include <algorithm>
// CHECK: #include <syclct/syclct_thrust.hpp>
#include <cstdio>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
int main() {

  int *mapsp1D, *mapspkeyD,*mapspvalD;
  int numsH=10;

  hipMalloc(&mapsp1D, numsH*sizeof(int));
  hipMalloc(&mapspkeyD, numsH*sizeof(int));
  hipMalloc(&mapspvalD, numsH*sizeof(int));

//  thrust::device_ptr<int> mapsp1T(mapsp1D);
//  thrust::device_ptr<int> mapspkeyT(mapspkeyD);
//  thrust::device_ptr<int> mapspvalT(mapspvalD);

//  thrust::copy(mapsp1T, mapsp1T + numsH, mapspkeyT);
//  thrust::sequence(mapspvalT, mapspvalT + numsH);
//  thrust::stable_sort_by_key(mapspkeyT, mapspkeyT + numsH, mapspvalT);

}
