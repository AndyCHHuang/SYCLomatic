// UNSUPPORTED: cuda-8.0
// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/thrust-header.sycl.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK-NEXT: #include <algorithm>
#include <cstdio>
#include <algorithm>
// CHECK: #include <dpstd/containers>
// CHECK-NEXT: #include <dpstd/algorithm>
// CHECK-NEXT: #include <dpstd/execution>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
int main() {

  int *mapsp1D, *mapspkeyD,*mapspvalD;
  int numsH=10;

  hipMalloc(&mapsp1D, numsH*sizeof(int));
  hipMalloc(&mapspkeyD, numsH*sizeof(int));
  hipMalloc(&mapspvalD, numsH*sizeof(int));

// CHECK:  dpstd::device_ptr<int> mapsp1T(mapsp1D);
  thrust::device_ptr<int> mapsp1T(mapsp1D);
// CHECK:  dpstd::device_ptr<int> mapspkeyT(mapspkeyD);
  thrust::device_ptr<int> mapspkeyT(mapspkeyD);
// CHECK:  dpstd::device_ptr<int> mapspvalT(mapspvalD);
  thrust::device_ptr<int> mapspvalT(mapspvalD);

// CHECK:  std::copy(dpstd::execution::sycl, mapsp1T, mapsp1T + numsH, mapspkeyT);
  thrust::copy(mapsp1T, mapsp1T + numsH, mapspkeyT);
// CHECK:  std::sequence(dpstd::execution::sycl, mapspvalT, mapspvalT + numsH);
  thrust::sequence(mapspvalT, mapspvalT + numsH);
// CHECK:  std::stable_sort_by_key(dpstd::execution::sycl, mapspkeyT, mapspkeyT + numsH, mapspvalT);
  thrust::stable_sort_by_key(mapspkeyT, mapspkeyT + numsH, mapspvalT);
}
