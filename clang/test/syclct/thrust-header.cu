// UNSUPPORTED: cuda-8.0
// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/thrust-header.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK-NEXT: #include <algorithm>
#include <cstdio>
#include <algorithm>
// CHECK: #include <dpstd/algorithm>
// CHECK-NEXT: #include <dpstd/execution>
// CHECK-NEXT: #include <dpct/dpct_dpstd_utils.hpp>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
int main() {

  int *mapsp1D, *mapspkeyD,*mapspvalD;
  int numsH=10;

  hipMalloc(&mapsp1D, numsH*sizeof(int));
  hipMalloc(&mapspkeyD, numsH*sizeof(int));
  hipMalloc(&mapspvalD, numsH*sizeof(int));

// CHECK:  dpct::device_ptr<int> mapsp1T(mapsp1D);
  thrust::device_ptr<int> mapsp1T(mapsp1D);
// CHECK:  dpct::device_ptr<int> mapspkeyT(mapspkeyD);
  thrust::device_ptr<int> mapspkeyT(mapspkeyD);
// CHECK:  dpct::device_ptr<int> mapspvalT(mapspvalD);
  thrust::device_ptr<int> mapspvalT(mapspvalD);

// CHECK:  std::copy(dpstd::execution::make_sycl_policy<class Policy_{{[0-9a-f]+}}>(dpstd::execution::sycl), mapsp1T, mapsp1T + numsH, mapspkeyT);
  thrust::copy(mapsp1T, mapsp1T + numsH, mapspkeyT);
// CHECK:  dpct::sequence(dpstd::execution::make_sycl_policy<class Policy_{{[0-9a-f]+}}>(dpstd::execution::sycl), mapspvalT, mapspvalT + numsH);
  thrust::sequence(mapspvalT, mapspvalT + numsH);
// CHECK:  dpct::stable_sort_by_key(dpstd::execution::make_sycl_policy<class Policy_{{[0-9a-f]+}}>(dpstd::execution::sycl), mapspkeyT, mapspkeyT + numsH, mapspvalT);
  thrust::stable_sort_by_key(mapspkeyT, mapspkeyT + numsH, mapspvalT);
}
