// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cublasLegacyHelper.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <sycl_types.hpp>
// CHECK: #include <complex>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CHECK: #define MACRO_A 0
#define MACRO_A cublasInit()

#define MACRO_B(status) (status)

// CHECK: #define MACRO_C(pointer) status = (dpct::dpct_free(d_A), 0)
#define MACRO_C(pointer) status = cublasFree(d_A)

void foo2(hipblasStatus_t){}

// CHECK: void foo(int, int, int, int, int, int, int, int, int, int) {}
void foo(hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t) {}

// CHECK: void bar(int, int, int, int, int, int, int, int, int, int) {}
void bar(hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t) {}

// CHECK: int foo(int m, int n) {
hipblasStatus_t foo(int m, int n) {
  return HIPBLAS_STATUS_SUCCESS;
}

int main() {
  // CHECK: foo(0, 1, 3, 7, 8, 11, 13, 14, 15, 16);
  foo(HIPBLAS_STATUS_SUCCESS, HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED, HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_ARCH_MISMATCH, HIPBLAS_STATUS_MAPPING_ERROR, HIPBLAS_STATUS_EXECUTION_FAILED, HIPBLAS_STATUS_INTERNAL_ERROR, HIPBLAS_STATUS_NOT_SUPPORTED, HIPBLAS_STATUS_UNKNOWN);
  // CHECK: bar(0, 1, 3, 7, 8, 11, 13, 14, 15, 16);
  bar(HIPBLAS_STATUS_SUCCESS, HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED, HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_ARCH_MISMATCH, HIPBLAS_STATUS_MAPPING_ERROR, HIPBLAS_STATUS_EXECUTION_FAILED, HIPBLAS_STATUS_INTERNAL_ERROR, HIPBLAS_STATUS_NOT_SUPPORTED, HIPBLAS_STATUS_UNKNOWN);

  // CHECK: int status;
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  status = cublasInit();
  cublasInit();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  status = MACRO_A;

  // CHECK: int a = sizeof(int);
  // CHECK-NEXT: a = sizeof(int);
  // CHECK-NEXT: a = sizeof(cl::sycl::queue);
  // CHECK-NEXT: a = sizeof(cl::sycl::float2);
  // CHECK-NEXT: a = sizeof(cl::sycl::double2);
  int a = sizeof(hipblasStatus_t);
  a = sizeof(hipblasStatus_t);
  a = sizeof(hipblasHandle_t);
  a = sizeof(hipComplex);
  a = sizeof(hipDoubleComplex);

  float *d_A = NULL;
  int n = 10;
  int elemSize = 4;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  //CHECK-NEXT: */
  // CHECK-NEXT: status = (dpct::dpct_malloc((void **)&d_A, (n)*(elemSize)), 0);
  // CHECK-NEXT: dpct::dpct_malloc((void **)&d_A, (n)*(elemSize));
  status = cublasAlloc(n, elemSize, (void **)&d_A);
  cublasAlloc(n, elemSize, (void **)&d_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: foo2((dpct::dpct_malloc((void **)&d_A, (n)*(elemSize)), 0));
  foo2(cublasAlloc(n, elemSize, (void **)&d_A));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (dpct::dpct_free(d_A), 0);
  // CHECK-NEXT: dpct::dpct_free(d_A);
  status = cublasFree(d_A);
  cublasFree(d_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: foo2((dpct::dpct_free(d_A), 0));
  foo2(cublasFree(d_A));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: MACRO_B((dpct::dpct_free(d_A), 0));
  MACRO_B(cublasFree(d_A));

  // CHECK: MACRO_B(0);
  MACRO_B(cublasGetError());

  MACRO_C(d_A);

  // CHECK: status = 0;
  cublasGetError();
  status = cublasGetError();

  // CHECK: foo2(0);
  foo2(cublasGetError());

  // CHECK: foo2(0);
  foo2(cublasShutdown());

  // CHECK: foo2(0);
  foo2(cublasInit());

  // CHECK: status = 0;
  // CHECK-NEXT: return 0;
  status = cublasShutdown();
  cublasShutdown();
  return 0;
}
