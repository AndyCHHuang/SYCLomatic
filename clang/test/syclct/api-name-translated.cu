// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --match-full-lines --input-file %T/api-name-translated.sycl.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  // CHECK: /*
  // CHECK-NEXT:SYCLCT1007:{{[0-9]+}}: cudaFreeHost: not support API, need manual porting.
  // CHECK-NEXT:*/
  hipHostFree(d_A);

  // CHECK: /*
  // CHECK-NEXT:SYCLCT1007:{{[0-9]+}}: cudaMallocHost: not support API, need manual porting.
  // CHECK-NEXT:*/
  hipHostMalloc(&src, bytes, hipHostMallocDefault);

  struct hipPitchedPtr srcGPU;

  // CHECK: /*
  // CHECK-NEXT:SYCLCT1007:{{[0-9]+}}: make_cudaExtent: not support API, need manual porting.
  // CHECK-NEXT:*/
  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  // CHECK: /*
  // CHECK-NEXT:SYCLCT1007:{{[0-9]+}}: cudaMalloc3D: not support API, need manual porting.
  // CHECK-NEXT:*/
  hipMalloc3D(&srcGPU, extent);
}
