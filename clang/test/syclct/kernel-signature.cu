
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -passes "FunctionAttrsRule,KernelIterationSpaceRule" -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/kernel-signature.sycl.cpp

// CHECK: void test_01(cl::sycl::nd_item<3> item_{{[a-f0-9]+}});
// CHECK: void test_06(cl::sycl::nd_item<3> item_{{[a-f0-9]+}});
// CHECK: void test_06(cl::sycl::nd_item<3> item_{{[a-f0-9]+}}, int *, int *);
// CHECK: void test_06(cl::sycl::nd_item<3> item_{{[a-f0-9]+}}, int *pA, int *pB) { }
__global__ void test_01();
__global__ void test_06();
__global__ void test_06(int *, int *);
__global__ void test_06(int *pA, int *pB) { }

// CHECK: void test_02();
__host__ void test_02();

// CHECK: void test_03();
__host__ __device__ void test_03();

// CHECK: void test_04() ;
void test_04() __device__;

// CHECK: void test_05() ;
__device__ void test_05() __device__;
