
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/max_min_windows.sycl.cpp

#if defined(_WIN32) || defined(WIN32)
#include <Windows.h>
#endif

__global__ void test_max_min(void) {
  float a = 2.0, b = 3.0;

  // CHECK: float c = cl::sycl::max(a, b);
  float c = max(a, b);

  // CHECK: float d = cl::sycl::min(a, b);
  float d = min(a, b);
}
