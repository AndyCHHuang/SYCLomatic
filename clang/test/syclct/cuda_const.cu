// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda_const.sycl.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_ELEMENTS 16
const unsigned num_elements = 16;
// CHECK: syclct::constant_memory<float, 1> const_angle(360);
// CHECK: syclct::constant_memory<float, 2> const_float(NUM_ELEMENTS, num_elements * 2);
__constant__ float const_angle[360], const_float[NUM_ELEMENTS][num_elements * 2];
// CHECK: syclct::constant_memory<cl::sycl::double2, 0> vec_d;
__constant__ double2 vec_d;

// CHECK:void simple_kernel(float *d_array, cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]], syclct::syclct_accessor<float, syclct::constant, 1> const_angle) {
// CHECK-NEXT:  int index;
// CHECK-NEXT:  index = [[ITEM]].get_group(0) * [[ITEM]].get_local_range().get(0) + [[ITEM]].get_local_id(0);
// CHECK-NEXT:  if (index < 360) {
// CHECK-NEXT:    d_array[index] = const_angle[index];
// CHECK-NEXT:  }
// CHECK-NEXT:  return;
// CHECK-NEXT:}
__global__ void simple_kernel(float *d_array) {
  int index;
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < 360) {
    d_array[index] = const_angle[index];
  }
  return;
}

// CHECK: syclct::constant_memory<float, 0> const_one;
__constant__ float const_one;

// CHECK:void simple_kernel_one(float *d_array, cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]], syclct::syclct_accessor<float, syclct::constant, 2> const_float, syclct::syclct_accessor<float, syclct::constant, 0> const_one) {
// CHECK-NEXT:  int index;
// CHECK-NEXT:  index = [[ITEM]].get_group(0) * [[ITEM]].get_local_range().get(0) + [[ITEM]].get_local_id(0);
// CHECK-NEXT:  if (index < 33) {
// CHECK-NEXT:    d_array[index] = (float)const_one + const_float[index][index];
// CHECK-NEXT:  }
// CHECK-NEXT:  return;
// CHECK-NEXT:}
__global__ void simple_kernel_one(float *d_array) {
  int index;
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < 33) {
    d_array[index] = const_one + const_float[index][index];
  }
  return;
}

int main(int argc, char **argv) {
  int size = 3200;
  float *d_array;
  float h_array[360];

  // CHECK: syclct::sycl_malloc((void **)&d_array, sizeof(float) * size);
  hipMalloc((void **)&d_array, sizeof(float) * size);

  // CHECK: syclct::sycl_memset((void*)(d_array), (int)(0), (size_t)(sizeof(float) * size));
  hipMemset(d_array, 0, sizeof(float) * size);

  for (int loop = 0; loop < 360; loop++)
    h_array[loop] = acos(-1.0f) * loop / 180.0f;

  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   (syclct::sycl_memcpy_to_symbol(const_angle.get_ptr(), (void*)(&h_array[0]), sizeof(float) * 360), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[0]), &h_array[0], sizeof(float) * 360);

  // CHECK:    {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> d_array_buf = syclct::get_buffer_and_offset(d_array);
  // CHECK-NEXT:    size_t d_array_offset = d_array_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto const_angle_acc = const_angle.get_access(cgh);
  // CHECK-NEXT:        auto d_array_acc = d_array_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class simple_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(size / 64, 1, 1) * cl::sycl::range<3>(64, 1, 1)), cl::sycl::range<3>(64, 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            float *d_array = (float*)(&d_array_acc[0] + d_array_offset);
  // CHECK-NEXT:            simple_kernel(d_array, [[ITEM]], syclct::syclct_accessor<float, syclct::constant, 1>(const_angle_acc));
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  simple_kernel<<<size / 64, 64>>>(d_array);

  float hangle_h[360];
  // CHECK:  syclct::sycl_memcpy((void*)(hangle_h), (void*)(d_array), 360 * sizeof(float), syclct::device_to_host);
  hipMemcpy(hangle_h, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 360; i++) {
    if (fabs(h_array[i] - hangle_h[i]) > 1e-5) {
      exit(-1);
    }
  }

  h_array[0] = 10.0f; // Just to test
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT:  (syclct::sycl_memcpy_to_symbol(const_one.get_ptr(), (void*)(&h_array[0]), sizeof(float) * 1), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_one), &h_array[0], sizeof(float) * 1);

  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> d_array_buf = syclct::get_buffer_and_offset(d_array);
  // CHECK-NEXT:    size_t d_array_offset = d_array_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto const_float_acc = const_float.get_access(cgh);
  // CHECK-NEXT:        auto const_one_acc = const_one.get_access(cgh);
  // CHECK-NEXT:        auto d_array_acc = d_array_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class simple_kernel_one_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(size / 64, 1, 1) * cl::sycl::range<3>(64, 1, 1)), cl::sycl::range<3>(64, 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            float *d_array = (float*)(&d_array_acc[0] + d_array_offset);
  // CHECK-NEXT:            simple_kernel_one(d_array, [[ITEM]], syclct::syclct_accessor<float, syclct::constant, 2>(const_float_acc), syclct::syclct_accessor<float, syclct::constant, 0>(const_one_acc));
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  simple_kernel_one<<<size / 64, 64>>>(d_array);

  hangle_h[360];
  // CHECK:  syclct::sycl_memcpy((void*)(hangle_h), (void*)(d_array), 360 * sizeof(float), syclct::device_to_host);
  hipMemcpy(hangle_h, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 1; i < 360; i++) {
    if (fabs(h_array[i] + 30.0f - hangle_h[i]) > 1e-5) {
      exit(-1);
    }
  }

  hipFree(d_array);

  printf("Test Passed!\n");
  return 0;
}
