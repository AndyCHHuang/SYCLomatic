#include "hip/hip_runtime.h"
// RUN: syclct -out-root %T %S/../.././test/./syclct/writing-files.cu -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/writing-files.sycl.cpp --match-full-lines %s
// RUN: FileCheck --input-file %T/test-header.sycl.hpp --match-full-lines %S/test-header.cuh

#include "test-header.cuh"

__global__ void foo() {
  // CHECK: size_t tix = item_{{[a-f0-9]+}}.get_local_id(0);
  // CHECK: size_t tiy = item_{{[a-f0-9]+}}.get_local_id(1);
  // CHECK: size_t tiz = item_{{[a-f0-9]+}}.get_local_id(2);

  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;

  // size_t bix = blockIdx.x;
  // size_t biy = blockIdx.y;
  // size_t biz = blockIdx.z;

  // CHECK: size_t bdx = item_{{[a-f0-9]+}}.get_local_range().get(0);
  // CHECK: size_t bdy = item_{{[a-f0-9]+}}.get_local_range().get(1);
  // CHECK: size_t bdz = item_{{[a-f0-9]+}}.get_local_range().get(2);

  size_t bdx = blockDim.x;
  size_t bdy = blockDim.y;
  size_t bdz = blockDim.z;

  // size_t gdx = gridDim.x;
  // size_t gdy = gridDim.y;
  // size_t gdz = gridDim.z;
}
