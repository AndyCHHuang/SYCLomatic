
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -std=c++11 -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda-get-error-string.sycl.cpp

int printf(const char* format, ...);

void test_function() {
  // CHECK:printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(0)*/);
  printf("%s\n", hipGetErrorString(hipGetLastError()));
 
  // CHECK:printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(0)*/);
  printf("%s\n", hipGetErrorString(hipSuccess));
}

