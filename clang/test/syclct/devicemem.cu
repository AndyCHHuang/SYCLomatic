// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/devicemem.sycl.cpp

#include <hip/hip_runtime.h>

#include <cassert>

#define NUM_ELEMENTS (/* Threads per block */16)

// TODO:
//   1. Multiple device variables used in a kernel function (usage analysis)
//   2. Initialized value for device variable
//   3. Muti-dimensional array

// CHECK: syclct::device_memory<float, 1> in(16);
__device__ float in[NUM_ELEMENTS];
// CHECK: syclct::device_memory<int, 1> init(syclct::syclct_range<1>(4), {1, 2, 3, 4});
__device__ int init[4] = {1, 2, 3, 4};

// CHECK: void kernel1(float *out, cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]], syclct::syclct_accessor<float, syclct::device, 1> in) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(0)] = in[{{.*}}[[ITEM]].get_local_id(0)];
// CHECK: }
__global__ void kernel1(float *out) {
  out[threadIdx.x] = in[threadIdx.x];
}

// CHECK: syclct::device_memory<int, 0> al;
__device__ int al;
// CHECK: syclct::device_memory<int, 0> ainit(syclct::syclct_range<0>(), (16));
__device__ int ainit = NUM_ELEMENTS;

// CHECK: syclct::device_memory<float, 1> fx(2);
// CHECK: syclct::device_memory<float, 2> fy(3, 4);
__device__ float fx[2], fy[3][4];

// CHECK: void kernel2(float *out, cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]], syclct::syclct_accessor<int, syclct::device, 0> al, syclct::syclct_accessor<float, syclct::device, 1> fx, syclct::syclct_accessor<float, syclct::device, 2> fy) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(0)] += (int)al;
// CHECK:   fx[{{.*}}[[ITEM]].get_local_id(0)] = fy[{{.*}}[[ITEM]].get_local_id(0)][{{.*}}[[ITEM]].get_local_id(0)];
// CHECK: }
__global__ void kernel2(float *out) {
  out[threadIdx.x] += al;
  fx[threadIdx.x] = fy[threadIdx.x][threadIdx.x];
}

int main() {
  float h_in[NUM_ELEMENTS] = { 0 };
  float h_out[NUM_ELEMENTS] = { 0 };

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    h_in[i] = i;
    h_out[i] = -i;
  }

  const size_t array_size = sizeof(float) * NUM_ELEMENTS;
  // CTST-50
  hipMemcpyToSymbol(HIP_SYMBOL(in), h_in, array_size);

  const int h_a = 3;
  // CTST-50
  hipMemcpyToSymbol(HIP_SYMBOL(al), &h_a, sizeof(int));

  float *d_out = NULL;
  hipMalloc((void **)&d_out, array_size);

  const int threads_per_block = NUM_ELEMENTS;
  // CHECK: {
  // CHECK:   std::pair<syclct::buffer_t, size_t> d_out_buf = syclct::get_buffer_and_offset(d_out);
  // CHECK:   size_t d_out_offset = d_out_buf.second;
  // CHECK:   syclct::get_default_queue().submit(
  // CHECK:     [&](cl::sycl::handler &cgh) {
  // CHECK:       auto in_acc = in.get_access(cgh);
  // CHECK:       auto d_out_acc = d_out_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK:       cgh.parallel_for<syclct_kernel_name<class kernel1_{{[a-f0-9]+}}>>(
  // CHECK:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(threads_per_block, 1, 1)), cl::sycl::range<3>(threads_per_block, 1, 1)),
  // CHECK:         [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK:           float *d_out = (float*)(&d_out_acc[0] + d_out_offset);
  // CHECK:           kernel1(d_out, [[ITEM]], syclct::syclct_accessor<float, syclct::device, 1>(in_acc));
  // CHECK:         });
  // CHECK:     });
  // CHECK: };
  kernel1<<<1, threads_per_block>>>(d_out);

  // CHECK: {
  // CHECK:   std::pair<syclct::buffer_t, size_t> d_out_buf = syclct::get_buffer_and_offset(d_out);
  // CHECK:   size_t d_out_offset = d_out_buf.second;
  // CHECK:   syclct::get_default_queue().submit(
  // CHECK:     [&](cl::sycl::handler &cgh) {
  // CHECK:       auto al_acc = al.get_access(cgh);
  // CHECK:       auto fx_acc = fx.get_access(cgh);
  // CHECK:       auto fy_acc = fy.get_access(cgh);
  // CHECK:       auto d_out_acc = d_out_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK:       cgh.parallel_for<syclct_kernel_name<class kernel2_{{[a-f0-9]+}}>>(
  // CHECK:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(threads_per_block, 1, 1)), cl::sycl::range<3>(threads_per_block, 1, 1)),
  // CHECK:         [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK:           float *d_out = (float*)(&d_out_acc[0] + d_out_offset);
  // CHECK:           kernel2(d_out, [[ITEM]], syclct::syclct_accessor<int, syclct::device, 0>(al_acc), syclct::syclct_accessor<float, syclct::device, 1>(fx_acc), syclct::syclct_accessor<float, syclct::device, 2>(fy_acc));
  // CHECK:         });
  // CHECK:     });
  // CHECK: };
  kernel2<<<1, threads_per_block>>>(d_out);

  hipMemcpy(h_out, d_out, array_size, hipMemcpyDeviceToHost);

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    assert(h_out[i] == i + h_a && "Value mis-calculated!");
  }

  return 0;
}
