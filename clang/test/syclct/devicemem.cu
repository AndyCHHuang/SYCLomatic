#include "hip/hip_runtime.h"
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/devicemem.sycl.cpp

#include <hip/hip_runtime.h>

#include <cassert>

#define NUM_ELEMENTS (/* Threads per block */ 16)

// CHECK: syclct::device_memory<float, 1> in(NUM_ELEMENTS);
__device__ float in[NUM_ELEMENTS];
// CHECK: syclct::device_memory<int, 1> init(syclct::syclct_range<1>(4), {1, 2, 3, 4});
__device__ int init[4] = {1, 2, 3, 4};

// CHECK: void kernel1(float *out, cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]], syclct::syclct_accessor<float, syclct::device, 1> in) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(0)] = in[{{.*}}[[ITEM]].get_local_id(0)];
// CHECK: }
__global__ void kernel1(float *out) {
  out[threadIdx.x] = in[threadIdx.x];
}

// CHECK: syclct::device_memory<int, 0> al;
__device__ int al;
// CHECK: syclct::device_memory<int, 0> ainit(syclct::syclct_range<0>(), NUM_ELEMENTS);
__device__ int ainit = NUM_ELEMENTS;

const int num_elements = 16;
// CHECK: syclct::device_memory<float, 1> fx(2);
// CHECK: syclct::device_memory<float, 2> fy(num_elements, 4 * num_elements);
__device__ float fx[2], fy[num_elements][4 * num_elements];

// CHECK: void kernel2(float *out, cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]], syclct::syclct_accessor<int, syclct::device, 0> al, syclct::syclct_accessor<float, syclct::device, 1> fx, syclct::syclct_accessor<float, syclct::device, 2> fy, syclct::syclct_accessor<float, syclct::device, 1> tmp) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(0)] += (int)al;
// CHECK:   fx[{{.*}}[[ITEM]].get_local_id(0)] = fy[{{.*}}[[ITEM]].get_local_id(0)][{{.*}}[[ITEM]].get_local_id(0)];
// CHECK: }
__global__ void kernel2(float *out) {
  const int size = 64;
  __device__ float tmp[size];
  out[threadIdx.x] += al;
  fx[threadIdx.x] = fy[threadIdx.x][threadIdx.x];
}

int main() {
  float h_in[NUM_ELEMENTS] = {0};
  float h_out[NUM_ELEMENTS] = {0};

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    h_in[i] = i;
    h_out[i] = -i;
  }

  const size_t array_size = sizeof(float) * NUM_ELEMENTS;
  // CTST-50
  hipMemcpyToSymbol(HIP_SYMBOL(in), h_in, array_size);

  const int h_a = 3;
  // CTST-50
  hipMemcpyToSymbol(HIP_SYMBOL(al), &h_a, sizeof(int));

  float *d_out = NULL;
  hipMalloc((void **)&d_out, array_size);

  const int threads_per_block = NUM_ELEMENTS;
  // CHECK: {
  // CHECK:   std::pair<syclct::buffer_t, size_t> d_out_buf = syclct::get_buffer_and_offset(d_out);
  // CHECK:   size_t d_out_offset = d_out_buf.second;
  // CHECK:   syclct::get_default_queue().submit(
  // CHECK:     [&](cl::sycl::handler &cgh) {
  // CHECK:       auto in_acc_[[HASH:[a-f0-9]+]] = in.get_access(cgh);
  // CHECK:       auto d_out_acc = d_out_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK:       cgh.parallel_for<syclct_kernel_name<class kernel1_{{[a-f0-9]+}}>>(
  // CHECK:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(threads_per_block, 1, 1)), cl::sycl::range<3>(threads_per_block, 1, 1)),
  // CHECK:         [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK:           float *d_out = (float*)(&d_out_acc[0] + d_out_offset);
  // CHECK:           kernel1(d_out, [[ITEM]], syclct::syclct_accessor<float, syclct::device, 1>(in_acc_[[HASH]]));
  // CHECK:         });
  // CHECK:     });
  // CHECK: }
  kernel1<<<1, threads_per_block>>>(d_out);

  // CHECK: {
  // CHECK:   std::pair<syclct::buffer_t, size_t> d_out_buf = syclct::get_buffer_and_offset(d_out);
  // CHECK:   size_t d_out_offset = d_out_buf.second;
  // CHECK:   syclct::get_default_queue().submit(
  // CHECK:     [&](cl::sycl::handler &cgh) {
  // CHECK:       syclct::device_memory<float, 1> tmp(64/*size*/);
  // CHECK:       auto tmp_acc_[[HASH]] = tmp.get_access(cgh);
  // CHECK:       auto al_acc_[[HASH]] = al.get_access(cgh);
  // CHECK:       auto fx_acc_[[HASH]] = fx.get_access(cgh);
  // CHECK:       auto fy_acc_[[HASH]] = fy.get_access(cgh);
  // CHECK:       auto d_out_acc = d_out_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK:       cgh.parallel_for<syclct_kernel_name<class kernel2_{{[a-f0-9]+}}>>(
  // CHECK:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(threads_per_block, 1, 1)), cl::sycl::range<3>(threads_per_block, 1, 1)),
  // CHECK:         [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK:           float *d_out = (float*)(&d_out_acc[0] + d_out_offset);
  // CHECK:           kernel2(d_out, [[ITEM]], syclct::syclct_accessor<int, syclct::device, 0>(al_acc_[[HASH]]), syclct::syclct_accessor<float, syclct::device, 1>(fx_acc_[[HASH]]), syclct::syclct_accessor<float, syclct::device, 2>(fy_acc_[[HASH]]), syclct::syclct_accessor<float, syclct::device, 1>(tmp_acc_[[HASH]]));
  // CHECK:         });
  // CHECK:     });
  // CHECK: }
  kernel2<<<1, threads_per_block>>>(d_out);

  hipMemcpy(h_out, d_out, array_size, hipMemcpyDeviceToHost);

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    assert(h_out[i] == i + h_a && "Value mis-calculated!");
  }

  return 0;
}
