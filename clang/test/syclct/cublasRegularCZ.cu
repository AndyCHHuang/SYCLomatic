// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cublasRegularCZ.sycl.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;

  int* result = 0;
  float* result_f = 0;
  double* result_d = 0;
  hipComplex* x_c = 0;
  hipDoubleComplex* x_z = 0;

  int incx = 1;
  int incy = 1;
  int n = 10;

  //level 1
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::icamax(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::icamax(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamax(handle, n, x_c, incx, result);
  hipblasIcamax(handle, n, x_c, incx, result);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::izamax(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::izamax(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamax(handle, n, x_z, incx, result);
  hipblasIzamax(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::icamin(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::icamin(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamin(handle, n, x_c, incx, result);
  hipblasIcamin(handle, n, x_c, incx, result);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::izamin(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> result_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(result_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::izamin(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: result_{{[0-9]+}}_buffer_{{[0-9a-z]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamin(handle, n, x_z, incx, result);
  hipblasIzamin(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: status = (mkl::scasum(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::scasum(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasScasum(handle, n, x_c, incx, result_f);
  hipblasScasum(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  // CHECK-NEXT: status = (mkl::dzasum(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  // CHECK-NEXT: mkl::dzasum(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasDzasum(handle, n, x_z, incx, result_d);
  hipblasDzasum(handle, n, x_z, incx, result_d);

  hipComplex* alpha_c = 0;
  hipComplex* beta_c = 0;
  hipDoubleComplex* alpha_z = 0;
  hipDoubleComplex* beta_z = 0;
  float* alpha_f = 0;
  double* alpha_d = 0;
  hipComplex* y_c = 0;
  hipDoubleComplex* y_z = 0;

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::caxpy(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::caxpy(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);
  hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zaxpy(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zaxpy(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);
  hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ccopy(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ccopy(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCcopy(handle, n, x_c, incx, y_c, incy);
  hipblasCcopy(handle, n, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zcopy(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zcopy(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZcopy(handle, n, x_z, incx, y_z, incy);
  hipblasZcopy(handle, n, x_z, incx, y_z, incy);

  hipComplex* result_c = 0;
  hipDoubleComplex* result_z = 0;

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cdotu(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cdotu(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cdotc(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cdotc(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zdotu(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zdotu(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zdotc(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zdotc(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: status = (mkl::scnrm2(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(result_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::scnrm2(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasScnrm2(handle, n, x_c, incx, result_f);
  hipblasScnrm2(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  // CHECK-NEXT: status = (mkl::dznrm2(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(result_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  // CHECK-NEXT: mkl::dznrm2(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasDznrm2(handle, n, x_z, incx, result_d);
  hipblasDznrm2(handle, n, x_z, incx, result_d);

  float* c_f = 0;
  float* s_f = 0;
  double* c_d = 0;
  double* s_d = 0;
  hipComplex* c_c = 0;
  hipComplex* s_c = 0;
  hipDoubleComplex* c_z = 0;
  hipDoubleComplex* s_z = 0;

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::csrot(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, *(c_f), *(s_f)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::csrot(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, *(c_f), *(s_f));
  // CHECK-NEXT: }
  status = hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);
  hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zdrot(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, *(c_d), *(s_d)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zdrot(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, *(c_d), *(s_d));
  // CHECK-NEXT: }
  status = hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);
  hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto c_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(c_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> c_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = c_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(c_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto s_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(s_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> s_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = s_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(s_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::crotg(handle, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, c_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, s_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto c_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(c_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> c_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = c_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(c_f_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto s_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(s_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> s_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = s_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(s_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::crotg(handle, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, c_f_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, s_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasCrotg(handle, x_c, y_c, c_f, s_c);
  hipblasCrotg(handle, x_c, y_c, c_f, s_c);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto c_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(c_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> c_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = c_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(c_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto s_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(s_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> s_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = s_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(s_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zrotg(handle, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, c_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, s_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto c_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(c_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> c_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = c_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(c_d_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto s_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(s_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> s_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = s_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(s_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zrotg(handle, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, c_d_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, s_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasZrotg(handle, x_z, y_z, c_d, s_z);
  hipblasZrotg(handle, x_z, y_z, c_d, s_z);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cscal(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cscal(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCscal(handle, n, alpha_c, x_c, incx);
  hipblasCscal(handle, n, alpha_c, x_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zscal(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zscal(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZscal(handle, n, alpha_z, x_z, incx);
  hipblasZscal(handle, n, alpha_z, x_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::csscal(handle, n, *(alpha_f), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::csscal(handle, n, *(alpha_f), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCsscal(handle, n, alpha_f, x_c, incx);
  hipblasCsscal(handle, n, alpha_f, x_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zdscal(handle, n, *(alpha_d), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zdscal(handle, n, *(alpha_d), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZdscal(handle, n, alpha_d, x_z, incx);
  hipblasZdscal(handle, n, alpha_d, x_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cswap(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cswap(handle, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCswap(handle, n, x_c, incx, y_c, incy);
  hipblasCswap(handle, n, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zswap(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zswap(handle, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZswap(handle, n, x_z, incx, y_z, incy);
  hipblasZswap(handle, n, x_z, incx, y_z, incy);

  //level 2
  int m=0;
  int kl=0;
  int ku=0;
  int lda = 10;
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cgbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cgbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zgbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zgbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cgemv(handle, mkl::transpose::nontrans, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cgemv(handle, mkl::transpose::nontrans, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zgemv(handle, mkl::transpose::nontrans, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zgemv(handle, mkl::transpose::nontrans, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cgeru(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cgeru(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cgerc(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cgerc(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zgeru(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zgeru(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zgerc(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zgerc(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  int k = 1;
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ctbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ctbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);
  hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::ztbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::ztbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);
  hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ctbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ctbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);
  hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::ztbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::ztbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);
  hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ctpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ctpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);
  hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::ztpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::ztpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);
  hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ctpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ctpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);
  hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::ztpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::ztpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);
  hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ctrmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ctrmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);
  hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::ztrmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::ztrmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);
  hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::ctrsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::ctrsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);
  hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::ztrsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::ztrsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);
  hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::chemv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::chemv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zhemv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zhemv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::chbmv(handle, mkl::uplo::lower, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::chbmv(handle, mkl::uplo::lower, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zhbmv(handle, mkl::uplo::lower, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zhbmv(handle, mkl::uplo::lower, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::chpmv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::chpmv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);
  hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zhpmv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zhpmv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);
  hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cher(handle, mkl::uplo::lower, n, *(alpha_f), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cher(handle, mkl::uplo::lower, n, *(alpha_f), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);
  hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zher(handle, mkl::uplo::lower, n, *(alpha_d), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zher(handle, mkl::uplo::lower, n, *(alpha_d), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);
  hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cher2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cher2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zher2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zher2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::chpr(handle, mkl::uplo::lower, n, *(alpha_f), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::chpr(handle, mkl::uplo::lower, n, *(alpha_f), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);
  hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zhpr(handle, mkl::uplo::lower, n, *(alpha_d), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zhpr(handle, mkl::uplo::lower, n, *(alpha_d), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);
  hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::chpr2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::chpr2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);
  hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zhpr2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zhpr2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incx, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, incy, result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);
  hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);

  int N = 100;
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, std::complex<float>((beta_c)->x(),(beta_c)->y()), result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(x_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(y_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(result_c_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, y_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, std::complex<float>((beta_c)->x(),(beta_c)->y()), result_c_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N);
  // CHECK-NEXT: }
  status = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, std::complex<double>((beta_z)->x(),(beta_z)->y()), result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(x_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(y_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}} = result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(result_z_{{[0-9]+}}_allocation_{{[0-9a-z]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, y_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N, std::complex<double>((beta_z)->x(),(beta_z)->y()), result_z_{{[0-9]+}}_buffer_{{[0-9a-z]+}}, N);
  // CHECK-NEXT: }
  status = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
}