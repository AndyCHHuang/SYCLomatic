// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/insert_sycl_header_anyway.sycl.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>
// CHECK-NEXT: #include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
int main(){
  return 0;
}

