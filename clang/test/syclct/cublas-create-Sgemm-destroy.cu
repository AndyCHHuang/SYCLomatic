// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy.sycl.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <DPCPP_blas_TEMP.h>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
int main() {
  // CHECK: int status;
  // CHECK-NEXT: cl::sycl::queue handle;
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }
  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  // CHECK: status = (cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N), 0);
  // CHECK-NEXT: mkl::Sgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: status = (cublasDgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N), 0);
  // CHECK-NEXT: mkl::Dgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  float2 *d_A_C = 0;
  float2 *d_B_C = 0;
  float2 *d_C_C = 0;
  float2 alpha_C = make_float2(1, 0);
  float2 beta_C = make_float2(0, 0);
  // CHECK: status = (cublasCgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N), 0);
  // CHECK-NEXT: mkl::Cgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  status = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  double2 *d_A_Z = 0;
  double2 *d_B_Z = 0;
  double2 *d_C_Z = 0;
  double2 alpha_Z = make_double2(1, 0);
  double2 beta_Z = make_double2(0, 0);
  // CHECK: status = (cublasZgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N), 0);
  // CHECK-NEXT: mkl::Zgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);
  status = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);
  // CHECK: status = 0;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}