// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy.sycl.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <sycl_types.hpp>
// CHECK: #include <complex>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

int main() {
  // CHECK: int status;
  // CHECK-NEXT: cl::sycl::queue handle;
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }
  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), d_A_S_BUFFER_{{[0-9,a-z]+}}, N, d_B_S_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_S), d_C_S_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK: mkl::sgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), d_A_S_BUFFER_{{[0-9,a-z]+}}, N, d_B_S_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_S), d_C_S_BUFFER_{{[0-9,a-z]+}}, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_D), d_A_D_BUFFER_{{[0-9,a-z]+}}, N, d_B_D_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_D), d_C_D_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK: mkl::dgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_D), d_A_D_BUFFER_{{[0-9,a-z]+}}, N, d_B_D_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_D), d_C_D_BUFFER_{{[0-9,a-z]+}}, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  float2 *d_A_C = 0;
  float2 *d_B_C = 0;
  float2 *d_C_C = 0;
  float2 alpha_C = make_float2(1, 0);
  float2 beta_C = make_float2(0, 0);
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::cgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_C), d_A_C_BUFFER_{{[0-9,a-z]+}}, N, d_B_C_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_C), d_C_C_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK: mkl::cgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_C), d_A_C_BUFFER_{{[0-9,a-z]+}}, N, d_B_C_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_C), d_C_C_BUFFER_{{[0-9,a-z]+}}, N);
  status = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  double2 *d_A_Z = 0;
  double2 *d_B_Z = 0;
  double2 *d_C_Z = 0;
  double2 alpha_Z = make_double2(1, 0);
  double2 beta_Z = make_double2(0, 0);
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::zgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_Z), d_A_Z_BUFFER_{{[0-9,a-z]+}}, N, d_B_Z_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_Z), d_C_Z_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK: mkl::zgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_Z), d_A_Z_BUFFER_{{[0-9,a-z]+}}, N, d_B_Z_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_Z), d_C_Z_BUFFER_{{[0-9,a-z]+}}, N);
  status = hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);
  hipComplex *d_A_C_2 = 0;
  hipComplex *d_B_C_2 = 0;
  hipComplex *d_C_C_2 = 0;
  hipComplex alpha_C_2 = make_float2(1, 0);
  hipComplex beta_C_2 = make_float2(0, 0);
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::cgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_C_2), d_A_C_2_BUFFER_{{[0-9,a-z]+}}, N, d_B_C_2_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_C_2), d_C_C_2_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK: mkl::cgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_C_2), d_A_C_2_BUFFER_{{[0-9,a-z]+}}, N, d_B_C_2_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_C_2), d_C_C_2_BUFFER_{{[0-9,a-z]+}}, N);
  status = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C_2, d_A_C_2, N, d_B_C_2, N, &beta_C_2, d_C_C_2, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C_2, d_A_C_2, N, d_B_C_2, N, &beta_C_2, d_C_C_2, N);
  hipDoubleComplex *d_A_Z_2 = 0;
  hipDoubleComplex *d_B_Z_2 = 0;
  hipDoubleComplex *d_C_Z_2 = 0;
  hipDoubleComplex alpha_Z_2 = make_double2(1, 0);
  hipDoubleComplex beta_Z_2 = make_double2(0, 0);
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::zgemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_Z_2), d_A_Z_2_BUFFER_{{[0-9,a-z]+}}, N, d_B_Z_2_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_Z_2), d_C_Z_2_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK: mkl::zgemm(handle, mkl::transpose::conjtrans, mkl::transpose::conjtrans, N, N, N, *(&alpha_Z_2), d_A_Z_2_BUFFER_{{[0-9,a-z]+}}, N, d_B_Z_2_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_Z_2), d_C_Z_2_BUFFER_{{[0-9,a-z]+}}, N);
  status = hipblasZgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_Z_2, d_A_Z_2, N, d_B_Z_2, N, &beta_Z_2, d_C_Z_2, N);
  hipblasZgemm(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_Z_2, d_A_Z_2, N, d_B_Z_2, N, &beta_Z_2, d_C_Z_2, N);


  // CHECK: for (;;) {
  // CHECK-NEXT: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_A_S_BUFFER_{{[0-9,a-z]+}} = d_A_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_A_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto d_B_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_B_S_BUFFER_{{[0-9,a-z]+}} = d_B_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_B_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto d_C_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_C_S_BUFFER_{{[0-9,a-z]+}} = d_C_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_C_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: status = (mkl::sgemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_S), d_A_S_BUFFER_{{[0-9,a-z]+}}, N, d_B_S_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_S), d_C_S_BUFFER_{{[0-9,a-z]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_A_S_BUFFER_{{[0-9,a-z]+}} = d_A_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_A_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto d_B_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_B_S_BUFFER_{{[0-9,a-z]+}} = d_B_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_B_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto d_C_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_C_S_BUFFER_{{[0-9,a-z]+}} = d_C_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_C_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::sgemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_S), d_A_S_BUFFER_{{[0-9,a-z]+}}, N, d_B_S_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_S), d_C_S_BUFFER_{{[0-9,a-z]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_A_S_BUFFER_{{[0-9,a-z]+}} = d_A_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_A_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto d_B_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_B_S_BUFFER_{{[0-9,a-z]+}} = d_B_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_B_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto d_C_S_ALLOCATION_{{[0-9,a-z]+}} = syclct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> d_C_S_BUFFER_{{[0-9,a-z]+}} = d_C_S_ALLOCATION_{{[0-9,a-z]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(d_C_S_ALLOCATION_{{[0-9,a-z]+}}.size/sizeof(float)));
  // CHECK-NEXT: foo(bar((mkl::sgemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), d_A_S_BUFFER_{{[0-9,a-z]+}}, N, d_B_S_BUFFER_{{[0-9,a-z]+}}, N, *(&beta_S), d_C_S_BUFFER_{{[0-9,a-z]+}}, N), 0)));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  // CHECK: status = 0;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}