#include "hip/hip_runtime.h"
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/replace-cuda-header-includes.sycl.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>
// CHECK-NEXT: #include "test-header.sycl.hpp"
// CHECK: // First function
#include "test-header.cuh"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hipfft/hipfft.h>
// First function
__global__ void foo() {
  // CHECK: size_t tix = item_{{[a-f0-9]+}}.get_local_id(0);
  // CHECK: size_t tiy = item_{{[a-f0-9]+}}.get_local_id(1);
  // CHECK: size_t tiz = item_{{[a-f0-9]+}}.get_local_id(2);

  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;

  // size_t bix = blockIdx.x;
  // size_t biy = blockIdx.y;
  // size_t biz = blockIdx.z;

  // CHECK: size_t bdx = item_{{[a-f0-9]+}}.get_local_range().get(0);
  // CHECK: size_t bdy = item_{{[a-f0-9]+}}.get_local_range().get(1);
  // CHECK: size_t bdz = item_{{[a-f0-9]+}}.get_local_range().get(2);

  size_t bdx = blockDim.x;
  size_t bdy = blockDim.y;
  size_t bdz = blockDim.z;

  // size_t gdx = gridDim.x;
  // size_t gdy = gridDim.y;
  // size_t gdz = gridDim.z;
}
