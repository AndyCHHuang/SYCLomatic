// RUN: syclct -out-root %T %s -- -std=c++14 -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cuda-event-api.sycl.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernelFunc()
{
}

int main(int argc, char* argv[]) {
  // CHECK: cl::sycl::event start, stop;
  // CHECK-EMPTY:
  // CHECK-EMPTY:
  // CHECK-NEXT: float elapsed_time;
  // CHECK-EMPTY:
  // CHECK-NEXT: syclct::get_device_manager().current_device().queues_wait_and_throw();
  // CHECK-EMPTY:
  // CHECK-NEXT: int blocks = 32, threads = 32;
  hipEvent_t start, stop;

  hipEventCreate(&start)  
    ;   
  hipEventCreate(&stop)  ;   

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  // kernel call without sync
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in DPC++. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: auto syclct_start_{{[a-f0-9]+}} = clock();
  hipEventRecord(start, 0);

  // kernel call without sync
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in DPC++. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: syclct_start_{{[a-f0-9]+}} = clock();
  hipEventRecord(start, 0);

  // kernel call with sync
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     }).wait();
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     }).wait();
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in DPC++. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: auto syclct_stop_{{[a-f0-9]+}} = clock();
  hipEventRecord(stop, 0);

  // kernel call without sync
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     }).wait();
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in DPC++. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: syclct_stop_{{[a-f0-9]+}} = clock();
  hipEventRecord(stop, 0);

  // CHECK: stop.wait_and_throw();
  hipEventSynchronize(stop);

  // kernel call without sync
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();

  // CHECK: *(&elapsed_time) = (float)(syclct_stop_{{[a-f0-9]+}} - syclct_start_{{[a-f0-9]+}}) / CLOCKS_PER_SEC * 1000;
  hipEventElapsedTime(&elapsed_time, start, stop);

  // kernel call without sync
  // CHECK: {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<syclct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(blocks, 1, 1) * cl::sycl::range<3>(threads, 1, 1)), cl::sycl::range<3>(threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  kernelFunc<<<blocks,threads>>>();

  // CHECK: syclct::get_device_manager().current_device().queues_wait_and_throw();
  // CHECK-EMPTY:
  // CHECK-NEXT: }
  hipDeviceSynchronize();

  hipEventDestroy(start)  ;   
  hipEventDestroy(stop)  
    ;   
}
