// RUN: syclct -report-type=apis -report-file-prefix=check-apis-report -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: echo "// `perl -e 'print "CH","ECK"'`: API name, Frequency" >%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaDeviceSynchronize,4" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFree,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemset,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemcpy,2" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFreeHost,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMallocHost,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc3D,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: make_cudaExtent,1" >>%T/check-apis-report_csv_check.txt
// RUN: cat %T/check-apis-report.apis.csv >>%T/check-apis-report_csv_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report_csv_check.txt %T/check-apis-report_csv_check.txt

// RUN: syclct -report-file-prefix=report -report-type=apis  -report-format=formatted -report-only  -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: echo "// `perl -e 'print "CH","ECK"'`: API name                                Frequency" >%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaDeviceSynchronize                        4" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFree                                     1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemset                                   1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemcpy                                   2" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc                                   1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFreeHost                                 1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMallocHost                               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc3D                                 1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: make_cudaExtent                              1" >>%T/check-apis-report_check.txt
// RUN: cat %T/report.apis.log >>%T/check-apis-report_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report_check.txt %T/check-apis-report_check.txt

#include <hip/hip_runtime.h>

void checkError(hipError_t err) {
}

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  hipHostFree(d_A);

  hipHostMalloc(&src, bytes, hipHostMallocDefault);

  struct hipPitchedPtr srcGPU;

  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  hipMalloc3D(&srcGPU, extent);
}

int cool() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  hipMemset(d_A, 0xf, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  free(h_A);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  checkError(hipDeviceSynchronize());
  return hipDeviceSynchronize();
}
