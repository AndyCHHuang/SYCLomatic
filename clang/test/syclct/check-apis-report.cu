// RUN: syclct -report-type=apis -report-file-prefix=check-apis-report -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: echo "// `perl -e 'print "CH","ECK"'`: API name, Frequency" >%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFree,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemcpy,2" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemset,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: longlong4,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc3D,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: dim3,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaDeviceSynchronize,4" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMallocHost,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: uint4,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaError_t,2" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFreeHost,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: int2,3" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: make_cudaExtent,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaDeviceProp,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaStream_t,1" >>%T/check-apis-report_csv_check.txt
// RUN: cat %T/check-apis-report.apis.csv >>%T/check-apis-report_csv_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report_csv_check.txt %T/check-apis-report_csv_check.txt

// RUN: syclct -report-file-prefix=report -report-type=apis  -report-format=formatted -report-only  -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: echo "// `perl -e 'print "CH","ECK"'`: API name                                Frequency" >%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFree                                     1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemcpy                                   2" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMemset                                   1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc                                   1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: longlong4                                    1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMalloc3D                                 1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: dim3                                         1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaDeviceSynchronize                        4" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaMallocHost                               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: uint4                                        1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaError_t                                  2" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaFreeHost                                 1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: int2                                         3" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: make_cudaExtent                              1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaDeviceProp                               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK-NEXT"'`: cudaStream_t                                 1" >>%T/check-apis-report_check.txt
// RUN: cat %T/report.apis.log >>%T/check-apis-report_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report_check.txt %T/check-apis-report_check.txt

#include <hip/hip_runtime.h>

void checkError(hipError_t err) {
}

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  hipHostFree(d_A);

  hipHostMalloc(&src, bytes, hipHostMallocDefault);

  struct hipPitchedPtr srcGPU;

  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  hipMalloc3D(&srcGPU, extent);

  int2 a;
  uint4 b;
  dim3 d3;
  hipDeviceProp_t cdp;
  hipStream_t cuSt;
  const int2 c = {0,0};
  int2 d[100];
  longlong4 ll4;
}

int cool() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  hipMemset(d_A, 0xf, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  free(h_A);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  checkError(hipDeviceSynchronize());
  return hipDeviceSynchronize();
}
