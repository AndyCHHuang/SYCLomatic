
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/kernel-function-info.sycl.cpp --match-full-lines %s

//CHECK: template<class T>
//CHECK-NEXT: void testTemplateKernel(cl::sycl::nd_item<3> item, T *data)
template<class T>
__global__ void testTemplateKernel(T *data)
{
}

//CHECK: void testKernel(cl::sycl::nd_item<3> item, void* data)
__global__ void testKernel(void* data)
{
}

template<class T>
void getTemplateFuncAttrs()
{
  //CHECK: sycl_kernel_function_info attrs;
  hipFuncAttributes attrs;

  //CHECK: getSyclKernelFunctionInfo(&attrs, (const void *)testTemplateKernel<T>);
  hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(testTemplateKernel<T>));

  //CHECK: int threadPerBlock = attrs.max_work_group_size;
  int threadPerBlock = attrs.maxThreadsPerBlock;
}

void getFuncAttrs()
{
  //CHECK: sycl_kernel_function_info attrs;
  hipFuncAttributes attrs;

  //CHECK: getSyclKernelFunctionInfo(&attrs, (const void *)testKernel);
  hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(testKernel));

  //CHECK: int threadPerBlock = attrs.max_work_group_size;
  int threadPerBlock = attrs.maxThreadsPerBlock;

}

int main()
{
  getTemplateFuncAttrs<int>();
  getFuncAttrs();
}
