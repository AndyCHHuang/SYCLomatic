// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cusolverDnEi.sycl.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    hipsolverEigMode_t jobz;

    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    signed char jobu;
    signed char jobvt;

    float S_f = 0;
    double S_d = 0.0;
    hipComplex S_c = make_hipComplex(1,0);
    hipDoubleComplex S_z = make_hipDoubleComplex(1,0);

    float U_f = 0;
    double U_d = 0.0;
    hipComplex U_c = make_hipComplex(1,0);
    hipDoubleComplex U_z = make_hipDoubleComplex(1,0);
    int ldu;

    float VT_f = 0;
    double VT_d = 0.0;
    hipComplex VT_c = make_hipComplex(1,0);
    hipDoubleComplex VT_z = make_hipDoubleComplex(1,0);
    int ldvt;

    float Rwork_f = 0;
    double Rwork_d = 0.0;
    hipComplex Rwork_c = make_hipComplex(1,0);
    hipDoubleComplex Rwork_z = make_hipDoubleComplex(1,0);

    float W_f = 0;
    double W_d = 0.0;
    hipComplex W_c = make_hipComplex(1,0);
    hipDoubleComplex W_z = make_hipDoubleComplex(1,0);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_d(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_e(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_tauq(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_taup(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgebrd_get_lwork(*cusolverH, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_d, buffer_ct_mkl_e, buffer_ct_mkl_tauq, buffer_ct_mkl_taup, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAUQ_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAUP_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgebrd(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, buffer_ct9, Lwork, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sorgbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sorgbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sorgbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sorgbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dorgbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dorgbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dorgbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dorgbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cungbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cungbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cungbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cungbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zungbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zungbr_get_lwork(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zungbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zungbr(*cusolverH, (mkl::vector)side, m, n, k, buffer_ct5, lda, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);





    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::ssytrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::ssytrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::ssytrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::ssytrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    status = hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dsytrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dsytrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dsytrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dsytrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    status = hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::chetrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::chetrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::chetrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::chetrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    status = hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zhetrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zhetrd_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zhetrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&D_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&E_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer10(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zhetrd(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, buffer_ct7, buffer_ct8, Lwork, result_temp_buffer10);
    // CHECK-NEXT: buffer_ct10.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    status = hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sormtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sormtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sormtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sormtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    status = hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dormtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dormtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dormtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dormtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    status = hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cunmtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cunmtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cunmtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cunmtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    status = hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zunmtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11), 0);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer11(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zunmtr_get_lwork(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, result_temp_buffer11);
    // CHECK-NEXT: buffer_ct11.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zunmtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = syclct::memory_manager::get_instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct13 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct13 = allocation_ct13.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zunmtr(*cusolverH, side, uplo, trans, m, n, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    status = hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sorgtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sorgtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sorgtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sorgtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dorgtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dorgtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dorgtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dorgtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cungtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cungtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cungtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cungtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zungtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zungtr_get_lwork(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, result_temp_buffer6);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zungtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zungtr(*cusolverH, uplo, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: status = (mkl::sgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: mkl::sgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::sgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork,  result_temp_buffer15), 0);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::sgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork,  result_temp_buffer15);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);
    hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: status = (mkl::dgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: mkl::dgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::dgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork,  result_temp_buffer15), 0);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::dgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork,  result_temp_buffer15);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);
    hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: status = (mkl::cgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: mkl::cgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct14 = syclct::memory_manager::get_instance().translate_ptr(&Rwork_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct14 = allocation_ct14.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct14.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::cgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork, buffer_ct14, result_temp_buffer15), 0);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct14 = syclct::memory_manager::get_instance().translate_ptr(&Rwork_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct14 = allocation_ct14.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct14.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::cgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork, buffer_ct14, result_temp_buffer15);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);
    hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: status = (mkl::zgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3), 0);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = syclct::memory_manager::get_instance().translate_ptr(&Lwork);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer3(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_a(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_lda;
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct_mkl_s(cl::sycl::range<1>(1));
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_u(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldu;
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct_mkl_vt(cl::sycl::range<1>(1));
    // CHECK-NEXT: int64_t int64_t_ct_mkl_ldvt;
    // CHECK-NEXT: mkl::zgesvd_get_lwork(*cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, buffer_ct_mkl_a, int64_t_ct_mkl_lda, buffer_ct_mkl_s, buffer_ct_mkl_u, int64_t_ct_mkl_ldu, buffer_ct_mkl_vt, int64_t_ct_mkl_ldvt, result_temp_buffer3);
    // CHECK-NEXT: buffer_ct3.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer3.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct14 = syclct::memory_manager::get_instance().translate_ptr(&Rwork_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct14 = allocation_ct14.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct14.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::zgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork, buffer_ct14, result_temp_buffer15), 0);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct5 = syclct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = syclct::memory_manager::get_instance().translate_ptr(&S_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = syclct::memory_manager::get_instance().translate_ptr(&U_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct10 = syclct::memory_manager::get_instance().translate_ptr(&VT_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct10 = allocation_ct10.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct10.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct12 = syclct::memory_manager::get_instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct12 = allocation_ct12.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct12.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct14 = syclct::memory_manager::get_instance().translate_ptr(&Rwork_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct14 = allocation_ct14.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct14.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct15 = syclct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct15 = allocation_ct15.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct15.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer15(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::zgesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, buffer_ct5, lda, buffer_ct7, buffer_ct8, ldu, buffer_ct10, ldvt, buffer_ct12, Lwork, buffer_ct14, result_temp_buffer15);
    // CHECK-NEXT: buffer_ct15.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);
    hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);

}
