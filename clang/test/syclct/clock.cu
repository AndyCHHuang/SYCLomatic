// RUN: syclct -out-root %T %s -- -std=c++14  -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/clock.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <time.h>

// CHECK: #include <time.h> // For clock_t, clock and CLOCKS_PER_SEC

__global__ static void timedReduction(const float *input, float *output, clock_t *timer)
{
    // CHECK: /*
    // CHECK-NEXT: DPCT1008:{{[0-9]+}}: Function clock is not defined in the SYCL specification. This is a hardware-specific feature. Consider consulting with hardware vendor to find a replacement.
    // CHECK-NEXT: */
    *timer = clock();

    // CHECK: /*
    // CHECK-NEXT: DPCT1008:{{[0-9]+}}: Function clock is not defined in the SYCL specification. This is a hardware-specific feature. Consider consulting with hardware vendor to find a replacement.
    // CHECK-NEXT: */
    clock();
}

int main(int argc, char **argv)
{
    float *dinput = NULL;
    float *doutput = NULL;
    clock_t *dtimer = NULL;

    timedReduction<<<64, 256, sizeof(float) * 2 * 256>>>(dinput, doutput, dtimer);

    return 0;
}
