// RUN: syclct -out-root %T %s -passes "MemoryTranslationRule" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/memory_management.sycl.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
    size_t size = 1234567 * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *d_A = NULL;
    // CHECK: syclct::sycl_malloc<char>((void **)&d_A, size);
    hipMalloc((void **)&d_A, size);
    // CHECK: syclct::sycl_memset((void*)(d_A), (void*)(15), (void*)(size));
    hipMemset(d_A, 0xf, size);
    // CHECK: syclct::sycl_memcpy<char>(d_A, h_A, size, cudaMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    // CHECK: syclct::sycl_memcpy<char>(h_A, d_A, size, cudaMemcpyDeviceToHost);
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    // CHECK: syclct::sycl_free<char>(d_A);
    hipFree(d_A);
    free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  // CHECK: return syclct::sycl_malloc<char>(buffer, size);
  return hipMalloc(buffer, size);
}
