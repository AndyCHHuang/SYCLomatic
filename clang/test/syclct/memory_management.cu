// RUN: syclct -out-root %T %s -passes "MemoryTranslationRule" -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --match-full-lines --input-file %T/memory_management.sycl.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;
  // CHECK: syclct::sycl_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);
  // CHECK: syclct::sycl_memset((void*)(d_A), (int)(0xf), (size_t)(size));
  hipMemset(d_A, 0xf, size);
  // CHECK: syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

  // CHECK: syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device, stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  // CHECK: syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK: syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, 0);
  // CHECK: syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host, stream);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, stream);

  // CHECK: syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK: syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice, 0);
  // CHECK: syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::host_to_device, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice, stream);

  // CHECK: syclct::sycl_memcpy_from_symbol((void*)(d_A), h_A.get_ptr(), size, 0, syclct::device_to_host);
  hipMemcpyFromSymbolAsync(d_A, HIP_SYMBOL(h_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK: syclct::sycl_memcpy_from_symbol((void*)(d_A), h_A.get_ptr(), size, 0, syclct::device_to_host);
  hipMemcpyFromSymbolAsync(d_A, HIP_SYMBOL(h_A), size, 0, hipMemcpyDeviceToHost, 0);
  // CHECK: syclct::sycl_memcpy_from_symbol((void*)(d_A), h_A.get_ptr(), size, 0, syclct::device_to_host, stream);
  hipMemcpyFromSymbolAsync(d_A, HIP_SYMBOL(h_A), size, 0, hipMemcpyDeviceToHost, stream);

  // CHECK: syclct::sycl_free(d_A);
  hipFree(d_A);
  free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  return (syclct::sycl_malloc(buffer, size), 0);
  return hipMalloc(buffer, size);
}

void checkError(hipError_t err) {
}

void testCommas() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  float *d_B = NULL;
  // CHECK:  syclct::sycl_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  cudaError_t err = (syclct::sycl_malloc((void **)&d_A, size), 0);
  hipError_t err = hipMalloc((void **)&d_A, size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_malloc((void **)&d_A, size), 0));
  checkError(hipMalloc((void **)&d_A, size));
  // CHECK:  syclct::sycl_memset((void*)(d_A), (int)(0xf), (size_t)(size));
  hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memset((void*)(d_A), (int)(0xf), (size_t)(size)), 0);
  err = hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memset((void*)(d_A), (int)(0xf), (size_t)(size)), 0));
  checkError(hipMemset(d_A, 0xf, size));

  ///////// Host to host
  // CHECK:  syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_host);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_host), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_host), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_device);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_device), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_device), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::automatic), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::automatic), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  ///////// Host to device
  // CHECK:  syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  syclct::sycl_memcpy_to_symbol(d_B.get_ptr(), (void*)(d_B), size, 0, syclct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_B), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy_to_symbol(d_B.get_ptr(), (void*)(d_B), size, 0, syclct::device_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_B), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((syclct::sycl_memcpy_to_symbol(d_A.get_ptr(), (void*)(h_A), size, 0, syclct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  syclct::sycl_memcpy_to_symbol(h_A.get_ptr(), (void*)(d_B), size, 0, syclct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (syclct::sycl_memcpy_to_symbol(h_A.get_ptr(), (void*)(d_B), size, 0, syclct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((syclct::sycl_memcpy_to_symbol(h_A.get_ptr(), (void*)(d_B), size, 0, syclct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  syclct::sycl_memcpy_to_symbol(h_A.get_ptr(), (void*)(d_B), size);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (syclct::sycl_memcpy_to_symbol(h_A.get_ptr(), (void*)(d_B), size), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((syclct::sycl_memcpy_to_symbol(h_A.get_ptr(), (void*)(d_B), size), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size));

  ///////// Device to host
  // CHECK:  syclct::sycl_memcpy_from_symbol((void*)(d_A), h_A.get_ptr(), size, 0, syclct::device_to_host);
  hipMemcpyFromSymbol(d_A, HIP_SYMBOL(h_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy_from_symbol((void*)(d_A), h_A.get_ptr(), size, 0, syclct::device_to_host), 0);
  err = hipMemcpyFromSymbol(d_A, HIP_SYMBOL(h_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_memcpy_from_symbol((void*)(d_A), h_A.get_ptr(), size, 0, syclct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(d_A, HIP_SYMBOL(h_A), size, 0, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  syclct::sycl_memcpy_from_symbol((void*)(d_B), d_B.get_ptr(), size, 0, syclct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_memcpy_from_symbol((void*)(d_B), d_B.get_ptr(), size, 0, syclct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((syclct::sycl_memcpy_from_symbol((void*)(d_B), d_B.get_ptr(), size, 0, syclct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  syclct::sycl_memcpy_from_symbol((void*)(h_A), d_B.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (syclct::sycl_memcpy_from_symbol((void*)(h_A), d_B.get_ptr(), size), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((syclct::sycl_memcpy_from_symbol((void*)(h_A), d_B.get_ptr(), size), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size));

  // CHECK: syclct::sycl_free(d_A);
  hipFree(d_A);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (syclct::sycl_free(d_A), 0);
  err = hipFree(d_A);
  // CHECK:/*
  // CHECK-NEXT:SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((syclct::sycl_free(d_A), 0));
  checkError(hipFree(d_A));
  // CHECK:  free(h_A);
  free(h_A);
}

#define N 1024
void test_segmentation_fault() {
  float *buffer;
  /*
  * Original code in getSizeString():
  * "SizeExpr->getBeginLoc()" cannot get the real SourceLocation of "N*sizeof(float)",
  * and results in boundary violation in "SyclctGlobalInfo::getSourceManager().getCharacterData(SizeBegin)"
  * and fails with segmentation fault.
  * https://jira.devtools.intel.com/browse/CTST-527
  * https://jira.devtools.intel.com/browse/CTST-528
  */
  hipMalloc(&buffer, N*sizeof(float));
}
