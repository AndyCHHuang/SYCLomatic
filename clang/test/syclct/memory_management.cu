// RUN: syclct -out-root %T %s -passes "MemoryTranslationRule" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_management.sycl.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
    size_t size = 1234567 * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *d_A = NULL;
    // CHECK: syclct::sycl_malloc<char>((void **)&d_A, size);
    hipMalloc((void **)&d_A, size);
    // CHECK: syclct::sycl_memset((void*)(d_A), (void*)(15), (void*)(size));
    hipMemset(d_A, 0xf, size);
    // CHECK: syclct::sycl_memcpy<char>(d_A, h_A, size, cudaMemcpyHostToDevice);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    // CHECK: syclct::sycl_memcpy<char>(h_A, d_A, size, cudaMemcpyDeviceToHost);
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    // CHECK: syclct::sycl_free<char>(d_A);
    hipFree(d_A);
    free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  // CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  return (syclct::sycl_malloc<char>(buffer, size), 0);
  return hipMalloc(buffer, size);
}

void checkError(hipError_t err) {

}

int testCommas() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
// CHECK:  syclct::sycl_malloc<char>((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  cudaError_t err = (syclct::sycl_malloc<char>((void **)&d_A, size), 0);
  hipError_t err = hipMalloc((void **)&d_A, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_malloc<char>((void **)&d_A, size), 0));
  checkError(hipMalloc((void **)&d_A, size));
// CHECK:  syclct::sycl_memset((void*)(d_A), (void*)(15), (void*)(size));
  hipMemset(d_A, 0xf, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memset((void*)(d_A), (void*)(15), (void*)(size)), 0);
  err = hipMemset(d_A, 0xf, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memset((void*)(d_A), (void*)(15), (void*)(size)), 0));
  checkError(hipMemset(d_A, 0xf, size));
// CHECK:  syclct::sycl_memcpy<char>(d_A, h_A, size, cudaMemcpyHostToDevice);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy<char>(d_A, h_A, size, cudaMemcpyHostToDevice), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy<char>(d_A, h_A, size, cudaMemcpyHostToDevice), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
// CHECK:  syclct::sycl_memcpy<char>(h_A, d_A, size, cudaMemcpyDeviceToHost);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy<char>(h_A, d_A, size, cudaMemcpyDeviceToHost), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy<char>(h_A, d_A, size, cudaMemcpyDeviceToHost), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));
// CHECK:  syclct::sycl_free<char>(d_A);
  hipFree(d_A);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_free<char>(d_A), 0);
  err = hipFree(d_A);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_free<char>(d_A), 0));
  checkError(hipFree(d_A));
// CHECK:  free(h_A);
  free(h_A);
}