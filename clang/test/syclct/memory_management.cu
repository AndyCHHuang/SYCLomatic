// RUN: syclct -out-root %T %s -passes "MemoryTranslationRule" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_management.sycl.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
    size_t size = 1234567 * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *d_A = NULL;
    // CHECK: syclct::sycl_malloc((void **)&d_A, size);
    hipMalloc((void **)&d_A, size);
    // CHECK: syclct::sycl_memset((void*)(d_A), (int)(15), (size_t)(size));
    hipMemset(d_A, 0xf, size);
    // CHECK: syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    // CHECK: syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host);
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    // CHECK: syclct::sycl_free(d_A);
    hipFree(d_A);
    free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  // CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  return (syclct::sycl_malloc(buffer, size), 0);
  return hipMalloc(buffer, size);
}

void checkError(hipError_t err) {

}

void testCommas() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
// CHECK:  syclct::sycl_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  cudaError_t err = (syclct::sycl_malloc((void **)&d_A, size), 0);
  hipError_t err = hipMalloc((void **)&d_A, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_malloc((void **)&d_A, size), 0));
  checkError(hipMalloc((void **)&d_A, size));
// CHECK:  syclct::sycl_memset((void*)(d_A), (int)(15), (size_t)(size));
  hipMemset(d_A, 0xf, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memset((void*)(d_A), (int)(15), (size_t)(size)), 0);
  err = hipMemset(d_A, 0xf, size);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memset((void*)(d_A), (int)(15), (size_t)(size)), 0));
  checkError(hipMemset(d_A, 0xf, size));

///////// Host to host
// CHECK:  syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_host);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_host), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_host), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost));

///////// Host to device
// CHECK:  syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(d_A), (void*)(h_A), size, syclct::host_to_device), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

///////// Device to host
// CHECK:  syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_host), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

///////// Device to Device
// CHECK:  syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_device);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_device), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::device_to_device), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice));

///////// Default
// CHECK:  syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::automatic), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_memcpy((void*)(h_A), (void*)(d_A), size, syclct::automatic), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

// CHECK:  syclct::sycl_free(d_A);
  hipFree(d_A);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  err = (syclct::sycl_free(d_A), 0);
  err = hipFree(d_A);
// CHECK:/*
// CHECK-NEXT:SYCLCT1003: Translated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:  checkError((syclct::sycl_free(d_A), 0));
  checkError(hipFree(d_A));
// CHECK:  free(h_A);
  free(h_A);
}
