// RUN: syclct -out-root %T %s -passes "MemoryTranslationRule" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_management.sycl.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
    size_t size = 1234567 * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *d_A = NULL;
    // CHECK: (syclct::sycl_malloc<char>((void **)&d_A, size), 0);
    hipMalloc((void **)&d_A, size);
    // CHECK: (syclct::sycl_memset((void*)(d_A), (void*)(15), (void*)(size)), 0);
    hipMemset(d_A, 0xf, size);
    // CHECK: (syclct::sycl_memcpy<char>(d_A, h_A, size, cudaMemcpyHostToDevice), 0);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    // CHECK: (syclct::sycl_memcpy<char>(h_A, d_A, size, cudaMemcpyDeviceToHost), 0);
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    // CHECK: (syclct::sycl_free<char>(d_A), 0);
    hipFree(d_A);
    free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  // CHECK: return (syclct::sycl_malloc<char>(buffer, size), 0);
  return hipMalloc(buffer, size);
}