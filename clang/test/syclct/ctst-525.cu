// RUN: syclct -out-root %T %s -- -std=c++11 -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/ctst-525.sycl.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
class C {
  int nDevices;
public:
  void problem() {
    // CHECK: nDevices = syclct::get_device_manager().device_count();
    hipGetDeviceCount(&nDevices);
  }
};
