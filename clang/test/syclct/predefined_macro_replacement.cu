// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path -D__NVCC__  -D __CUDA_ARCH__ -D__CUDACC__
// RUN: FileCheck --input-file %T/predefined_macro_replacement.sycl.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <stdio.h>
//CHECK: #ifdef DPCPP_COMPATIBILITY_TEMP
//CHECK-NEXT: void hello() { printf("intel"); }
#ifdef __CUDA_ARCH__
__global__ void hello() { printf("intel"); }
#else
void hello() { printf("other"); }
#endif

//CHECK: #ifndef DPCPP_COMPATIBILITY_TEMP
#ifndef __NVCC__
void hello2() { printf("hello2"); }
#endif
//CHECK: #if defined(DPCPP_COMPATIBILITY_TEMP)
#if defined(__HIPCC__)
void hello3() { printf("hello2"); }
#endif

#if defined(xxx)
void hello4() { printf("hello2"); }
//CHECK: #elif defined(DPCPP_COMPATIBILITY_TEMP)
#elif defined(__CUDA_ARCH__)
void hello5() { printf("hello2"); }
#endif
int main() {
//CHECK: #if defined(DPCPP_COMPATIBILITY_TEMP)
//CHECK-NEXT:   {
//CHECK-NEXT:     syclct::get_default_queue().submit(
#if defined(__NVCC__)
  hello<<<1,1>>>();
#else
  hello();
#endif
  return 0;
}
