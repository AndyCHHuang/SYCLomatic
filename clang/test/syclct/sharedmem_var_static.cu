// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/sharedmem_var_static.sycl.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 100
// CHECK: void staticReverse(cl::sycl::nd_item<3> item_{{[a-f0-9]+}}, cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s, int *d, int n) {
// CHECK-NEXT:  // the size of s is static
__global__ void staticReverse(int *d, int n) {
  __shared__ int s[64]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
    printf("s[%d]=%d\n", t, s[t]);
  }
}

// CHECK: template<typename TData>
// CHECK-NEXT: void templateReverse(cl::sycl::nd_item<3> item_{{[a-f0-9]+}}, cl::sycl::accessor<TData, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s, TData *d, TData n) {
template <class TData>
__global__ void templateReverse(TData *d, TData n) {
  __shared__ TData s[64][128]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t][0] = d[t];
    printf("s[%d][0]=%d\n", t, s[t][0]);
  }
}

template <typename T>
void testTemplate() {
  const int n = 64;
  T a[n], r[n], d[n];
  T *d_d;
  int mem_size = n * sizeof(T);
  hipMalloc((void **)&d_d, mem_size);
  hipMemcpy(d_d, a, mem_size, hipMemcpyHostToDevice);

  // CHECK: {
  // CHECK-NEXT:  std::pair<syclct::buffer_t, size_t> d_d_buf = syclct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:  size_t d_d_offset = d_d_buf.second;
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:	[&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:	  auto d_d_acc = d_d_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:	  cl::sycl::accessor<T, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s(cl::sycl::range<2>(64, 128), cgh);
  // CHECK-NEXT:	  cgh.parallel_for<SyclKernelName<class templateReverse_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:		cl::sycl::nd_range<1>((cl::sycl::range<1>(1) * cl::sycl::range<1>(n)), cl::sycl::range<1>(n)),
  // CHECK-NEXT:		[=](cl::sycl::nd_item<1> it) {
  // CHECK-NEXT:		  T *d_d = (T*)(&d_d_acc[0] + d_d_offset);
  // CHECK-NEXT:		  templateReverse<T>(it, s, d_d, n);
  // CHECK-NEXT:		});
  // CHECK-NEXT:	});
  // CHECK-NEXT:};
  templateReverse<T><<<1, n>>>(d_d, n);
}

int main(void) {
  const int n = 64;
  int a[n], r[n], d[n];
  int *d_d;
  hipMalloc((void **)&d_d, n * sizeof(int));
  hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
  // CHECK: {
  // CHECK-NEXT:  std::pair<syclct::buffer_t, size_t> d_d_buf = syclct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:  size_t d_d_offset = d_d_buf.second;
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:	[&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:	  auto d_d_acc = d_d_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:	  cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s(cl::sycl::range<1>(64), cgh);
  // CHECK-NEXT:	  cgh.parallel_for<SyclKernelName<class staticReverse_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:		cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1)), cl::sycl::range<3>(n, 1, 1)),
  // CHECK-NEXT:		[=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:		  int *d_d = (int*)(&d_d_acc[0] + d_d_offset);
  // CHECK-NEXT:		  staticReverse(it, s, d_d, n);
  // CHECK-NEXT:		});
  // CHECK-NEXT:	});
  // CHECK-NEXT:};
  staticReverse<<<1, n>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);

  // CHECK: {
  // CHECK-NEXT:  std::pair<syclct::buffer_t, size_t> d_d_buf = syclct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:  size_t d_d_offset = d_d_buf.second;
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:	[&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:	  auto d_d_acc = d_d_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:	  cl::sycl::accessor<int, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s(cl::sycl::range<2>(64, 128), cgh);
  // CHECK-NEXT:	  cgh.parallel_for<SyclKernelName<class templateReverse_{{[a-f0-9]+}}, int>>(
  // CHECK-NEXT:		cl::sycl::nd_range<1>((cl::sycl::range<1>(1) * cl::sycl::range<1>(n)), cl::sycl::range<1>(n)),
  // CHECK-NEXT:		[=](cl::sycl::nd_item<1> it) {
  // CHECK-NEXT:		  int *d_d = (int*)(&d_d_acc[0] + d_d_offset);
  // CHECK-NEXT:		  templateReverse<int>(it, s, d_d, n);
  // CHECK-NEXT:		});
  // CHECK-NEXT:	});
  // CHECK-NEXT:};
  templateReverse<int><<<1, n>>>(d_d, n);
}
