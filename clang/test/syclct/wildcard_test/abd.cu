
#include <hip/hip_runtime.h>
// RUN: syclct -out-root=%T/abd -in-root=%S %S/ab*.cu -- -std=c++11 -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/abd/abd.sycl.cpp --match-full-lines %S/abd.cu
// RUN: FileCheck --input-file %T/abd/abc.sycl.cpp --match-full-lines %S/abc.cu

int printf(const char *format, ...);

const char *test_function() {

//CHECK:/*
//CHECK-NEXT:SYCLCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:/*
//CHECK-NEXT:SYCLCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(0)*/);
  printf("%s\n", hipGetErrorString(hipGetLastError()));
}
