
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/kernel-call.sycl.cpp --match-full-lines %s

// CHECK: void testKernel(int L, int M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_[a-f0-9]+]]);
__global__ void testKernel(int L, int M, int N);

// CHECK: void testKernelPtr(const int *L, const int *M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_[a-f0-9]+]]) {
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

// CHECK: void testKernel(int L, int M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_[a-f0-9]+]]) {
__global__ void testKernel(int L, int M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU (int i, cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
// CHECK-NEXT: printf("Hello World from GPU! -- tid=%u = (blockId.x=%u * blockDim.x=%u)\
// CHECK-NEXT:     + threadId.x=%u\n", (item_{{[a-f0-9]+}}.get_group(0)*item_{{[a-f0-9]+}}.get_local_range().get(0))+item_{{[a-f0-9]+}}.get_local_id(0), item_{{[a-f0-9]+}}.get_group(0),
// CHECK-NEXT:     item_{{[a-f0-9]+}}.get_local_range().get(0), item_{{[a-f0-9]+}}.get_local_id(0));
// CHECK-NEXT: }
__global__ void helloFromGPU (int i) {
  printf("Hello World from GPU! -- tid=%u = (blockId.x=%u * blockDim.x=%u)\
    + threadId.x=%u\n", (blockIdx.x*blockDim.x)+threadIdx.x, blockIdx.x,
    blockDim.x, threadIdx.x);
}

// CHECK: void helloFromGPU (cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
// CHECK-NEXT: printf("Hello World from GPU! -- tid=%u = (blockId.x=%u * blockDim.x=%u)\
// CHECK-NEXT:     + threadId.x=%u\n", (item_{{[a-f0-9]+}}.get_group(0)*item_{{[a-f0-9]+}}.get_local_range().get(0))+item_{{[a-f0-9]+}}.get_local_id(0), item_{{[a-f0-9]+}}.get_group(0),
// CHECK-NEXT:     item_{{[a-f0-9]+}}.get_local_range().get(0), item_{{[a-f0-9]+}}.get_local_id(0));
// CHECK-NEXT: }
__global__ void helloFromGPU  (void) {
  printf("Hello World from GPU! -- tid=%u = (blockId.x=%u * blockDim.x=%u)\
    + threadId.x=%u\n", (blockIdx.x*blockDim.x)+threadIdx.x, blockIdx.x,
    blockDim.x, threadIdx.x);
}

// CHECK: void helloFromGPU2 (cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
// CHECK-NEXT: printf("Hello World from GPU! -- tid=%u = (blockId.x=%u * blockDim.x=%u)\
// CHECK-NEXT:     + threadId.x=%u\n", (item_{{[a-f0-9]+}}.get_group(0)*item_{{[a-f0-9]+}}.get_local_range().get(0))+item_{{[a-f0-9]+}}.get_local_id(0), item_{{[a-f0-9]+}}.get_group(0),
// CHECK-NEXT:     item_{{[a-f0-9]+}}.get_local_range().get(0), item_{{[a-f0-9]+}}.get_local_id(0));
// CHECK-NEXT: }
__global__ void helloFromGPU2 () {
  printf("Hello World from GPU! -- tid=%u = (blockId.x=%u * blockDim.x=%u)\
    + threadId.x=%u\n", (blockIdx.x*blockDim.x)+threadIdx.x, blockIdx.x,
    blockDim.x, threadIdx.x);
}

void testReference(const int &i) {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  // CHECK:  {
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto [[DEREF:i_deref_[a-f0-9]+]] = i;
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            helloFromGPU([[DEREF]], [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  helloFromGPU<<<griddim, threaddim>>>(i);
}

int main() {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  void *karg1 = 0;
  const int *karg2 = 0;
  int karg3 = 80;
  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg2_buf = syclct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:    size_t karg2_offset = karg2_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg2_acc = karg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            void *karg1 = (void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            const int *karg2 = (const int*)(&karg2_acc[0] + karg2_offset);
  // CHECK-NEXT:            testKernelPtr((const int *)karg1, karg2, karg3, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(10, 1, 1) * cl::sycl::range<3>(intvar, 1, 1)), cl::sycl::range<3>(intvar, 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            testKernel(karg1int, karg2int, karg3int, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  int karg1int = 1;
  int karg2int = 2;
  int karg3int = 3;
  int intvar = 20;
  testKernel<<<10, intvar>>>(karg1int, karg2int, karg3int);

  struct KernelPointer {
    const int *arg1, *arg2;
  } args;
  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> arg1_buf = syclct::get_buffer_and_offset(args.arg1);
  // CHECK-NEXT:    size_t arg1_offset = arg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> arg2_buf = syclct::get_buffer_and_offset(args.arg2);
  // CHECK-NEXT:    size_t arg2_offset = arg2_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto arg1_acc = arg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto arg2_acc = arg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 2, 1)), cl::sycl::range<3>(1, 2, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            const int *arg1 = (const int*)(&arg1_acc[0] + arg1_offset);
  // CHECK-NEXT:            const int *arg2 = (const int*)(&arg2_acc[0] + arg2_offset);
  // CHECK-NEXT:            testKernelPtr(arg1, arg2, karg3int, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<<<dim3(1), dim3(1, 2)>>>(args.arg1, args.arg2, karg3int);

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 2, 1) * cl::sycl::range<3>(1, 2, 3)), cl::sycl::range<3>(1, 2, 3)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            testKernel(karg1int, karg2int, karg3int, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernel<<<dim3(1, 2), dim3(1, 2, 3)>>>(karg1int, karg2int, karg3int);

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:      cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:        cl::sycl::nd_range<3>((cl::sycl::range<3>(griddim[0], 1, 1) * cl::sycl::range<3>(griddim[1] + 2, 1, 1)), cl::sycl::range<3>(griddim[1] + 2, 1, 1)),
  // CHECK-NEXT:        [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:        testKernel(karg1int, karg2int, karg3int, [[ITEM]]);
  // CHECK-NEXT:      });
  // CHECK-NEXT:    });
  // CHECK-NEXT:  }
  testKernel <<<griddim.x, griddim.y + 2 >>>(karg1int, karg2int, karg3int);

  // CHECK: {
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:    [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:     cgh.parallel_for<syclct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:       cl::sycl::nd_range<3>((cl::sycl::range<3>(2, 1, 1) * cl::sycl::range<3>(4, 1, 1)), cl::sycl::range<3>(4, 1, 1)),
  // CHECK-NEXT:       [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:         helloFromGPU(23, item_{{[a-f0-9]+}});
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  // CHECK-NEXT: }
  helloFromGPU <<<2, 4>>>(23);

  // CHECK: {
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:    [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:      cgh.parallel_for<syclct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:        cl::sycl::nd_range<3>((cl::sycl::range<3>(2, 1, 1) * cl::sycl::range<3>(4, 1, 1)), cl::sycl::range<3>(4, 1, 1)),
  // CHECK-NEXT:        [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:          helloFromGPU(item_{{[a-f0-9]+}});
  // CHECK-NEXT:        });
  // CHECK-NEXT:    });
  // CHECK-NEXT: }
  helloFromGPU <<<2, 4>>>();

  // CHECK: {
  // CHECK-NEXT:  syclct::get_default_queue().submit(
  // CHECK-NEXT:    [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:      cgh.parallel_for<syclct_kernel_name<class helloFromGPU2_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:        cl::sycl::nd_range<3>((cl::sycl::range<3>(2, 1, 1) * cl::sycl::range<3>(3, 1, 1)), cl::sycl::range<3>(3, 1, 1)),
  // CHECK-NEXT:        [=](cl::sycl::nd_item<3> item_{{[a-f0-9]+}}) {
  // CHECK-NEXT:          helloFromGPU2(item_{{[a-f0-9]+}});
  // CHECK-NEXT:        });
  // CHECK-NEXT:    });
  // CHECK-NEXT: }
  helloFromGPU2 <<<2, 3>>>();
}
