// RUN: syclct -out-root %T %s -- -std=c++14 -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cusolverHelper.sycl.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <syclct/syclct.hpp>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

// CHECK: #define MACRO_A cusolverDnCreate
#define MACRO_A hipsolverDnCreate

// CHECK: void foo(int, int, int, int, int, int, int, int) {}
void foo(hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t) {}

// CHECK: void foo2(int){}
void foo2(hipsolverStatus_t){}

// CHECK: int foo3(int m, int n)
hipsolverStatus_t foo3(int m, int n)
{
    // CHECK: return 0;
    return HIPSOLVER_STATUS_SUCCESS;
}

int main(int argc, char *argv[])
{
    // CHECK: cl::sycl::queue * cusolverH = NULL;
    // CHECK-NEXT: int status = 0;
    // CHECK-NEXT: status = 1;
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;

    // CHECK: foo(0, 1, 2, 3, 4, 6, 7, 8);
    // CHECK-NEXT: status = 0;
    foo(HIPSOLVER_STATUS_SUCCESS, HIPSOLVER_STATUS_NOT_INITIALIZED, HIPSOLVER_STATUS_ALLOC_FAILED, HIPSOLVER_STATUS_INVALID_VALUE, HIPSOLVER_STATUS_ARCH_MISMATCH, HIPSOLVER_STATUS_EXECUTION_FAILED, HIPSOLVER_STATUS_INTERNAL_ERROR, HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED);
    hipsolverDnCreate(cusolverH);
    status = hipsolverDnCreate(cusolverH);

    // CHECK: status = 0;
    status = MACRO_A(cusolverH);

    // CHECK: status = 0;
    status = hipsolverDnDestroy(*cusolverH);

    // CHECK: int a = sizeof(int);
    // CHECK-NEXT: int b = sizeof(cl::sycl::queue);
    int a = sizeof(hipblasStatus_t);
    int b = sizeof(hipsolverHandle_t);
}
