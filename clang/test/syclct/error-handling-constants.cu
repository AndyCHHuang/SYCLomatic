
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -passes "ErrorConstantsRule" -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/error-handling-constants.sycl.cpp --match-full-lines %s


// CHECK:const char *switch_test(cudaError_t error)
// CHECK-NEXT:{
// CHECK-NEXT:    switch (error)
// CHECK-NEXT:    {
// CHECK-NEXT:        case 0:
// CHECK-NEXT:            return "cudaSuccess";
// CHECK-NEXT:        case 1:
// CHECK-NEXT:            return "cudaErrorMissingConfiguration";
// CHECK-NEXT:        case 2:
// CHECK-NEXT:            return "cudaErrorMemoryAllocation";
// CHECK-NEXT:        case 3:
// CHECK-NEXT:            return "cudaErrorInitializationError";
// CHECK-NEXT:        case 4:
// CHECK-NEXT:            return "cudaErrorLaunchFailure";
// CHECK-NEXT:        case 5:
// CHECK-NEXT:            return "cudaErrorPriorLaunchFailure";
// CHECK-NEXT:        case 6:
// CHECK-NEXT:            return "cudaErrorLaunchTimeout";
// CHECK-NEXT:        case 7:
// CHECK-NEXT:            return "cudaErrorLaunchOutOfResources";
// CHECK-NEXT:        case 8:
// CHECK-NEXT:            return "cudaErrorInvalidDeviceFunction";
// CHECK-NEXT:        case 9:
// CHECK-NEXT:            return "cudaErrorInvalidConfiguration";
// CHECK-NEXT:        case 10:
// CHECK-NEXT:            return "cudaErrorInvalidDevice";
// CHECK-NEXT:        case 11:
// CHECK-NEXT:            return "cudaErrorInvalidValue";
// CHECK-NEXT:        case 12:
// CHECK-NEXT:            return "cudaErrorInvalidPitchValue";
// CHECK-NEXT:        case 13:
// CHECK-NEXT:            return "cudaErrorInvalidSymbol";
// CHECK-NEXT:    }
// CHECK-NEXT:    return 0;
// CHECK-NEXT:}
const char *switch_test(hipError_t error)
{
    switch (error)
    {
        case hipSuccess:
            return "hipSuccess";
        case hipErrorMissingConfiguration:
            return "hipErrorMissingConfiguration";
        case hipErrorOutOfMemory:
            return "hipErrorOutOfMemory";
        case hipErrorNotInitialized:
            return "hipErrorNotInitialized";
        case hipErrorLaunchFailure:
            return "hipErrorLaunchFailure";
        case hipErrorPriorLaunchFailure:
            return "hipErrorPriorLaunchFailure";
        case hipErrorLaunchTimeOut:
            return "hipErrorLaunchTimeOut";
        case hipErrorLaunchOutOfResources:
            return "hipErrorLaunchOutOfResources";
        case hipErrorInvalidDeviceFunction:
            return "hipErrorInvalidDeviceFunction";
        case hipErrorInvalidConfiguration:
            return "hipErrorInvalidConfiguration";
        case hipErrorInvalidDevice:
            return "hipErrorInvalidDevice";
        case hipErrorInvalidValue:
            return "hipErrorInvalidValue";
        case hipErrorInvalidPitchValue:
            return "hipErrorInvalidPitchValue";
        case hipErrorInvalidSymbol:
            return "hipErrorInvalidSymbol";
    }
    return 0;
}

// CHECK:int test_simple_ifs() {
// CHECK-NEXT:  cudaError_t err = 13;
// CHECK-NEXT:  if (err != 0) {
// CHECK-NEXT:  }
// CHECK-NEXT:  if (switch_test(1)) {
// CHECK-NEXT:  }
// CHECK-NEXT:  if (0 != err) {
// CHECK-NEXT:  }
// CHECK-NEXT:  if (err == 0) {
// CHECK-NEXT:    return (int)0;
// CHECK-NEXT:  }
// CHECK-NEXT:}
int test_simple_ifs() {
  hipError_t err = hipErrorInvalidSymbol;
  if (err != hipSuccess) {
  }
  if (switch_test(hipErrorMissingConfiguration)) {
  }
  if (hipSuccess != err) {
  }
  if (err == hipSuccess) {
    return (int)hipSuccess;
  }
}
