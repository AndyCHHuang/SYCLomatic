
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck %s --match-full-lines --input-file %T/types001.sycl.cpp

// CHECK: syclct::sycl_device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const syclct::sycl_device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile syclct::sycl_device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHDCK: cl::sycl::event events[23];
hipEvent_t events[23];
// CHECK: const cl::sycl::event *pevents[23];
const hipEvent_t *pevents[23];
// CHECK: const cl::sycl::event **ppevents[23];
const hipEvent_t **ppevents[23];

// CHECK: int errors[23];
hipError_t errors[23];
// CHECK: const int *perrors[23];
const hipError_t *perrors[23];
// CHECK: const int **pperrors[23];
const hipError_t **pperrors[23];

// CHECK: int errors1[23];
hipError_t errors1[23];
// CHECK: const int *perrors1[23];
const hipError_t *perrors1[23];
// CHECK: const int **pperrors1[23];
const hipError_t **pperrors1[23];

// CHECK: cl::sycl::range<3> dims[23];
dim3 dims[23];
// CHECK: const cl::sycl::range<3> *pdims[23];
const dim3 *pdims[23];
// CHECK: const cl::sycl::range<3> **ppdims[23];
const dim3 **ppdims[23];

struct s {
  // CHDCK: cl::sycl::event events[23];
  hipEvent_t events[23];
  // CHECK: const cl::sycl::event *pevents[23];
  const hipEvent_t *pevents[23];
  // CHECK: const cl::sycl::event **ppevents[23];
  const hipEvent_t **ppevents[23];

  // CHECK: int errors[23];
  hipError_t errors[23];
  // CHECK: const int *perrors[23];
  const hipError_t *perrors[23];
  // CHECK: const int **pperrors[23];
  const hipError_t **pperrors[23];

  // CHECK: int errors1[23];
  hipError_t errors1[23];
  // CHECK: const int *perrors1[23];
  const hipError_t *perrors1[23];
  // CHECK: const int **pperrors1[23];
  const hipError_t **pperrors1[23];

  // CHECK: cl::sycl::range<3> dims[23];
  dim3 dims[23];
  // CHECK: const cl::sycl::range<3> *pdims[23];
  const dim3 *pdims[23];
  // CHECK: const cl::sycl::range<3> **ppdims[23];
  const dim3 **ppdims[23];
};

// CHECK:  void foo(syclct::sycl_device_info p) {
void foo(hipDeviceProp_t p) {
  return;
}

// CHECK: int e;
hipError_t e;

// CHECK: int ee;
hipError_t ee;

// CHECK: int foo_0(int);
hipError_t foo_0(hipError_t);

// CHECK: int foo_1(int);
hipError_t foo_1(hipError_t);

// CHECK: int apicall(int i) {
hipError_t apicall(int i) {
  return hipSuccess;
};

// CHECK: int err = apicall(0);
hipError_t err = apicall(0);

template <typename T>
// CHECK: void check(T result, char const *const func) {
void check(T result, char const *const func) {
}

#define checkCudaErrors(val) check((val), #val)

int main(int argc, char **argv) {

  checkCudaErrors(apicall(0));
  return 0;
}
