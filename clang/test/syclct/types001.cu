
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types001.sycl.cpp

// CHECK: syclct::sycl_device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const syclct::sycl_device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile syclct::sycl_device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHECK:  void foo(syclct::sycl_device_info p) try {
void foo(hipDeviceProp_t p) {
  return;
}

// CHECK: int e;
hipError_t e;

// CHECK: int ee;
hipError_t ee;

// CHECK: int foo_0(int);
hipError_t foo_0(hipError_t);

// CHECK: int foo_1(int);
hipError_t foo_1(hipError_t);

// CHECK: int apicall(int i) try {
hipError_t apicall(int i) {
  return hipSuccess;
};

// CHECK: int err = apicall(0);
hipError_t err = apicall(0);

template <typename T>
// CHECK: void check(T result, char const *const func) try {
void check(T result, char const *const func) {
}

#define checkCudaErrors(val) check((val), #val)

int main(int argc, char **argv) {

  checkCudaErrors(apicall(0));
  return 0;
}
