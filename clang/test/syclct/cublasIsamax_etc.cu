// RUN: syclct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/cublasIsamax_etc.sycl.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int k = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;

  const float *x_S = 0;
  const double *x_D = 0;
  const float *y_S = 0;
  const double *y_D = 0;
  int incx = 1;
  int incy = 1;
  int *result = 0;
  float *result_S = 0;
  double *result_D = 0;
  //level1
  //cublasI<t>amax
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::isamax(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::isamax(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}});
  status = hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::idamax(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::idamax(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}});
  status = hipblasIdamax(handle, n, x_D, incx, result);
  hipblasIdamax(handle, n, x_D, incx, result);

  //cublasI<t>amin
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::isamin(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::isamin(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}});
  status = hipblasIsamin(handle, n, x_S, incx, result);
  hipblasIsamin(handle, n, x_S, incx, result);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::idamin(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::idamin(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_BUFFER_{{[0-9,a-z]+}});
  status = hipblasIdamin(handle, n, x_D, incx, result);
  hipblasIdamin(handle, n, x_D, incx, result);

  //cublas<t>asum
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sasum(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::sasum(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSasum(handle, n, x_S, incx, result_S);
  hipblasSasum(handle, n, x_S, incx, result_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dasum(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::dasum(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDasum(handle, n, x_D, incx, result_D);
  hipblasDasum(handle, n, x_D, incx, result_D);

  //cublas<t>axpy
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::saxpy(handle, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::saxpy(handle, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);
  hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::daxpy(handle, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::daxpy(handle, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);
  hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);

  //cublas<t>copy
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::scopy(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::scopy(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasScopy(handle, n, x_S, incx, result_S, incy);
  hipblasScopy(handle, n, x_S, incx, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dcopy(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dcopy(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDcopy(handle, n, x_D, incx, result_D, incy);
  hipblasDcopy(handle, n, x_D, incx, result_D, incy);

  //cublas<t>dot
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sdot(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::sdot(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);
  hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ddot(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::ddot(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);
  hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);

  //cublas<t>nrm2
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::snrm2(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::snrm2(handle, n, x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSnrm2(handle, n, x_S, incx, result_S);
  hipblasSnrm2(handle, n, x_S, incx, result_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dnrm2(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::dnrm2(handle, n, x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDnrm2(handle, n, x_D, incx, result_D);
  hipblasDnrm2(handle, n, x_D, incx, result_D);

  float *x_f = 0;
  float *y_f = 0;
  double *x_d = 0;
  double *y_d = 0;
  //cublas<t>rot
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::srot(handle, n, x_f_BUFFER_{{[0-9,a-z]+}}, incx, y_f_BUFFER_{{[0-9,a-z]+}}, incy, *(x_S), *(y_S)), 0);
  // CHECK: mkl::srot(handle, n, x_f_BUFFER_{{[0-9,a-z]+}}, incx, y_f_BUFFER_{{[0-9,a-z]+}}, incy, *(x_S), *(y_S));
  status = hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);
  hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::drot(handle, n, x_d_BUFFER_{{[0-9,a-z]+}}, incx, y_d_BUFFER_{{[0-9,a-z]+}}, incy, *(x_D), *(y_D)), 0);
  // CHECK: mkl::drot(handle, n, x_d_BUFFER_{{[0-9,a-z]+}}, incx, y_d_BUFFER_{{[0-9,a-z]+}}, incy, *(x_D), *(y_D));
  status = hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);
  hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);

  //cublas<t>rotg
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::srotg(handle, x_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}, x_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::srotg(handle, x_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}, x_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSrotg(handle, x_f, y_f, x_f, y_f);
  hipblasSrotg(handle, x_f, y_f, x_f, y_f);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::drotg(handle, x_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}, x_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::drotg(handle, x_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}, x_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDrotg(handle, x_d, y_d, x_d, y_d);
  hipblasDrotg(handle, x_d, y_d, x_d, y_d);

  //cublas<t>rotm
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::srotm(handle, n, x_f_BUFFER_{{[0-9,a-z]+}}, incx, y_f_BUFFER_{{[0-9,a-z]+}}, incy, x_S_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::srotm(handle, n, x_f_BUFFER_{{[0-9,a-z]+}}, incx, y_f_BUFFER_{{[0-9,a-z]+}}, incy, x_S_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);
  hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::drotm(handle, n, x_d_BUFFER_{{[0-9,a-z]+}}, incx, y_d_BUFFER_{{[0-9,a-z]+}}, incy, x_D_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::drotm(handle, n, x_d_BUFFER_{{[0-9,a-z]+}}, incx, y_d_BUFFER_{{[0-9,a-z]+}}, incy, x_D_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);
  hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);

  //cublas<t>rotmg
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::srotmg(handle, x_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}, x_S, y_f_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::srotmg(handle, x_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}, y_f_BUFFER_{{[0-9,a-z]+}}, x_S, y_f_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);
  hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::drotmg(handle, x_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}, x_D, y_d_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::drotmg(handle, x_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}, y_d_BUFFER_{{[0-9,a-z]+}}, x_D, y_d_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);
  hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);

  //cublas<t>scal
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sscal(handle, n, *(&alpha_S), x_f_BUFFER_{{[0-9,a-z]+}}, incx), 0);
  // CHECK: mkl::sscal(handle, n, *(&alpha_S), x_f_BUFFER_{{[0-9,a-z]+}}, incx);
  status = hipblasSscal(handle, n, &alpha_S, x_f, incx);
  hipblasSscal(handle, n, &alpha_S, x_f, incx);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dscal(handle, n, *(&alpha_D), x_d_BUFFER_{{[0-9,a-z]+}}, incx), 0);
  // CHECK: mkl::dscal(handle, n, *(&alpha_D), x_d_BUFFER_{{[0-9,a-z]+}}, incx);
  status = hipblasDscal(handle, n, &alpha_D, x_d, incx);
  hipblasDscal(handle, n, &alpha_D, x_d, incx);

  //cublas<t>swap
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sswap(handle, n, x_f_BUFFER_{{[0-9,a-z]+}}, incx, y_f_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::sswap(handle, n, x_f_BUFFER_{{[0-9,a-z]+}}, incx, y_f_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSswap(handle, n, x_f, incx, y_f, incy);
  hipblasSswap(handle, n, x_f, incx, y_f, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dswap(handle, n, x_d_BUFFER_{{[0-9,a-z]+}}, incx, y_d_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dswap(handle, n, x_d_BUFFER_{{[0-9,a-z]+}}, incx, y_d_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDswap(handle, n, x_d, incx, y_d, incy);
  hipblasDswap(handle, n, x_d, incx, y_d, incy);

  //level2
  //cublas<t>gbmv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sgbmv(handle, mkl::transpose::nontrans, m, n, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::sgbmv(handle, mkl::transpose::nontrans, m, n, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dgbmv(handle, mkl::transpose::nontrans, m, n, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dgbmv(handle, mkl::transpose::nontrans, m, n, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>gemv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sgemv(handle, mkl::transpose::nontrans, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::sgemv(handle, mkl::transpose::nontrans, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dgemv(handle, mkl::transpose::nontrans, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dgemv(handle, mkl::transpose::nontrans, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>ger
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sger(handle, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}}, lda), 0);
  // CHECK: mkl::sger(handle, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}}, lda);
  status = hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dger(handle, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}}, lda), 0);
  // CHECK: mkl::dger(handle, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}}, lda);
  status = hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  //cublas<t>sbmv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssbmv(handle, mkl::uplo::upper, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::ssbmv(handle, mkl::uplo::upper, m, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsbmv(handle, mkl::uplo::upper, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dsbmv(handle, mkl::uplo::upper, m, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spmv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sspmv(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::sspmv(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);
  hipblasSspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dspmv(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dspmv(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);
  hipblasDspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spr
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sspr(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::sspr(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S);
  hipblasSspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dspr(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::dspr(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D);
  hipblasDspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D);

  //cublas<t>spr2
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::sspr2(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::sspr2(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}});
  status = hipblasSspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S);
  hipblasSspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dspr2(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}}), 0);
  // CHECK: mkl::dspr2(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}});
  status = hipblasDspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D);
  hipblasDspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D);

  //cublas<t>symv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssymv(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::ssymv(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, lda, y_S_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_S), result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasSsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsymv(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dsymv(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, lda, y_D_BUFFER_{{[0-9,a-z]+}}, incx, *(&beta_D), result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>syr
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssyr(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}, lda), 0);
  // CHECK: mkl::ssyr(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, result_S_BUFFER_{{[0-9,a-z]+}}, lda);
  status = hipblasSsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S, lda);
  hipblasSsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsyr(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}, lda), 0);
  // CHECK: mkl::dsyr(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, result_D_BUFFER_{{[0-9,a-z]+}}, lda);
  status = hipblasDsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D, lda);
  hipblasDsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D, lda);

  //cublas<t>syr2
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssyr2(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}}, lda), 0);
  // CHECK: mkl::ssyr2(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_BUFFER_{{[0-9,a-z]+}}, incx, y_S_BUFFER_{{[0-9,a-z]+}}, incy, result_S_BUFFER_{{[0-9,a-z]+}}, lda);
  status = hipblasSsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsyr2(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}}, lda), 0);
  // CHECK: mkl::dsyr2(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_BUFFER_{{[0-9,a-z]+}}, incx, y_D_BUFFER_{{[0-9,a-z]+}}, incy, result_D_BUFFER_{{[0-9,a-z]+}}, lda);
  status = hipblasDsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  //cublas<t>tbmv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::stbmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::stbmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasStbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);
  hipblasStbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtbmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dtbmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDtbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);
  hipblasDtbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tbsv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::stbsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::stbsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasStbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);
  hipblasStbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtbsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dtbsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDtbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);
  hipblasDtbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tpmv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::stpmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::stpmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasStpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);
  hipblasStpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtpmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dtpmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDtpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);
  hipblasDtpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>tpsv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::stpsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::stpsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasStpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);
  hipblasStpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtpsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dtpsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDtpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);
  hipblasDtpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>trmv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::strmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::strmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasStrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);
  hipblasStrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtrmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dtrmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDtrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);
  hipblasDtrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //cublas<t>trsv
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::strsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::strsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_BUFFER_{{[0-9,a-z]+}}, lda, result_S_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasStrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);
  hipblasStrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtrsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy), 0);
  // CHECK: mkl::dtrsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_BUFFER_{{[0-9,a-z]+}}, lda, result_D_BUFFER_{{[0-9,a-z]+}}, incy);
  status = hipblasDtrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);
  hipblasDtrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //level3

  // cublas<T>symm
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssymm(handle, mkl::side::left, mkl::uplo::upper,  m, n, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, B_S_BUFFER_{{[0-9,a-z]+}},  ldb, *(&beta_S), C_S_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::ssymm(handle, mkl::side::right, mkl::uplo::lower,  m, n, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, B_S_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_S), C_S_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasSsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsymm(handle, mkl::side::left, mkl::uplo::upper,  m, n, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, B_D_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_D), C_D_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::dsymm(handle, mkl::side::right, mkl::uplo::lower,  m, n, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, B_D_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_D), C_D_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasDsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>syrk
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssyrk(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, *(&beta_S), C_S_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::ssyrk(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, *(&beta_S), C_S_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasSsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);
  hipblasSsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsyrk(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, *(&beta_D), C_D_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::dsyrk(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, *(&beta_D), C_D_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasDsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);

  // cublas<T>syr2k
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::ssyr2k(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, B_S_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_S), C_S_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::ssyr2k(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, B_S_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_S), C_S_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dsyr2k(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, B_D_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_D), C_D_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::dsyr2k(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, B_D_BUFFER_{{[0-9,a-z]+}}, ldb, *(&beta_D), C_D_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasDsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>trsm
  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::strsm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, m, n, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, C_S_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::strsm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, *(&alpha_S), A_S_BUFFER_{{[0-9,a-z]+}}, lda, C_S_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, m, n, &alpha_S, A_S, lda, C_S, ldc);
  hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: SYCLCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may want to rewrite this code
  // CHECK-NEXT: */
  // CHECK: status = (mkl::dtrsm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, m, n, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, C_D_BUFFER_{{[0-9,a-z]+}}, ldc), 0);
  // CHECK: mkl::dtrsm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, *(&alpha_D), A_D_BUFFER_{{[0-9,a-z]+}}, lda, C_D_BUFFER_{{[0-9,a-z]+}}, ldc);
  status = hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, m, n, &alpha_D, A_D, lda, C_D, ldc);
  hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, C_D, ldc);

  return 0;
}
