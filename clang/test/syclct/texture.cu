#include "hip/hip_runtime.h"
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/texture.sycl.cpp --match-full-lines %s

// CHECK: syclct::syclct_texture<cl::sycl::float4, 2> tex42;
static texture<float4, 2> tex42;
// CHECK: syclct::syclct_texture<cl::sycl::uint2, 1> tex21;
static texture<uint2, 1> tex21;
/// TODO: Expect to support 3D array in future.
// TODO-CHECK: syclct::syclct_texture<int, 3> tex13;
// static texture<int, 3> tex13;

// CHECK: void device01(syclct::syclct_texture_accessor<cl::sycl::uint2, 1> tex21) {
// CHECK-NEXT: cl::sycl::uint2 u21 = syclct::syclct_read_texture(tex21, 1.0f);
// CHECK-NEXT: cl::sycl::uint2 u21_fetch = syclct::syclct_read_texture(tex21, 1);
__device__ void device01() {
  uint2 u21 = tex1D(tex21, 1.0f);
  uint2 u21_fetch = tex1Dfetch(tex21, 1);
}

// CHECK: void kernel(syclct::syclct_texture_accessor<cl::sycl::float4, 2> tex42, syclct::syclct_texture_accessor<cl::sycl::uint2, 1> tex21) {
// CHECK-NEXT: device01(tex21);
// CHECK-NEXT: cl::sycl::float4 f42 = syclct::syclct_read_texture(tex42, 1.0f, 1.0f);
/// Texture accessors should be passed down to __global__/__device__ function if used.
__global__ void kernel() {
  device01();
  float4 f42 = tex2D(tex42, 1.0f, 1.0f);
}

int main() {

  // CHECK: syclct::syclct_array a42;
  // CHECK-NEXT: cl::sycl::float4 *d_data42;
  // CHECK-NEXT: syclct::sycl_malloc(&d_data42, sizeof(cl::sycl::float4) * 32 * 32);
  // CHECK-NEXT: syclct::syclct_channel_desc desc42 = syclct::create_channel_desc(32, 32, 32, 32, syclct::channel_float);
  // CHECK-NEXT: syclct::syclct_malloc_array(&a42, &desc42, 32, 32);
  // CHECK-NEXT: syclct::syclct_memcpy_to_array(a42, d_data42);
  // CHECK-NEXT: tex42.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex42.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex42.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex42.set_filter_mode( cl::sycl::filtering_mode::nearest);
  // CHECK-NEXT: syclct::syclct_bind_texture(tex42, a42);
  hipArray_t a42;
  float4 *d_data42;
  hipMalloc(&d_data42, sizeof(float4) * 32 * 32);
  hipChannelFormatDesc desc42 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipMallocArray(&a42, &desc42, 32, 32);
  hipMemcpyToArray(a42, 0, 0, d_data42, 32 * 32, hipMemcpyDeviceToDevice);
  tex42.addressMode[0] = hipAddressModeClamp;
  tex42.addressMode[1] = hipAddressModeClamp;
  tex42.addressMode[2] = hipAddressModeClamp;
  tex42.filterMode = hipFilterModePoint;
  hipBindTextureToArray(tex42, a42, desc42);

  // CHECK: syclct::syclct_array a21;
  // CHECK-NEXT: cl::sycl::uint2 *d_data21;
  // CHECK-NEXT: syclct::sycl_malloc(&d_data21, sizeof(cl::sycl::uint2) * 32);
  // CHECK-NEXT: syclct::syclct_channel_desc desc21 = syclct::create_channel_desc(32, 32, 0, 0, syclct::channel_unsigned);
  // CHECK-NEXT: syclct::syclct_malloc_array(&a21, &desc21, 32, 0);
  // CHECK-NEXT: syclct::syclct_memcpy_to_array(a21, d_data21);
  // CHECK-NEXT: tex21.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex21.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex21.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex21.set_filter_mode( cl::sycl::filtering_mode::linear);
  // CHECK-NEXT: syclct::syclct_bind_texture(tex21, a21);
  hipArray *a21;
  uint2 *d_data21;
  hipMalloc(&d_data21, sizeof(uint2) * 32);
  hipChannelFormatDesc desc21 = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindUnsigned);
  hipMallocArray(&a21, &desc21, 32);
  hipMemcpyToArray(a21, 0, 0, d_data21, 32, hipMemcpyDeviceToDevice);
  tex21.addressMode[0] = hipAddressModeClamp;
  tex21.addressMode[1] = hipAddressModeClamp;
  tex21.addressMode[2] = hipAddressModeClamp;
  tex21.filterMode = hipFilterModeLinear;
  hipBindTextureToArray(tex21, a21, desc21);

  // CHECK:   {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:       [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:         auto tex42_acc = tex42.get_access(cgh);
  // CHECK-NEXT:         auto tex21_acc = tex21.get_access(cgh);
  // CHECK-NEXT:         cgh.parallel_for<syclct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:             cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, 1)), cl::sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:             [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:               kernel(tex42_acc, tex21_acc);
  // CHECK-NEXT:             });
  // CHECK-NEXT:       });
  // CHECK-NEXT: }
  kernel<<<1, 1>>>();

  // CHECK: syclct::syclct_unbind_texture(tex42);
  // CHECK-NEXT: syclct::syclct_unbind_texture(tex21);
  hipUnbindTexture(tex42);
  hipUnbindTexture(tex21);

  // CHECK: syclct::syclct_free_array(a42);
  // CHECK-NEXT: syclct::syclct_free_array(a21);
  hipFreeArray(a42);
  hipFreeArray(a21);
 
  // CHECK: syclct::sycl_free(d_data42);
  // CHECK-NEXT: syclct::sycl_free(d_data21);
  hipFree(d_data42);
  hipFree(d_data21);
}
