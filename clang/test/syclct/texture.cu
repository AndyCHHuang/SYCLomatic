#include "hip/hip_runtime.h"
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/texture.sycl.cpp --match-full-lines %s

// CHECK: syclct::dpct_image<cl::sycl::float4, 2> tex42;
static texture<float4, 2> tex42;
// CHECK: syclct::dpct_image<cl::sycl::uint2, 1> tex21;
static texture<uint2, 1> tex21;
/// TODO: Expect to support 3D array in future.
// TODO-CHECK: syclct::dpct_image<int, 3> tex13;
// static texture<int, 3> tex13;

// CHECK: void device01(syclct::dpct_image_accessor<cl::sycl::uint2, 1> tex21) {
// CHECK-NEXT: cl::sycl::uint2 u21 = syclct::dpct_read_image(tex21, 1.0f);
// CHECK-NEXT: cl::sycl::uint2 u21_fetch = syclct::dpct_read_image(tex21, 1);
__device__ void device01() {
  uint2 u21 = tex1D(tex21, 1.0f);
  uint2 u21_fetch = tex1Dfetch(tex21, 1);
}

// CHECK: void kernel(syclct::dpct_image_accessor<cl::sycl::float4, 2> tex42, syclct::dpct_image_accessor<cl::sycl::uint2, 1> tex21) {
// CHECK-NEXT: device01(tex21);
// CHECK-NEXT: cl::sycl::float4 f42 = syclct::dpct_read_image(tex42, 1.0f, 1.0f);
/// Texture accessors should be passed down to __global__/__device__ function if used.
__global__ void kernel() {
  device01();
  float4 f42 = tex2D(tex42, 1.0f, 1.0f);
}

int main() {

  // CHECK: cl::sycl::float4 *d_data42;
  // CHECK-NEXT: syclct::dpct_image_data a42;
  // CHECK-NEXT: syclct::sycl_malloc(&d_data42, sizeof(cl::sycl::float4) * 32 * 32);
  // CHECK-NEXT: syclct::dpct_image_channel desc42 = syclct::create_image_channel(32, 32, 32, 32, syclct::channel_float);
  // CHECK-NEXT: syclct::dpct_malloc_image(&a42, &desc42, 32, 32);
  // CHECK-NEXT: syclct::dpct_memcpy_to_image(a42, 0, 0, d_data42, 32 * 32 * sizeof(cl::sycl::float4));
  // CHECK-NEXT: tex42.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex42.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex42.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex42.set_filter_mode( cl::sycl::filtering_mode::nearest);
  // CHECK-NEXT: syclct::dpct_attach_image(tex42, a42);
  float4 *d_data42;
  hipArray_t a42;
  hipMalloc(&d_data42, sizeof(float4) * 32 * 32);
  hipChannelFormatDesc desc42 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipMallocArray(&a42, &desc42, 32, 32);
  hipMemcpyToArray(a42, 0, 0, d_data42, 32 * 32 * sizeof(float4), hipMemcpyDeviceToDevice);
  tex42.addressMode[0] = hipAddressModeClamp;
  tex42.addressMode[1] = hipAddressModeClamp;
  tex42.addressMode[2] = hipAddressModeClamp;
  tex42.filterMode = hipFilterModePoint;
  hipBindTextureToArray(tex42, a42, desc42);

  // CHECK: cl::sycl::uint2 *d_data21;
  // CHECK-NEXT: syclct::sycl_malloc(&d_data21, sizeof(cl::sycl::uint2) * 32);
  // CHECK-NEXT: syclct::dpct_image_channel desc21 = syclct::create_image_channel(32, 32, 0, 0, syclct::channel_unsigned);
  // CHECK-NEXT: tex21.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex21.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex21.set_addr_mode( cl::sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: tex21.set_filter_mode( cl::sycl::filtering_mode::linear);
  // CHECK-NEXT: syclct::dpct_attach_image(tex21, d_data21, desc21, 32 * sizeof(cl::sycl::uint2));
  uint2 *d_data21;
  hipMalloc(&d_data21, sizeof(uint2) * 32);
  hipChannelFormatDesc desc21 = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindUnsigned);
  tex21.addressMode[0] = hipAddressModeClamp;
  tex21.addressMode[1] = hipAddressModeClamp;
  tex21.addressMode[2] = hipAddressModeClamp;
  tex21.filterMode = hipFilterModeLinear;
  hipBindTexture(0, tex21, d_data21, desc21, 32 * sizeof(uint2));

  // CHECK:   {
  // CHECK-NEXT:   syclct::get_default_queue().submit(
  // CHECK-NEXT:       [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:         auto tex42_acc = tex42.get_access(cgh);
  // CHECK-NEXT:         auto tex21_acc = tex21.get_access(cgh);
  // CHECK-NEXT:         cgh.parallel_for<syclct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:             cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, 1)), cl::sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:             [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:               kernel(tex42_acc, tex21_acc);
  // CHECK-NEXT:             });
  // CHECK-NEXT:       });
  // CHECK-NEXT: }
  kernel<<<1, 1>>>();

  // CHECK: syclct::dpct_detach_image(tex42);
  // CHECK-NEXT: syclct::dpct_detach_image(tex21);
  hipUnbindTexture(tex42);
  hipUnbindTexture(tex21);

  // CHECK: syclct::dpct_free(a42);
  hipFreeArray(a42);
 
  // CHECK: syclct::sycl_free(d_data42);
  // CHECK-NEXT: syclct::sycl_free(d_data21);
  hipFree(d_data42);
  hipFree(d_data21);
}
