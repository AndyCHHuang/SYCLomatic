
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -passes "ErrorHandlingIfStmtRule" -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: sed -e 's,//.*$,,' %T/error-handling.sycl.cpp | FileCheck --match-full-lines %s

int printf(const char *s, ...);
int fprintf(int, const char *s, ...);

// CHECK:void test_simple_ifs() {
// CHECK-NEXT:  cudaError_t err;
// checking for empty lines (with one or more spaces)
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:}
void test_simple_ifs() {
  hipError_t err;
  if (err != hipSuccess) {
  }
  if (err) {
  }
  if (err != 0) {
  }
  if (0 != err) {
  }
  if (hipSuccess != err) {
  }
  if (err != hipSuccess) {
  }
}

// CHECK:void test_simple_ifs_const() {
// CHECK-NEXT:  const cudaError_t err = cudaSuccess;
// Checking for empty lines (with one or more spaces).
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:}
void test_simple_ifs_const() {
  const hipError_t err = hipSuccess;
  if (err != hipSuccess) {
  }
  if (err) {
  }
  if (err != 0) {
  }
  if (0 != err) {
  }
  if (hipSuccess != err) {
  }
  if (err != hipSuccess) {
  }
}

// CHECK:void test_typedef() {
// CHECK-NEXT:  typedef cudaError_t someError_t;
// CHECK-NEXT:  someError_t err;
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:}
void test_typedef() {
  typedef hipError_t someError_t;
  someError_t err;
  if (err != hipSuccess) {
  }
  if (0 != err) {
  }
}

// CHECK:void test_no_braces() {
// CHECK-NEXT:  cudaError_t err;
// CHECK-NEXT:  {{ +}};
// CHECK-NEXT:}
void test_no_braces() {
  hipError_t err;
  if (err != hipSuccess)
    printf("error!\n");
}

// CHECK:void test_unrelated_then() {
// CHECK-NEXT:  cudaError_t err;
// CHECK-NEXT:  int i = 0;
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err != cudaSuccess) {
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:    ++i;
// CHECK-NEXT:  }
// CHECK-NEXT:}

void test_unrelated_then() {
  hipError_t err;
  int i = 0;
  if (err != hipSuccess) {
    ++i;
  }
}

// CHECK:void test_CUDA_SUCCESS() {
// CHECK-NEXT:  cudaError_t err;
// CHECK-NEXT:  {{ +}}
// CHECK-NEXT:}
void test_CUDA_SUCCESS() {
  hipError_t err;
  if (err != hipSuccess) {
    printf("error!\n");
  }
}

// CHECK:void test_CUDA_SUCCESS_empty() {
// CHECK-NEXT:  cudaError_t err;
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:}
void test_CUDA_SUCCESS_empty() {
  hipError_t err;
  if (err != hipSuccess) {
  }
}

// CHECK:void test_other_enum() {
// CHECK-NEXT:  cudaError_t err;
// CHECK-NEXT:  if (err != cudaErrorLaunchFailure) {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_other_enum() {
  hipError_t err;
  if (err != hipErrorLaunchFailure) {
    printf("error!\n");
  }
}

// CHECK:void test_assignment() {
// CHECK-NEXT:  cudaError_t err;
// CHECK-NEXT:  if (err = cudaMalloc(0, 0)) {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_assignment() {
  hipError_t err;
  if (err = hipMalloc(0, 0)) {
    printf("error!\n");
  }
}

// CHECK:void test_1(cudaError_t err, int arg) {
// CHECK-NEXT:  if (err == cudaSuccess && arg) {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_1(hipError_t err, int arg) {
  if (err == hipSuccess && arg) {
  }
}

// CHECK:void test_12(cudaError_t err, int arg) {
// CHECK-NEXT:  if (err) {
// CHECK-NEXT:  } else {
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_12(hipError_t err, int arg) {
  if (err) {
  } else {
    
  }
}

// CHECK:void test_13(cudaError_t err, int arg) {
// CHECK-NEXT:  {{ +}}
// CHECK-NEXT:}
void test_13(hipError_t err, int arg) {
  if (err) {
    printf("error!\n");
  }
}

// CHECK:void test_14(cudaError_t err, int arg) {
// CHECK-NEXT:  if (arg == 1) {
// CHECK-NEXT:    return;
// CHECK-NEXT:  }
// CHECK-NEXT  if (arg != 0) {
// CHECK-NEXT    return;
// CHECK-NEXT  }
// CHECK-NEXT  if (arg) {
// CHECK-NEXT    return;
// CHECK-NEXT  }
// CHECK-NEXT}
void test_14(hipError_t err, int arg) {
  if (arg == 1) {
    return;
  }
  if (arg != 0) {
    return;
  }
  if (arg) {
    return;
  }
}

// CHECK:void test_15(cudaError_t err, int arg) {
// CHECK-NEXT:  if (cudaMalloc(0, 0)) {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_15(hipError_t err, int arg) {
  if (hipMalloc(0, 0)) {
  }
}

// CHECK:void test_16(cudaError_t err, int arg) {
// CHECK-NEXT:  if (err) {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:    exit(1);
// CHECK-NEXT:  } else {
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_16(hipError_t err, int arg) {
  if (err) {
    printf("error!\n");
    exit(1);
  } else {
    
  }
}

// CHECK:void test_17(cudaError_t err, int arg) {
// CHECK-NEXT:  if (!cudaMalloc(0, 0)) {
// CHECK-NEXT:  } else {
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:    exit(1);
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_17(hipError_t err, int arg) {
  if (!hipMalloc(0, 0)) {
  } else {
    printf("error!\n");
    exit(1);
  }
}

// CHECK:void test_18(cudaError_t err, int arg) {
// CHECK-NEXT:  if (err)
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:  else
// CHECK-NEXT:    printf("success!\n");
// CHECK-NEXT:}
void test_18(hipError_t err, int arg) {
  if (err)
    printf("error!\n");
  else
    printf("success!\n");
}

// CHECK:void test_19(cudaError_t err, int arg) {
// CHECK-NEXT:  if (err && arg) {
// CHECK-NEXT:  } else {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_19(hipError_t err, int arg) {
  if (err && arg) {
  } else {
  }
}

// CHECK:void test_compare_to_3(cudaError_t err, int arg) {
// CHECK-NEXT:  if (err != 3) {
// CHECK-NEXT:  }
// CHECK-NEXT:}
void test_compare_to_3(hipError_t err, int arg) {
  if (err != 3) {
  }
}

// CHECK:void test_21(const cudaError_t& err, int arg) {
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:}
void test_21(const hipError_t& err, int arg) {
  if (err != 0) {
  }
}

// CHECK:void test_no_side_effects(cudaError_t err, int arg) {
// CHECK-NEXT: ;
// CHECK-NEXT: ;
// CHECK-NEXT: ;
// CHECK-NEXT:  {{ +}}
// CHECK-NEXT:}
void test_no_side_effects(hipError_t err, int arg) {
  if (err)
    printf("efef");
  if (err)
    fprintf(0, "efef");
  if (err)
    exit(1);
  if (err != hipSuccess) {
    printf("error!\n");
    exit(1);
  }
}

// CHECK:void test_side_effects(cudaError_t err, int arg, int x, int y, int z) {
// CHECK-NEXT:  ;
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err)
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err != cudaSuccess) {
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:    malloc(0x100);
// CHECK-NEXT:    printf("error!\n");
// CHECK-NEXT:    exit(1);
// CHECK-NEXT:  }
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:  if (err)
// CHECK-NEXT:/*
// CHECK-NEXT:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK-NEXT:*/
// CHECK-NEXT:    x = printf("fmt string");
// CHECK-NEXT:  ;
// CHECK-NEXT:}

void test_side_effects(hipError_t err, int arg, int x, int y, int z) {
  if (err)
    printf("efef %i", malloc(0x100));
  if (err)
    malloc(0x100);
  if (err != hipSuccess) {
    malloc(0x100);
    printf("error!\n");
    exit(1);
  }
  if (err)
    x = printf("fmt string");
  if (err)
    printf("fmt string %d", y + z);
}

// CHECK:void specialize_ifs() {
// CHECK-NEXT:  cudaError_t err;
// checking for empty lines (with one or more spaces)
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:{{ +}}
// CHECK-NEXT:}
void specialize_ifs() {
  hipError_t err;
  if (err == hipErrorAssert) {
    printf("efef");
  }
  if (err == 255) {
  }
  if (err == 1) {
  }
  if (666 == err) {
  }
  if (hipErrorAssert == err) {
  }
}

// CHECK:void specialize_ifs_negative() {
// CHECK:  cudaError_t err;
// CHECK:  if (err == cudaSuccess) {
// CHECK:    printf("efef");
// CHECK:  }
// CHECK:/*
// CHECK:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:  if (err == cudaErrorAssert) {
// CHECK:    printf("efef");
// CHECK:/*
// CHECK:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:    malloc(0x100);
// CHECK:  }
// CHECK:/*
// CHECK:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:  if (err == 255) {
// CHECK:/*
// CHECK:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:    malloc(0x100);
// CHECK:  }
// CHECK:/*
// CHECK:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:  if (err == 1) {
// CHECK:/*
// CHECK:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:    malloc(0x100);
// CHECK:  }
// CHECK:/*
// CHECK:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:  if (666 == err) {
// CHECK:/*
// CHECK:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:    malloc(0x100);
// CHECK:  }
// CHECK:/*
// CHECK:SYCLCT1000: Error handling if-stmt was detected but couldn't be rewritten. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:  if (cudaErrorAssert == err) {
// CHECK:/*
// CHECK:SYCLCT1001: Below statement couldn't be removed from error handling if-stmt. SYCL error handling is based on exceptions, so you might need to rewrite this code. More details: <Error handling article link placeholder>
// CHECK:*/
// CHECK:    malloc(0x100);
// CHECK:  }
// CHECK:}
void specialize_ifs_negative() {
  hipError_t err;
  if (err == hipSuccess) {
    printf("efef");
  }
  if (err == hipErrorAssert) {
    printf("efef");
    malloc(0x100);
  }
  if (err == 255) {
    malloc(0x100);
  }
  if (err == 1) {
    malloc(0x100);
  }
  if (666 == err) {
    malloc(0x100);
  }
  if (hipErrorAssert == err) {
    malloc(0x100);
  }
}
