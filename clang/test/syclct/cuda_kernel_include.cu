#include "hip/hip_runtime.h"
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"  -I ./
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda_kernel_include.dp.cpp

// CHECK:#include <CL/sycl.hpp>
// CHECK-NEXT:#include <dpct/dpct.hpp>
#include <stdio.h>

// CHECK:#include "simple_kernel.dp.hpp"
#include "simple_kernel.cuh"

int main(int argc, char **argv) {
  int size = 360;
  float *d_array;
  float h_array[360];

  // CHECK: dpct::dpct_malloc((void **)&d_array, sizeof(float) * size);
  hipMalloc((void **)&d_array, sizeof(float) * size);

  // CHECK: dpct::dpct_memset((void*)(d_array), 0, sizeof(float) * size);
  hipMemset(d_array, 0, sizeof(float) * size);

  // CHECK:  {
  // CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> d_array_buf = dpct::get_buffer_and_offset(d_array);
  // CHECK-NEXT:    size_t d_array_offset = d_array_buf.second;
  // CHECK-NEXT:    dpct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto d_array_acc = d_array_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class simple_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(size / 64, 1, 1) * cl::sycl::range<3>(64, 1, 1)), cl::sycl::range<3>(64, 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_ct1]]) {
  // CHECK-NEXT:            float *d_array = (float*)(&d_array_acc[0] + d_array_offset);
  // CHECK-NEXT:            simple_kernel(d_array, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  simple_kernel<<<size / 64, 64>>>(d_array);

  // CHECK:  dpct::dpct_memcpy((void*)(h_array), (void*)(d_array), 360 * sizeof(float), dpct::device_to_host);
  hipMemcpy(h_array, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 1; i < 360; i++) {
    if (fabs(h_array[i] - 10.0) > 1e-5) {
      exit(-1);
    }
  }

  hipFree(d_array);

  printf("Test Passed!\n");
  return 0;
}
