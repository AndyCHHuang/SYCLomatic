
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/sycl_style_double2.sycl.cpp --match-full-lines %s

// CHECK: void func3(cl::sycl::double2 a, cl::sycl::double2 b, cl::sycl::double2 c) try {
void func3(double2 a, double2 b, double2 c) {
}
// CHECK: void fun(cl::sycl::double2 a) try {}
void fun(double2 a) {}

int main() {
  // range default constructor does the right thing.
  // CHECK: cl::sycl::double2 deflt;
  double2 deflt;

  // CHECK: cl::sycl::double2 copyctor1 = cl::sycl::double2(1, 2);
  double2 copyctor1 = make_double2(1, 2);

  // CHECK: cl::sycl::double2 copyctor2 = cl::sycl::double2(copyctor1);
  double2 copyctor2 = double2(copyctor1);

  // CHECK: cl::sycl::double2 copyctor3(copyctor1);
  double2 copyctor3(copyctor1);

  // CHECK: func3(deflt, cl::sycl::double2(deflt), (cl::sycl::double2)deflt);
  func3(deflt, double2(deflt), (double2)deflt);

  // CHECK: cl::sycl::double2 *i4;
  double2 *i4;
  // CHECK: cl::sycl::double2 *i5;
  double2 *i5;
  // CHECK: cl::sycl::double2 i6;
  double2 i6;
  // CHECK: cl::sycl::double2 i7;
  double2 i7;
  // CHECK: int i = i6.x();
  int i = i6.x;
  // CHECK: i6.x() = i7.x();
  i6.x = i7.x;
  // CHECK: if (i6.x() == i7.x()) {
  if (i6.x == i7.x) {
  }
  // CHECK: cl::sycl::double2 i2_array[10];
  double2 i2_array[10];
  // CHECK: cl::sycl::double2 i2_array2[10];
  double2 i2_array2[10];
  // CHECK: if (i2_array[1].x() == i2_array2[1].x()) {
  if (i2_array[1].x == i2_array2[1].x) {
  }
  // CHECK: cl::sycl::double2 x = cl::sycl::double2(1, 2);
  double2 x = make_double2(1, 2);
  // CHECK: i4 = (cl::sycl::double2 *)i2_array;
  i4 = (double2 *)i2_array;
  // CHECK: i7 = (cl::sycl::double2)i6;
  i7 = (double2)i6;
  // CHECK: i7 = cl::sycl::double2(i6);
  i7 = double2(i6);
}
