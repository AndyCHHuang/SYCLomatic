// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/nested_device_call.sycl.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <cstdio>

// CHECK: void test0(cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]],
// CHECK:         int a) {
__device__ void test0(int a) {
  // CHECK: printf("Hello World %d\n", a);
  printf("Hello World %d\n", a);
  // CHECK: cl::sycl::sqrt(10.0);
  sqrt(10.0);
}

// CHECK: void test1(cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]],
// CHECK:         int a) {
__device__ void test1(int a) {
  // CHECK: test0([[ITEM]], a);
  test0(a);
  // CHECK: test0([[ITEM]], a + 1);
  test0(a + 1);
}

// CHECK: void test2(cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]],
// CHECK:         int a) {
__device__ void test2(int a) {
  // CHECK: test1([[ITEM]], a);
  test1(a);
  // CHECK: test1([[ITEM]], a + 1);
  test1(a + 1);
}

// CHECK: void test3(cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]],
// CHECK:         int a) {
__device__ void test3(int a) {
  // CHECK: test2([[ITEM]], a);
  test2(a);
  // CHECK: test2([[ITEM]], a + 1);
  test2(a + 1);
}

// CHECK: void kernel(cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
__global__ void kernel() {
  // CHECK: test3([[ITEM]], 1);
  test3(1);
  // CHECK: test3([[ITEM]], 2);
  test3(2);
}

int main() { kernel<<<1, 1>>>(); }
