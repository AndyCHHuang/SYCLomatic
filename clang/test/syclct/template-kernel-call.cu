
#include <hip/hip_runtime.h>
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path=%cuda-path
// RUN: FileCheck --input-file %T/template-kernel-call.sycl.cpp --match-full-lines %s

void printf(const char *format, unsigned char data);

template <class TName, unsigned N, class TData>
// CHECK: void testKernelPtr(cl::sycl::nd_item<3> item, const TData *L, const TData *M) {
__global__ void testKernelPtr(const TData *L, const TData *M) {
  // CHECK: int gtid = item.get_group(0) * item.get_local_range().get(0) + item.get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

template<class TData>
// CHECK: void testKernel(cl::sycl::nd_item<3> item, TData L, TData M, int N) {
__global__ void testKernel(TData L, TData M, int N) {
  // CHECK: int gtid = item.get_group(0) * item.get_local_range().get(0) + item.get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  L = M;
}

// CHECK: struct __sycl_align__(8) LA {
struct __align__(8) LA {
  unsigned int l, a;
};

template<class T>
class TestTemplate {
public:
  T data;
};

const unsigned ktarg = 80;
dim3 griddim = 2;
dim3 threaddim = 32;

template<class T>
void runTest() {
  typedef TestTemplate<T> TT;
  const void *karg1 = 0;
  const T *karg2 = 0;
  T *karg3 = 0;
  const TestTemplate<T> *karg4 = 0;
  TT *karg5 = 0;

  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg2_buf = syclct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:    size_t karg2_offset = karg2_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg2_acc = karg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernelPtr_{{[a-f0-9]+}}, class TestName, ktarg, T>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            const void *karg1 = (const void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            const T *karg2 = (const T*)(&karg2_acc[0] + karg2_offset);
  // CHECK-NEXT:            testKernelPtr<class TestName, ktarg, T>(it, (const T *)karg1, karg2);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernelPtr<class TestName, ktarg, T><<<griddim, threaddim>>>((const T*)karg1, karg2);

  
  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg3_buf = syclct::get_buffer_and_offset(karg3);
  // CHECK-NEXT:    size_t karg3_offset = karg3_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg3_acc = karg3_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernelPtr_{{[a-f0-9]+}}, class TestTemplate<T>, ktarg, T>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            const void *karg1 = (const void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            T *karg3 = (T*)(&karg3_acc[0] + karg3_offset);
  // CHECK-NEXT:            testKernelPtr<class TestTemplate<T>, ktarg, T>(it, karg1, karg3);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernelPtr<class TestTemplate<T>, ktarg, T><<<griddim, threaddim>>>(karg1, karg3);

  
  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg4_buf = syclct::get_buffer_and_offset(karg4);
  // CHECK-NEXT:    size_t karg4_offset = karg4_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg5_buf = syclct::get_buffer_and_offset(karg5);
  // CHECK-NEXT:    size_t karg5_offset = karg5_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg4_acc = karg4_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg5_acc = karg5_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernelPtr_{{[a-f0-9]+}}, T, ktarg, TestTemplate<T> >>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            const TestTemplate<T> *karg4 = (const TestTemplate<T>*)(&karg4_acc[0] + karg4_offset);
  // CHECK-NEXT:            TT *karg5 = (TT*)(&karg5_acc[0] + karg5_offset);
  // CHECK-NEXT:            testKernelPtr<T, ktarg, TestTemplate<T> >(it, karg4, karg5);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernelPtr<T, ktarg, TestTemplate<T> ><<<griddim, threaddim>>>(karg4, karg5);

  T karg1T, karg2T;
  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernel_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            testKernel<T>(it, karg1T, karg2T, ktarg);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernel<T><<<griddim, threaddim>>>(karg1T, karg2T, ktarg);

  TestTemplate<T> karg3TT;
  TT karg4TT;

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernel_{{[a-f0-9]+}}, TestTemplate<T> >>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            testKernel<TestTemplate<T> >(it, karg3TT, karg4TT, ktarg);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernel<TestTemplate<T> ><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernel_{{[a-f0-9]+}}, TT>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            testKernel<TT>(it, karg3TT, karg4TT, ktarg);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernel<TT><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);
}

int main() {
  void *karg1 = 0;
  LA *karg2 = 0;
  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg2_buf = syclct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:    size_t karg2_offset = karg2_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg2_acc = karg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernelPtr_{{[a-f0-9]+}}, class TestName, ktarg, LA>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> it) {
  // CHECK-NEXT:            void *karg1 = (void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            struct LA *karg2 = (struct LA*)(&karg2_acc[0] + karg2_offset);
  // CHECK-NEXT:            testKernelPtr<class TestName, ktarg, LA>(it, (const LA *)karg1, karg2);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernelPtr<class TestName, ktarg, LA><<<griddim, threaddim>>>((const LA *)karg1, karg2);

  LA karg1LA, karg2LA;
  int intvar = 20;
  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<SyclKernelName<class testKernel_{{[a-f0-9]+}}, LA>>(
  // CHECK-NEXT:          cl::sycl::nd_range<1>((cl::sycl::range<1>(10) * cl::sycl::range<1>(intvar)), cl::sycl::range<1>(intvar)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<1> it) {
  // CHECK-NEXT:            testKernel<LA>(it, karg1LA, karg2LA, ktarg);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  };
  testKernel<LA><<<10, intvar>>>(karg1LA, karg2LA, ktarg);
}
