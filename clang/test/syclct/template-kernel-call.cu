
#include <hip/hip_runtime.h>
// FIXME
// UNSUPPORTED: -windows-
// RUN: syclct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/template-kernel-call.sycl.cpp --match-full-lines %s

void printf(const char *format, unsigned char data);

template <class TName, unsigned N, class TData>
// CHECK: void testKernelPtr(const TData *L, const TData *M, cl::sycl::nd_item<3> [[ITEMNAME:item_[a-f0-9]+]]) {
__global__ void testKernelPtr(const TData *L, const TData *M) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

template<class TData>
// CHECK: void testKernel(TData L, TData M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_[a-f0-9]+]]) {
__global__ void testKernel(TData L, TData M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  L = M;
}

// CHECK: struct __sycl_align__(8) LA {
struct __align__(8) LA {
  unsigned int l, a;
};

template<class T>
class TestTemplate {
public:
  T data;
};

const unsigned ktarg = 80;
dim3 griddim = 2;
dim3 threaddim = 32;

template<class T>
void runTest() {
  typedef TestTemplate<T> TT;
  const void *karg1 = 0;
  const T *karg2 = 0;
  T *karg3 = 0;
  const TestTemplate<T> *karg4 = 0;
  TT *karg5 = 0;

  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg2_buf = syclct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:    size_t karg2_offset = karg2_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg2_acc = karg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestName, syclct_kernel_scalar<ktarg>, T>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            const void *karg1 = (const void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            const T *karg2 = (const T*)(&karg2_acc[0] + karg2_offset);
  // CHECK-NEXT:            testKernelPtr<class TestName, ktarg, T>((const T *)karg1, karg2, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<class TestName, ktarg, T><<<griddim, threaddim>>>((const T *)karg1, karg2);

  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg3_buf = syclct::get_buffer_and_offset(karg3);
  // CHECK-NEXT:    size_t karg3_offset = karg3_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg3_acc = karg3_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestTemplate<T>, syclct_kernel_scalar<ktarg>, T>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            const void *karg1 = (const void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            T *karg3 = (T*)(&karg3_acc[0] + karg3_offset);
  // CHECK-NEXT:            testKernelPtr<class TestTemplate<T>, ktarg, T>(karg1, karg3, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<class TestTemplate<T>, ktarg, T><<<griddim, threaddim>>>(karg1, karg3);

  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg4_buf = syclct::get_buffer_and_offset(karg4);
  // CHECK-NEXT:    size_t karg4_offset = karg4_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg5_buf = syclct::get_buffer_and_offset(karg5);
  // CHECK-NEXT:    size_t karg5_offset = karg5_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg4_acc = karg4_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg5_acc = karg5_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, T, syclct_kernel_scalar<ktarg>, TestTemplate<T>>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            const TestTemplate<T> *karg4 = (const TestTemplate<T>*)(&karg4_acc[0] + karg4_offset);
  // CHECK-NEXT:            TT *karg5 = (TT*)(&karg5_acc[0] + karg5_offset);
  // CHECK-NEXT:            testKernelPtr<T, ktarg, TestTemplate<T>>(karg4, karg5, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<T, ktarg, TestTemplate<T> ><<<griddim, threaddim>>>(karg4, karg5);

  T karg1T, karg2T;
  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            testKernel<T>(karg1T, karg2T, ktarg, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernel<T><<<griddim, threaddim>>>(karg1T, karg2T, ktarg);

  TestTemplate<T> karg3TT;
  TT karg4TT;

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}, TestTemplate<T>>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            testKernel<TestTemplate<T>>(karg3TT, karg4TT, ktarg, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernel<TestTemplate<T> ><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);

  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}, TT>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            testKernel<TT>(karg3TT, karg4TT, ktarg, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernel<TT><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);
}

int main() {
  void *karg1 = 0;
  LA *karg2 = 0;
  // CHECK:  {
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg1_buf = syclct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:    size_t karg1_offset = karg1_buf.second;
  // CHECK-NEXT:    std::pair<syclct::buffer_t, size_t> karg2_buf = syclct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:    size_t karg2_offset = karg2_buf.second;
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg1_acc = karg1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        auto karg2_acc = karg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestName, syclct_kernel_scalar<ktarg>, LA>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            void *karg1 = (void*)(&karg1_acc[0] + karg1_offset);
  // CHECK-NEXT:            LA *karg2 = (LA*)(&karg2_acc[0] + karg2_offset);
  // CHECK-NEXT:            testKernelPtr<class TestName, ktarg, LA>((const LA *)karg1, karg2, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<class TestName, ktarg, LA><<<griddim, threaddim>>>((const LA *)karg1, karg2);

  LA karg1LA, karg2LA;
  int intvar = 20;
  // CHECK:  {
  // CHECK-NEXT:    syclct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<syclct_kernel_name<class testKernel_{{[a-f0-9]+}}, LA>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(10, 1, 1) * cl::sycl::range<3>(intvar, 1, 1)), cl::sycl::range<3>(intvar, 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_[a-f0-9]+]]) {
  // CHECK-NEXT:            testKernel<LA>(karg1LA, karg2LA, ktarg, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernel<LA><<<10, intvar>>>(karg1LA, karg2LA, ktarg);
}
