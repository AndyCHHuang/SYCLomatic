// RUN: cd %T
// RUN: mkdir foo
// RUN: cat %s > %T/foo/use_format_file_with_p1.cu
// RUN: echo "ColumnLimit: 50" > %T/foo/.clang-format
// RUN: dpct -p=./foo ./foo/use_format_file_with_p1.cu --out-root=%T/out --cuda-include-path="%cuda-path/include" -- --cuda-host-only
// RUN: FileCheck -strict-whitespace %s --match-full-lines --input-file %T/out/use_format_file_with_p1.dp.cpp
// RUN: rm -rf ./*
#include "hip/hip_runtime.h"

void bar();
#define SIZE 100

size_t size = 1234567 * sizeof(float);
float *h_A = (float *)malloc(size);
float *d_A = NULL;

     //CHECK:void foo1() try {
//CHECK-NEXT:  for(;;)
//CHECK-NEXT:    /*
//CHECK-NEXT:    DPCT1003:0: Migrated api does not return error
//CHECK-NEXT:    code. (*, 0) is inserted. You may need to
//CHECK-NEXT:    rewrite this code.
//CHECK-NEXT:    */
//CHECK-NEXT:    int a =
//CHECK-NEXT:        (dpct::get_default_queue_wait()
//CHECK-NEXT:             .memcpy(d_A, h_A,
//CHECK-NEXT:                     sizeof(double) * SIZE * SIZE)
//CHECK-NEXT:             .wait(),
//CHECK-NEXT:         0);
//CHECK-NEXT:}
//CHECK-NEXT:catch (sycl::exception const &exc) {
//CHECK-NEXT:  std::cerr << exc.what()
//CHECK-NEXT:            << "Exception caught at file:"
//CHECK-NEXT:            << __FILE__ << ", line:" << __LINE__
//CHECK-NEXT:            << std::endl;
//CHECK-NEXT:  std::exit(1);
//CHECK-NEXT:}
void foo1() {
  for(;;)
    int a = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
}