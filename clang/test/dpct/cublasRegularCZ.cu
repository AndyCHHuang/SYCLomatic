// RUN: dpct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cublasRegularCZ.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;

  int* result = 0;
  float* result_f = 0;
  double* result_d = 0;
  hipComplex* x_c = 0;
  hipDoubleComplex* x_z = 0;

  int incx = 1;
  int incy = 1;
  int n = 10;

  //level 1
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::iamax(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::iamax(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamax(handle, n, x_c, incx, result);
  hipblasIcamax(handle, n, x_c, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::iamax(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::iamax(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamax(handle, n, x_z, incx, result);
  hipblasIzamax(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::iamin(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::iamin(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamin(handle, n, x_c, incx, result);
  hipblasIcamin(handle, n, x_c, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::iamin(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer), 0);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result);
  // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(int)));
  // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer(cl::sycl::range<1>(1));
  // CHECK-NEXT: mkl::iamin(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer);
  // CHECK-NEXT: buffer_ct{{[0-9]+}}.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<cl::sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamin(handle, n, x_z, incx, result);
  hipblasIzamin(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: status = (mkl::asum(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::asum(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasScasum(handle, n, x_c, incx, result_f);
  hipblasScasum(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: status = (mkl::asum(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: mkl::asum(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDzasum(handle, n, x_z, incx, result_d);
  hipblasDzasum(handle, n, x_z, incx, result_d);

  hipComplex* alpha_c = 0;
  hipComplex* beta_c = 0;
  hipDoubleComplex* alpha_z = 0;
  hipDoubleComplex* beta_z = 0;
  float* alpha_f = 0;
  double* alpha_d = 0;
  hipComplex* y_c = 0;
  hipDoubleComplex* y_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::axpy(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::axpy(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);
  hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::axpy(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::axpy(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);
  hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::copy(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::copy(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCcopy(handle, n, x_c, incx, y_c, incy);
  hipblasCcopy(handle, n, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::copy(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::copy(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZcopy(handle, n, x_z, incx, y_z, incy);
  hipblasZcopy(handle, n, x_z, incx, y_z, incy);

  hipComplex* result_c = 0;
  hipDoubleComplex* result_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::dotu(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::dotu(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::dotc(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::dotc(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::dotu(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::dotu(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::dotc(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::dotc(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: status = (mkl::nrm2(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::nrm2(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasScnrm2(handle, n, x_c, incx, result_f);
  hipblasScnrm2(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: status = (mkl::nrm2(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: mkl::nrm2(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDznrm2(handle, n, x_z, incx, result_d);
  hipblasDznrm2(handle, n, x_z, incx, result_d);

  float* c_f = 0;
  float* s_f = 0;
  double* c_d = 0;
  double* s_d = 0;
  hipComplex* c_c = 0;
  hipComplex* s_c = 0;
  hipDoubleComplex* c_z = 0;
  hipDoubleComplex* s_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::rot(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, *(c_f), *(s_f)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::rot(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, *(c_f), *(s_f));
  // CHECK-NEXT: }
  status = hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);
  hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::rot(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, *(c_d), *(s_d)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::rot(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, *(c_d), *(s_d));
  // CHECK-NEXT: }
  status = hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);
  hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(c_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(s_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::rotg(handle, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(c_f);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(s_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::rotg(handle, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasCrotg(handle, x_c, y_c, c_f, s_c);
  hipblasCrotg(handle, x_c, y_c, c_f, s_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(c_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(s_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::rotg(handle, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(c_d);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(s_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::rotg(handle, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZrotg(handle, x_z, y_z, c_d, s_z);
  hipblasZrotg(handle, x_z, y_z, c_d, s_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::scal(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::scal(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCscal(handle, n, alpha_c, x_c, incx);
  hipblasCscal(handle, n, alpha_c, x_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::scal(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::scal(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZscal(handle, n, alpha_z, x_z, incx);
  hipblasZscal(handle, n, alpha_z, x_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::scal(handle, n, *(alpha_f), buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::scal(handle, n, *(alpha_f), buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCsscal(handle, n, alpha_f, x_c, incx);
  hipblasCsscal(handle, n, alpha_f, x_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::scal(handle, n, *(alpha_d), buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::scal(handle, n, *(alpha_d), buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZdscal(handle, n, alpha_d, x_z, incx);
  hipblasZdscal(handle, n, alpha_d, x_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::swap(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::swap(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCswap(handle, n, x_c, incx, y_c, incy);
  hipblasCswap(handle, n, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::swap(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::swap(handle, n, buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZswap(handle, n, x_z, incx, y_z, incy);
  hipblasZswap(handle, n, x_z, incx, y_z, incy);

  //level 2
  int m=0;
  int kl=0;
  int ku=0;
  int lda = 10;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::gbmv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, kl, ku, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::gbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCgbmv(handle, (hipblasOperation_t)trans0, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans1;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::gbmv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, kl, ku, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::gbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZgbmv(handle, (hipblasOperation_t)trans1, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans2;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::gemv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::gemv(handle, mkl::transpose::nontrans, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCgemv(handle, (hipblasOperation_t)trans2, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = 0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::gemv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::gemv(handle, mkl::transpose::nontrans, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZgemv(handle, (hipblasOperation_t)0, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::geru(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::geru(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::gerc(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::gerc(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::geru(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::geru(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::gerc(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::gerc(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  int k = 1;
  int fill0 = 0;
  int fill1 = 1;
  int diag0 = 0;
  int diag1 = 1;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = 1;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::tbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::tbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, k, x_c, lda, result_c, incx);
  hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = 2;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::tbmv(handle, (((int)fill1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag1, n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::tbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbmv(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, k, x_z, lda, result_z, incx);
  hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::tbsv(handle, (((int)0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)0,  n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::tbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbsv(handle, (hipblasFillMode_t)0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0,  n, k, x_c, lda, result_c, incx);
  hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::tbsv(handle, (((int)1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)1,  n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::tbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbsv(handle, (hipblasFillMode_t)1, (hipblasOperation_t)trans0, (hipblasDiagType_t)1,  n, k, x_z, lda, result_z, incx);
  hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::tpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::tpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::tpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::tpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::tpsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::tpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::tpsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::tpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::trmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::trmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::trmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::trmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::trsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::trsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::trsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::trsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::hemv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::hemv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChemv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::hemv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::hemv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhemv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::hbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::hbmv(handle, mkl::uplo::lower, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::hbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::hbmv(handle, mkl::uplo::lower, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::hpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::hpmv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChpmv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);
  hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::hpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::hpmv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, buffer_ct{{[0-9]+}}, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhpmv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);
  hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::her(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_f), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::her(handle, mkl::uplo::lower, n, *(alpha_f), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c, lda);
  hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::her(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_d), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::her(handle, mkl::uplo::lower, n, *(alpha_d), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z, lda);
  hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::her2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::her2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::her2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::her2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::hpr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_f), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::hpr(handle, mkl::uplo::lower, n, *(alpha_f), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasChpr(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c);
  hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::hpr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_d), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::hpr(handle, mkl::uplo::lower, n, *(alpha_d), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z);
  hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::hpr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::hpr2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasChpr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c);
  hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::hpr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::hpr2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, incx, buffer_ct{{[0-9]+}}, incy, buffer_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z);
  hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);

  int N = 100;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  status = hipblasCgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(x_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(y_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(result_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  status = hipblasZgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);

  hipComplex* A_c = 0;
  hipDoubleComplex* A_z = 0;
  hipComplex* B_c = 0;
  hipDoubleComplex* B_z = 0;
  hipComplex* C_c = 0;
  hipDoubleComplex* C_z = 0;


  int ldb = 10;
  int ldc = 10;


  const float alpha_s = 1;
  const float beta_s = 1;
  const double beta_d = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:62: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::cgemm3m(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), m, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::cgemm3m(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, m, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = cublasCgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:63: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::zgemm3m(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), m, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::zgemm3m(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, m, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = cublasZgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  int side0 = 0;
  int side1 = 1;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:64: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::symm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::symm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:65: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::symm(handle, (mkl::side)side1, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::symm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:66: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::syrk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::syrk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  hipblasCsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:67: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::syrk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::syrk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);
  hipblasZsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:68: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::syr2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::syr2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:69: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::syr2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::syr2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:70: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::trsm(handle, (mkl::side)0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::trsm(handle, mkl::side::left, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT: }
  status = hipblasCtrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_c, A_c, lda, B_c, ldb);
  hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_c, A_c, lda, B_c, ldb);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:71: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::trsm(handle, (mkl::side)1, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::trsm(handle, mkl::side::left, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT: }
  status = hipblasZtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_z, A_z, lda, B_z, ldb);
  hipblasZtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_z, A_z, lda, B_z, ldb);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:72: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::hemm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::hemm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasChemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasChemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:73: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::hemm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::hemm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZhemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZhemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:74: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::herk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(&alpha_s), buffer_ct{{[0-9]+}}, lda, *(&beta_s), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::herk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, *(&alpha_s), buffer_ct{{[0-9]+}}, lda, *(&beta_s), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);
  hipblasCherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:75: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::herk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(alpha_d), buffer_ct{{[0-9]+}}, lda, *(&beta_d), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::herk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, *(alpha_d), buffer_ct{{[0-9]+}}, lda, *(&beta_d), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);
  hipblasZherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:76: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: status = (mkl::her2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, *(&beta_s), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_c);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: mkl::her2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, *(&beta_s), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);
  hipblasCher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:77: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: status = (mkl::her2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, *(&beta_d), buffer_ct{{[0-9]+}}, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(A_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(B_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(C_z);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: mkl::her2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb, *(&beta_d), buffer_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
  hipblasZher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
}
