// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasRegularCZ.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;

  int* result = 0;
  float* result_f = 0;
  double* result_d = 0;
  hipComplex* x_c = 0;
  hipDoubleComplex* x_z = 0;

  int incx = 1;
  int incy = 1;
  int n = 10;

  //level 1
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamax(handle, n, x_c_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamax(handle, n, x_c_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamax(handle, n, x_c, incx, result);
  hipblasIcamax(handle, n, x_c, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamax(handle, n, x_z_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamax(handle, n, x_z_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamax(handle, n, x_z, incx, result);
  hipblasIzamax(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamin(handle, n, x_c_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamin(handle, n, x_c_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamin(handle, n, x_c, incx, result);
  hipblasIcamin(handle, n, x_c, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamin(handle, n, x_z_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::mem_mgr::instance().get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamin(handle, n, x_z_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamin(handle, n, x_z, incx, result);
  hipblasIzamin(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(result_f);
  // CHECK-NEXT: status = (mkl::blas::asum(handle, n, x_c_buff_ct1, incx, result_f_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(result_f);
  // CHECK-NEXT: mkl::blas::asum(handle, n, x_c_buff_ct1, incx, result_f_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasScasum(handle, n, x_c, incx, result_f);
  hipblasScasum(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buff_ct1 = dpct::mem_mgr::instance().get_buffer<double>(result_d);
  // CHECK-NEXT: status = (mkl::blas::asum(handle, n, x_z_buff_ct1, incx, result_d_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buff_ct1 = dpct::mem_mgr::instance().get_buffer<double>(result_d);
  // CHECK-NEXT: mkl::blas::asum(handle, n, x_z_buff_ct1, incx, result_d_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDzasum(handle, n, x_z, incx, result_d);
  hipblasDzasum(handle, n, x_z, incx, result_d);

  hipComplex* alpha_c = 0;
  hipComplex* beta_c = 0;
  hipDoubleComplex* alpha_z = 0;
  hipDoubleComplex* beta_z = 0;
  float* alpha_f = 0;
  double* alpha_d = 0;
  hipComplex* y_c = 0;
  hipDoubleComplex* y_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::axpy(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::axpy(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);
  hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::axpy(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::axpy(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);
  hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::copy(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::copy(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasCcopy(handle, n, x_c, incx, y_c, incy);
  hipblasCcopy(handle, n, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::copy(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::copy(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZcopy(handle, n, x_z, incx, y_z, incy);
  hipblasZcopy(handle, n, x_z, incx, y_z, incy);

  hipComplex* result_c = 0;
  hipDoubleComplex* result_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::dotu(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::dotu(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::dotc(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::dotc(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::dotu(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::dotu(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::dotc(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::dotc(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(result_f);
  // CHECK-NEXT: status = (mkl::blas::nrm2(handle, n, x_c_buff_ct1, incx, result_f_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(result_f);
  // CHECK-NEXT: mkl::blas::nrm2(handle, n, x_c_buff_ct1, incx, result_f_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasScnrm2(handle, n, x_c, incx, result_f);
  hipblasScnrm2(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buff_ct1 = dpct::mem_mgr::instance().get_buffer<double>(result_d);
  // CHECK-NEXT: status = (mkl::blas::nrm2(handle, n, x_z_buff_ct1, incx, result_d_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buff_ct1 = dpct::mem_mgr::instance().get_buffer<double>(result_d);
  // CHECK-NEXT: mkl::blas::nrm2(handle, n, x_z_buff_ct1, incx, result_d_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDznrm2(handle, n, x_z, incx, result_d);
  hipblasDznrm2(handle, n, x_z, incx, result_d);

  float* c_f = 0;
  float* s_f = 0;
  double* c_d = 0;
  double* s_d = 0;
  hipComplex* c_c = 0;
  hipComplex* s_c = 0;
  hipDoubleComplex* c_z = 0;
  hipDoubleComplex* s_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::rot(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy, *(c_f), *(s_f)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::rot(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy, *(c_f), *(s_f));
  // CHECK-NEXT: }
  status = hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);
  hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::rot(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy, *(c_d), *(s_d)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::rot(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy, *(c_d), *(s_d));
  // CHECK-NEXT: }
  status = hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);
  hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto c_f_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(c_f);
  // CHECK-NEXT: auto s_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(s_c);
  // CHECK-NEXT: status = (mkl::blas::rotg(handle, x_c_buff_ct1, y_c_buff_ct1, c_f_buff_ct1, s_c_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto c_f_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(c_f);
  // CHECK-NEXT: auto s_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(s_c);
  // CHECK-NEXT: mkl::blas::rotg(handle, x_c_buff_ct1, y_c_buff_ct1, c_f_buff_ct1, s_c_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasCrotg(handle, x_c, y_c, c_f, s_c);
  hipblasCrotg(handle, x_c, y_c, c_f, s_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto c_d_buff_ct1 = dpct::mem_mgr::instance().get_buffer<double>(c_d);
  // CHECK-NEXT: auto s_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(s_z);
  // CHECK-NEXT: status = (mkl::blas::rotg(handle, x_z_buff_ct1, y_z_buff_ct1, c_d_buff_ct1, s_z_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto c_d_buff_ct1 = dpct::mem_mgr::instance().get_buffer<double>(c_d);
  // CHECK-NEXT: auto s_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(s_z);
  // CHECK-NEXT: mkl::blas::rotg(handle, x_z_buff_ct1, y_z_buff_ct1, c_d_buff_ct1, s_z_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasZrotg(handle, x_z, y_z, c_d, s_z);
  hipblasZrotg(handle, x_z, y_z, c_d, s_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: status = (mkl::blas::scal(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: mkl::blas::scal(handle, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCscal(handle, n, alpha_c, x_c, incx);
  hipblasCscal(handle, n, alpha_c, x_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: status = (mkl::blas::scal(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: mkl::blas::scal(handle, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZscal(handle, n, alpha_z, x_z, incx);
  hipblasZscal(handle, n, alpha_z, x_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: status = (mkl::blas::scal(handle, n, *(alpha_f), x_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: mkl::blas::scal(handle, n, *(alpha_f), x_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCsscal(handle, n, alpha_f, x_c, incx);
  hipblasCsscal(handle, n, alpha_f, x_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: status = (mkl::blas::scal(handle, n, *(alpha_d), x_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: mkl::blas::scal(handle, n, *(alpha_d), x_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZdscal(handle, n, alpha_d, x_z, incx);
  hipblasZdscal(handle, n, alpha_d, x_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::swap(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::swap(handle, n, x_c_buff_ct1, incx, y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasCswap(handle, n, x_c, incx, y_c, incy);
  hipblasCswap(handle, n, x_c, incx, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::swap(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::swap(handle, n, x_z_buff_ct1, incx, y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZswap(handle, n, x_z, incx, y_z, incy);
  hipblasZswap(handle, n, x_z, incx, y_z, incy);

  //level 2
  int m=0;
  int kl=0;
  int ku=0;
  int lda = 10;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::gbmv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, kl, ku, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::gbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasCgbmv(handle, (hipblasOperation_t)trans0, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans1;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::gbmv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, kl, ku, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::gbmv(handle, mkl::transpose::nontrans, m, n, kl, ku, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZgbmv(handle, (hipblasOperation_t)trans1, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans2;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::gemv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::gemv(handle, mkl::transpose::nontrans, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasCgemv(handle, (hipblasOperation_t)trans2, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = 0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::gemv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::gemv(handle, mkl::transpose::nontrans, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZgemv(handle, (hipblasOperation_t)0, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::geru(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::geru(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::gerc(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::gerc(handle, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::geru(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::geru(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::gerc(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::gerc(handle, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  int k = 1;
  int fill0 = 0;
  int fill1 = 1;
  int diag0 = 0;
  int diag1 = 1;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = 1;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::tbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, k, x_c_buff_ct1, lda, result_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::tbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, x_c_buff_ct1, lda, result_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, k, x_c, lda, result_c, incx);
  hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = 2;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::tbmv(handle, (((int)fill1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag1, n, k, x_z_buff_ct1, lda, result_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::tbmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, k, x_z_buff_ct1, lda, result_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZtbmv(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, k, x_z, lda, result_z, incx);
  hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::tbsv(handle, (((int)0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)0,  n, k, x_c_buff_ct1, lda, result_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::tbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, x_c_buff_ct1, lda, result_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCtbsv(handle, (hipblasFillMode_t)0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0,  n, k, x_c, lda, result_c, incx);
  hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::tbsv(handle, (((int)1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)1,  n, k, x_z_buff_ct1, lda, result_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::tbsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit,  n, k, x_z_buff_ct1, lda, result_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZtbsv(handle, (hipblasFillMode_t)1, (hipblasOperation_t)trans0, (hipblasDiagType_t)1,  n, k, x_z, lda, result_z, incx);
  hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::tpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_c_buff_ct1, result_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::tpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_buff_ct1, result_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::tpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_z_buff_ct1, result_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::tpmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_buff_ct1, result_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::tpsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_c_buff_ct1, result_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::tpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_buff_ct1, result_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::tpsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_z_buff_ct1, result_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::tpsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_buff_ct1, result_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::trmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_c_buff_ct1, lda, result_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::trmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_buff_ct1, lda, result_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::trmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_z_buff_ct1, lda, result_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::trmv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_buff_ct1, lda, result_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::trsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_c_buff_ct1, lda, result_c_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::trsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_c_buff_ct1, lda, result_c_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasCtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::trsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_z_buff_ct1, lda, result_z_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::trsv(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, n, x_z_buff_ct1, lda, result_z_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasZtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::hemv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::hemv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasChemv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::hemv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::hemv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZhemv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::hbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::hbmv(handle, mkl::uplo::lower, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, lda, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasChbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::hbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::hbmv(handle, mkl::uplo::lower, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, lda, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZhbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = (mkl::blas::hpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: mkl::blas::hpmv(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, x_c_buff_ct2, incx, std::complex<float>((beta_c)->x(),(beta_c)->y()), y_c_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasChpmv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);
  hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = (mkl::blas::hpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buff_ct2 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: mkl::blas::hpmv(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, x_z_buff_ct2, incx, std::complex<double>((beta_z)->x(),(beta_z)->y()), y_z_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasZhpmv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);
  hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::her(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_f), x_c_buff_ct1, incx, result_c_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::her(handle, mkl::uplo::lower, n, *(alpha_f), x_c_buff_ct1, incx, result_c_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasCher(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c, lda);
  hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::her(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_d), x_z_buff_ct1, incx, result_z_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::her(handle, mkl::uplo::lower, n, *(alpha_d), x_z_buff_ct1, incx, result_z_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasZher(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z, lda);
  hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::her2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::her2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasCher2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::her2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::her2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasZher2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::hpr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_f), x_c_buff_ct1, incx, result_c_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::hpr(handle, mkl::uplo::lower, n, *(alpha_f), x_c_buff_ct1, incx, result_c_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasChpr(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c);
  hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::hpr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(alpha_d), x_z_buff_ct1, incx, result_z_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::hpr(handle, mkl::uplo::lower, n, *(alpha_d), x_z_buff_ct1, incx, result_z_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasZhpr(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z);
  hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::hpr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::hpr2(handle, mkl::uplo::lower, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, incx, y_c_buff_ct1, incy, result_c_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasChpr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c);
  hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::hpr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::hpr2(handle, mkl::uplo::lower, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, incx, y_z_buff_ct1, incy, result_z_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasZhpr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z);
  hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);

  int N = 100;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = (mkl::blas::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, N, y_c_buff_ct1, N, std::complex<float>((beta_c)->x(),(beta_c)->y()), result_c_buff_ct1, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), x_c_buff_ct1, N, y_c_buff_ct1, N, std::complex<float>((beta_c)->x(),(beta_c)->y()), result_c_buff_ct1, N);
  // CHECK-NEXT: }
  status = hipblasCgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = (mkl::blas::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, N, y_z_buff_ct1, N, std::complex<double>((beta_z)->x(),(beta_z)->y()), result_z_buff_ct1, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), x_z_buff_ct1, N, y_z_buff_ct1, N, std::complex<double>((beta_z)->x(),(beta_z)->y()), result_z_buff_ct1, N);
  // CHECK-NEXT: }
  status = hipblasZgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);

  hipComplex* A_c = 0;
  hipDoubleComplex* A_z = 0;
  hipComplex* B_c = 0;
  hipDoubleComplex* B_z = 0;
  hipComplex* C_c = 0;
  hipDoubleComplex* C_z = 0;


  int ldb = 10;
  int ldc = 10;


  const float alpha_s = 1;
  const float beta_s = 1;
  const double beta_d = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:62: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::cgemm3m(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), m, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::cgemm3m(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, m, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = cublasCgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:63: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::zgemm3m(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), m, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::zgemm3m(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, m, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = cublasZgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  int side0 = 0;
  int side1 = 1;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:64: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::symm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::symm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasCsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:65: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::symm(handle, (mkl::side)side1, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::symm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasZsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:66: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::syrk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::syrk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasCsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  hipblasCsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:67: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::syrk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::syrk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasZsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);
  hipblasZsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:68: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::syr2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::syr2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasCsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:69: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::syr2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::syr2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasZsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:70: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: status = (mkl::blas::trsm(handle, (mkl::side)0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: mkl::blas::trsm(handle, mkl::side::left, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb);
  // CHECK-NEXT: }
  status = hipblasCtrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_c, A_c, lda, B_c, ldb);
  hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_c, A_c, lda, B_c, ldb);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:71: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: status = (mkl::blas::trsm(handle, (mkl::side)1, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: mkl::blas::trsm(handle, mkl::side::left, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb);
  // CHECK-NEXT: }
  status = hipblasZtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_z, A_z, lda, B_z, ldb);
  hipblasZtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_z, A_z, lda, B_z, ldb);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:72: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::hemm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::hemm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, std::complex<float>((beta_c)->x(),(beta_c)->y()), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasChemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasChemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:73: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::hemm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::hemm(handle, mkl::side::left, mkl::uplo::lower, m, n, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, std::complex<double>((beta_z)->x(),(beta_z)->y()), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasZhemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZhemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:74: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::herk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(&alpha_s), A_c_buff_ct1, lda, *(&beta_s), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::herk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, *(&alpha_s), A_c_buff_ct1, lda, *(&beta_s), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasCherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);
  hipblasCherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:75: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::herk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(alpha_d), A_z_buff_ct1, lda, *(&beta_d), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::herk(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, *(alpha_d), A_z_buff_ct1, lda, *(&beta_d), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasZherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);
  hipblasZherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:76: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = (mkl::blas::her2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, *(&beta_s), C_c_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: mkl::blas::her2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<float>((alpha_c)->x(),(alpha_c)->y()), A_c_buff_ct1, lda, B_c_buff_ct1, ldb, *(&beta_s), C_c_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasCher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);
  hipblasCher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:77: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = (mkl::blas::her2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, *(&beta_d), C_z_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buff_ct1 = dpct::mem_mgr::instance().get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: mkl::blas::her2k(handle, mkl::uplo::lower, mkl::transpose::nontrans, n, k, std::complex<double>((alpha_z)->x(),(alpha_z)->y()), A_z_buff_ct1, lda, B_z_buff_ct1, ldb, *(&beta_d), C_z_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasZher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
  hipblasZher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
}
