// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.2, v10.0
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn_cuda10-1.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int m = 0;
    int n = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);
    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;

    //CHECK: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnSpotri_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnDpotri_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnCpotri_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnZpotri_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    status = hipsolverDnSpotri_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotri_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotri_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotri_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }

    status = hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::memory_manager::get_instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::memory_manager::get_instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int,1> buffer_ct7 = allocation_ct7.buffer.reinterpret<int, 1>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t,1> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
}
