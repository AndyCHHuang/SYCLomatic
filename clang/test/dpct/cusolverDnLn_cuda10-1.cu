// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.2, v10.0
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn_cuda10-1.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int m = 0;
    int n = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);
    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;

    //CHECK: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnSpotri_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnDpotri_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnCpotri_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnZpotri_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    status = hipsolverDnSpotri_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotri_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotri_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotri_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buff_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, A_f_buff_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buff_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, A_f_buff_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }

    status = hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buff_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, A_d_buff_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buff_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, A_d_buff_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buff_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, A_c_buff_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buff_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, A_c_buff_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buff_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potri(*cusolverH, uplo, n, A_z_buff_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buff_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devInfo_buff_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potri(*cusolverH, uplo, n, A_z_buff_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
}
