#include "hip/hip_runtime.h"
// UNSUPPORTED: -windows-
// RUN: dpct -report-type=apis -report-file-prefix=check-apis-report -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: echo "// `perl -e 'print "CH","ECK"'`: API name, Frequency" >%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMemGetInfo(size_t * free,size_t * total),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipDeviceProp_t,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t,3" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipDeviceSynchronize(),4" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipFree(void * devPtr),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipHostFree(void * ptr),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipFuncGetAttributes(struct hipFuncAttributes * attr,reinterpret_cast<const void*>(void (*))(float *, const float *, const float *) entry),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMalloc(void ** devPtr,size_t size),2" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMalloc3D(struct hipPitchedPtr * pitchedDevPtr,struct hipExtent extent),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipHostMalloc(float ** ptr,size_t size,unsigned int flags),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMemcpy(void * dst,const void * src,size_t count,enum hipMemcpyKind kind),2" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMemset(void * devPtr,int value,size_t count),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipStream_t,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: dim3,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: float max(float a,float b),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: int2,3" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: longlong4,1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: struct hipChannelFormatDesc hipCreateChannelDesc(),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: struct hipExtent make_hipExtent(size_t w,size_t h,size_t d),2" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: struct hipPitchedPtr make_hipPitchedPtr(void * d,size_t p,size_t xsz,size_t ysz),1" >>%T/check-apis-report_csv_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: uint4,1" >>%T/check-apis-report_csv_check.txt
// RUN: cat %T/check-apis-report.apis.csv >>%T/check-apis-report_csv_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report_csv_check.txt %T/check-apis-report_csv_check.txt

// RUN: dpct -report-file-prefix=report -report-type=apis  -report-format=formatted -report-only  -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: echo "// `perl -e 'print "CH","ECK"'`: API name				Frequency" >%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMemGetInfo(size_t * free,size_t * total)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipDeviceProp_t                               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t                                  3" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipDeviceSynchronize()               4" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipFree(void * devPtr)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipHostFree(void * ptr)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipFuncGetAttributes(struct hipFuncAttributes * attr,reinterpret_cast<const void*>(void (*))(float *, const float *, const float *) entry)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMalloc(void ** devPtr,size_t size)               2" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMalloc3D(struct hipPitchedPtr * pitchedDevPtr,struct hipExtent extent)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipHostMalloc(float ** ptr,size_t size,unsigned int flags)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMemcpy(void * dst,const void * src,size_t count,enum hipMemcpyKind kind)               2" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipError_t hipMemset(void * devPtr,int value,size_t count)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: hipStream_t                                 1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: dim3                                         1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: float max(float a,float b)                   1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: int2                                         3" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: longlong4                                    1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: struct hipChannelFormatDesc hipCreateChannelDesc()               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: struct hipExtent make_hipExtent(size_t w,size_t h,size_t d)               2" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: struct hipPitchedPtr make_hipPitchedPtr(void * d,size_t p,size_t xsz,size_t ysz)               1" >>%T/check-apis-report_check.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: uint4                                        1" >>%T/check-apis-report_check.txt
// RUN: cat %T/report.apis.log >>%T/check-apis-report_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report_check.txt %T/check-apis-report_check.txt

// RUN: dpct -output-file=output-file.txt -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: echo "// `perl -e 'print "CH","ECK"'`: Processing: {{(.+)/([^/]+)}}" >%T/check_output-file.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: {{(.+)/([^/]+)}}:{{[0-9]+}}:{{[0-9]+}} warning: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code." >>%T/check_output-file.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`:   hipError_t err = hipDeviceSynchronize();" >>%T/check_output-file.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: {{(.+)/([^/]+)}}:{{[0-9]+}}:{{[0-9]+}} warning: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code." >>%T/check_output-file.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`:   checkError(hipDeviceSynchronize());" >>%T/check_output-file.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`: {{(.+)/([^/]+)}}:{{[0-9]+}}:{{[0-9]+}} warning: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code." >>%T/check_output-file.txt
// RUN: echo "// `perl -e 'print "CH","ECK"'`:   return hipDeviceSynchronize();" >>%T/check_output-file.txt
// RUN: cat %T/output-file.txt >>%T/check_output-file.txt
// RUN: FileCheck --match-full-lines --input-file %T/check_output-file.txt %T/check_output-file.txt

// NOMATCH-CHECK-NOT: '{{.}}'

// RUN: dpct -output-verbosity=silent  -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only  2>&1  \
// RUN: | FileCheck -check-prefix=NOMATCH-CHECK -allow-empty %s


// FAKE-FILE-CHECK-NOT:Processing: {{(.+)/([^/]+)}}
// FAKE-FILE-STDERR: Processing: {{(.+)/([^/]+)}}

// RUN: dpct -output-verbosity=normal  -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only  2>&1  \
// RUN: | FileCheck -check-prefix=FAKE-FILE-CHECK -allow-empty %s

// RUN: dpct -output-verbosity=detailed  -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only  2>&1  \
// RUN: | FileCheck -check-prefix=FAKE-FILE-STDERR -allow-empty %s

#include <hip/hip_runtime.h>

void checkError(hipError_t err) {
}

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  hipHostFree(d_A);

  hipHostMalloc(&src, bytes);

  struct hipPitchedPtr srcGPU;

  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  hipMalloc3D(&srcGPU, extent);

  int2 a;
  uint4 b;
  dim3 d3;
  hipDeviceProp_t cdp;
  hipStream_t cuSt;
  const int2 c = {0,0};
  int2 d[100];
  longlong4 ll4;
}

int cool() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  hipMemset(d_A, 0xf, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  free(h_A);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  checkError(hipDeviceSynchronize());
  return hipDeviceSynchronize();
}

static texture<float, 3, hipReadModeElementType>
    NoiseTextures[4]; // texture Array

void CreateTexture();

__global__ void
AccesTexture(texture<float, 3, hipReadModeElementType> *NoiseTextures) {
  int test = tex3D(NoiseTextures[0], threadIdx.x, threadIdx.y,
                   threadIdx.z); // by using this the error occurs
}

int main(int argc, char **argv) {
  AccesTexture<<<1, dim3(4, 4, 4)>>>(NoiseTextures);
}


template<typename T>
__global__ void addKernel(T *c, const T *a, const T *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

#define  SIZE_X 128 //numbers in elements
#define  SIZE_Y 128
#define  SIZE_Z 128
void bar(){
  typedef float  VolumeType;
  hipExtent volumeSize = make_hipExtent(SIZE_X, SIZE_Y, SIZE_Z);
  float d_volumeMem[100];
  hipMalloc((void**)&d_volumeMem[0], SIZE_X*SIZE_Y*SIZE_Z*sizeof(float));

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
  make_hipPitchedPtr((void*)d_volumeMem, SIZE_X*sizeof(VolumeType), SIZE_X, SIZE_Y);
  hipFuncAttributes attrib;
  hipError_t err;
  err = hipFuncGetAttributes(&attrib, reinterpret_cast<const void*>(addKernel<float>));
}

namespace libsvm {
extern "C"
void SVMTrain(void){
    float* d_value_inter;
    size_t free_mem, total;
    hipMemGetInfo(&free_mem, &total);
    int a = max(1, 3);
}
}
