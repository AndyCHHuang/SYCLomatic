#include "hip/hip_runtime.h"
//==============================================================
// Copyright � 2019 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "dpct/code_pin/code_pin.hpp"
#include "generated_schema.hpp"
#define VECTOR_SIZE 256

__global__ void VectorAddKernel(float* A, float* B, float* C)
{
    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

int main()
{
    float *d_A, *d_B, *d_C;
    hipError_t status;

    hipMalloc(&d_A, VECTOR_SIZE * sizeof(float));
    hipMalloc(&d_B, VECTOR_SIZE * sizeof(float));
    hipMalloc(&d_C, VECTOR_SIZE * sizeof(float));
    dpct::experimental::gen_prolog_API_CP("vectorAdd:vecotr.cu:[29]:", 0, TYPE_SHCEMA_005, (long *)&d_A, dpct::experimental::get_size_of_schema(TYPE_SHCEMA_005), TYPE_SHCEMA_006, (long *)&d_B, dpct::experimental::get_size_of_schema(TYPE_SHCEMA_006), TYPE_SHCEMA_007, (long *)&d_C, dpct::experimental::get_size_of_schema(TYPE_SHCEMA_007));
    VectorAddKernel<<<1, VECTOR_SIZE>>>(d_A, d_B, d_C);
    dpct::experimental::gen_epilog_API_CP("vectorAdd:vecotr.cu:[29]]:", 0, TYPE_SHCEMA_005, (long *)&d_A, dpct::experimental::get_size_of_schema(TYPE_SHCEMA_005), TYPE_SHCEMA_006, (long *)&d_B, dpct::experimental::get_size_of_schema(TYPE_SHCEMA_006), TYPE_SHCEMA_007, (long *)&d_C, dpct::experimental::get_size_of_schema(TYPE_SHCEMA_007));

    float Result[VECTOR_SIZE] = {};

    // dpct::experimental::gen_prolog_API_CP("hipMemcpy:vecotr.cu:[237]:", 0, TYPE_SHCEMA_004, (long *)&h_C, (size_t)size, TYPE_SHCEMA_007, (long *)&d_C, (size_t)size);
    status = hipMemcpy(Result, d_C, VECTOR_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    // dpct::experimental::gen_epilog_API_CP("hipMemcpy:vecotr.cu:[237]:", 0, TYPE_SHCEMA_004, (long *)&h_C, (size_t)size, TYPE_SHCEMA_007, (long *)&d_C, (size_t)size);

    if (status != hipSuccess) {
        printf("Could not copy result to host\n");
        exit(EXIT_FAILURE);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        if (i % 16 == 0) {
            printf("\n");
        }
        printf("%3.0f ", Result[i]);    
    }
    printf("\n");
	
    return 0;
}
