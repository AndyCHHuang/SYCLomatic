#include "hip/hip_runtime.h"
// RUN: dpct -keep-original-code -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/kernel-call-origcode-embedded.dp.cpp --match-full-lines %s

#include <iostream>
// includes CUDA
// CHECK:  /* DPCT_ORIG #include <hip/hip_runtime.h>*/
#include <hip/hip_runtime.h>

// CHECK:   /* DPCT_ORIG __global__ void testKernelPtr(const int *L, const int *M, int N) {*/
// CHECK-NEXT:void testKernelPtr(const int *L, const int *M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const int *L, const int *M, int N) {

  // CHECK: /* DPCT_ORIG   int gtid = blockIdx.x  * blockDim.x */
  // CHECK-NEXT:   int gtid = item_ct1.get_group(0) /*comments*/ * item_ct1.get_local_range().get(0) /*comments
  // CHECK-NEXT:  comments*/
  // CHECK-NEXT: /* DPCT_ORIG   + threadIdx.x;*/
  // CHECK-NEXT:  + item_ct1.get_local_id(0);
  int gtid = blockIdx.x /*comments*/ * blockDim.x /*comments
  comments*/
             + threadIdx.x;
}

// CHECK:     /* DPCT_ORIG __global__ void testKernel(int L, int M, int N) {*/
// CHECK-NEXT: void testKernel(int L, int M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernel(int L, int M, int N) {
  // CHECK:      /* DPCT_ORIG   int gtid = blockIdx.x*/
  // CHECK-NEXT:  int gtid = item_ct1.get_group(0)
  // CHECK-NEXT: /* DPCT_ORIG              * blockDim.x*/
  // CHECK-NEXT:                * item_ct1.get_local_range().get(0)
  // CHECK-NEXT: /* DPCT_ORIG              + threadIdx.x;*/
  // CHECK-NEXT:                + item_ct1.get_local_id(0);
  int gtid = blockIdx.x
             * blockDim.x
             + threadIdx.x;
}

// Error handling macro

// CHECK: #define CUDA_CHECK(call) \
// CHECK-NEXT:  /* DPCT_ORIG     if ((call) != hipSuccess) { \*/ \
// CHECK-NEXT:      if ((call) != 0) { \
// CHECK-NEXT:  /* DPCT_ORIG         hipError_t err = hipGetLastError(); \*/ \
// CHECK-NEXT:          int err = 0; \
// CHECK-NEXT:          std::cout << "CUDA error calling \"" #call "\", code is " << err << std::endl; \
// CHECK-NEXT:          exit(err); \
// CHECK-NEXT:       }
#define CUDA_CHECK(call)                                                           \
    if ((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        std::cout << "CUDA error calling \"" #call "\", code is " << err << std::endl; \
        exit(err); \
    }

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {}

int main() {
  // CHECK:  /* DPCT_ORIG   dim3 griddim = 2;*/
  // CHECK-NEXT:  cl::sycl::range<3> griddim = cl::sycl::range<3>(2, 1, 1);
  dim3 griddim = 2;

  // CHECK:  /* DPCT_ORIG   dim3 threaddim = 32;*/
  // CHECK-NEXT:   cl::sycl::range<3> threaddim = cl::sycl::range<3>(32, 1, 1);
  dim3 threaddim = 32;

  void *karg1 = 0;
  const int *karg2 = 0;
  int karg3 = 80;
  // CHECK:  /* DPCT_ORIG   testKernelPtr<<<griddim, threaddim>>>((const int *)karg1,
  // CHECK-NEXT:  karg2, karg3);*/
  // CHECK-NEXT: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset((const int *)karg1);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const int *arg_ct0 = (const int *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           const int *arg_ct1 = (const int *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr(arg_ct0, arg_ct1, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1,
                                        karg2, karg3);


  int karg1int = 1;
  int karg2int = 2;
  int karg3int = 3;
  int intvar = 20;
  // CHECK: /* DPCT_ORIG   testKernel<<<10, intvar>>>(karg1int, karg2int,
  // CHECK:  karg3int);*/
  // CHECK-NEXT: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(10, 1, 1) * cl::sycl::range<3>(intvar, 1, 1)), cl::sycl::range<3>(intvar, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<<<10, intvar>>>(karg1int, karg2int, // comments
                             // comments.
                             karg3int);

  // CHECK: /* DPCT_ORIG   testKernel<<<dim3(1), dim3(1, 2)>>>(karg1int,
  // CHECK:  karg2int,
  // CHECK:  karg3int);*/
  // CHECK-NEXT: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 2, 1)), cl::sycl::range<3>(1, 2, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<<<dim3(1), dim3(1, 2)>>>(karg1int,
                                      /* comments */
                                      karg2int, // comments
                                      /*
                                      comments
                                      */
                                      karg3int);

  // CHECK: /* DPCT_ORIG   testKernel<<<dim3(1, 2), dim3(1, 2, 3)>>>(karg1int,
  // CHECK-NEXT:  karg2int, karg3int); */
  // CHECK-NEXT: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 2, 1) * cl::sycl::range<3>(1, 2, 3)), cl::sycl::range<3>(1, 2, 3)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<<<dim3(1, 2), dim3(1, 2, 3)>>>(karg1int,
	  karg2int, /* comments */karg3int/* comments */); // comments

  // CHECK: /* DPCT_ORIG   testKernel<<<griddim.x, griddim.y + 2>>>(karg1int, karg2int, karg3int);*/
  // CHECK-NEXT: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(griddim[0], 1, 1) * cl::sycl::range<3>(griddim[1] + 2, 1, 1)), cl::sycl::range<3>(griddim[1] + 2, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<<<griddim.x, griddim.y + 2>>>(karg1int, karg2int, karg3int);

  float *deviceOutputData = NULL;

  // CHECK: /* DPCT_ORIG   CUDA_CHECK(hipMalloc((void **)&deviceOutputData, 10 * sizeof(float)));*/
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  CUDA_CHECK(hipMalloc((void **)&deviceOutputData, 10 * sizeof(float)));

  // copy result from device to host
  float *h_odata = NULL;
  float *d_odata = NULL;
  // CHECK: /* DPCT_ORIG   checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * 4, hipMemcpyDeviceToHost));*/
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * 4, hipMemcpyDeviceToHost));

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of this CUDA API is not supported by the oneAPI DPC++ Compatibility Tool.
  // CHECK-NEXT:*/
  hipDeviceGetCacheConfig(NULL);

  // CHECK: /* DPCT_ORIG   hipDeviceGetCacheConfig(NULL);hipMalloc((void **)&deviceOutputData, 10 * sizeof(float));*/
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1007:{{[0-9]+}}: Migration of this CUDA API is not supported by the oneAPI DPC++ Compatibility Tool.
  // CHECK-NEXT: */
  hipDeviceGetCacheConfig(NULL);hipMalloc((void **)&deviceOutputData, 10 * sizeof(float));

  // CHECK: /* DPCT_ORIG   hipEventCreate(NULL);checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * 4, hipMemcpyDeviceToHost));checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * 4, hipMemcpyDeviceToHost));*/
  // CHECK-NEXT:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  hipEventCreate(NULL);checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * 4, hipMemcpyDeviceToHost));checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * 4, hipMemcpyDeviceToHost));
}
