// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasIsamax_etc.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int k = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;

  const float *x_S = 0;
  const double *x_D = 0;
  const float *y_S = 0;
  const double *y_D = 0;
  int incx = 1;
  int incy = 1;
  int *result = 0;
  float *result_S = 0;
  double *result_D = 0;
  //level1
  //cublasI<t>amax
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamax(handle, n, x_S_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamax(handle, n, x_S_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamax(handle, n, x_D_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamax(handle, n, x_D_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIdamax(handle, n, x_D, incx, result);
  hipblasIdamax(handle, n, x_D, incx, result);

  //cublasI<t>amin
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamin(handle, n, x_S_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamin(handle, n, x_S_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIsamin(handle, n, x_S, incx, result);
  hipblasIsamin(handle, n, x_S, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: status = (mkl::blas::iamin(handle, n, x_D_buff_ct1, incx, result_temp_buffer), 0);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_buff_ct1 = dpct::get_buffer<int>(result);
  // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer(sycl::range<1>(1));
  // CHECK-NEXT: mkl::blas::iamin(handle, n, x_D_buff_ct1, incx, result_temp_buffer);
  // CHECK-NEXT: result_buff_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer.get_access<sycl::access::mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIdamin(handle, n, x_D, incx, result);
  hipblasIdamin(handle, n, x_D, incx, result);

  //cublas<t>asum
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::asum(handle, n, x_S_buff_ct1, incx, result_S_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::asum(handle, n, x_S_buff_ct1, incx, result_S_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasSasum(handle, n, x_S, incx, result_S);
  hipblasSasum(handle, n, x_S, incx, result_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::asum(handle, n, x_D_buff_ct1, incx, result_D_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::asum(handle, n, x_D_buff_ct1, incx, result_D_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDasum(handle, n, x_D, incx, result_D);
  hipblasDasum(handle, n, x_D, incx, result_D);

  //cublas<t>axpy
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::axpy(handle, n, *(&alpha_S), x_S_buff_ct1, incx, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::axpy(handle, n, *(&alpha_S), x_S_buff_ct1, incx, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);
  hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::axpy(handle, n, *(&alpha_D), x_D_buff_ct1, incx, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::axpy(handle, n, *(&alpha_D), x_D_buff_ct1, incx, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);
  hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);

  //cublas<t>copy
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::copy(handle, n, x_S_buff_ct1, incx, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::copy(handle, n, x_S_buff_ct1, incx, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasScopy(handle, n, x_S, incx, result_S, incy);
  hipblasScopy(handle, n, x_S, incx, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::copy(handle, n, x_D_buff_ct1, incx, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::copy(handle, n, x_D_buff_ct1, incx, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDcopy(handle, n, x_D, incx, result_D, incy);
  hipblasDcopy(handle, n, x_D, incx, result_D, incy);

  //cublas<t>dot
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::dot(handle, n, x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::dot(handle, n, x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);
  hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::dot(handle, n, x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::dot(handle, n, x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);
  hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);

  //cublas<t>nrm2
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::nrm2(handle, n, x_S_buff_ct1, incx, result_S_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::nrm2(handle, n, x_S_buff_ct1, incx, result_S_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasSnrm2(handle, n, x_S, incx, result_S);
  hipblasSnrm2(handle, n, x_S, incx, result_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::nrm2(handle, n, x_D_buff_ct1, incx, result_D_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::nrm2(handle, n, x_D_buff_ct1, incx, result_D_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDnrm2(handle, n, x_D, incx, result_D);
  hipblasDnrm2(handle, n, x_D, incx, result_D);

  float *x_f = 0;
  float *y_f = 0;
  double *x_d = 0;
  double *y_d = 0;
  //cublas<t>rot
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: status = (mkl::blas::rot(handle, n, x_f_buff_ct1, incx, y_f_buff_ct1, incy, *(x_S), *(y_S)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: mkl::blas::rot(handle, n, x_f_buff_ct1, incx, y_f_buff_ct1, incy, *(x_S), *(y_S));
  // CHECK-NEXT: }
  status = hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);
  hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: status = (mkl::blas::rot(handle, n, x_d_buff_ct1, incx, y_d_buff_ct1, incy, *(x_D), *(y_D)), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: mkl::blas::rot(handle, n, x_d_buff_ct1, incx, y_d_buff_ct1, incy, *(x_D), *(y_D));
  // CHECK-NEXT: }
  status = hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);
  hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);

  //cublas<t>rotg
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto x_f_buff_ct2 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct2 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: status = (mkl::blas::rotg(handle, x_f_buff_ct1, y_f_buff_ct1, x_f_buff_ct2, y_f_buff_ct2), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto x_f_buff_ct2 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct2 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: mkl::blas::rotg(handle, x_f_buff_ct1, y_f_buff_ct1, x_f_buff_ct2, y_f_buff_ct2);
  // CHECK-NEXT: }
  status = hipblasSrotg(handle, x_f, y_f, x_f, y_f);
  hipblasSrotg(handle, x_f, y_f, x_f, y_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto x_d_buff_ct2 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct2 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: status = (mkl::blas::rotg(handle, x_d_buff_ct1, y_d_buff_ct1, x_d_buff_ct2, y_d_buff_ct2), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto x_d_buff_ct2 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct2 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: mkl::blas::rotg(handle, x_d_buff_ct1, y_d_buff_ct1, x_d_buff_ct2, y_d_buff_ct2);
  // CHECK-NEXT: }
  status = hipblasDrotg(handle, x_d, y_d, x_d, y_d);
  hipblasDrotg(handle, x_d, y_d, x_d, y_d);

  //cublas<t>rotm
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: status = (mkl::blas::rotm(handle, n, x_f_buff_ct1, incx, y_f_buff_ct1, incy, x_S_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: mkl::blas::rotm(handle, n, x_f_buff_ct1, incx, y_f_buff_ct1, incy, x_S_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);
  hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: status = (mkl::blas::rotm(handle, n, x_d_buff_ct1, incx, y_d_buff_ct1, incy, x_D_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: mkl::blas::rotm(handle, n, x_d_buff_ct1, incx, y_d_buff_ct1, incy, x_D_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);
  hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);

  //cublas<t>rotmg
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto y_f_buff_ct2 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto y_f_buff_ct3 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: status = (mkl::blas::rotmg(handle, x_f_buff_ct1, y_f_buff_ct1, y_f_buff_ct2, *(x_S), y_f_buff_ct3), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto y_f_buff_ct2 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto y_f_buff_ct3 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: mkl::blas::rotmg(handle, x_f_buff_ct1, y_f_buff_ct1, y_f_buff_ct2, *(x_S), y_f_buff_ct3);
  // CHECK-NEXT: }
  status = hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);
  hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto y_d_buff_ct2 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto y_d_buff_ct3 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: status = (mkl::blas::rotmg(handle, x_d_buff_ct1, y_d_buff_ct1, y_d_buff_ct2, *(x_D), y_d_buff_ct3), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto y_d_buff_ct2 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto y_d_buff_ct3 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: mkl::blas::rotmg(handle, x_d_buff_ct1, y_d_buff_ct1, y_d_buff_ct2, *(x_D), y_d_buff_ct3);
  // CHECK-NEXT: }
  status = hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);
  hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);

  //cublas<t>scal
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: status = (mkl::blas::scal(handle, n, *(&alpha_S), x_f_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: mkl::blas::scal(handle, n, *(&alpha_S), x_f_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasSscal(handle, n, &alpha_S, x_f, incx);
  hipblasSscal(handle, n, &alpha_S, x_f, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: status = (mkl::blas::scal(handle, n, *(&alpha_D), x_d_buff_ct1, incx), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: mkl::blas::scal(handle, n, *(&alpha_D), x_d_buff_ct1, incx);
  // CHECK-NEXT: }
  status = hipblasDscal(handle, n, &alpha_D, x_d, incx);
  hipblasDscal(handle, n, &alpha_D, x_d, incx);

  //cublas<t>swap
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: status = (mkl::blas::swap(handle, n, x_f_buff_ct1, incx, y_f_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buff_ct1 = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buff_ct1 = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: mkl::blas::swap(handle, n, x_f_buff_ct1, incx, y_f_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSswap(handle, n, x_f, incx, y_f, incy);
  hipblasSswap(handle, n, x_f, incx, y_f, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: status = (mkl::blas::swap(handle, n, x_d_buff_ct1, incx, y_d_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buff_ct1 = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buff_ct1 = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: mkl::blas::swap(handle, n, x_d_buff_ct1, incx, y_d_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDswap(handle, n, x_d, incx, y_d, incy);
  hipblasDswap(handle, n, x_d, incx, y_d, incy);

  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  //level2
  //cublas<t>gbmv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans0;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::gbmv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, m, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::gbmv(handle, mkl::transpose::nontrans, m, n, m, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSgbmv(handle, (hipblasOperation_t)trans0, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans1;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::gbmv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, m, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::gbmv(handle, mkl::transpose::nontrans, m, n, m, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDgbmv(handle, (hipblasOperation_t)trans1, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>gemv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = trans2;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::gemv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::gemv(handle, mkl::transpose::nontrans, m, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSgemv(handle, (hipblasOperation_t)trans2, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct1 = 0;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::gemv(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), m, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::gemv(handle, mkl::transpose::nontrans, m, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDgemv(handle, (hipblasOperation_t)0, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>ger
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::ger(handle, m, n, *(&alpha_S), x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::ger(handle, m, n, *(&alpha_S), x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::ger(handle, m, n, *(&alpha_D), x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::ger(handle, m, n, *(&alpha_D), x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  int fill0 = 0;
  int fill1 = 1;
  //cublas<t>sbmv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::sbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::sbmv(handle, mkl::uplo::upper, m, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSsbmv(handle, (hipblasFillMode_t)fill0, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::sbmv(handle, (((int)fill1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::sbmv(handle, mkl::uplo::upper, m, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDsbmv(handle, (hipblasFillMode_t)fill1, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spmv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::spmv(handle, (((int)0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_S), x_S_buff_ct1, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::spmv(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_buff_ct1, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSspmv(handle, (hipblasFillMode_t)0, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);
  hipblasSspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::spmv(handle, (((int)1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_D), x_D_buff_ct1, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::spmv(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_buff_ct1, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDspmv(handle, (hipblasFillMode_t)1, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);
  hipblasDspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spr
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::spr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_S), x_S_buff_ct1, incx, result_S_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::spr(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_buff_ct1, incx, result_S_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasSspr(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, result_S);
  hipblasSspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::spr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_D), x_D_buff_ct1, incx, result_D_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::spr(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_buff_ct1, incx, result_D_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDspr(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, result_D);
  hipblasDspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D);

  //cublas<t>spr2
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::spr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_S), x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::spr2(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasSspr2(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, y_S, incy, result_S);
  hipblasSspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::spr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_D), x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::spr2(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1);
  // CHECK-NEXT: }
  status = hipblasDspr2(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, y_D, incy, result_D);
  hipblasDspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D);

  //cublas<t>symv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::symv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::symv(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_buff_ct1, lda, y_S_buff_ct1, incx, *(&beta_S), result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasSsymv(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::symv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::symv(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_buff_ct1, lda, y_D_buff_ct1, incx, *(&beta_D), result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDsymv(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>syr
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::syr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_S), x_S_buff_ct1, incx, result_S_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::syr(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_buff_ct1, incx, result_S_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasSsyr(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, result_S, lda);
  hipblasSsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::syr(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_D), x_D_buff_ct1, incx, result_D_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::syr(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_buff_ct1, incx, result_D_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasDsyr(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, result_D, lda);
  hipblasDsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D, lda);

  //cublas<t>syr2
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::syr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_S), x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buff_ct1 = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::syr2(handle, mkl::uplo::upper, n, *(&alpha_S), x_S_buff_ct1, incx, y_S_buff_ct1, incy, result_S_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasSsyr2(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::syr2(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), n, *(&alpha_D), x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1, lda), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buff_ct1 = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::syr2(handle, mkl::uplo::upper, n, *(&alpha_D), x_D_buff_ct1, incx, y_D_buff_ct1, incy, result_D_buff_ct1, lda);
  // CHECK-NEXT: }
  status = hipblasDsyr2(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  int diag0 = 0;
  int diag1 = 1;
  //cublas<t>tbmv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = 1;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::tbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::tbmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasStbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, n, x_S, lda, result_S, incy);
  hipblasStbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = 2;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::tbmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag1, n, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::tbmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, n, x_D, lda, result_D, incy);
  hipblasDtbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tbsv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::tbsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)0, n, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::tbsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasStbsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0, n, n, x_S, lda, result_S, incy);
  hipblasStbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::tbsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)1, n, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::tbsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDtbsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)1, n, n, x_D, lda, result_D, incy);
  hipblasDtbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tpmv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::tpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_S_buff_ct1, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::tpmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_buff_ct1, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasStpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, result_S, incy);
  hipblasStpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::tpmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_D_buff_ct1, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::tpmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_buff_ct1, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, result_D, incy);
  hipblasDtpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>tpsv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::tpsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_S_buff_ct1, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::tpsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_buff_ct1, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasStpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, result_S, incy);
  hipblasStpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::tpsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_D_buff_ct1, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::tpsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_buff_ct1, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, result_D, incy);
  hipblasDtpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>trmv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::trmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::trmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasStrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, lda, result_S, incy);
  hipblasStrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::trmv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::trmv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, lda, result_D, incy);
  hipblasDtrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //cublas<t>trsv
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = (mkl::blas::trsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buff_ct1 = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buff_ct1 = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: mkl::blas::trsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_S_buff_ct1, lda, result_S_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasStrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, lda, result_S, incy);
  hipblasStrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = (mkl::blas::trsv(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), (mkl::diag)diag0, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buff_ct1 = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buff_ct1 = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: mkl::blas::trsv(handle, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::unit, n, x_D_buff_ct1, lda, result_D_buff_ct1, incy);
  // CHECK-NEXT: }
  status = hipblasDtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, lda, result_D, incy);
  hipblasDtrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //level3
  int side0 = 0;
  int side1 = 1;
  // cublas<T>symm
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buff_ct1 = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: status = (mkl::blas::symm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, *(&alpha_S), A_S_buff_ct1, lda, B_S_buff_ct1, ldb, *(&beta_S), C_S_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buff_ct1 = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: mkl::blas::symm(handle, mkl::side::right, mkl::uplo::lower, m, n, *(&alpha_S), A_S_buff_ct1, lda, B_S_buff_ct1, ldb, *(&beta_S), C_S_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasSsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buff_ct1 = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: status = (mkl::blas::symm(handle, (mkl::side)side1, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), m, n, *(&alpha_D), A_D_buff_ct1, lda, B_D_buff_ct1, ldb, *(&beta_D), C_D_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buff_ct1 = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: mkl::blas::symm(handle, mkl::side::right, mkl::uplo::lower, m, n, *(&alpha_D), A_D_buff_ct1, lda, B_D_buff_ct1, ldb, *(&beta_D), C_D_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasDsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>syrk
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: status = (mkl::blas::syrk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(&alpha_S), A_S_buff_ct1, lda, *(&beta_S), C_S_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: mkl::blas::syrk(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_S), A_S_buff_ct1, lda, *(&beta_S), C_S_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasSsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);
  hipblasSsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: status = (mkl::blas::syrk(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(&alpha_D), A_D_buff_ct1, lda, *(&beta_D), C_D_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: mkl::blas::syrk(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_D), A_D_buff_ct1, lda, *(&beta_D), C_D_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasDsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);

  // cublas<T>syr2k
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buff_ct1 = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: status = (mkl::blas::syr2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(&alpha_S), A_S_buff_ct1, lda, B_S_buff_ct1, ldb, *(&beta_S), C_S_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buff_ct1 = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: mkl::blas::syr2k(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_S), A_S_buff_ct1, lda, B_S_buff_ct1, ldb, *(&beta_S), C_S_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasSsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct2 = trans0;
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buff_ct1 = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: status = (mkl::blas::syr2k(handle, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), n, k, *(&alpha_D), A_D_buff_ct1, lda, B_D_buff_ct1, ldb, *(&beta_D), C_D_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buff_ct1 = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: mkl::blas::syr2k(handle, mkl::uplo::upper, mkl::transpose::nontrans, n, k, *(&alpha_D), A_D_buff_ct1, lda, B_D_buff_ct1, ldb, *(&beta_D), C_D_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasDsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>trsm
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: status = (mkl::blas::trsm(handle, (mkl::side)0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m, n, *(&alpha_S), A_S_buff_ct1, lda, C_S_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_S_buff_ct1 = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto C_S_buff_ct1 = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: mkl::blas::trsm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, *(&alpha_S), A_S_buff_ct1, lda, C_S_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasStrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_S, A_S, lda, C_S, ldc);
  hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, C_S, ldc);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: status = (mkl::blas::trsm(handle, (mkl::side)1, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m, n, *(&alpha_D), A_D_buff_ct1, lda, C_D_buff_ct1, ldc), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_D_buff_ct1 = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto C_D_buff_ct1 = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: mkl::blas::trsm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m, n, *(&alpha_D), A_D_buff_ct1, lda, C_D_buff_ct1, ldc);
  // CHECK-NEXT: }
  status = hipblasDtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_D, A_D, lda, C_D, ldc);
  hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, C_D, ldc);

  return 0;
}
