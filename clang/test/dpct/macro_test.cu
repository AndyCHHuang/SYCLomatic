
#include <hip/hip_runtime.h>
// RUN: cat %s > %T/macro_test.cu
// RUN: cd %T
// RUN: dpct -out-root %T macro_test.cu --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/macro_test.dp.cpp --match-full-lines macro_test.cu

#define CUDA_NUM_THREADS 1024+32
#define GET_BLOCKS(n,t)  1+n+t-1
#define GET_BLOCKS2(n,t) 1+n+t
#define GET_BLOCKS3(n,t) n+t-1
#define GET_BLOCKS4(n,t) n+t

class DDD{
public:
  dim3* A;
  dim3 B;
};

#define EMPTY_MACRO(x) x
//CHECK:#define GET_MEMBER_MACRO(x) x[1] = 5
#define GET_MEMBER_MACRO(x) x.y = 5

__global__ void foo_kernel() {}

__global__ void foo2(){
  // CHECK: #define IMUL(a, b) sycl::mul24(a, b)
  // CHECK-NEXT: int vectorBase = IMUL(1, 2);
  #define IMUL(a, b) __mul24(a, b)
  int vectorBase = IMUL(1, 2);
}

__global__ void foo3(int x, int y) {}

void foo() {
  DDD d3;

  // CHECK: (*d3.A)[0] = 3;
  // CHECK-NEXT: d3.B[0] = 2;
  // CHECK-NEXT: EMPTY_MACRO(d3.B[0]);
  // CHECK-NEXT: GET_MEMBER_MACRO(d3.B);
  d3.A->x = 3;
  d3.B.x = 2;
  EMPTY_MACRO(d3.B.x);
  GET_MEMBER_MACRO(d3.B);

  int outputThreadCount = 512;

  // CHECK: dpct::get_default_queue().submit([&](sycl::handler &cgh) {
  // CHECK-NEXT:   cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS(outputThreadCount, outputThreadCount)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 2),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 2)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });
  // CHECK-NEXT: });
  foo_kernel<<<GET_BLOCKS(outputThreadCount, outputThreadCount), 2, 0>>>();

  // CHECK: dpct::get_default_queue().submit([&](sycl::handler &cgh) {
  // CHECK-NEXT:   cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS2(CUDA_NUM_THREADS, CUDA_NUM_THREADS)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 0),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 0)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });
  // CHECK-NEXT: });
  foo_kernel<<<GET_BLOCKS2(CUDA_NUM_THREADS, CUDA_NUM_THREADS), 0, 0>>>();

  // CHECK: dpct::get_default_queue().submit([&](sycl::handler &cgh) {
  // CHECK-NEXT:   cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS3(CUDA_NUM_THREADS, outputThreadCount)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 0),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 0)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });
  // CHECK-NEXT: });
  foo_kernel<<<GET_BLOCKS3(CUDA_NUM_THREADS, outputThreadCount), 0, 0>>>();

  // CHECK: dpct::get_default_queue().submit([&](sycl::handler &cgh) {
  // CHECK-NEXT:   cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS4(outputThreadCount, CUDA_NUM_THREADS)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 2),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 2)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });
  // CHECK-NEXT: });
  foo_kernel<<<GET_BLOCKS4(outputThreadCount, CUDA_NUM_THREADS), 2, 0>>>();

  // Test if SIGABRT.
  // No check here because the generated code need further fine tune.
  #define MACRO_CALL(a, b) foo_kernel<<<a, b, 0>>>();
  MACRO_CALL(0,0)

// CHECK: #define HANDLE_GPU_ERROR(err) \
// CHECK-NEXT: do \
// CHECK-NEXT: { \
// CHECK-NEXT:     if (err != 0) \
// CHECK-NEXT:     { \
// CHECK-NEXT:         int currentDevice; \
// CHECK-NEXT:         currentDevice = dpct::dev_mgr::instance().current_device_id(); \
// CHECK-NEXT:     } \
// CHECK-NEXT: } while (0)
#define HANDLE_GPU_ERROR(err) \
do \
{ \
    if(err != hipSuccess) \
    { \
        int currentDevice; \
        hipGetDevice(&currentDevice); \
    } \
} \
while(0)

HANDLE_GPU_ERROR(0);

// CHECK: #define cbrt(x) pow((double)x,(double)(1.0/3.0))
// CHECK-NEXT: double DD = sqrt(cbrt(5.9)) / sqrt(cbrt(3.2));
#define cbrt(x) pow((double)x,(double)(1.0/3.0))
  double DD = sqrt(cbrt(5.9)) / sqrt(cbrt(3.2));

// CHECK: #define NNBI(x) floor(x+0.5)
// CHECK-NEXT: NNBI(3.0);
#define NNBI(x) floor(x+0.5)
NNBI(3.0);

// CHECK: #define PI acos(-1)
#define PI acos(-1)
// CHECK: double cosine = cos(2 * PI);
double cosine = cos(2 * PI);

//CHECK: #define MACRO_KC                                                               \
//CHECK-NEXT:   dpct::get_default_queue().submit([&](sycl::handler &cgh) {                   \
//CHECK-NEXT:     cgh.parallel_for(                                                          \
//CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 2) * sycl::range<3>(1, 1, 2),   \
//CHECK-NEXT:                           sycl::range<3>(1, 1, 2)),                            \
//CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });                     \
//CHECK-NEXT:   });
#define MACRO_KC foo_kernel<<<2, 2, 0>>>();

//CHECK: MACRO_KC
MACRO_KC

// CHECK: #define HARD_KC(NAME)                                                          \
// CHECK-NEXT:   NAME<<<sycl::range<3>(2, 1, 1), sycl::range<3>(2, 1, 1), 0>>>();
#define HARD_KC(NAME) NAME<<<2,2,0>>>();
// CHECK: /*
// CHECK-NEXT: DPCT1038:0: Kernel calls with kernel function name in macro arguments are not
// CHECK-NEXT: supported. Try to rewrite this code using DPC++ kernel.
// CHECK-NEXT: */
//CHECK-NEXT: HARD_KC(foo_kernel)
HARD_KC(foo_kernel)


// CHECK: #define MACRO_KC2(a, b, c, d)                                                  \
// CHECK-NEXT:   dpct::get_default_queue().submit([&](sycl::handler &cgh) {                   \
// CHECK-NEXT:     auto dpct_global_range = a * b;                                            \
// CHECK-NEXT:                                                                                \
// CHECK-NEXT:     auto c_ct0 = c;                                                            \
// CHECK-NEXT:     auto d_ct1 = d;                                                            \
// CHECK-NEXT:                                                                                \
// CHECK-NEXT:     cgh.parallel_for(                                                          \
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(dpct_global_range.get(2),             \
// CHECK-NEXT:                                          dpct_global_range.get(1),             \
// CHECK-NEXT:                                          dpct_global_range.get(0)),            \
// CHECK-NEXT:                           sycl::range<3>(b.get(2), b.get(1), b.get(0))),       \
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) { foo3(c_ct0, d_ct1); });               \
// CHECK-NEXT:   });
#define MACRO_KC2(a,b,c,d) foo3<<<a, b, 0>>>(c,d);

dim3 griddim = 2;
dim3 threaddim = 32;

// CHECK: MACRO_KC2(griddim,threaddim,1,0)
MACRO_KC2(griddim,threaddim,1,0)

// [Note] Since 3 and 2 are migrated to sycl::range<3>, if they are used in macro as native numbers,
// there might be some issues in the migrated code.
// Since this is a corner case, not to emit warning message here.
// CHECK: MACRO_KC2(sycl::range<3>(3, 1, 1), sycl::range<3>(2, 1, 1), 1, 0)
MACRO_KC2(3,2,1,0)

// CHECK: MACRO_KC2(sycl::range<3>(5, 4, 3), sycl::range<3>(2, 1, 1), 1, 0)
MACRO_KC2(dim3(5,4,3),2,1,0)

}


