// RUN: dpct --cuda-include-path="%cuda-path/include" -out-root %T %s  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/check-cuda-include-path-option.dp.cpp --match-full-lines %s

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CHECK: void foo (int s){
void foo (hipblasStatus_t s){
}