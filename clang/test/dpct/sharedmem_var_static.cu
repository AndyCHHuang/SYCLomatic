// FIXME
// UNSUPPORTED: -windows-

// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sharedmem_var_static.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 64

class TestObject{
public:
  // CHECK: static void run(int *in, int *out, cl::sycl::nd_item<3> item_ct1, dpct::accessor<int, dpct::local, 0> a0) {
  __device__ static void run(int *in, int *out) {
    // CHECK:  // the size of s is static
    __shared__ int a0; // the size of s is static
    a0 = threadIdx.x;
  }
};

// CHECK: void nonTypeTemplateReverse(int *d, int n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::accessor<int, dpct::local, 1> s) {
// CHECK-NEXT:  // the size of s is dependent on parameter
template <int ArraySize>
__global__ void nonTypeTemplateReverse(int *d, int n) {
  __shared__ int s[2*ArraySize*ArraySize]; // the size of s is dependent on parameter
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
}

// CHECK: void staticReverse(int *d, int n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::accessor<int, dpct::local, 0> a0, dpct::accessor<int, dpct::local, 1> s) {
__global__ void staticReverse(int *d, int n) {
  const int size = 64;
  // CHECK:  // the size of s is static
  __shared__ int s[size]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
  // CHECK: TestObject::run(d, d, item_ct1, a0);
  TestObject::run(d, d);
}

// CHECK: template<typename TData>
// CHECK-NEXT: void templateReverse(TData *d, TData n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::accessor<TData, dpct::local, 2> s) {
template<typename TData>
__global__ void templateReverse(TData *d, TData n) {
  const int size = 32;
  // CHECK:  // the size of s is static
  __shared__ TData s[size * 2][size * 4]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t][0] = d[t];
  }
}

template <typename T>
void testTemplate() {
  const int n = 64;
  T a[n], r[n], d[n];
  T *d_d;
  int mem_size = n * sizeof(T);
  hipMalloc((void **)&d_d, mem_size);
  hipMemcpy(d_d, a, mem_size, hipMemcpyHostToDevice);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_d_buf_ct0 = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t d_d_offset_ct0 = d_d_buf_ct0.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::range<2> s_range_ct1(64/*size * 2*/, 128/*size * 4*/);
  // CHECK-NEXT:       cl::sycl::accessor<T, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, n), cl::sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           T *d_d_ct0 = (T *)(&d_d_acc_ct0[0] + d_d_offset_ct0);
  // CHECK-NEXT:           templateReverse<T>(d_d_ct0, n, item_ct1, dpct::accessor<T, dpct::local, 2>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<T><<<1, n>>>(d_d, n);
}

int main(void) {
  const int n = 64;
  int a[n], r[n], d[n];
  int *d_d;
  hipMalloc((void **)&d_d, n * sizeof(int));
  hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_d_buf_ct0 = dpct::get_buffer(d_d);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::range<0> a0_range_ct1;
  // CHECK-NEXT:       cl::sycl::accessor<int, 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> a0_acc_ct1(cgh);
  // CHECK-NEXT:       dpct::range<1> s_range_ct1(64/*size*/);
  // CHECK-NEXT:       cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class staticReverse_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, n), cl::sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           staticReverse((int *)(&d_d_acc_ct0[0]), n, item_ct1, dpct::accessor<int, dpct::local, 0>(a0_acc_ct1, a0_range_ct1), dpct::accessor<int, dpct::local, 1>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  staticReverse<<<1, n>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);

  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_d_buf_ct0 = dpct::get_buffer(d_d);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::range<2> s_range_ct1(64/*size * 2*/, 128/*size * 4*/);
  // CHECK-NEXT:       cl::sycl::accessor<int, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, int>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, n), cl::sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           templateReverse<int>((int *)(&d_d_acc_ct0[0]), n, item_ct1, dpct::accessor<int, dpct::local, 2>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<int><<<1, n>>>(d_d, n);

  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_d_buf_ct0 = dpct::get_buffer(d_d);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::range<1> s_range_ct1(2*SIZE*SIZE);
  // CHECK-NEXT:       cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class nonTypeTemplateReverse_{{[a-f0-9]+}}, dpct_kernel_scalar<SIZE>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, n), cl::sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           nonTypeTemplateReverse<SIZE>((int *)(&d_d_acc_ct0[0]), n, item_ct1, dpct::accessor<int, dpct::local, 1>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  nonTypeTemplateReverse<SIZE><<<1, n>>>(d_d, n);
}
