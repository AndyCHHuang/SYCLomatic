// FIXME
// UNSUPPORTED: -windows-

// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/sharedmem_var_static.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 64
// CHECK: void nonTypeTemplateReverse(int *d, int n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<int, dpct::local, 1> s) {
// CHECK-NEXT:  // the size of s is dependent on parameter
template <int ArraySize>
__global__ void nonTypeTemplateReverse(int *d, int n) {
  __shared__ int s[2*ArraySize*ArraySize]; // the size of s is dependent on parameter
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
}

// CHECK: void staticReverse(int *d, int n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<int, dpct::local, 1> s) {
__global__ void staticReverse(int *d, int n) {
  const int size = 64;
  // CHECK:  // the size of s is static
  __shared__ int s[size]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
}

// CHECK: template<typename TData>
// CHECK-NEXT: void templateReverse(TData *d, TData n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<TData, dpct::local, 2> s) {
template<typename TData>
__global__ void templateReverse(TData *d, TData n) {
  const int size = 32;
  // CHECK:  // the size of s is static
  __shared__ TData s[size * 2][size * 4]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t][0] = d[t];
  }
}

template <typename T>
void testTemplate() {
  const int n = 64;
  T a[n], r[n], d[n];
  T *d_d;
  int mem_size = n * sizeof(T);
  hipMalloc((void **)&d_d, mem_size);
  hipMemcpy(d_d, a, mem_size, hipMemcpyHostToDevice);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<2> s_range_ct1(64/*size * 2*/, 128/*size * 4*/);
  // CHECK-NEXT:       cl::sycl::accessor<TData, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1)), cl::sycl::range<3>(n, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           T *arg_ct0 = (T *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           templateReverse<T>(arg_ct0, n, item_ct1, dpct::dpct_accessor<T, dpct::local, 2>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<T><<<1, n>>>(d_d, n);
}

int main(void) {
  const int n = 64;
  int a[n], r[n], d[n];
  int *d_d;
  hipMalloc((void **)&d_d, n * sizeof(int));
  hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<1> s_range_ct1(64/*size*/);
  // CHECK-NEXT:       cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class staticReverse_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1)), cl::sycl::range<3>(n, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           int *arg_ct0 = (int *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           staticReverse(arg_ct0, n, item_ct1, dpct::dpct_accessor<int, dpct::local, 1>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  staticReverse<<<1, n>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<2> s_range_ct1(64/*size * 2*/, 128/*size * 4*/);
  // CHECK-NEXT:       cl::sycl::accessor<TData, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, int>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1)), cl::sycl::range<3>(n, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           int *arg_ct0 = (int *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           templateReverse<int>(arg_ct0, n, item_ct1, dpct::dpct_accessor<int, dpct::local, 2>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<int><<<1, n>>>(d_d, n);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<1> s_range_ct1(2*SIZE*SIZE);
  // CHECK-NEXT:       cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class nonTypeTemplateReverse_{{[a-f0-9]+}}, dpct_kernel_scalar<SIZE>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1)), cl::sycl::range<3>(n, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           int *arg_ct0 = (int *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           nonTypeTemplateReverse<SIZE>(arg_ct0, n, item_ct1, dpct::dpct_accessor<int, dpct::local, 1>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  nonTypeTemplateReverse<SIZE><<<1, n>>>(d_d, n);
}
