// FIXME
// UNSUPPORTED: -windows-

// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sharedmem_var_static.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 64

class TestObject{
public:
  // CHECK: static void run(int *in, int *out, cl::sycl::nd_item<3> item_ct1, dpct::dpct_accessor<int, dpct::local, 0> a0) {
  __device__ static void run(int *in, int *out) {
    // CHECK:  // the size of s is static
    __shared__ int a0; // the size of s is static
    a0 = threadIdx.x;
  }
};

// CHECK: void nonTypeTemplateReverse(int *d, int n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<int, dpct::local, 1> s) {
// CHECK-NEXT:  // the size of s is dependent on parameter
template <int ArraySize>
__global__ void nonTypeTemplateReverse(int *d, int n) {
  __shared__ int s[2*ArraySize*ArraySize]; // the size of s is dependent on parameter
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
}

// CHECK: void staticReverse(int *d, int n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<int, dpct::local, 0> a0, dpct::dpct_accessor<int, dpct::local, 1> s) {
__global__ void staticReverse(int *d, int n) {
  const int size = 64;
  // CHECK:  // the size of s is static
  __shared__ int s[size]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
  // CHECK: TestObject::run(d, d, item_ct1, a0);
  TestObject::run(d, d);
}

// CHECK: template<typename TData>
// CHECK-NEXT: void templateReverse(TData *d, TData n, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<TData, dpct::local, 2> s) {
template<typename TData>
__global__ void templateReverse(TData *d, TData n) {
  const int size = 32;
  // CHECK:  // the size of s is static
  __shared__ TData s[size * 2][size * 4]; // the size of s is static
  int t = threadIdx.x;
  if (t < 64) {
    s[t][0] = d[t];
  }
}

template <typename T>
void testTemplate() {
  const int n = 64;
  T a[n], r[n], d[n];
  T *d_d;
  int mem_size = n * sizeof(T);
  hipMalloc((void **)&d_d, mem_size);
  hipMemcpy(d_d, a, mem_size, hipMemcpyHostToDevice);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_d_buf_ct0 = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t d_d_offset_ct0 = d_d_buf_ct0.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<2> s_range_ct1(64/*size * 2*/, 128/*size * 4*/);
  // CHECK-NEXT:       cl::sycl::accessor<T, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           T *d_d_ct0 = (T *)(&d_d_acc_ct0[0] + d_d_offset_ct0);
  // CHECK-NEXT:           templateReverse<T>(d_d_ct0, n, item_ct1, dpct::dpct_accessor<T, dpct::local, 2>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<T><<<1, n>>>(d_d, n);
}

int main(void) {
  const int n = 64;
  int a[n], r[n], d[n];
  int *d_d;
  hipMalloc((void **)&d_d, n * sizeof(int));
  hipMemcpy(d_d, a, n * sizeof(int), hipMemcpyHostToDevice);
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_d_buf_ct0 = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t d_d_offset_ct0 = d_d_buf_ct0.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<0> a0_range_ct1;
  // CHECK-NEXT:       cl::sycl::accessor<int, 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> a0_acc_ct1(cgh);
  // CHECK-NEXT:       dpct::dpct_range<1> s_range_ct1(64/*size*/);
  // CHECK-NEXT:       cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class staticReverse_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           int *d_d_ct0 = (int *)(&d_d_acc_ct0[0] + d_d_offset_ct0);
  // CHECK-NEXT:           staticReverse(d_d_ct0, n, item_ct1, dpct::dpct_accessor<int, dpct::local, 0>(a0_acc_ct1, a0_range_ct1), dpct::dpct_accessor<int, dpct::local, 1>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  staticReverse<<<1, n>>>(d_d, n);
  hipMemcpy(d, d_d, n * sizeof(int), hipMemcpyDeviceToHost);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_d_buf_ct0 = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t d_d_offset_ct0 = d_d_buf_ct0.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<2> s_range_ct1(64/*size * 2*/, 128/*size * 4*/);
  // CHECK-NEXT:       cl::sycl::accessor<int, 2, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, int>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           int *d_d_ct0 = (int *)(&d_d_acc_ct0[0] + d_d_offset_ct0);
  // CHECK-NEXT:           templateReverse<int>(d_d_ct0, n, item_ct1, dpct::dpct_accessor<int, dpct::local, 2>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<int><<<1, n>>>(d_d, n);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_d_buf_ct0 = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t d_d_offset_ct0 = d_d_buf_ct0.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::dpct_range<1> s_range_ct1(2*SIZE*SIZE);
  // CHECK-NEXT:       cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s_acc_ct1(s_range_ct1, cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(n, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class nonTypeTemplateReverse_{{[a-f0-9]+}}, dpct_kernel_scalar<SIZE>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           int *d_d_ct0 = (int *)(&d_d_acc_ct0[0] + d_d_offset_ct0);
  // CHECK-NEXT:           nonTypeTemplateReverse<SIZE>(d_d_ct0, n, item_ct1, dpct::dpct_accessor<int, dpct::local, 1>(s_acc_ct1, s_range_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  nonTypeTemplateReverse<SIZE><<<1, n>>>(d_d, n);
}
