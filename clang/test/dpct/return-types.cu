// RUN: dpct --format-range=none  --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/return-types.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <stdio.h>
// CHECK-EMPTY:
// CHECK-NEXT: using queue_p = cl::sycl::queue *;

#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: #define DEF_BAR queue_p bar() { \
// CHECK-NEXT:   return 0; \
// CHECK-NEXT: }
#define DEF_BAR hipStream_t bar() { \
  return 0; \
}
// CHECK: #define DEF_BAR2 cl::sycl::event bar2() { \
// CHECK-NEXT:   return 0; \
// CHECK-NEXT: }
#define DEF_BAR2 hipEvent_t bar2() { \
  return 0; \
}

DEF_BAR
DEF_BAR2

// CHECK: template <typename T>
// CHECK-NEXT: queue_p bar() {
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
template <typename T>
hipStream_t bar() {
  return 0;
}

// CHECK: template <typename T>
// CHECK-NEXT: cl::sycl::event bar2() {
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
template <typename T>
hipEvent_t bar2() {
  return 0;
}

// CHECK: queue_p foo() {
hipStream_t foo() {
  return 0;
}

// CHECK: cl::sycl::event foo2() {
hipEvent_t foo2() {
  return 0;
}

class S {
  // CHECK: queue_p foo() {
  hipStream_t foo() {
    return 0;
  }

  // CHECK: cl::sycl::event foo2() {
  hipEvent_t foo2() {
    return 0;
  }
};

class C {
  // CHECK: queue_p foo() {
  hipStream_t foo() {
    return 0;
  }

  // CHECK: cl::sycl::event foo2() {
  hipEvent_t foo2() {
    return 0;
  }
};

// CHECK: queue_p *foo(int i) {
hipStream_t *foo(int i) {
  return 0;
}

// CHECK: const queue_p *foo(unsigned i) {
const hipStream_t *foo(unsigned i) {
  return 0;
}

// CHECK: queue_p **foo(char i) {
hipStream_t **foo(char i) {
  return 0;
}

// CHECK: queue_p &foo(short i) {
hipStream_t &foo(short i) {
  hipStream_t s;
  return s;
}

// CHECK: const queue_p &foo(long i) {
const hipStream_t &foo(long i) {
  hipStream_t s;
  return s;
}

// CHECK: cl::sycl::event *bar(int i) {
hipEvent_t *bar(int i) {
  return 0;
}

// CHECK: const cl::sycl::event *bar(unsigned i) {
const hipEvent_t *bar(unsigned i) {
  return 0;
}

// CHECK: cl::sycl::event **bar(char i) {
hipEvent_t **bar(char i) {
  return 0;
}

// CHECK: cl::sycl::event &bar(short i) {
hipEvent_t &bar(short i) {
  hipEvent_t e;
  return e;
}

// CHECK: const cl::sycl::event &bar(long i) {
const hipEvent_t &bar(long i) {
  hipEvent_t e;
  return e;
}
