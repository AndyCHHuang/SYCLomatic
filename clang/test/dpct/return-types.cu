// RUN: c2s --format-range=none --usm-level=none -out-root %T/return-types %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/return-types/return-types.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <c2s/c2s.hpp>
// CHECK-NEXT: #include <stdio.h>
// CHECK-EMPTY:

#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: #define DEF_BAR sycl::queue * bar() { \
// CHECK-NEXT:   return 0; \
// CHECK-NEXT: }
#define DEF_BAR hipStream_t bar() { \
  return 0; \
}
// CHECK: #define DEF_BAR2 sycl::event bar2() { \
// CHECK-NEXT:   return 0; \
// CHECK-NEXT: }
#define DEF_BAR2 hipEvent_t bar2() { \
  return 0; \
}

DEF_BAR
DEF_BAR2

// CHECK: template <typename T>
// CHECK-NEXT: sycl::queue * bar() {
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
template <typename T>
hipStream_t bar() {
  return 0;
}

// CHECK: template <typename T>
// CHECK-NEXT: sycl::event bar2() {
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
template <typename T>
hipEvent_t bar2() {
  return 0;
}

// CHECK: sycl::queue * foo() {
hipStream_t foo() {
  return 0;
}

// CHECK: sycl::event foo2() {
hipEvent_t foo2() {
  return 0;
}

class S {
  // CHECK: sycl::queue * foo() {
  hipStream_t foo() {
    return 0;
  }

  // CHECK: sycl::event foo2() {
  hipEvent_t foo2() {
    return 0;
  }
};

class C {
  // CHECK: sycl::queue * foo() {
  hipStream_t foo() {
    return 0;
  }

  // CHECK: sycl::event foo2() {
  hipEvent_t foo2() {
    return 0;
  }
};

// CHECK: sycl::queue * *foo(int i) {
hipStream_t *foo(int i) {
  return 0;
}

// CHECK: const sycl::queue * *foo(unsigned i) {
const hipStream_t *foo(unsigned i) {
  return 0;
}

// CHECK: sycl::queue * **foo(char i) {
hipStream_t **foo(char i) {
  return 0;
}

// CHECK: sycl::queue * &foo(short i) {
hipStream_t &foo(short i) {
  hipStream_t s;
  return s;
}

// CHECK: const sycl::queue * &foo(long i) {
const hipStream_t &foo(long i) {
  hipStream_t s;
  return s;
}

// CHECK: sycl::event *bar(int i) {
hipEvent_t *bar(int i) {
  return 0;
}

// CHECK: const sycl::event *bar(unsigned i) {
const hipEvent_t *bar(unsigned i) {
  return 0;
}

// CHECK: sycl::event **bar(char i) {
hipEvent_t **bar(char i) {
  return 0;
}

// CHECK: sycl::event &bar(short i) {
hipEvent_t &bar(short i) {
  hipEvent_t e;
  return e;
}

// CHECK: const sycl::event &bar(long i) {
const hipEvent_t &bar(long i) {
  hipEvent_t e;
  return e;
}

