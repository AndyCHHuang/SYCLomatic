// RUN: dpct --usm-level=none -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --match-full-lines --input-file %T/api-name-migrated.dp.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  // CHECK: free(d_A);
  hipHostFree(d_A);

  // CHECK: *(&src) = malloc(bytes);
  hipHostMalloc(&src, bytes, hipHostMallocDefault);

  struct hipPitchedPtr srcGPU;

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: make_cudaExtent: Migration of this API is not supported.
  // CHECK-NEXT:*/
  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: cudaMalloc3D: Migration of this API is not supported.
  // CHECK-NEXT:*/
  hipMalloc3D(&srcGPU, extent);
}
