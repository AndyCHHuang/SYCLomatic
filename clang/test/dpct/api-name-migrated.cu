// RUN: dpct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --match-full-lines --input-file %T/api-name-migrated.dp.cpp %s

#include <hip/hip_runtime.h>

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: cudaFreeHost: Migration of this API is not supported.
  // CHECK-NEXT:*/
  hipHostFree(d_A);

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: cudaMallocHost: Migration of this API is not supported.
  // CHECK-NEXT:*/
  hipHostMalloc(&src, bytes, hipHostMallocDefault);

  struct hipPitchedPtr srcGPU;

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: make_cudaExtent: Migration of this API is not supported.
  // CHECK-NEXT:*/
  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  // CHECK: /*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: cudaMalloc3D: Migration of this API is not supported.
  // CHECK-NEXT:*/
  hipMalloc3D(&srcGPU, extent);
}
