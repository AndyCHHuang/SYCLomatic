// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-restricted.dp.cpp %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipStream_t stream;

  /// malloc

  // CHECK: d_A = (float *)cl::sycl::malloc_device(size, dpct::get_current_device(), dpct::get_default_context());
  hipMalloc((void **)&d_A, size);
  // CHECK: errorCode = (d_A = (float *)cl::sycl::malloc_device(size, dpct::get_current_device(), dpct::get_default_context()), 0);
  errorCode = hipMalloc((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((d_A = (float *)cl::sycl::malloc_device(size, dpct::get_current_device(), dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_A, size));

  // CHECK: d_A = (float *)cl::sycl::malloc_device(sizeof(cl::sycl::double2) + size, dpct::get_current_device(), dpct::get_default_context());
  // CHECK-NEXT: d_A = (float *)cl::sycl::malloc_device(sizeof(cl::sycl::uchar4) + size, dpct::get_current_device(), dpct::get_default_context());
  // CHECK-NEXT: d_A = (float *)cl::sycl::malloc_device(sizeof(d_A[0]), dpct::get_current_device(), dpct::get_default_context());
  hipMalloc((void **)&d_A, sizeof(double2) + size);
  hipMalloc((void **)&d_A, sizeof(uchar4) + size);
  hipMalloc((void **)&d_A, sizeof(d_A[0]));

  // CHECK: h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context());
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context()), 0);
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: h_A = (float *)cl::sycl::malloc_host(sizeof(cl::sycl::double2) - size, dpct::get_default_context());
  // CHECK-NEXT: h_A = (float *)cl::sycl::malloc_host(sizeof(cl::sycl::uchar4) - size, dpct::get_default_context());
  hipHostAlloc((void **)&h_A, sizeof(double2) - size, hipHostMallocDefault);
  hipHostAlloc((void **)&h_A, sizeof(uchar4) - size, hipHostMallocDefault);

  // CHECK: h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context());
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context()), 0);
  errorCode = hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: h_A = (float *)cl::sycl::malloc_host(sizeof(cl::sycl::double2) * size, dpct::get_default_context());
  // CHECK-NEXT: h_A = (float *)cl::sycl::malloc_host(sizeof(cl::sycl::uchar4) * size, dpct::get_default_context());
  hipHostMalloc((void **)&h_A, sizeof(double2) * size, hipHostMallocDefault);
  hipHostMalloc((void **)&h_A, sizeof(uchar4) * size, hipHostMallocDefault);

  // CHECK: h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context());
  hipHostMalloc(&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context()), 0);
  errorCode = hipHostMalloc(&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)cl::sycl::malloc_host(size, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostMalloc(&h_A, size, hipHostMallocDefault));

  // CHECK: h_A = (float *)cl::sycl::malloc_host(sizeof(cl::sycl::double2) / size, dpct::get_default_context());
  // CHECK-NEXT: h_A = (float *)cl::sycl::malloc_host(sizeof(cl::sycl::uchar4) / size, dpct::get_default_context());
  hipHostMalloc(&h_A, sizeof(double2) / size, hipHostMallocDefault);
  hipHostMalloc(&h_A, sizeof(uchar4) / size, hipHostMallocDefault);

  // CHECK: d_A = (float *)cl::sycl::malloc_shared(size, dpct::get_current_device(), dpct::get_default_context());
  hipMallocManaged((void **)&d_A, size);
  // CHECK: errorCode = (d_A = (float *)cl::sycl::malloc_shared(size, dpct::get_current_device(), dpct::get_default_context()), 0);
  errorCode = hipMallocManaged((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((d_A = (float *)cl::sycl::malloc_shared(size, dpct::get_current_device(), dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipMallocManaged((void **)&d_A, size));

  // CHECK: d_A = (float *)cl::sycl::malloc_shared(sizeof(cl::sycl::double2) + size + sizeof(cl::sycl::uchar4), dpct::get_current_device(), dpct::get_default_context());
  // CHECK-NEXT: d_A = (float *)cl::sycl::malloc_shared(sizeof(cl::sycl::double2) * size * sizeof(cl::sycl::uchar4), dpct::get_current_device(), dpct::get_default_context());
  hipMallocManaged((void **)&d_A, sizeof(double2) + size + sizeof(uchar4));
  hipMallocManaged((void **)&d_A, sizeof(double2) * size * sizeof(uchar4));

  /// memcpy

  // CHECK: dpct::get_default_queue_wait().memcpy(d_A, h_A, size).wait();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode  = (dpct::get_default_queue_wait().memcpy(d_A, h_A, size).wait(), 0);
  errorCode  = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(d_A, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

#define SIZE 100
  // CHECK: dpct::get_default_queue_wait().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );

  /// memcpy async

  // CHECK: dpct::get_default_queue_wait().memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue_wait().memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));

  /// memcpy from symbol

  // CHECK: dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  /// memcpy from symbol async

  // CHECK: dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = (stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol

  // CHECK: dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  /// memcpy to symbol async

  // CHECK: dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));

  /// memset

  // CHECK: dpct::get_default_queue_wait().memset(d_A, 23, size).wait();
  hipMemset(d_A, 23, size);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memset(d_A, 23, size).wait(), 0);
  errorCode = hipMemset(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memset(d_A, 23, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemset(d_A, 23, size));

  /// memset async

  // CHECK: dpct::get_default_queue_wait().memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size));

  // CHECK: dpct::get_default_queue_wait().memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, 0));

  // CHECK: stream->memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: errorCode = (stream->memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, stream));

  // CHECK: cl::sycl::free(h_A, dpct::get_default_context());
  hipHostFree(h_A);
  // CHECK: errorCode = (cl::sycl::free(h_A, dpct::get_default_context()), 0);
  errorCode = hipHostFree(h_A);
  // CHECK: CUDA_SAFE_CALL((cl::sycl::free(h_A, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostFree(h_A));

  // CHECK: *(&d_A) = h_A;
  hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: errorCode = (*(&d_A) = h_A, 0);
  errorCode = hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: CUDA_SAFE_CALL((*(&d_A) = h_A, 0));
  CUDA_SAFE_CALL(hipHostGetDevicePointer(&d_A, h_A, 0));

  hipHostRegister(h_A, size, 0);
  // CHECK: errorCode = 0;
  errorCode = hipHostRegister(h_A, size, 0);
  // CHECK: CUDA_SAFE_CALL(0);
  CUDA_SAFE_CALL(hipHostRegister(h_A, size, 0));

  hipHostUnregister(h_A);
  // CHECK: errorCode = 0;
  errorCode = hipHostUnregister(h_A);
  // CHECK: CUDA_SAFE_CALL(0);
  CUDA_SAFE_CALL(hipHostUnregister(h_A));
}
