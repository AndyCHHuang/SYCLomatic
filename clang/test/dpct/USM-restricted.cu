#include "hip/hip_runtime.h"
// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-restricted.dp.cpp %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <memory>

#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)

__constant__ float constData[1234567 * 4];

int foo_b(int a){
  return 0;
}

void foo() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  hipStream_t stream;

  /// malloc
  // CHECK: d_A = (float *)sycl::malloc_device(size, q_ct1);
  hipMalloc((void **)&d_A, size);
  // CHECK: errorCode = (d_A = (float *)sycl::malloc_device(size, q_ct1), 0);
  errorCode = hipMalloc((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((d_A = (float *)sycl::malloc_device(size, q_ct1), 0));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_A, size));

  // CHECK: d_A = (float *)sycl::malloc_device(sizeof(sycl::double2) + size, q_ct1);
  // CHECK-NEXT: d_A = (float *)sycl::malloc_device(sizeof(sycl::uchar4) + size, q_ct1);
  // CHECK-NEXT: d_A = (float *)sycl::malloc_device(sizeof(d_A[0]), q_ct1);
  hipMalloc((void **)&d_A, sizeof(double2) + size);
  hipMalloc((void **)&d_A, sizeof(uchar4) + size);
  hipMalloc((void **)&d_A, sizeof(d_A[0]));
  
  // CHECK: dpct::dpct_malloc((void **)&d_A, &size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK: dpct::dpct_malloc(&p_A, e);
  hipMalloc3D(&p_A, e);

  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)sycl::malloc_host(size, q_ct1), 0);
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)sycl::malloc_host(size, q_ct1), 0));
  CUDA_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocDefault is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) - size, q_ct1);
  hipHostAlloc((void **)&h_A, sizeof(double2) - size, hipHostMallocDefault);
  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocDefault is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) - size, q_ct1);
  hipHostAlloc((void **)&h_A, sizeof(uchar4) - size, hipHostMallocDefault);

  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostMalloc((void **)&h_A, size);
  // CHECK: errorCode = (h_A = (float *)sycl::malloc_host(size, q_ct1), 0);
  errorCode = hipHostMalloc((void **)&h_A, size);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)sycl::malloc_host(size, q_ct1), 0));
  CUDA_SAFE_CALL(hipHostMalloc((void **)&h_A, size));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) * size, q_ct1);
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) * size, q_ct1);
  hipHostMalloc((void **)&h_A, sizeof(double2) * size);
  hipHostMalloc((void **)&h_A, sizeof(uchar4) * size);

  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostMalloc(&h_A, size);
  // CHECK: errorCode = (h_A = (float *)sycl::malloc_host(size, q_ct1), 0);
  errorCode = hipHostMalloc(&h_A, size);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)sycl::malloc_host(size, q_ct1), 0));
  CUDA_SAFE_CALL(hipHostMalloc(&h_A, size));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) / size, q_ct1);
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) / size, q_ct1);
  hipHostMalloc(&h_A, sizeof(double2) / size);
  hipHostMalloc(&h_A, sizeof(uchar4) / size);

  // CHECK: d_A = (float *)sycl::malloc_shared(size, q_ct1);
  hipMallocManaged((void **)&d_A, size);
  // CHECK: errorCode = (d_A = (float *)sycl::malloc_shared(size, q_ct1), 0);
  errorCode = hipMallocManaged((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((d_A = (float *)sycl::malloc_shared(size, q_ct1), 0));
  CUDA_SAFE_CALL(hipMallocManaged((void **)&d_A, size));

  // CHECK: d_A = (float *)sycl::malloc_shared(sizeof(sycl::double2) + size + sizeof(sycl::uchar4), q_ct1);
  // CHECK-NEXT: d_A = (float *)sycl::malloc_shared(sizeof(sycl::double2) * size * sizeof(sycl::uchar4), q_ct1);
  hipMallocManaged((void **)&d_A, sizeof(double2) + size + sizeof(uchar4));
  hipMallocManaged((void **)&d_A, sizeof(double2) * size * sizeof(uchar4));

  /// memcpy

  // CHECK: q_ct1.memcpy(d_A, h_A, size).wait();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode  = (q_ct1.memcpy(d_A, h_A, size).wait(), 0);
  errorCode  = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(d_A, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
#define MACRO_A(x) size
#define MACRO_A2(x) MACRO_A(x)
#define MACRO_B size
#define MACOR_C(x) hipMemcpyDeviceToHost
#define CUDA_SAFE_CALL2(x) CUDA_SAFE_CALL(x)
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((q_ct1.memcpy(d_A, h_A, size).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((q_ct1.memcpy(d_A, h_A, MACRO_B).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, MACRO_B, hipMemcpyDeviceToHost));
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((q_ct1.memcpy(d_A, h_A, MACRO_A2(1)).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, MACRO_A2(1), MACOR_C(1)));
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((q_ct1.memcpy(d_A, h_A, foo_b(1)).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, foo_b(1), MACOR_C(1)));

#define SIZE 100
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );

  /// memcpy async

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct1.memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (q_ct1.memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));

  // CHECK: dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3D(&parms);

  struct hipMemcpy3DParms *parms_pointer;
  // Followed call can't be processed.
  hipMemcpy3D(parms_pointer);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device, *stream);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host, *stream);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms, 0);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1, *stream);
  hipMemcpy3DAsync(&parms, stream);
  /// memcpy from symbol

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  /// memcpy from symbol async

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = (stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  /// memcpy to symbol async

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));

  /// memset

  // CHECK: q_ct1.memset(d_A, 23, size).wait();
  hipMemset(d_A, 23, size);
  // CHECK: errorCode = (q_ct1.memset(d_A, 23, size).wait(), 0);
  errorCode = hipMemset(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memset(d_A, 23, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemset(d_A, 23, size));

  /// memset async

  // CHECK: q_ct1.memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: errorCode = (q_ct1.memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size));

  // CHECK: q_ct1.memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: errorCode = (q_ct1.memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, 0));

  // CHECK: stream->memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: errorCode = (stream->memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, stream));
  
  // CHECK: dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK: dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);

  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size, *stream);
  hipMemset2DAsync(d_A, size, 0xf, size, size, stream);

  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e, 0);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e, *stream);
  hipMemset3DAsync(p_A, 0xf, e, stream);

  // CHECK: sycl::free(h_A, q_ct1);
  hipHostFree(h_A);
  // CHECK: errorCode = (sycl::free(h_A, q_ct1), 0);
  errorCode = hipHostFree(h_A);
  // CHECK: CUDA_SAFE_CALL((sycl::free(h_A, q_ct1), 0));
  CUDA_SAFE_CALL(hipHostFree(h_A));

  // CHECK: *(&d_A) = h_A;
  hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: errorCode = (*(&d_A) = h_A, 0);
  errorCode = hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: CUDA_SAFE_CALL((*(&d_A) = h_A, 0));
  CUDA_SAFE_CALL(hipHostGetDevicePointer(&d_A, h_A, 0));

  hipHostRegister(h_A, size, 0);
  // CHECK: errorCode = 0;
  errorCode = hipHostRegister(h_A, size, 0);
  // CHECK: CUDA_SAFE_CALL(0);
  CUDA_SAFE_CALL(hipHostRegister(h_A, size, 0));

  hipHostUnregister(h_A);
  // CHECK: errorCode = 0;
  errorCode = hipHostUnregister(h_A);
  // CHECK: CUDA_SAFE_CALL(0);
  CUDA_SAFE_CALL(hipHostUnregister(h_A));
}


template <typename T>
int foo2() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipStream_t stream;
  /// memcpy from symbol

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: q_ct1.memcpy(h_A, constData.get_ptr(), size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, constData.get_ptr(), size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, constData.get_ptr(), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size));

  /// memcpy from symbol async

  // CHECK: q_ct1.memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, constData.get_ptr(), size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, constData.get_ptr(), size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size));
  
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = (stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy(constData.get_ptr(), h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: errorCode = (q_ct1.memcpy(constData.get_ptr(), h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(constData.get_ptr(), h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size));

  /// memcpy to symbol async
  // CHECK: q_ct1.memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: errorCode = (q_ct1.memcpy(constData.get_ptr(), h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(constData.get_ptr(), h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));
}

template int foo2<float>();
template int foo2<int>();

void foo3() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;
  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  int *data;
  size_t width, height, depth, pitch, woffset, hoffset;
  hipArray_t a1;
  int deviceID = 0;

  // CHECK: auto s1 = std::make_shared<sycl::queue *>(&q_ct1);
  // CHECK: auto s2 = std::make_shared<sycl::queue *>(&q_ct1);
  // CHECK: auto s3 = std::make_shared<sycl::queue *>(&q_ct1);
  auto s1 = std::make_shared<hipStream_t>(hipStreamDefault);
  auto s2 = std::make_shared<hipStream_t>(hipStreamLegacy);
  auto s3 = std::make_shared<hipStream_t>(hipStreamPerThread);

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  // CHECK: errorCode = (q_ct1.memcpy(d_A, h_A, size), 0);
  // CHECK: errorCode = (q_ct1.memcpy(d_A, h_A, size), 0);
  // CHECK: errorCode = (q_ct1.memcpy(d_A, h_A, size), 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(d_A, h_A, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(d_A, h_A, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(d_A, h_A, size), 0));
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread));


  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  // CHECK: errorCode = (q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0));
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  hipMemcpy3DAsync(&parms, hipStreamDefault);
  hipMemcpy3DAsync(&parms, hipStreamLegacy);
  hipMemcpy3DAsync(&parms, hipStreamPerThread);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamDefault);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamLegacy);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  CUDA_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault));
  CUDA_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  CUDA_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  CUDA_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault));
  CUDA_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  CUDA_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: q_ct1.memset(d_A, 23, size);
  // CHECK: q_ct1.memset(d_A, 23, size);
  // CHECK: q_ct1.memset(d_A, 23, size);
  // CHECK: errorCode = (q_ct1.memset(d_A, 23, size), 0);
  // CHECK: errorCode = (q_ct1.memset(d_A, 23, size), 0);
  // CHECK: errorCode = (q_ct1.memset(d_A, 23, size), 0);
  // CHECK: CUDA_SAFE_CALL((q_ct1.memset(d_A, 23, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memset(d_A, 23, size), 0));
  // CHECK: CUDA_SAFE_CALL((q_ct1.memset(d_A, 23, size), 0));
  hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0));
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: errorCode = (dpct::async_dpct_memset(p_A, 0xf, e), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(p_A, 0xf, e), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(p_A, 0xf, e), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memset(p_A, 0xf, e), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memset(p_A, 0xf, e), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::async_dpct_memset(p_A, 0xf, e), 0));
  hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread));


  // CHECK: dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100);
  // CHECK: dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100);
  // CHECK: dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100);
  // CHECK: errorCode = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100), 0);
  // CHECK: errorCode = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100), 0);
  // CHECK: errorCode = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100), 0);
  // CHECK: CUDA_SAFE_CALL((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100), 0));
  // CHECK: CUDA_SAFE_CALL((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(d_A,100), 0));
  hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamDefault);
  hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamLegacy);
  hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamPerThread);
  errorCode = hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamDefault);
  errorCode = hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamLegacy);
  errorCode = hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamPerThread);
  CUDA_SAFE_CALL(hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamDefault));
  CUDA_SAFE_CALL(hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamLegacy));
  CUDA_SAFE_CALL(hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamPerThread));
}
