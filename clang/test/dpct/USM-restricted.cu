// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-restricted.dp.cpp %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipStream_t stream;

  /// malloc

  // CHECK: *((void **)&d_A) = cl::sycl::malloc_device(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  hipMalloc((void **)&d_A, size);
  // CHECK: errorCode = (*((void **)&d_A) = cl::sycl::malloc_device(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context()), 0);
  errorCode = hipMalloc((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((*((void **)&d_A) = cl::sycl::malloc_device(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context()), 0));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_A, size));

  // CHECK: *((void **)&d_A) = cl::sycl::malloc_device(sizeof(cl::sycl::double2) + size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  // CHECK-NEXT: *((void **)&d_A) = cl::sycl::malloc_device(sizeof(cl::sycl::uchar4) + size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  // CHECK-NEXT: *((void **)&d_A) = cl::sycl::malloc_device(sizeof(d_A[0]), dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  hipMalloc((void **)&d_A, sizeof(double2) + size);
  hipMalloc((void **)&d_A, sizeof(uchar4) + size);
  hipMalloc((void **)&d_A, sizeof(d_A[0]));

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context());
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (*((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context()), 0);
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((*((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context()), 0));
  CUDA_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(sizeof(cl::sycl::double2) - size, dpct::get_default_queue().get_context());
  // CHECK-NEXT: *((void **)&h_A) = cl::sycl::malloc_host(sizeof(cl::sycl::uchar4) - size, dpct::get_default_queue().get_context());
  hipHostAlloc((void **)&h_A, sizeof(double2) - size, hipHostMallocDefault);
  hipHostAlloc((void **)&h_A, sizeof(uchar4) - size, hipHostMallocDefault);

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context());
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (*((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context()), 0);
  errorCode = hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((*((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context()), 0));
  CUDA_SAFE_CALL(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(sizeof(cl::sycl::double2) * size, dpct::get_default_queue().get_context());
  // CHECK-NEXT: *((void **)&h_A) = cl::sycl::malloc_host(sizeof(cl::sycl::uchar4) * size, dpct::get_default_queue().get_context());
  hipHostMalloc((void **)&h_A, sizeof(double2) * size, hipHostMallocDefault);
  hipHostMalloc((void **)&h_A, sizeof(uchar4) * size, hipHostMallocDefault);

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context());
  hipHostMalloc(&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (*((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context()), 0);
  errorCode = hipHostMalloc(&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((*((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context()), 0));
  CUDA_SAFE_CALL(hipHostMalloc(&h_A, size, hipHostMallocDefault));

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(sizeof(cl::sycl::double2) / size, dpct::get_default_queue().get_context());
  // CHECK-NEXT: *((void **)&h_A) = cl::sycl::malloc_host(sizeof(cl::sycl::uchar4) / size, dpct::get_default_queue().get_context());
  hipHostMalloc(&h_A, sizeof(double2) / size, hipHostMallocDefault);
  hipHostMalloc(&h_A, sizeof(uchar4) / size, hipHostMallocDefault);

  // CHECK: *((void **)&d_A) = cl::sycl::malloc_shared(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  hipMallocManaged((void **)&d_A, size);
  // CHECK: errorCode = (*((void **)&d_A) = cl::sycl::malloc_shared(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context()), 0);
  errorCode = hipMallocManaged((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((*((void **)&d_A) = cl::sycl::malloc_shared(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context()), 0));
  CUDA_SAFE_CALL(hipMallocManaged((void **)&d_A, size));

  // CHECK: *((void **)&d_A) = cl::sycl::malloc_shared(sizeof(cl::sycl::double2) + size + sizeof(cl::sycl::uchar4), dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  // CHECK-NEXT: *((void **)&d_A) = cl::sycl::malloc_shared(sizeof(cl::sycl::double2) * size * sizeof(cl::sycl::uchar4), dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  hipMallocManaged((void **)&d_A, sizeof(double2) + size + sizeof(uchar4));
  hipMallocManaged((void **)&d_A, sizeof(double2) * size * sizeof(uchar4));

  /// memcpy

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size).wait();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode  = (dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size).wait(), 0);
  errorCode  = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

#define SIZE 100
  // CHECK: dpct::get_default_queue_wait().memcpy( (void*)(d_A), (void*)(h_A), sizeof(double)*SIZE*SIZE ).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );

  /// memcpy async

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(d_A), (void*)(h_A), size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((void*)(d_A), (void*)(h_A), size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((void*)(d_A), (void*)(h_A), size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((void*)(d_A), (void*)(h_A), size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));

  /// memcpy from symbol

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  /// memcpy from symbol async

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 2), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 2), size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 2), size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 3), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = (stream->memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 3), size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 3), size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol

  // CHECK: dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  /// memcpy to symbol async

  // CHECK: dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 2), (void*)(h_A), size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 2), (void*)(h_A), size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memcpy((void *)((char *)(constData.get_ptr()) + 2), (void*)(h_A), size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((void *)((char *)(constData.get_ptr()) + 3), (void*)(h_A), size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((void *)((char *)(constData.get_ptr()) + 3), (void*)(h_A), size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((void *)((char *)(constData.get_ptr()) + 3), (void*)(h_A), size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));

  /// memset

  // CHECK: dpct::get_default_queue_wait().memset((void*)(d_A), 23, size).wait();
  hipMemset(d_A, 23, size);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memset((void*)(d_A), 23, size).wait(), 0);
  errorCode = hipMemset(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memset((void*)(d_A), 23, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemset(d_A, 23, size));

  /// memset async

  // CHECK: dpct::get_default_queue_wait().memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memset((void*)(d_A), 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memset((void*)(d_A), 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size));

  // CHECK: dpct::get_default_queue_wait().memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: errorCode = (dpct::get_default_queue_wait().memset((void*)(d_A), 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue_wait().memset((void*)(d_A), 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, 0));

  // CHECK: stream->memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: errorCode = (stream->memset((void*)(d_A), 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memset((void*)(d_A), 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, stream));

  // CHECK: cl::sycl::free(h_A, dpct::get_default_queue().get_context());
  hipHostFree(h_A);
  // CHECK: errorCode = (cl::sycl::free(h_A, dpct::get_default_queue().get_context()), 0);
  errorCode = hipHostFree(h_A);
  // CHECK: CUDA_SAFE_CALL((cl::sycl::free(h_A, dpct::get_default_queue().get_context()), 0));
  CUDA_SAFE_CALL(hipHostFree(h_A));

  // CHECK: *(&d_A) = h_A;
  hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: errorCode = (*(&d_A) = h_A, 0);
  errorCode = hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: CUDA_SAFE_CALL((*(&d_A) = h_A, 0));
  CUDA_SAFE_CALL(hipHostGetDevicePointer(&d_A, h_A, 0));

  hipHostRegister(h_A, size, 0);
  // CHECK: errorCode = (0, 0);
  errorCode = hipHostRegister(h_A, size, 0);
  // CHECK: CUDA_SAFE_CALL((0, 0));
  CUDA_SAFE_CALL(hipHostRegister(h_A, size, 0));

  hipHostUnregister(h_A);
  // CHECK: errorCode = (0, 0);
  errorCode = hipHostUnregister(h_A);
  // CHECK: CUDA_SAFE_CALL((0, 0));
  CUDA_SAFE_CALL(hipHostUnregister(h_A));
}
