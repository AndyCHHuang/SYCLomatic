// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-restricted.dp.cpp %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)

__constant__ float constData[1234567 * 4];

int foo_b(int a){
  return 0;
}

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  hipStream_t stream;

  /// malloc
  // CHECK: d_A = (float *)sycl::malloc_device(size, dpct::get_current_device(), dpct::get_default_context());
  hipMalloc((void **)&d_A, size);
  // CHECK: errorCode = (d_A = (float *)sycl::malloc_device(size, dpct::get_current_device(), dpct::get_default_context()), 0);
  errorCode = hipMalloc((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((d_A = (float *)sycl::malloc_device(size, dpct::get_current_device(), dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_A, size));

  // CHECK: d_A = (float *)sycl::malloc_device(sizeof(sycl::double2) + size, dpct::get_current_device(), dpct::get_default_context());
  // CHECK-NEXT: d_A = (float *)sycl::malloc_device(sizeof(sycl::uchar4) + size, dpct::get_current_device(), dpct::get_default_context());
  // CHECK-NEXT: d_A = (float *)sycl::malloc_device(sizeof(d_A[0]), dpct::get_current_device(), dpct::get_default_context());
  hipMalloc((void **)&d_A, sizeof(double2) + size);
  hipMalloc((void **)&d_A, sizeof(uchar4) + size);
  hipMalloc((void **)&d_A, sizeof(d_A[0]));
  
  // CHECK: dpct::dpct_malloc((void **)&d_A, &size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK: dpct::dpct_malloc(&p_A, e);
  hipMalloc3D(&p_A, e);

  // CHECK: h_A = (float *)sycl::malloc_host(size, dpct::get_default_context());
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)sycl::malloc_host(size, dpct::get_default_context()), 0);
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)sycl::malloc_host(size, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) - size, dpct::get_default_context());
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) - size, dpct::get_default_context());
  hipHostAlloc((void **)&h_A, sizeof(double2) - size, hipHostMallocDefault);
  hipHostAlloc((void **)&h_A, sizeof(uchar4) - size, hipHostMallocDefault);

  // CHECK: h_A = (float *)sycl::malloc_host(size, dpct::get_default_context());
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)sycl::malloc_host(size, dpct::get_default_context()), 0);
  errorCode = hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)sycl::malloc_host(size, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) * size, dpct::get_default_context());
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) * size, dpct::get_default_context());
  hipHostMalloc((void **)&h_A, sizeof(double2) * size, hipHostMallocDefault);
  hipHostMalloc((void **)&h_A, sizeof(uchar4) * size, hipHostMallocDefault);

  // CHECK: h_A = (float *)sycl::malloc_host(size, dpct::get_default_context());
  hipHostMalloc(&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = (h_A = (float *)sycl::malloc_host(size, dpct::get_default_context()), 0);
  errorCode = hipHostMalloc(&h_A, size, hipHostMallocDefault);
  // CHECK: CUDA_SAFE_CALL((h_A = (float *)sycl::malloc_host(size, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostMalloc(&h_A, size, hipHostMallocDefault));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) / size, dpct::get_default_context());
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) / size, dpct::get_default_context());
  hipHostMalloc(&h_A, sizeof(double2) / size, hipHostMallocDefault);
  hipHostMalloc(&h_A, sizeof(uchar4) / size, hipHostMallocDefault);

  // CHECK: d_A = (float *)sycl::malloc_shared(size, dpct::get_current_device(), dpct::get_default_context());
  hipMallocManaged((void **)&d_A, size);
  // CHECK: errorCode = (d_A = (float *)sycl::malloc_shared(size, dpct::get_current_device(), dpct::get_default_context()), 0);
  errorCode = hipMallocManaged((void **)&d_A, size);
  // CHECK: CUDA_SAFE_CALL((d_A = (float *)sycl::malloc_shared(size, dpct::get_current_device(), dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipMallocManaged((void **)&d_A, size));

  // CHECK: d_A = (float *)sycl::malloc_shared(sizeof(sycl::double2) + size + sizeof(sycl::uchar4), dpct::get_current_device(), dpct::get_default_context());
  // CHECK-NEXT: d_A = (float *)sycl::malloc_shared(sizeof(sycl::double2) * size * sizeof(sycl::uchar4), dpct::get_current_device(), dpct::get_default_context());
  hipMallocManaged((void **)&d_A, sizeof(double2) + size + sizeof(uchar4));
  hipMallocManaged((void **)&d_A, sizeof(double2) * size * sizeof(uchar4));

  /// memcpy

  // CHECK: sycl::queue& q_ct0 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct0.wait();
  // CHECK-NEXT: q_ct0.memcpy(d_A, h_A, size).wait();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode  = (q_ct0.memcpy(d_A, h_A, size).wait(), 0);
  errorCode  = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(d_A, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
#define MACRO_A(x) size
#define MACRO_A2(x) MACRO_A(x)
#define MACRO_B size
#define MACOR_C(x) hipMemcpyDeviceToHost
#define CUDA_SAFE_CALL2(x) CUDA_SAFE_CALL(x)
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((dpct::get_default_queue().memcpy(d_A, h_A, size).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((dpct::get_default_queue().memcpy(d_A, h_A, MACRO_B).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, MACRO_B, hipMemcpyDeviceToHost));
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((dpct::get_default_queue().memcpy(d_A, h_A, MACRO_A2(1)).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, MACRO_A2(1), MACOR_C(1)));
  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: CUDA_SAFE_CALL2((dpct::get_default_queue().memcpy(d_A, h_A, foo_b(1)).wait(), 0));
  CUDA_SAFE_CALL2(hipMemcpy(d_A, h_A, foo_b(1), MACOR_C(1)));

#define SIZE 100
  // CHECK: dpct::get_default_queue().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );

  /// memcpy async

  // CHECK: dpct::get_default_queue().memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue().memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy(d_A, h_A, size), 0);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(d_A, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));

  // CHECK: dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3D(&parms);

  struct hipMemcpy3DParms *parms_pointer;
  // Followed call can't be processed.
  hipMemcpy3D(parms_pointer);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device, *stream);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host, *stream);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms, 0);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1, *stream);
  hipMemcpy3DAsync(&parms, stream);
  /// memcpy from symbol

  // CHECK: sycl::queue& q_ct1 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct1.wait();
  // CHECK-NEXT: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: sycl::queue& q_ct2 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct2.wait();
  // CHECK-NEXT: q_ct2.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (q_ct2.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  /// memcpy from symbol async

  // CHECK: dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = (stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol
  // CHECK: sycl::queue& q_ct3 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct3.wait();
  // CHECK-NEXT: q_ct3.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (q_ct3.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: sycl::queue& q_ct4 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct4.wait();
  // CHECK-NEXT: q_ct4.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct4.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  /// memcpy to symbol async

  // CHECK: dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));

  /// memset

  // CHECK: sycl::queue& q_ct5 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct5.wait();
  // CHECK-NEXT: q_ct5.memset(d_A, 23, size).wait();
  hipMemset(d_A, 23, size);
  // CHECK: errorCode = (q_ct5.memset(d_A, 23, size).wait(), 0);
  errorCode = hipMemset(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memset(d_A, 23, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemset(d_A, 23, size));

  /// memset async

  // CHECK: dpct::get_default_queue().memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: errorCode = (dpct::get_default_queue().memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size));

  // CHECK: dpct::get_default_queue().memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: errorCode = (dpct::get_default_queue().memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, 0));

  // CHECK: stream->memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: errorCode = (stream->memset(d_A, 23, size), 0);
  errorCode = hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memset(d_A, 23, size), 0));
  CUDA_SAFE_CALL(hipMemsetAsync(d_A, 23, size, stream));
  
  // CHECK: dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK: dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);

  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size, *stream);
  hipMemset2DAsync(d_A, size, 0xf, size, size, stream);

  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e, 0);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e, *stream);
  hipMemset3DAsync(p_A, 0xf, e, stream);

  // CHECK: sycl::free(h_A, dpct::get_default_context());
  hipHostFree(h_A);
  // CHECK: errorCode = (sycl::free(h_A, dpct::get_default_context()), 0);
  errorCode = hipHostFree(h_A);
  // CHECK: CUDA_SAFE_CALL((sycl::free(h_A, dpct::get_default_context()), 0));
  CUDA_SAFE_CALL(hipHostFree(h_A));

  // CHECK: *(&d_A) = h_A;
  hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: errorCode = (*(&d_A) = h_A, 0);
  errorCode = hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: CUDA_SAFE_CALL((*(&d_A) = h_A, 0));
  CUDA_SAFE_CALL(hipHostGetDevicePointer(&d_A, h_A, 0));

  hipHostRegister(h_A, size, 0);
  // CHECK: errorCode = 0;
  errorCode = hipHostRegister(h_A, size, 0);
  // CHECK: CUDA_SAFE_CALL(0);
  CUDA_SAFE_CALL(hipHostRegister(h_A, size, 0));

  hipHostUnregister(h_A);
  // CHECK: errorCode = 0;
  errorCode = hipHostUnregister(h_A);
  // CHECK: CUDA_SAFE_CALL(0);
  CUDA_SAFE_CALL(hipHostUnregister(h_A));
}


template <typename T>
int foo2() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipStream_t stream;
  /// memcpy from symbol

  // CHECK: sycl::queue& q_ct6 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct6.wait();
  // CHECK-NEXT: q_ct6.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (q_ct6.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: sycl::queue& q_ct7 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct7.wait();
  // CHECK-NEXT: q_ct7.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (q_ct7.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: sycl::queue& q_ct8 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct8.wait();
  // CHECK-NEXT: q_ct8.memcpy(h_A, constData.get_ptr(), size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: errorCode = (q_ct8.memcpy(h_A, constData.get_ptr(), size).wait(), 0);
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, constData.get_ptr(), size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size));

  /// memcpy from symbol async

  // CHECK: dpct::get_default_queue().memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(h_A, constData.get_ptr(), size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, constData.get_ptr(), size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size));
  
  // CHECK: dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 1, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(h_A, (char *)(constData.get_ptr()) + 2, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = (stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy(h_A, (char *)(constData.get_ptr()) + 3, size), 0));
  CUDA_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol
  // CHECK: sycl::queue& q_ct9 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct9.wait();
  // CHECK-NEXT: q_ct9.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (q_ct9.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: sycl::queue& q_ct10 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct10.wait();
  // CHECK-NEXT: q_ct10.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (q_ct10.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: sycl::queue& q_ct11 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct11.wait();
  // CHECK-NEXT: q_ct11.memcpy(constData.get_ptr(), h_A, size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: errorCode = (q_ct11.memcpy(constData.get_ptr(), h_A, size).wait(), 0);
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(constData.get_ptr(), h_A, size).wait(), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size));

  /// memcpy to symbol async
  // CHECK: dpct::get_default_queue().memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy(constData.get_ptr(), h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy(constData.get_ptr(), h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size));

  // CHECK: dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = (dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: CUDA_SAFE_CALL((dpct::get_default_queue().memcpy((char *)(constData.get_ptr()) + 2, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = (stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size), 0);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: CUDA_SAFE_CALL((stream->memcpy((char *)(constData.get_ptr()) + 3, h_A, size), 0));
  CUDA_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));
}

template int foo2<float>();
template int foo2<int>();
