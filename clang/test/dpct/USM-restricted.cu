// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=restricted -out-root %T %s -- -std=c++14 -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --match-full-lines --input-file %T/USM-restricted.dp.cpp %s

#include <hip/hip_runtime.h>

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  hipStream_t stream;

  // CHECK: *((void **)&d_A) = cl::sycl::malloc_device(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  hipMalloc((void **)&d_A, size);

  /// memcpy
  // CHECK: dpct::get_default_queue().memcpy((void*)(d_A), (void*)(h_A), size).wait();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  /// memcpy async
  // CHECK: dpct::get_default_queue().memcpy((void*)(d_A), (void*)(h_A), size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::get_default_queue().memcpy((void*)(d_A), (void*)(h_A), size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: stream.memcpy((void*)(d_A), (void*)(h_A), size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  /// memcpy from symbol
  // CHECK: dpct::get_default_queue().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: dpct::get_default_queue().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: dpct::get_default_queue().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::get_default_queue().memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 2), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: stream.memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 3), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: dpct::get_default_queue().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: dpct::get_default_queue().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size).wait();
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: dpct::get_default_queue().memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::get_default_queue().memcpy((void *)((char *)(constData.get_ptr()) + 2), (void*)(h_A), size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: stream.memcpy((void *)((char *)(constData.get_ptr()) + 3), (void*)(h_A), size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  /// memset
  // CHECK: dpct::get_default_queue().memset((void*)(d_A), 23, size).wait();
  hipMemset(d_A, 23, size);

  /// memset async
  // CHECK: dpct::get_default_queue().memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: dpct::get_default_queue().memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: stream.memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size, stream);

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context());
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);

  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context());
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: *((void **)&h_A) = cl::sycl::malloc_host(size, dpct::get_default_queue().get_context());
  hipHostMalloc(&h_A, size, hipHostMallocDefault);

  // CHECK: *((void **)&d_A) = cl::sycl::malloc_shared(size, dpct::get_device_manager().current_device(), dpct::get_default_queue().get_context());
  hipMallocManaged((void **)&d_A, size);

  // CHECK: cl::sycl::free(h_A, dpct::get_default_queue().get_context());
  hipHostFree(h_A);

  // CHECK: *(&d_A) = h_A;
  hipHostGetDevicePointer(&d_A, h_A, 0);

  hipHostRegister(h_A, size, 0);
  hipHostUnregister(h_A);
}
