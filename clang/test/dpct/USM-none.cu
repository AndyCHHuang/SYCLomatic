// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=none -out-root %T %s -- -std=c++14 -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --match-full-lines --input-file %T/USM-none.dp.cpp %s

#include <hip/hip_runtime.h>

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;

  // CHECK: dpct::dpct_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);

  /// memcpy
  // CHECK: dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  /// memcpy async
  // CHECK: dpct::async_dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device, stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  /// memcpy from symbol
  // CHECK: dpct::dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: dpct::dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 2), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 3), size, dpct::device_to_host, stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: dpct::async_dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((void *)((char *)(constData.get_ptr()) + 2), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((void *)((char *)(constData.get_ptr()) + 3), (void*)(h_A), size, dpct::host_to_device, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  /// memset
  // CHECK: dpct::dpct_memset((void*)(d_A), 23, size);
  hipMemset(d_A, 23, size);

  /// memset async
  // CHECK: dpct::async_dpct_memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: dpct::async_dpct_memset((void*)(d_A), 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: dpct::async_dpct_memset((void*)(d_A), 23, size, stream);
  hipMemsetAsync(d_A, 23, size, stream);

  // CHECK: *((void **)&h_A) = malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: *((void **)&h_A) = malloc(size);
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipMallocManaged((void **)&d_A, size);

  // CHECK: free(h_A);
  hipHostFree(h_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipHostGetDevicePointer(&d_A, h_A, 0);

  hipHostRegister(h_A, size, 0);
  hipHostUnregister(h_A);
}
