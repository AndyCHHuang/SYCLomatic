// FIXME
// UNSUPPORTED: -windows-
// RUN: c2s --format-range=none --usm-level=none -out-root %T/USM-none %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-none/USM-none.dp.cpp %s

// CHECK: #define C2S_USM_LEVEL_NONE
// CHECK-NEXT: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <c2s/c2s.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <memory>

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;

  // CHECK: d_A = (float *)c2s::c2s_malloc(size);
  hipMalloc((void **)&d_A, size);

  /// memcpy
  // CHECK: c2s::c2s_memcpy(d_A, h_A, size, c2s::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  /// memcpy async
  // CHECK: c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device, *stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  /// memcpy from symbol
  // CHECK: c2s::c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: c2s::c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: c2s::c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, c2s::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, c2s::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, c2s::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToHost);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, c2s::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, c2s::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: c2s::c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: c2s::c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: c2s::c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, c2s::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  /// memset
  // CHECK: c2s::c2s_memset(d_A, 23, size);
  hipMemset(d_A, 23, size);

  /// memset async
  // CHECK: c2s::async_c2s_memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: c2s::async_c2s_memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: c2s::async_c2s_memset(d_A, 23, size, *stream);
  hipMemsetAsync(d_A, 23, size, stream);

  // CHECK: h_A = (float *)malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: h_A = (float *)malloc(size);
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: d_A = (float *)c2s::c2s_malloc(size);
  hipMallocManaged((void **)&d_A, size);

  // CHECK: h_A = (float *)malloc(size);
  hipMemAllocHost((void **)&h_A, size);

  hipDeviceptr_t* D_ptr;
  // CHECK: *D_ptr = c2s::c2s_malloc(size);
  hipMallocManaged(D_ptr, size, hipMemAttachHost);

  float* buffer[2];
#define SIZE_1 (128 * 1024 * 1024)
  // CHECK: *buffer = (float *)malloc(SIZE_1 * sizeof(float));
  // CHECK-NEXT: *(buffer + 1) = (float *)malloc(SIZE_1 * sizeof(float));
  hipHostMalloc((void**)buffer, SIZE_1 * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void**)(buffer + 1), SIZE_1 * sizeof(float), hipHostMallocDefault);
#undef SIZE_1


  // CHECK: free(h_A);
  hipHostFree(h_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cudaHostGetDevicePointer is not supported.
  // CHECK-NEXT: */
  hipHostGetDevicePointer(&d_A, h_A, 0);

  hipHostRegister(h_A, size, 0);
  hipHostUnregister(h_A);
}

template <typename T>
int foo2() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;
  /// memcpy from symbol
  // CHECK: c2s::c2s_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: c2s::c2s_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size);
  // CHECK: c2s::c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: c2s::c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: c2s::c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, c2s::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: c2s::async_c2s_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: c2s::async_c2s_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 2);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, c2s::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, c2s::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, c2s::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, c2s::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: c2s::c2s_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: c2s::c2s_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size);
  // CHECK: c2s::c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: c2s::c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: c2s::c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: c2s::async_c2s_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: c2s::async_c2s_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 2);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, c2s::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, c2s::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, c2s::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 3, hipMemcpyHostToDevice, stream);
}

template int foo2<float>();
template int foo2<int>();

#define MY_SAFE_CALL(CALL) do {    \
  int Error = CALL;                \
} while (0)

void foo3() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;
  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  int *data;
  size_t width, height, depth, pitch, woffset, hoffset;
  hipArray_t a1;
  int deviceID = 0;

  // CHECK: auto s1 = std::make_shared<sycl::queue *>((sycl::queue *)&q_ct1);
  // CHECK: auto s2 = std::make_shared<sycl::queue *>(&q_ct1);
  // CHECK: auto s3 = std::make_shared<sycl::queue *>(&q_ct1);
  auto s1 = std::make_shared<hipStream_t>((hipStream_t)hipStreamDefault);
  auto s2 = std::make_shared<hipStream_t>(hipStreamLegacy);
  auto s3 = std::make_shared<hipStream_t>(hipStreamPerThread);

  // CHECK: c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device);
  // CHECK: c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device);
  // CHECK: c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(d_A, h_A, size, c2s::host_to_device), 0));
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  // CHECK: c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device);
  // CHECK: errorCode = (c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, c2s::host_to_device), 0));
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host);
  // CHECK: c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, c2s::device_to_host), 0));
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread));

  // CHECK: c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device);
  // CHECK: c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device);
  // CHECK: c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(d_A, size, h_A, size, size, size, c2s::host_to_device), 0));
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  hipMemcpy3DAsync(&parms, hipStreamDefault);
  hipMemcpy3DAsync(&parms, hipStreamLegacy);
  hipMemcpy3DAsync(&parms, hipStreamPerThread);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamDefault);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamLegacy);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = (c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = (c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = (c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = (c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memcpy(c2s::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memset(d_A, 23, size);
  // CHECK: c2s::async_c2s_memset(d_A, 23, size);
  // CHECK: c2s::async_c2s_memset(d_A, 23, size);
  // CHECK: errorCode = (c2s::async_c2s_memset(d_A, 23, size), 0);
  // CHECK: errorCode = (c2s::async_c2s_memset(d_A, 23, size), 0);
  // CHECK: errorCode = (c2s::async_c2s_memset(d_A, 23, size), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(d_A, 23, size), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(d_A, 23, size), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(d_A, 23, size), 0));
  hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamDefault));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamLegacy));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memset(d_A, size, 0xf, size, size);
  // CHECK: c2s::async_c2s_memset(d_A, size, 0xf, size, size);
  // CHECK: c2s::async_c2s_memset(d_A, size, 0xf, size, size);
  // CHECK: errorCode = (c2s::async_c2s_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: errorCode = (c2s::async_c2s_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: errorCode = (c2s::async_c2s_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(d_A, size, 0xf, size, size), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(d_A, size, 0xf, size, size), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(d_A, size, 0xf, size, size), 0));
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault));
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy));
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread));


  // CHECK: c2s::async_c2s_memset(p_A, 0xf, e);
  // CHECK: c2s::async_c2s_memset(p_A, 0xf, e);
  // CHECK: c2s::async_c2s_memset(p_A, 0xf, e);
  // CHECK: errorCode = (c2s::async_c2s_memset(p_A, 0xf, e), 0);
  // CHECK: errorCode = (c2s::async_c2s_memset(p_A, 0xf, e), 0);
  // CHECK: errorCode = (c2s::async_c2s_memset(p_A, 0xf, e), 0);
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(p_A, 0xf, e), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(p_A, 0xf, e), 0));
  // CHECK: MY_SAFE_CALL((c2s::async_c2s_memset(p_A, 0xf, e), 0));
  hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault));
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy));
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread));
}

/// cuda driver memory api
void foo4(){
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  int errorCode;
  // CHECK: h_A = (float *)malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocPortable);
  // CHECK: errorCode = (h_A = (float *)malloc(size), 0);
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocPortable);
  // CHECK: MY_SAFE_CALL((h_A = (float *)malloc(size), 0));
  MY_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocPortable));
}
