// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=none -out-root %T %s -- -std=c++14 -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --match-full-lines --input-file %T/USM-none.dp.cpp %s

#include <hip/hip_runtime.h>

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  // CHECK: dpct::dpct_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);

  // CHECK: *((void **)&h_A) = malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: *((void **)&h_A) = malloc(size);
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipMallocManaged((void **)&d_A, size);

  // CHECK: free(h_A);
  hipHostFree(h_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipHostGetDevicePointer(&d_A, h_A, 0);

  hipHostRegister(h_A, size, 0);
  hipHostUnregister(h_A);
}
