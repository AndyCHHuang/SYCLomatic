// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-none.dp.cpp %s

// CHECK: #define DPCT_USM_LEVEL_NONE
// CHECK-NEXT: #define DPCT_NAMED_LAMBDA
// CHECK-NEXT: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;

  // CHECK: dpct::dpct_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);

  /// memcpy
  // CHECK: dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  /// memcpy async
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  /// memcpy from symbol
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  /// memset
  // CHECK: dpct::dpct_memset(d_A, 23, size);
  hipMemset(d_A, 23, size);

  /// memset async
  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, 23, size, *stream);
  hipMemsetAsync(d_A, 23, size, stream);

  // CHECK: h_A = (float *)malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: h_A = (float *)malloc(size);
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipMallocManaged((void **)&d_A, size);

  // CHECK: free(h_A);
  hipHostFree(h_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipHostGetDevicePointer(&d_A, h_A, 0);

  hipHostRegister(h_A, size, 0);
  hipHostUnregister(h_A);
}

template <typename T>
int foo2() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;
  /// memcpy from symbol
  // CHECK: dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: dpct::dpct_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
}

template int foo2<float>();
template int foo2<int>();
