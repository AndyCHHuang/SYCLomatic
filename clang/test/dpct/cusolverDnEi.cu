// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnEi.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    hipsolverEigMode_t jobz;

    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    signed char jobu;
    signed char jobvt;

    float S_f = 0;
    double S_d = 0.0;
    hipComplex S_c = make_hipComplex(1,0);
    hipDoubleComplex S_z = make_hipDoubleComplex(1,0);

    float U_f = 0;
    double U_d = 0.0;
    hipComplex U_c = make_hipComplex(1,0);
    hipDoubleComplex U_z = make_hipDoubleComplex(1,0);
    int ldu;

    float VT_f = 0;
    double VT_d = 0.0;
    hipComplex VT_c = make_hipComplex(1,0);
    hipDoubleComplex VT_z = make_hipDoubleComplex(1,0);
    int ldvt;

    float Rwork_f = 0;
    double Rwork_d = 0.0;
    hipComplex Rwork_c = make_hipComplex(1,0);
    hipDoubleComplex Rwork_z = make_hipDoubleComplex(1,0);

    float W_f = 0;
    double W_d = 0.0;
    hipComplex W_c = make_hipComplex(1,0);
    hipDoubleComplex W_z = make_hipDoubleComplex(1,0);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::gebrd_get_lwork<float>((*cusolverH).get_device(), m, n, m, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::gebrd_get_lwork<float>((*cusolverH).get_device(), m, n, m, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAUQ_f_buf_ct1 = dpct::get_buffer<float>(&TAUQ_f);
    // CHECK-NEXT: auto TAUP_f_buf_ct1 = dpct::get_buffer<float>(&TAUP_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gebrd(*cusolverH, m, n, A_f_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAUQ_f_buf_ct1, TAUP_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAUQ_f_buf_ct1 = dpct::get_buffer<float>(&TAUQ_f);
    // CHECK-NEXT: auto TAUP_f_buf_ct1 = dpct::get_buffer<float>(&TAUP_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gebrd(*cusolverH, m, n, A_f_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAUQ_f_buf_ct1, TAUP_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer11);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::gebrd_get_lwork<double>((*cusolverH).get_device(), m, n, m, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::gebrd_get_lwork<double>((*cusolverH).get_device(), m, n, m, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAUQ_d_buf_ct1 = dpct::get_buffer<double>(&TAUQ_d);
    // CHECK-NEXT: auto TAUP_d_buf_ct1 = dpct::get_buffer<double>(&TAUP_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gebrd(*cusolverH, m, n, A_d_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAUQ_d_buf_ct1, TAUP_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAUQ_d_buf_ct1 = dpct::get_buffer<double>(&TAUQ_d);
    // CHECK-NEXT: auto TAUP_d_buf_ct1 = dpct::get_buffer<double>(&TAUP_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gebrd(*cusolverH, m, n, A_d_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAUQ_d_buf_ct1, TAUP_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer11);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::gebrd_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n, m, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::gebrd_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n, m, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAUQ_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAUQ_c);
    // CHECK-NEXT: auto TAUP_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAUP_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gebrd(*cusolverH, m, n, A_c_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAUQ_c_buf_ct1, TAUP_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAUQ_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAUQ_c);
    // CHECK-NEXT: auto TAUP_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAUP_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gebrd(*cusolverH, m, n, A_c_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAUQ_c_buf_ct1, TAUP_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer11);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::gebrd_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n, m, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::gebrd_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n, m, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAUQ_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAUQ_z);
    // CHECK-NEXT: auto TAUP_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAUP_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gebrd(*cusolverH, m, n, A_z_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAUQ_z_buf_ct1, TAUP_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer11), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAUQ_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAUQ_z);
    // CHECK-NEXT: auto TAUP_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAUP_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer11(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gebrd(*cusolverH, m, n, A_z_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAUQ_z_buf_ct1, TAUP_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer11);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer11.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgbr_get_lwork<float>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgbr_get_lwork<float>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgbr(*cusolverH, (mkl::generate)side, m, n, k, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgbr(*cusolverH, (mkl::generate)side, m, n, k, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgbr_get_lwork<double>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgbr_get_lwork<double>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgbr(*cusolverH, (mkl::generate)side, m, n, k, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgbr(*cusolverH, (mkl::generate)side, m, n, k, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungbr_get_lwork<std::complex<float>>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungbr_get_lwork<std::complex<float>>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungbr(*cusolverH, (mkl::generate)side, m, n, k, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungbr(*cusolverH, (mkl::generate)side, m, n, k, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungbr_get_lwork<std::complex<double>>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungbr_get_lwork<std::complex<double>>((*cusolverH).get_device(), (mkl::generate)side, m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungbr(*cusolverH, (mkl::generate)side, m, n, k, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungbr(*cusolverH, (mkl::generate)side, m, n, k, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::sytrd_get_lwork<float>((*cusolverH).get_device(), uplo, n,  lda,    lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::sytrd_get_lwork<float>((*cusolverH).get_device(), uplo, n,  lda,    lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrd(*cusolverH, uplo, n, A_f_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrd(*cusolverH, uplo, n, A_f_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    status = hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::sytrd_get_lwork<double>((*cusolverH).get_device(), uplo, n,  lda,    lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::sytrd_get_lwork<double>((*cusolverH).get_device(), uplo, n,  lda,    lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrd(*cusolverH, uplo, n, A_d_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrd(*cusolverH, uplo, n, A_d_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    status = hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::hetrd_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo, n,  lda,    lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::hetrd_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo, n,  lda,    lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::hetrd(*cusolverH, uplo, n, A_c_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto D_f_buf_ct1 = dpct::get_buffer<float>(&D_f);
    // CHECK-NEXT: auto E_f_buf_ct1 = dpct::get_buffer<float>(&E_f);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::hetrd(*cusolverH, uplo, n, A_c_buf_ct1, lda, D_f_buf_ct1, E_f_buf_ct1, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    status = hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::hetrd_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo, n,  lda,    lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::hetrd_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo, n,  lda,    lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::hetrd(*cusolverH, uplo, n, A_z_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer10), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto D_d_buf_ct1 = dpct::get_buffer<double>(&D_d);
    // CHECK-NEXT: auto E_d_buf_ct1 = dpct::get_buffer<double>(&E_d);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer10(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::hetrd(*cusolverH, uplo, n, A_z_buf_ct1, lda, D_d_buf_ct1, E_d_buf_ct1, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer10);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer10.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    status = hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ormtr_get_lwork<float>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ormtr_get_lwork<float>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ormtr(*cusolverH, side, uplo, trans, m, n, A_f_buf_ct1, lda, TAU_f_buf_ct1, B_f_buf_ct1, ldb, workspace_f_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ormtr(*cusolverH, side, uplo, trans, m, n, A_f_buf_ct1, lda, TAU_f_buf_ct1, B_f_buf_ct1, ldb, workspace_f_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    status = hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ormtr_get_lwork<double>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ormtr_get_lwork<double>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ormtr(*cusolverH, side, uplo, trans, m, n, A_d_buf_ct1, lda, TAU_d_buf_ct1, B_d_buf_ct1, ldb, workspace_d_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ormtr(*cusolverH, side, uplo, trans, m, n, A_d_buf_ct1, lda, TAU_d_buf_ct1, B_d_buf_ct1, ldb, workspace_d_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    status = hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::unmtr_get_lwork<std::complex<float>>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::unmtr_get_lwork<std::complex<float>>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::unmtr(*cusolverH, side, uplo, trans, m, n, A_c_buf_ct1, lda, TAU_c_buf_ct1, B_c_buf_ct1, ldb, workspace_c_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::unmtr(*cusolverH, side, uplo, trans, m, n, A_c_buf_ct1, lda, TAU_c_buf_ct1, B_c_buf_ct1, ldb, workspace_c_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    status = hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::unmtr_get_lwork<std::complex<double>>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::unmtr_get_lwork<std::complex<double>>((*cusolverH).get_device(), side, uplo, trans, m, n,  lda,   ldb, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::unmtr(*cusolverH, side, uplo, trans, m, n, A_z_buf_ct1, lda, TAU_z_buf_ct1, B_z_buf_ct1, ldb, workspace_z_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::unmtr(*cusolverH, side, uplo, trans, m, n, A_z_buf_ct1, lda, TAU_z_buf_ct1, B_z_buf_ct1, ldb, workspace_z_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    status = hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgtr_get_lwork<float>((*cusolverH).get_device(), uplo, n,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgtr_get_lwork<float>((*cusolverH).get_device(), uplo, n,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgtr(*cusolverH, uplo, n, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgtr(*cusolverH, uplo, n, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgtr_get_lwork<double>((*cusolverH).get_device(), uplo, n,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgtr_get_lwork<double>((*cusolverH).get_device(), uplo, n,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgtr(*cusolverH, uplo, n, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgtr(*cusolverH, uplo, n, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungtr_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo, n,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungtr_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo, n,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungtr(*cusolverH, uplo, n, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungtr(*cusolverH, uplo, n, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungtr_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo, n,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungtr_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo, n,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungtr(*cusolverH, uplo, n, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungtr(*cusolverH, uplo, n, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: status = (mkl::lapack::gesvd_get_lwork<float>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: mkl::lapack::gesvd_get_lwork<float>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto S_f_buf_ct1 = dpct::get_buffer<float>(&S_f);
    // CHECK-NEXT: auto U_f_buf_ct1 = dpct::get_buffer<float>(&U_f);
    // CHECK-NEXT: auto VT_f_buf_ct1 = dpct::get_buffer<float>(&VT_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_f_buf_ct1, lda, S_f_buf_ct1, U_f_buf_ct1, ldu, VT_f_buf_ct1, ldvt, workspace_f_buf_ct1, Lwork,  result_temp_buffer15), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto S_f_buf_ct1 = dpct::get_buffer<float>(&S_f);
    // CHECK-NEXT: auto U_f_buf_ct1 = dpct::get_buffer<float>(&U_f);
    // CHECK-NEXT: auto VT_f_buf_ct1 = dpct::get_buffer<float>(&VT_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_f_buf_ct1, lda, S_f_buf_ct1, U_f_buf_ct1, ldu, VT_f_buf_ct1, ldvt, workspace_f_buf_ct1, Lwork,  result_temp_buffer15);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);
    hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: status = (mkl::lapack::gesvd_get_lwork<double>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: mkl::lapack::gesvd_get_lwork<double>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto S_d_buf_ct1 = dpct::get_buffer<double>(&S_d);
    // CHECK-NEXT: auto U_d_buf_ct1 = dpct::get_buffer<double>(&U_d);
    // CHECK-NEXT: auto VT_d_buf_ct1 = dpct::get_buffer<double>(&VT_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_d_buf_ct1, lda, S_d_buf_ct1, U_d_buf_ct1, ldu, VT_d_buf_ct1, ldvt, workspace_d_buf_ct1, Lwork,  result_temp_buffer15), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto S_d_buf_ct1 = dpct::get_buffer<double>(&S_d);
    // CHECK-NEXT: auto U_d_buf_ct1 = dpct::get_buffer<double>(&U_d);
    // CHECK-NEXT: auto VT_d_buf_ct1 = dpct::get_buffer<double>(&VT_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_d_buf_ct1, lda, S_d_buf_ct1, U_d_buf_ct1, ldu, VT_d_buf_ct1, ldvt, workspace_d_buf_ct1, Lwork,  result_temp_buffer15);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);
    hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: status = (mkl::lapack::gesvd_get_lwork<std::complex<float>>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: mkl::lapack::gesvd_get_lwork<std::complex<float>>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto S_f_buf_ct1 = dpct::get_buffer<float>(&S_f);
    // CHECK-NEXT: auto U_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&U_c);
    // CHECK-NEXT: auto VT_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&VT_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto Rwork_f_buf_ct1 = dpct::get_buffer<float>(&Rwork_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_c_buf_ct1, lda, S_f_buf_ct1, U_c_buf_ct1, ldu, VT_c_buf_ct1, ldvt, workspace_c_buf_ct1, Lwork, Rwork_f_buf_ct1, result_temp_buffer15), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto S_f_buf_ct1 = dpct::get_buffer<float>(&S_f);
    // CHECK-NEXT: auto U_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&U_c);
    // CHECK-NEXT: auto VT_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&VT_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto Rwork_f_buf_ct1 = dpct::get_buffer<float>(&Rwork_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_c_buf_ct1, lda, S_f_buf_ct1, U_c_buf_ct1, ldu, VT_c_buf_ct1, ldvt, workspace_c_buf_ct1, Lwork, Rwork_f_buf_ct1, result_temp_buffer15);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);
    hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: status = (mkl::lapack::gesvd_get_lwork<std::complex<double>>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::job job_ct_mkl_jobu;
    // CHECK-NEXT: mkl::job job_ct_mkl_jobvt;
    // CHECK-NEXT: mkl::lapack::gesvd_get_lwork<std::complex<double>>((*cusolverH).get_device(), job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto S_d_buf_ct1 = dpct::get_buffer<double>(&S_d);
    // CHECK-NEXT: auto U_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&U_z);
    // CHECK-NEXT: auto VT_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&VT_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto Rwork_d_buf_ct1 = dpct::get_buffer<double>(&Rwork_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_z_buf_ct1, lda, S_d_buf_ct1, U_z_buf_ct1, ldu, VT_z_buf_ct1, ldvt, workspace_z_buf_ct1, Lwork, Rwork_d_buf_ct1, result_temp_buffer15), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto S_d_buf_ct1 = dpct::get_buffer<double>(&S_d);
    // CHECK-NEXT: auto U_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&U_z);
    // CHECK-NEXT: auto VT_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&VT_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto Rwork_d_buf_ct1 = dpct::get_buffer<double>(&Rwork_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer15(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::gesvd (*cusolverH, (mkl::job)jobu, (mkl::job)jobvt, m, n, A_z_buf_ct1, lda, S_d_buf_ct1, U_z_buf_ct1, ldu, VT_z_buf_ct1, ldvt, workspace_z_buf_ct1, Lwork, Rwork_d_buf_ct1, result_temp_buffer15);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer15.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);
    hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);

}
