#include "hip/hip_runtime.h"
// RUN: dpct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/nestedqueue.dp.cpp --match-full-lines %s

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

__host__ __device__ void foo1(){
  // CHECK: /*
  // CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipblasHandle_t handle;
  hipblasHandle_t handle;
  int n=1;
  float* x_S=0;
  int incx=1;
  int* result =0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1020:{{[0-9]+}}: Migration of hipblasIsamax if called from __global__ or __device__ function is not supported. You may need to redesign the code to use host-side mkl::iamax instead, which will submit this call to DPC++ queue automatically.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);
}

__device__ void foo2(){
  // CHECK: /*
  // CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipblasHandle_t handle;
  hipblasHandle_t handle;
  int n=1;
  float* x_S=0;
  int incx=1;
  int* result =0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1020:{{[0-9]+}}: Migration of hipblasIsamax if called from __global__ or __device__ function is not supported. You may need to redesign the code to use host-side mkl::iamax instead, which will submit this call to DPC++ queue automatically.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);
}

__global__ void foo3(){
  // CHECK: /*
  // CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipblasHandle_t handle;
  hipblasHandle_t handle;
  int n=1;
  float* x_S=0;
  int incx=1;
  int* result =0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1020:{{[0-9]+}}: Migration of hipblasIsamax if called from __global__ or __device__ function is not supported. You may need to redesign the code to use host-side mkl::iamax instead, which will submit this call to DPC++ queue automatically.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);
}

__host__ void foo4(){
  // CHECK: cl::sycl::queue handle;
  hipblasHandle_t handle;
  int n=1;
  float* x_S=0;
  int incx=1;
  int* result =0;
  // CHECK: mkl::iamax(handle, n, buffer_ct{{[0-9]+}}, incx, result_temp_buffer);
  hipblasIsamax(handle, n, x_S, incx, result);
}
