// RUN: dpct --format-range=none -out-root %T/cufft-reuse-handle %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-reuse-handle/cufft-reuse-handle.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

int main() {
  hipfftHandle plan_mmany64_Z2Z;
  size_t* work_size_mmany64_Z2Z;
  long long int odist_mmany64_Z2Z;
  long long int ostride_mmany64_Z2Z;
  long long int * onembed_mmany64_Z2Z;
  long long int idist_mmany64_Z2Z;
  long long int istride_mmany64_Z2Z;
  long long int* inembed_mmany64_Z2Z;
  long long int * n_mmany64_Z2Z;
  double2* odata_mmany64_Z2Z;
  double2* idata_mmany64_Z2Z;

  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size_mmany64_Z2Z is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed_mmany64_Z2Z[2] * inembed_mmany64_Z2Z[1] * istride_mmany64_Z2Z, inembed_mmany64_Z2Z[2] * istride_mmany64_Z2Z, istride_mmany64_Z2Z};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed_mmany64_Z2Z[2] * onembed_mmany64_Z2Z[1] * ostride_mmany64_Z2Z, onembed_mmany64_Z2Z[2] * ostride_mmany64_Z2Z, ostride_mmany64_Z2Z};
  //CHECK-NEXT:plan_mmany64_Z2Z = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::COMPLEX>>(std::vector<std::int64_t>{n_mmany64_Z2Z[0], n_mmany64_Z2Z[1], n_mmany64_Z2Z[2]});
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist_mmany64_Z2Z);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist_mmany64_Z2Z);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan_mmany64_Z2Z->commit(q_ct1);
  hipfftMakePlanMany64(plan_mmany64_Z2Z, 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany64_Z2Z);

  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size_mmany64_Z2Z is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed_mmany64_Z2Z[2] * inembed_mmany64_Z2Z[1] * istride_mmany64_Z2Z, inembed_mmany64_Z2Z[2] * istride_mmany64_Z2Z, istride_mmany64_Z2Z};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed_mmany64_Z2Z[2] * onembed_mmany64_Z2Z[1] * ostride_mmany64_Z2Z, onembed_mmany64_Z2Z[2] * ostride_mmany64_Z2Z, ostride_mmany64_Z2Z};
  //CHECK-NEXT:plan_mmany64_Z2Z = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::COMPLEX>>(std::vector<std::int64_t>{n_mmany64_Z2Z[0], n_mmany64_Z2Z[1], n_mmany64_Z2Z[2]});
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist_mmany64_Z2Z);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist_mmany64_Z2Z);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan_mmany64_Z2Z->commit(q_ct1);
  hipfftMakePlanMany64(plan_mmany64_Z2Z, 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany64_Z2Z);

  //CHECK:if ((void *)idata_mmany64_Z2Z == (void *)odata_mmany64_Z2Z) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan_mmany64_Z2Z, (double*)idata_mmany64_Z2Z);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan_mmany64_Z2Z, (double*)idata_mmany64_Z2Z, (double*)odata_mmany64_Z2Z);
  //CHECK-NEXT:}
  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_BACKWARD);

  return 0;
}
