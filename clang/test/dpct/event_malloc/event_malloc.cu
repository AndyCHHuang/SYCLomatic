// RUN: c2s -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/event_malloc.dp.cpp --match-full-lines %s
// RUN: FileCheck --input-file %T/event_malloc.h --match-full-lines %S/event_malloc.h


#include "event_malloc.h"

// CHECK:C::~C(void) { delete[] kernelEvent; }
C::~C(void) { free(kernelEvent); }

void foo_1() {
  int n_streams = 4;

  // CHECK:  sycl::event *kernelEvent = new sycl::event[n_streams];
  // CHECK-NEXT:  delete[] kernelEvent;
  hipEvent_t *kernelEvent = (hipEvent_t *)malloc(n_streams * sizeof(hipEvent_t));
  free(kernelEvent);
}

void foo_2() {
  int n_streams = 4;

  // CHECK:  sycl::event *kernelEvent;
  // CHECK-NEXT:  kernelEvent = new sycl::event[n_streams];
  // CHECK-NEXT:  delete[] kernelEvent;
  hipEvent_t *kernelEvent;
  kernelEvent = (hipEvent_t *)malloc(n_streams * sizeof(hipEvent_t));
  free(kernelEvent);
}

void foo_3() {
  int n_streams = 4;

  // CHECK:  sycl::event *kernelEvent;
  // CHECK-NEXT:  int size = n_streams * sizeof(sycl::event);
  // CHECK-NEXT:  kernelEvent = new sycl::event[(size + sizeof(sycl::event)) / sizeof(sycl::event)];
  // CHECK-NEXT:  delete[] kernelEvent;
  hipEvent_t *kernelEvent;
  int size = n_streams * sizeof(hipEvent_t);
  kernelEvent = (hipEvent_t *)malloc(size + sizeof(hipEvent_t));
  free(kernelEvent);
}


hipEvent_t *kernelEvent = NULL;

void foo_4() {
  int n_streams = 4;
  // CHECK:  sycl::event *kernelEvent = new sycl::event[n_streams];
  hipEvent_t *kernelEvent = (hipEvent_t *)malloc(n_streams * sizeof(hipEvent_t));
}

inline void free(){}

void foo_5() {
  // CHECK:   delete[] kernelEvent;
   free(kernelEvent);
   free();
}
