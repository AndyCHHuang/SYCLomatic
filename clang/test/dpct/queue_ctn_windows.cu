// UNSUPPORTED: -linux-
// RUN: dpct --format-range=none  -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/queue_ctn_windows.dp.cpp


#include "hip/hip_runtime.h"

void bar();
#define SIZE 100

size_t size = 1234567 * sizeof(float);
float *h_A = (float *)malloc(size);
float *d_A = NULL;
__constant__ float constData[1234567 * 4];

void foo1() {
  // CHECK: sycl::queue& q_ct0 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct0.wait();
  // CHECK-NEXT: q_ct0.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct0.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct0.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  // CHECK-NEXT: q_ct0.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct0.memset(d_A, 23, size).wait();
  // CHECK-NEXT: bar();
  // CHECK-NEXT: sycl::queue& q_ct1 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct1.wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  bar();
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}


void foo2() {
  // CHECK: sycl::queue& q_ct2 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct2.wait();
  // CHECK-NEXT: q_ct2.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct2.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct2.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct2.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct2.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct2.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct2.memset(d_A, 23, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}

void foo3() {
  // CHECK: sycl::queue& q_ct3 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct3.wait();
  // CHECK-NEXT: q_ct3.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct3.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemset(d_A, 23, size);
}

void foo4() {
  // CHECK: dpct::get_default_queue().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: bar();
  // CHECK-NEXT: dpct::get_default_queue().memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  bar();
  hipMemset(d_A, 23, size);
}

void foo5() {
  // CHECK: sycl::queue& q_ct4 = dpct::get_default_queue();
  // CHECK-NEXT: q_ct4.wait();
  // CHECK-NEXT: q_ct4.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: int Err = (q_ct4.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait(), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: Err = (q_ct4.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait(), 0);
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  int Err = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  Err = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
}

#define CUDA_CALL( call) call

// unsupported conditions
void foo6() {
  // CHECK: dpct::get_default_queue().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: // call in macro
  // CHECK-NEXT: CUDA_CALL(dpct::get_default_queue().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait());
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  // call in macro
  CUDA_CALL(hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost ));
}

// On Windows, migration is not supported in normal case
//template <typename T>
//void foo7() {
//  T* d_A_unresolved, h_A_unresolved;
//  // Types of d_A_unresolved and h_A_unresolved are unresolved
//  cudaMemcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE, cudaMemcpyDeviceToHost );
//  cudaMemcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE, cudaMemcpyDeviceToHost );
//}