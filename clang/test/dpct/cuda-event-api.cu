// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-event-api.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
// CHECK: void check(T result, char const *const func) {
void check(T result, char const *const func) {
}

#define checkCudaErrors(val) check((val), #val)

#define CudaEvent(X)\
  hipEventCreate(&X)

#define cudaCheck(stmt) do {                         \
  hipError_t err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

__global__ void kernelFunc()
{
}

int main(int argc, char* argv[]) {
  // CHECK: cl::sycl::event start, stop;
  // CHECK-EMPTY:
  // CHECK-EMPTY:
  // CHECK-NEXT: float elapsed_time;
  // CHECK-EMPTY:
  // CHECK-NEXT: dpct::get_device_manager().current_device().queues_wait_and_throw();
  // CHECK-EMPTY:
  // CHECK-NEXT: int blocks = 32, threads = 32;
  hipEvent_t start, stop;

  hipEventCreate(&start)  
    ;   
  hipEventCreate(&stop)  ;   

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  CudaEvent(start);

  // CHECK: checkCudaErrors(0);
  // CHECK-NEXT: int et = 0;
  checkCudaErrors(hipEventCreate(&start));
  hipError_t et = hipEventCreate(&stop);


  // kernel call without sync
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: auto start_ct1 = clock();
  hipEventRecord(start, 0);

  // kernel call without sync
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: start_ct1 = clock();
  hipEventRecord(start, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[0-9a-f]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT: */
  // CHECK-NEXT: start_ct1 = clock(), checkCudaErrors(0);
  checkCudaErrors(hipEventRecord(start, 0));

  // CHECK: if (0)
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   start_ct1 = clock(), checkCudaErrors(0);
  if (0)
    checkCudaErrors(hipEventRecord(start, 0));

  // kernel call with sync
  // CHECK:   stop = dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: stop.wait();
  kernelFunc<<<blocks,threads>>>();
  // CHECK:   stop = dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: stop.wait();
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: auto stop_ct1 = clock();
  hipEventRecord(stop, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = clock(), checkCudaErrors(0);
  checkCudaErrors(hipEventRecord(stop, 0));

  // CHECK: if (1)
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   stop_ct1 = clock(), checkCudaErrors(0);
  if (1)
    checkCudaErrors(hipEventRecord(stop, 0));

  // kernel call without sync
  // CHECK:   stop = dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: stop.wait();
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = clock();
  hipEventRecord(stop, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = clock(), checkCudaErrors(0);
  checkCudaErrors(hipEventRecord(stop, 0));

  // CHECK: if (0)
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   stop_ct1 = clock(), checkCudaErrors(0);
  if (0)
    checkCudaErrors(hipEventRecord(stop, 0));

  // CHECK: stop.wait_and_throw();
  hipEventSynchronize(stop);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9a-z]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((stop.wait_and_throw(), 0));
  checkCudaErrors(hipEventSynchronize(stop));

  // kernel call without sync
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: *(&elapsed_time) = (float)(stop_ct1 - start_ct1) / CLOCKS_PER_SEC * 1000;
  hipEventElapsedTime(&elapsed_time, start, stop);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9a-z]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((*(&elapsed_time) = (float)(stop_ct1 - start_ct1) / CLOCKS_PER_SEC * 1000, 0));
  checkCudaErrors(hipEventElapsedTime(&elapsed_time, start, stop));

  // kernel call without sync
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, blocks) * cl::sycl::range<3>(1, 1, threads), cl::sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: dpct::get_device_manager().current_device().queues_wait_and_throw();
  // CHECK-EMPTY:
  // CHECK-NEXT: checkCudaErrors(0);
  // CHECK-NEXT: et = 0;
  // CHECK-NEXT: }
  hipDeviceSynchronize();

  hipEventDestroy(start)  ;   
  hipEventDestroy(stop)  
    ;   
  checkCudaErrors(hipEventDestroy(start));
  et = hipEventDestroy(stop);
}

void foo() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  // CHECK: auto start_ct1 = clock(), cudaCheck(0);
  cudaCheck(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
  // CHECK: auto stop_ct1 = clock(), cudaCheck(0);
  cudaCheck(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);

  // CHECK: cudaCheck((*(&elapsed_time) = (float)(stop_ct1 - start_ct1) / CLOCKS_PER_SEC * 1000, 0));
  cudaCheck(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void fun(int) {}

void bar() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  // CHECK: auto start_ct1 = clock(), fun(0);
  fun(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
  // CHECK: auto stop_ct1 = clock(), fun(0);
  fun(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);

  // CHECK: fun((*(&elapsed_time) = (float)(stop_ct1 - start_ct1) / CLOCKS_PER_SEC * 1000, 0));
  fun(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);
}
