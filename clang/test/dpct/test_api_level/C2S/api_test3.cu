
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/C2S/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/C2S/api_test3_out/MainSourceFiles.yaml | wc -l > %T/C2S/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/C2S/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/C2S/api_test3_out

// CHECK: 2

// TEST_FEATURE: C2S_c2s_compatibility_temp

#define AAA __CUDA_ARCH__

int main() {
  return 0;
}
