
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test9_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test9_out

// CHECK: 7
// TEST_FEATURE: Image_image_channel
// TEST_FEATURE: Image_image_channel_create

int main() {
  hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  return 0;
}
