// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test28_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test28_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test28_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test28_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test28_out

// CHECK: 21
// TEST_FEATURE: Image_image_data_get_data_type

void foo(HIPresourcetype res21) {}

int main() {
  hipResourceDesc res21;
  foo(res21.resType);
  return 0;
}
