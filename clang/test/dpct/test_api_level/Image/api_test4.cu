
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test4_out

// CHECK: 2
// TEST_FEATURE: Image_image_channel_data_type

int main() {
  hipChannelFormatKind a;
  return 0;
}
