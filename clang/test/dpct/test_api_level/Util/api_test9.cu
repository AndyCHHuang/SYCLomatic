
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test9_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test9_out

// CHECK: 15

// TEST_FEATURE: Util_make_index_sequence

int main() {
  hipArray_t a42;
  return 0;
}
