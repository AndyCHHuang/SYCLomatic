// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/RngUtils/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/RngUtils/api_test1_out/MainSourceFiles.yaml | wc -l > %T/RngUtils/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/RngUtils/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/RngUtils/api_test1_out

// CHECK: 5
// TEST_FEATURE: RngUtils_rng_generator
// TEST_FEATURE: RngUtils_rng_generator_generate


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
__device__ void foo() {
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(1, 2, 3, &rng);
  hiprand(&rng);
}

int main() {
  return 0;
}
