
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test10_out

// CHECK: 3
// TEST_FEATURE: Memory_pitched_data

int main() {
  hipPitchedPtr a;
  return 0;
}
