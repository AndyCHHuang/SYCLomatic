
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test1_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test1_out

// CHECK: 2
// TEST_FEATURE: Atomic_atomic_compare_exchange_strong

__global__ void test(int *data) {
  int inc = 1;


  atomicCAS(&data[7], inc - 1, inc);
}
int main() {
  return 0;
}
