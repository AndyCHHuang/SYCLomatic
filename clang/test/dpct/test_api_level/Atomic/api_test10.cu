
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test10_out

// CHECK: 2
// TEST_FEATURE: Atomic_atomic_exchange

__global__ void test(int *data) {
  int inc = 1;


  atomicExch(&data[2], inc);
}
int main() {
  return 0;
}
