
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test5_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test5_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test5_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test5_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test5_out

// CHECK: 2
// TEST_FEATURE: Atomic_atomic_fetch_or

__global__ void test(int *data) {
  int inc = 1;


  atomicOr(&data[9], inc);

}
int main() {
  return 0;
}
