
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test4_out

// CHECK: 2

// TEST_FEATURE: Atomic_atomic_fetch_and

__global__ void test(int *data) {
  int inc = 1;


  atomicAnd(&data[8], inc);

}
int main() {
  return 0;
}
