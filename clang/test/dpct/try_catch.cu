
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/try_catch.dp.cpp

namespace Test {
enum class AA : int { ONE,
                      TWO,
                      THREE };
}

__global__ void fun() {}

class B {
public:
// CHECK: B() : data_(Test::AA::ONE) {}
  B() : data_(Test::AA::ONE) {}

private:
  Test::AA data_;
};
