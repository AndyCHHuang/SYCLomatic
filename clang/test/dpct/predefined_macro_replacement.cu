#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -D__NVCC__ -D__CUDACC__
// RUN: FileCheck --input-file %T/predefined_macro_replacement.dp.cpp --match-full-lines %s
#include <stdio.h>
//CHECK: #ifdef DPCPP_COMPATIBILITY_TEMP
//CHECK-NEXT: void hello(sycl::stream [[STREAM:stream_ct1]]) { [[STREAM]] << "foo"; }
#ifdef __CUDA_ARCH__
__global__ void hello() { printf("foo"); }
#else
__global__ void hello() { printf("other"); }
#endif

//CHECK: #ifndef DPCPP_COMPATIBILITY_TEMP
#ifndef __NVCC__
__global__ void hello2() { printf("hello2"); }
#endif
//CHECK: #if defined(CL_SYCL_LANGUAGE_VERSION)
#if defined(__HIPCC__)
__global__ void hello3() { printf("hello2"); }
#endif

#if defined(xxx)
__global__ void hello4() { printf("hello2"); }
//CHECK: #elif defined(DPCPP_COMPATIBILITY_TEMP)
//CHECK-NEXT: void hello4(sycl::stream [[STREAM]]) { [[STREAM]] << "hello2"; }
#elif defined(__CUDA_ARCH__)
__global__ void hello4() { printf("hello2"); }
#endif

#if defined(xxx)
__global__ void hello5() { printf("hello2"); }
//CHECK: #elif (DPCPP_COMPATIBILITY_TEMP >= 400)
//CHECK-NEXT: void hello5(sycl::stream [[STREAM]]) { [[STREAM]] << "hello2"; }
#elif (__CUDA_ARCH__ >= 400)
__global__ void hello5() { printf("hello2"); }
#endif

//CHECK: #if defined(DPCPP_COMPATIBILITY_TEMP)
//CHECK-NEXT: void hello6(sycl::stream [[STREAM]]) { [[STREAM]] << "hello2"; }
#if defined(__CUDA_ARCH__)
__global__ void hello6() { printf("hello2"); }
#endif

//CHECK: #ifndef DPCPP_COMPATIBILITY_TEMP
//CHECK-NEXT: __global__ void hello7() { printf("hello2"); }
//CHECK-NEXT: #else
//CHECK-NEXT: void hello7(sycl::stream [[STREAM]]) { [[STREAM]] << "hello2"; }
#ifndef __CUDA_ARCH__
__global__ void hello7() { printf("hello2"); }
#else
__global__ void hello7() { printf("hello2"); }
#endif

__global__ void test(){
//CHECK:#if (DPCPP_COMPATIBILITY_TEMP >= 400) &&  (DPCPP_COMPATIBILITY_TEMP >= 400)
//CHECK-NEXT:[[STREAM]] << ">400, \n";
//CHECK-NEXT:#elif (DPCPP_COMPATIBILITY_TEMP >200)
//CHECK-NEXT:printf(">200, \n");
//CHECK-NEXT:#else
//CHECK-NEXT:printf("<200, \n");
//CHECK-NEXT:#endif
#if (__CUDA_ARCH__ >= 400) &&  (__CUDA_ARCH__ >= 400)
printf(">400, \n");
#elif (__CUDA_ARCH__ >200)
printf(">200, \n");
#else
printf("<200, \n");
#endif
}


int main() {
//CHECK: #if defined(DPCPP_COMPATIBILITY_TEMP)
//CHECK-NEXT:     dpct::get_default_queue().submit(
#if defined(__NVCC__)
  hello<<<1,1>>>();
#else
  hello();
#endif
  return 0;
}

//CHECK: #define AAA DPCPP_COMPATIBILITY_TEMP
//CHECK-NEXT: #define BBB CL_SYCL_LANGUAGE_VERSION
//CHECK-NEXT: #define CCC DPCPP_COMPATIBILITY_TEMP
#define AAA __CUDA_ARCH__
#define BBB __HIPCC__
#define CCC __NVCC__
