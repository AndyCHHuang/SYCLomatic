// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sharedmem_var_dynamic.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 100
// CHECK: void staticReverse(int *d, int n,
// CHECK-NEXT:  sycl::nd_item<3> item_ct1,
// CHECK-NEXT:  dpct::byte_t *dpct_local) {
// CHECK-NEXT:  auto s = (int *)dpct_local; // the size of s is dynamic
__global__ void staticReverse(int *d, int n) {
  extern __shared__ int s[]; // the size of s is dynamic
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
}

// CHECK: template<typename TData>
// CHECK-NEXT: void templateReverse(TData *d, TData n,
// CHECK-NEXT: sycl::nd_item<3> item_ct1,
// CHECK-NEXT: dpct::byte_t *dpct_local) {
template<typename TData>
__global__ void templateReverse(TData *d, TData n) {

  // CHECK: auto s = (TData *)dpct_local; // the size of s is dynamic
  extern __shared__ TData s[]; // the size of s is dynamic
  int t = threadIdx.x;
  if (t < 64) {
    s[t] = d[t];
  }
}

template<typename T>
void testTemplate(){
  const int n = 64;
  T a[n], r[n], d[n];
  T *d_d;
  int mem_size = n * sizeof(T);
  hipMalloc((void **)&d_d, mem_size);
  hipMemcpy(d_d, a, mem_size, hipMemcpyHostToDevice);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_d_buf_ct0 = dpct::get_buffer_and_offset(d_d);
  // CHECK-NEXT:   size_t d_d_offset_ct0 = d_d_buf_ct0.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::accessor<dpct::byte_t, 1, sycl::access::mode::read_write, sycl::access::target::local> dpct_local_acc_ct1(sycl::range<1>(mem_size), cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.first.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, n), sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           T *d_d_ct0 = (T *)(&d_d_acc_ct0[0] + d_d_offset_ct0);
  // CHECK-NEXT:           templateReverse<T>(d_d_ct0, n, item_ct1, dpct_local_acc_ct1.get_pointer());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<T><<<1, n, mem_size>>>(d_d, n);
}

int main(void) {
  const int n = 64;
  int a[n], r[n], d[n];
  int *d_d;
  int mem_size = n * sizeof(int);
  hipMalloc((void **)&d_d, mem_size);
  hipMemcpy(d_d, a, mem_size, hipMemcpyHostToDevice);
  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_d_buf_ct0 = dpct::get_buffer(d_d);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::accessor<dpct::byte_t, 1, sycl::access::mode::read_write, sycl::access::target::local> dpct_local_acc_ct1(sycl::range<1>(mem_size), cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class staticReverse_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, n), sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           staticReverse((int *)(&d_d_acc_ct0[0]), n, item_ct1, dpct_local_acc_ct1.get_pointer());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  staticReverse<<<1, n, mem_size>>>(d_d, n);
  hipMemcpy(d, d_d, mem_size, hipMemcpyDeviceToHost);

  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_d_buf_ct0 = dpct::get_buffer(d_d);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::accessor<dpct::byte_t, 1, sycl::access::mode::read_write, sycl::access::target::local> dpct_local_acc_ct1(sycl::range<1>(sizeof(int)), cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class staticReverse_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, n), sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           staticReverse((int *)(&d_d_acc_ct0[0]), n, item_ct1, dpct_local_acc_ct1.get_pointer());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  staticReverse<<<1, n, sizeof(int)>>>(d_d, n);

  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_d_buf_ct0 = dpct::get_buffer(d_d);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::accessor<dpct::byte_t, 1, sycl::access::mode::read_write, sycl::access::target::local> dpct_local_acc_ct1(sycl::range<1>(4), cgh);
  // CHECK-NEXT:       auto d_d_acc_ct0 = d_d_buf_ct0.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class templateReverse_{{[a-f0-9]+}}, int>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, n), sycl::range<3>(1, 1, n)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           templateReverse<int>((int *)(&d_d_acc_ct0[0]), n, item_ct1, dpct_local_acc_ct1.get_pointer());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  templateReverse<int><<<1, n, 4>>>(d_d, n);
}

