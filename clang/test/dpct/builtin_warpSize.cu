// RUN: dpct --format-range=none -out-root %T/builtin_warpSize %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/builtin_warpSize/builtin_warpSize.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


__global__ void foo(){
  // CHECK: int a = item_ct1.get_sub_group().get_local_range().get(0);
  // CHECK-NEXT: int warpSize = 1;
  // CHECK-NEXT: warpSize = 2;
  // CHECK-NEXT: int c= warpSize;
  int a = warpSize;
  int warpSize = 1;
  warpSize = 2;
  int c= warpSize;
}

