// UNSUPPORTED: -linux-
// RUN: cat %S/proj_c.vcxproj > %T/proj_c.vcxproj
// RUN: cd %T

// RUN: dpct --format-range=none  --vcxprojfile=%T/proj_c.vcxproj  -in-root=%S -out-root=%T  %s %S/CuTmp_1.cu --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only

// RUN: cat %S/CuTmp_1.cu > %T/CuTmp_1.dp.cpp
// RUN: cat %S/check_compilation_ref.txt  >%T/check_compilation_db.txt
// RUN: cat %T/compile_commands.json >>%T/check_compilation_db.txt
// RUN: FileCheck --match-full-lines --input-file %T/check_compilation_db.txt %T/check_compilation_db.txt
// RUN: FileCheck %S/CuTmp_1.cu --match-full-lines --input-file %T/CuTmp_1.dp.cpp

#include "hip/hip_runtime.h"
#include <stdio.h>

// CHECK: void addKernel(int *c, const int *a, const int *b, sycl::nd_item<3> item_ct1)
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // CHECK: int i = item_ct1.get_local_id(0);
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
