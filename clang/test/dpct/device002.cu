// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/device002.dp.cpp

#include <stdio.h>

void checkError(hipError_t err) {

}

int main(int argc, char **argv)
{
int devID = atoi(argv[1]);
hipDeviceProp_t cdp;
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int error_code = (dpct::get_device_manager().get_device(devID).get_device_info(cdp), 0);
hipError_t error_code = hipGetDeviceProperties(&cdp, devID);

if (error_code == hipSuccess) {
// CHECK: /*
// CHECK-NEXT:  DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: /*
// CHECK-NEXT:  DPCT1006:{{[0-9]+}}: DPC++ does not provide a standard API to differentiate between integrated/ discrete GPU devices.
// CHECK-NEXT: */
// CHECK-NEXT:if (cdp.get_major_version() < 3 && cdp.get_integrated() != 1) {
    if (cdp.major < 3 && cdp.integrated != 1) {
            printf("do_complex_compute requires compute capability 3.0 or later and not integrated\n");
    }
}

int deviceCount = 0;
// CHECK: deviceCount = dpct::get_device_manager().device_count();
hipGetDeviceCount(&deviceCount);

int dev_id;
// CHECK: dev_id = dpct::get_device_manager().current_device_id();
hipGetDevice(&dev_id);

hipDeviceProp_t deviceProp;
// CHECK: dpct::get_device_manager().get_device(0).get_device_info(deviceProp);
hipGetDeviceProperties(&deviceProp, 0);

int atomicSupported;
// CHECK: atomicSupported = dpct::get_device_manager().get_device(dev_id).is_native_atomic_supported();
hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);

int val;
// CHECK: val = dpct::get_device_manager().get_device(dev_id).get_major_version();
hipDeviceGetAttribute(&val, hipDeviceAttributeComputeCapabilityMajor, dev_id);

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: accessSupported = 0;
hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2);

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: perfRank = 0;
hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2);

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: atomicSupported = 0;
hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2);


char pciBusId[80];
// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
hipDeviceGetPCIBusId(pciBusId, 80, 0);


// CHECK: dpct::get_device_manager().current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::get_device_manager().current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK: dpct::get_device_manager().current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::get_device_manager().current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::get_device_manager().select_device(device2), 0);
error_code = hipSetDevice(device2);
// CHECK: dpct::get_device_manager().select_device(device2);
hipSetDevice(device2);

// CHECK:dpct::get_device_manager().current_device().queues_wait_and_throw();
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int err = (dpct::get_device_manager().current_device().queues_wait_and_throw(), 0);
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkError((dpct::get_device_manager().current_device().queues_wait_and_throw(), 0));
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:return (dpct::get_device_manager().current_device().queues_wait_and_throw(), 0);
hipDeviceSynchronize();
hipError_t err = hipDeviceSynchronize();
checkError(hipDeviceSynchronize());
return hipDeviceSynchronize();
// CHECK:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e = 0;
int e = hipGetLastError();
// CHECK:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e1 = 0;
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: 0;
int e1 = hipPeekAtLastError();
hipPeekAtLastError();
// CHECK:dpct::get_device_manager().current_device().queues_wait_and_throw();
hipDeviceSynchronize();
return 0;
}
