// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device002.dp.cpp

#include <stdio.h>

void checkError(hipError_t err) {

}

int main(int argc, char **argv)
{
int devID = atoi(argv[1]);
hipDeviceProp_t cdp;
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int error_code = (dpct::device_manager::get_instance().get_device(devID).get_device_info(cdp), 0);
hipError_t error_code = hipGetDeviceProperties(&cdp, devID);

if (error_code == hipSuccess) {
// CHECK: /*
// CHECK-NEXT:  DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: /*
// CHECK-NEXT:  DPCT1006:{{[0-9]+}}: DPC++ does not provide a standard API to differentiate between integrated/ discrete GPU devices.
// CHECK-NEXT: */
// CHECK-NEXT:if (cdp.get_major_version() < 3 && cdp.get_integrated() != 1) {
    if (cdp.major < 3 && cdp.integrated != 1) {
            printf("do_complex_compute requires compute capability 3.0 or later and not integrated\n");
    }
}

int deviceCount = 0;
// CHECK: deviceCount = dpct::device_manager::get_instance().device_count();
hipGetDeviceCount(&deviceCount);

int dev_id;
// CHECK: dev_id = dpct::device_manager::get_instance().current_device_id();
hipGetDevice(&dev_id);

hipDeviceProp_t deviceProp;
// CHECK: dpct::device_manager::get_instance().get_device(0).get_device_info(deviceProp);
hipGetDeviceProperties(&deviceProp, 0);

int atomicSupported;
// CHECK: atomicSupported = dpct::device_manager::get_instance().get_device(dev_id).is_native_atomic_supported();
hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);

int val;
// CHECK: val = dpct::device_manager::get_instance().get_device(dev_id).get_major_version();
hipDeviceGetAttribute(&val, hipDeviceAttributeComputeCapabilityMajor, dev_id);

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: accessSupported = 0;
hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2);

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: perfRank = 0;
hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2);

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: atomicSupported = 0;
hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2);


char pciBusId[80];
// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
hipDeviceGetPCIBusId(pciBusId, 80, 0);


// CHECK: dpct::device_manager::get_instance().current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::device_manager::get_instance().current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK: dpct::device_manager::get_instance().current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::device_manager::get_instance().current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::device_manager::get_instance().select_device(device2), 0);
error_code = hipSetDevice(device2);
// CHECK: dpct::device_manager::get_instance().select_device(device2);
hipSetDevice(device2);

// CHECK:dpct::device_manager::get_instance().current_device().queues_wait_and_throw();
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int err = (dpct::device_manager::get_instance().current_device().queues_wait_and_throw(), 0);
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkError((dpct::device_manager::get_instance().current_device().queues_wait_and_throw(), 0));
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:return (dpct::device_manager::get_instance().current_device().queues_wait_and_throw(), 0);
hipDeviceSynchronize();
hipError_t err = hipDeviceSynchronize();
checkError(hipDeviceSynchronize());
return hipDeviceSynchronize();
// CHECK:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e = 0;
int e = hipGetLastError();
// CHECK:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e1 = 0;
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: 0;
int e1 = hipPeekAtLastError();
hipPeekAtLastError();
// CHECK:dpct::device_manager::get_instance().current_device().queues_wait_and_throw();
hipDeviceSynchronize();
return 0;
}
