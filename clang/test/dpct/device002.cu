// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device002.dp.cpp

#include <stdio.h>

void checkError(hipError_t err) {

}

int main(int argc, char **argv)
{
int devID = atoi(argv[1]);
hipDeviceProp_t cdp;
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int error_code = (dpct::dev_mgr::instance().get_device(devID).get_device_info(cdp), 0);
hipError_t error_code = hipGetDeviceProperties(&cdp, devID);

if (error_code == hipSuccess) {
// CHECK: /*
// CHECK-NEXT:  DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: /*
// CHECK-NEXT:  DPCT1006:{{[0-9]+}}: DPC++ does not provide a standard API to differentiate between integrated/ discrete GPU devices.
// CHECK-NEXT: */
// CHECK-NEXT:if (cdp.get_major_version() < 3 && cdp.get_integrated() != 1) {
    if (cdp.major < 3 && cdp.integrated != 1) {
            printf("do_complex_compute requires compute capability 3.0 or later and not integrated\n");
    }
}

int deviceCount = 0;
// CHECK: deviceCount = dpct::dev_mgr::instance().device_count();
hipGetDeviceCount(&deviceCount);

int dev_id;
// CHECK: dev_id = dpct::dev_mgr::instance().current_device_id();
hipGetDevice(&dev_id);

hipDeviceProp_t deviceProp;
// CHECK: dpct::dev_mgr::instance().get_device(0).get_device_info(deviceProp);
hipGetDeviceProperties(&deviceProp, 0);

int atomicSupported;
// CHECK: atomicSupported = dpct::dev_mgr::instance().get_device(dev_id).is_native_atomic_supported();
hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);

int val;
// CHECK: val = dpct::dev_mgr::instance().get_device(dev_id).get_major_version();
hipDeviceGetAttribute(&val, hipDeviceAttributeComputeCapabilityMajor, dev_id);

int computeMode = -1, minor = 0;
// CHECK: /*
// CHECK-NEXT: DPCT1035:{{[0-9]+}}: All DPC++ devices can be used by host to submit tasks. You may need to adjust this code.
// CHECK-NEXT: */
// CHECK-NEXT: checkError((computeMode = 1, 0));
checkError(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, dev_id));
// CHECK: checkError((minor = dpct::dev_mgr::instance().get_device(dev_id).get_minor_version(), 0));
checkError(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, dev_id));

int multiProcessorCount = 0, clockRate = 0;
// CHECK: checkError((multiProcessorCount = dpct::dev_mgr::instance().get_device(dev_id).get_max_compute_units(), 0));
checkError(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, dev_id));
// CHECK: checkError((clockRate = dpct::dev_mgr::instance().get_device(dev_id).get_max_clock_frequency(), 0));
checkError(hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, dev_id));

int integrated = -1;
// CHECK: checkError((integrated = dpct::dev_mgr::instance().get_device(dev_id).get_integrated(), 0));
checkError(hipDeviceGetAttribute(&integrated, hipDeviceAttributeIntegrated, dev_id));

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: accessSupported = 0;
hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2);

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: perfRank = 0;
hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2);

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: atomicSupported = 0;
hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2);


char pciBusId[80];
// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
hipDeviceGetPCIBusId(pciBusId, 80, 0);


// CHECK: dpct::get_current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::get_current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK: dpct::get_current_device().reset();
hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::get_current_device().reset(), 0);
error_code = hipDeviceReset();

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:error_code = (dpct::dev_mgr::instance().select_device(device2), 0);
error_code = hipSetDevice(device2);
// CHECK: dpct::dev_mgr::instance().select_device(device2);
hipSetDevice(device2);

// CHECK:dpct::get_current_device().queues_wait_and_throw();
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int err = (dpct::get_current_device().queues_wait_and_throw(), 0);
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkError((dpct::get_current_device().queues_wait_and_throw(), 0));
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:return (dpct::get_current_device().queues_wait_and_throw(), 0);
hipDeviceSynchronize();
hipError_t err = hipDeviceSynchronize();
checkError(hipDeviceSynchronize());
return hipDeviceSynchronize();
// CHECK:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e = 0;
int e = hipGetLastError();
// CHECK:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: int e1 = 0;
// CHECK-NEXT:/*
// CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: 0;
int e1 = hipPeekAtLastError();
hipPeekAtLastError();
// CHECK:dpct::get_current_device().queues_wait_and_throw();
hipDeviceSynchronize();
return 0;
}
