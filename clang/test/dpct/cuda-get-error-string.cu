
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s -- -std=c++14 -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda-get-error-string.dp.cpp

int printf(const char *format, ...);

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR(X) printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(X)*/)
#define PRINT_ERROR_STR(X) printf("%s\n", hipGetErrorString(X))

// CHECK:  /*
// CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT: #define PRINT_ERROR_STR2(X)\
// CHECK-NEXT:  printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(X)*/)
#define PRINT_ERROR_STR2(X)\
  printf("%s\n", hipGetErrorString(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR3(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          \n", "cudaGetErrorString not supported"/*cudaGetErrorString(X)*/)
#define PRINT_ERROR_STR3(X)\
  printf("%s\
         \n", hipGetErrorString(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorName is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_NAME(X) printf("%s\n", "cudaGetErrorName not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_NAME(X) printf("%s\n", hipGetErrorName(X))

// CHECK:   /*
// CHECK-NEXT:   DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorName is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT:   */
// CHECK-NEXT: #define PRINT_ERROR_NAME2(X)\
// CHECK-NEXT:   printf("%s\n", "cudaGetErrorName not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_NAME2(X)\
  printf("%s\n", hipGetErrorName(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorName is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_NAME3(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          \n", "cudaGetErrorName not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_NAME3(X)\
  printf("%s\
         \n", hipGetErrorName(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorName is commented out and a warning string is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR_NAME(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          %s\
// CHECK-NEXT:          \n", "cudaGetErrorString not supported"/*cudaGetErrorString(X)*/,\
// CHECK-NEXT:          "cudaGetErrorName not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_STR_NAME(X)\
  printf("%s\
         %s\
         \n", hipGetErrorString(X),\
         hipGetErrorName(X))

const char *test_function() {
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME(hipGetLastError());
  PRINT_ERROR_STR(hipSuccess);
  PRINT_ERROR_NAME(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR2(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME2(hipGetLastError());
  PRINT_ERROR_STR2(hipSuccess);
  PRINT_ERROR_NAME2(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR3(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME3(hipGetLastError());
  PRINT_ERROR_STR3(hipSuccess);
  PRINT_ERROR_NAME3(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR_NAME(hipGetLastError());
  PRINT_ERROR_STR_NAME(hipSuccess);

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetLastError was replaced with 0. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(0)*/);
  printf("%s\n", hipGetErrorString(hipGetLastError()));


//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorString is commented out and a warning string is inserted. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", "cudaGetErrorString not supported"/*cudaGetErrorString(0)*/);
  printf("%s\n", hipGetErrorString(hipSuccess));

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorName is commented out and a warning string is inserted. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:printf("%s\n", "cudaGetErrorName not supported"/*cudaGetErrorName(0)*/);
  printf("%s\n", hipGetErrorName(hipSuccess));

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL API uses exceptions to report errors and doesn't use the error codes. Hence, cudaGetErrorName is commented out and a warning string is inserted. You may need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  return "cudaGetErrorName not supported"/*cudaGetErrorName(0)*/;
  return hipGetErrorName(hipSuccess);
}
