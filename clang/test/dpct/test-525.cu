// RUN: c2s --format-range=none -out-root %T/test-525 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/test-525/test-525.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
class C {
  int nDevices;
public:
  void problem() {
    // CHECK: nDevices = c2s::dev_mgr::instance().device_count();
    hipGetDeviceCount(&nDevices);
  }
};

