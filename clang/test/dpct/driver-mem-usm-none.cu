// RUN: dpct --usm-level=none --format-range=none -out-root %T/driver-mem-usm-none %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem-usm-none/driver-mem-usm-none.dp.cpp %s

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
int main(){
    size_t result1, result2;
    int size = 32;
    float* f_A;
    // CHECK: f_A = (float *)malloc(size);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);


    // CHECK: void * f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: f_D = dpct::dpct_malloc(size);
    hipMalloc(&f_D, size);

    // CHECK: sycl::queue * stream;
    hipStream_t stream;
    // CHECK: dpct::async_dpct_memcpy(f_D, f_A, size, stream);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_D, f_A, size, 0);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: dpct::dpct_memcpy(f_D, f_A, size);
    hipMemcpyHtoD(f_D, f_A, size);

    // CHECK: dpct::async_dpct_memcpy(f_A, f_D, size, stream);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_A, f_D, size, 0);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: dpct::dpct_memcpy(f_A, f_D, size);
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: dpct::pitched_data cpy_from_data_ct1, cpy_to_data_ct1;
    // CHECK: sycl::id<3> cpy_from_pos_ct1(0, 0, 0), cpy_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy_size_ct1(1, 1, 1);
    hip_Memcpy2D cpy;
    //
    cpy.dstMemoryType = hipMemoryTypeHost;
    // CHECK: cpy_to_data_ct1.set_data_ptr(f_A);
    cpy.dstHost = f_A;
    // CHECK: cpy_to_data_ct1.set_pitch(20);
    cpy.dstPitch = 20;
    // CHECK: cpy_to_pos_ct1[1] = 10;
    cpy.dstY = 10;
    // CHECK: cpy_to_pos_ct1[0] = 15;
    cpy.dstXInBytes = 15;

    //
    cpy.srcMemoryType = hipMemoryTypeDevice;
    // CHECK: cpy_from_data_ct1.set_data_ptr(f_D);
    cpy.srcDevice = f_D;
    // CHECK: cpy_from_data_ct1.set_pitch(20);
    cpy.srcPitch = 20;
    // CHECK: cpy_from_pos_ct1[1] = 10;
    cpy.srcY = 10;
    // CHECK: cpy_from_pos_ct1[0] = 15;
    cpy.srcXInBytes = 15;

    // CHECK: cpy_size_ct1[0] = 4;
    cpy.WidthInBytes = 4;
    // CHECK: cpy_size_ct1[1] = 7;
    cpy.Height = 7;

    // CHECK: dpct::dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1);
    hipMemcpyParam2D(&cpy);
    // CHECK: dpct::async_dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1, dpct::automatic, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    // CHECK: dpct::pitched_data cpy2_from_data_ct1, cpy2_to_data_ct1;
    // CHECK: sycl::id<3> cpy2_from_pos_ct1(0, 0, 0), cpy2_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy2_size_ct1(1, 1, 1);
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    //
    cpy2.dstMemoryType = hipMemoryTypeArray;
    // CHECK: cpy2_to_data_ct1 = ca->to_pitched_data();
    cpy2.dstArray = ca;
    // CHECK: cpy2_to_data_ct1.set_pitch(5);
    cpy2.dstPitch = 5;
    // CHECK: cpy2_to_data_ct1.set_y(4);
    cpy2.dstHeight = 4;
    // CHECK: cpy2_to_pos_ct1[1] = 3;
    cpy2.dstY = 3;
    // CHECK: cpy2_to_pos_ct1[2] = 2;
    cpy2.dstZ = 2;
    // CHECK: cpy2_to_pos_ct1[0] = 1;
    cpy2.dstXInBytes = 1;
    //
    cpy2.dstLOD = 0;

    //
    cpy2.srcMemoryType = hipMemoryTypeHost;
    // CHECK: cpy2_from_data_ct1.set_data_ptr(f_A);
    cpy2.srcHost = f_A;
    // CHECK: cpy2_from_data_ct1.set_pitch(5);
    cpy2.srcPitch = 5;
    // CHECK: cpy2_from_data_ct1.set_y(4);
    cpy2.srcHeight = 4;
    // CHECK: cpy2_from_pos_ct1[1] = 3;
    cpy2.srcY = 3;
    // CHECK: cpy2_from_pos_ct1[2] = 2;
    cpy2.srcZ = 2;
    // CHECK: cpy2_from_pos_ct1[0] = 1;
    cpy2.srcXInBytes = 1;
    //
    cpy2.srcLOD = 0;

    // CHECK: cpy2_size_ct1[0] = 3;
    cpy2.WidthInBytes = 3;
    // CHECK: cpy2_size_ct1[1] = 2;
    cpy2.Height = 2;
    // CHECK: cpy2_size_ct1[2] = 1;
    cpy2.Depth = 1;

    // CHECK: dpct::dpct_memcpy(cpy2_to_data_ct1, cpy2_to_pos_ct1, cpy2_from_data_ct1, cpy2_from_pos_ct1, cpy2_size_ct1);
    hipDrvMemcpy3D(&cpy2);

    return 0;
}