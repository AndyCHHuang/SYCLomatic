// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/ctst-525.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
class C {
  int nDevices;
public:
  void problem() {
    // CHECK: nDevices = dpct::get_device_manager().device_count();
    hipGetDeviceCount(&nDevices);
  }
};
