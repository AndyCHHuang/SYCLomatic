// RUN: dpct --format-range=none -out-root %T/ctst-525 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/ctst-525/ctst-525.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
class C {
  int nDevices;
public:
  void problem() {
    // CHECK: nDevices = dpct::dev_mgr::instance().device_count();
    hipGetDeviceCount(&nDevices);
  }
};

