// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/ctst-525.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
class C {
  int nDevices;
public:
  void problem() {
    // CHECK: nDevices = dpct::device_manager::get_instance().device_count();
    hipGetDeviceCount(&nDevices);
  }
};
