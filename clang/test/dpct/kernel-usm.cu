// RUN: dpct --format-range=none -out-root %T %s --usm-level=restricted --cuda-include-path="%cuda-path/include" --sycl-named-lambda  -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/kernel-usm.dp.cpp

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <vector>

// CHECK: void testDevice(const int *K) {
// CHECK-NEXT: int t = K[0];
// CHECK-NEXT: }
__device__ void testDevice(const int *K) {
  int t = K[0];
}

// CHECK: void testKernelPtr(const int *L, const int *M, int N, sycl::nd_item<3> item_ct1) {
// CHECK-NEXT: testDevice(L);
// CHECK-NEXT: int gtid = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  testDevice(L);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  dim3 griddim = 2;
  dim3 threaddim = 32;
  int *karg1, *karg2;
  // CHECK: karg1 = sycl::malloc_device<int>(32 , q_ct1);
  // CHECK-NEXT: karg2 = sycl::malloc_device<int>(32 , q_ct1);
  hipMalloc(&karg1, 32 * sizeof(int));
  hipMalloc(&karg2, 32 * sizeof(int));

  int karg3 = 80;
  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(dpct_global_range, threaddim),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernelPtr((const int *)karg1, karg2, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);
}

// CHECK:dpct::shared_memory<float, 1> result(32);
// CHECK-NEXT:void my_kernel(float* result, sycl::nd_item<3> item_ct1, float *resultInGroup) {
// CHECK-NEXT:  // __shared__ variable
// CHECK-NEXT:  resultInGroup[item_ct1.get_local_id(2)] = item_ct1.get_group(2);
// CHECK-NEXT:  memcpy(&result[item_ct1.get_group(2)*8], resultInGroup, sizeof(float)*8);
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo5 () {
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      auto result_ct0 = result.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result[10]);
// CHECK-NEXT:}
__managed__ __device__ float result[32];
__global__ void my_kernel(float* result) {
  __shared__ float resultInGroup[8]; // __shared__ variable
  resultInGroup[threadIdx.x] = blockIdx.x;
  memcpy(&result[blockIdx.x*8], resultInGroup, sizeof(float)*8);
}
int run_foo5 () {
  my_kernel<<<4, 8>>>(result);
  printf("%f ", result[10]);
}

// CHECK:dpct::shared_memory<float, 1> result2(32);
// CHECK-NEXT:int run_foo6 () {
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      auto result2_ct0 = result2.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result2_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result2[10]);
// CHECK-NEXT:}
__managed__ float result2[32];
int run_foo6 () {
  my_kernel<<<4, 8>>>(result2);
  printf("%f ", result2[10]);
}

// CHECK:dpct::shared_memory<float, 0> result3;
// CHECK-NEXT:int run_foo7 () {
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      auto result3_ct0 = result3.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result3_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result3[0]);
// CHECK-NEXT:}
__managed__ float result3;
int run_foo7 () {
  my_kernel<<<4, 8>>>(&result3);
  printf("%f ", result3);
}

// CHECK:dpct::shared_memory<float, 0> in;
// CHECK-NEXT:dpct::shared_memory<float, 0> out;
// CHECK-NEXT:void my_kernel2(float in, float *out, sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:  if (item_ct1.get_local_id(2) == 0) {
// CHECK-NEXT:    memcpy(out, &in, sizeof(float));
// CHECK-NEXT:  }
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo8() {
// CHECK-NEXT:  in[0] = 42;
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      auto in_ct0 = in[0];
// CHECK-NEXT:      auto out_ct1 = out.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel2_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel2(in_ct0, out_ct1, item_ct1);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", out[0]);
// CHECK-NEXT:}
__managed__ float in;
__managed__ float out;
__global__ void my_kernel2(float in, float *out) {
  if (threadIdx.x == 0) {
    memcpy(out, &in, sizeof(float));
  }
}
int run_foo8() {
  in = 42;
  my_kernel2<<<4, 8>>>(in, &out);
  printf("%f ", out);
}

struct A{
  int a;
  int* get_pointer(){
    return &a;
  }
};

__global__ void k(int *p){}

// CHECK:int run_foo9() {
// CHECK-NEXT:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT:  sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT:  std::vector<A> vec(10);
// CHECK-NEXT:  A aa;
// CHECK-NEXT:  q_ct1.submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      auto aa_get_pointer_ct0 = aa.get_pointer();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class k_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          k(aa_get_pointer_ct0);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  q_ct1.submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      auto vec_get_pointer_ct0 = vec[2].get_pointer();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class k_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          k(vec_get_pointer_ct0);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:}
int run_foo9() {
  std::vector<A> vec(10);
  A aa;
  k<<<1,1>>>(aa.get_pointer());
  k<<<1,1>>>(vec[2].get_pointer());
}

// CHECK:void cuda_pme_forces_dev(float **afn_s) {
// CHECK-NEXT:  // __shared__ variable
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo10() {
// CHECK-NEXT: dpct::get_default_queue().submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::accessor<float *, 1, sycl::access::mode::read_write, sycl::access::target::local> afn_s_acc_ct1(sycl::range<1>(3), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class cuda_pme_forces_dev_{{[0-9a-z]+}}>>(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         cuda_pme_forces_dev(afn_s_acc_ct1.get_pointer());
// CHECK-NEXT:       });
// CHECK-NEXT:   });
// CHECK-NEXT:}
__global__ void cuda_pme_forces_dev() {
  __shared__ float *afn_s[3]; // __shared__ variable
}
int run_foo10() {
  cuda_pme_forces_dev<<<1,1>>>();
}

__global__ void my_kernel3(){}
int run_foo11() {
  // CHECK:q_ct1.submit(
  // CHECK-NEXT:  [&](sycl::handler &cgh) {
  // CHECK-NEXT:    cgh.parallel_for<dpct_kernel_name<class my_kernel3_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        my_kernel3();
  // CHECK-NEXT:      });
  // CHECK-NEXT:  });
  // CHECK-NEXT:q_ct1.submit(
  // CHECK-NEXT:  [&](sycl::handler &cgh) {
  // CHECK-NEXT:    cgh.parallel_for<dpct_kernel_name<class my_kernel3_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        my_kernel3();
  // CHECK-NEXT:      });
  // CHECK-NEXT:  });
  // CHECK-NEXT:q_ct1.submit(
  // CHECK-NEXT:  [&](sycl::handler &cgh) {
  // CHECK-NEXT:    cgh.parallel_for<dpct_kernel_name<class my_kernel3_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        my_kernel3();
  // CHECK-NEXT:      });
  // CHECK-NEXT:  });
  my_kernel3<<<1,1,1,hipStreamDefault>>>();
  my_kernel3<<<1,1,1,hipStreamPerThread>>>();
  my_kernel3<<<1,1,1,hipStreamLegacy>>>();
}

int *g_a;

__global__ void foo_kernel3(int *d) {
}
//CHECK:void run_foo(sycl::range<3> c, sycl::range<3> d) {
//CHECK-NEXT:  if (1)
//CHECK-NEXT:      dpct::get_default_queue().submit(
//CHECK-NEXT:        [&](sycl::handler &cgh) {
//CHECK-NEXT:          auto g_a_ct0 = &g_a[0];
//CHECK-EMPTY:
//CHECK-NEXT:          cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:            sycl::nd_range<3>(c, sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:            [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:              foo_kernel3(g_a_ct0);
//CHECK-NEXT:            });
//CHECK-NEXT:        });
//CHECK-NEXT:    }
void run_foo(dim3 c, dim3 d) {
  if (1)
    foo_kernel3<<<c, 1>>>(&g_a[0]);
}
