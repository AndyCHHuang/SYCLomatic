// RUN: dpct --format-range=none -out-root %T %s --usm-level=restricted --cuda-include-path="%cuda-path/include" --sycl-named-lambda  -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/kernel-usm.dp.cpp

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>

// CHECK: void testDevice(const int *K) {
// CHECK-NEXT: int t = K[0];
// CHECK-NEXT: }
__device__ void testDevice(const int *K) {
  int t = K[0];
}

// CHECK: void testKernelPtr(const int *L, const int *M, int N, sycl::nd_item<3> item_ct1) {
// CHECK-NEXT: testDevice(L);
// CHECK-NEXT: int gtid = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  testDevice(L);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  int *karg1, *karg2;
  // CHECK: karg1 = sycl::malloc_device<int>(32 , dpct::get_default_queue());
  // CHECK-NEXT: karg2 = sycl::malloc_device<int>(32 , dpct::get_default_queue());
  hipMalloc(&karg1, 32 * sizeof(int));
  hipMalloc(&karg2, 32 * sizeof(int));

  int karg3 = 80;
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernelPtr((const int *)karg1, karg2, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);
}

// CHECK:dpct::shared_memory<float, 1> result(32);
// CHECK-NEXT:void my_kernel(float* result, sycl::nd_item<3> item_ct1, float *resultInGroup) {
// CHECK-NEXT:  // __shared__ variable
// CHECK-NEXT:  resultInGroup[item_ct1.get_local_id(2)] = item_ct1.get_group(2);
// CHECK-NEXT:  memcpy(&result[item_ct1.get_group(2)*8], resultInGroup, sizeof(float)*8);
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo5 () {
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      auto result_ct0 = result.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result[10]);
// CHECK-NEXT:}
__managed__ __device__ float result[32];
__global__ void my_kernel(float* result) {
  __shared__ float resultInGroup[8]; // __shared__ variable
  resultInGroup[threadIdx.x] = blockIdx.x;
  memcpy(&result[blockIdx.x*8], resultInGroup, sizeof(float)*8);
}
int run_foo5 () {
  my_kernel<<<4, 8>>>(result);
  printf("%f ", result[10]);
}

// CHECK:dpct::shared_memory<float, 1> result2(32);
// CHECK-NEXT:int run_foo6 () {
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      auto result2_ct0 = result2.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result2_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result2[10]);
// CHECK-NEXT:}
__managed__ float result2[32];
int run_foo6 () {
  my_kernel<<<4, 8>>>(result2);
  printf("%f ", result2[10]);
}

// CHECK:dpct::shared_memory<float, 0> result3;
// CHECK-NEXT:int run_foo7 () {
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      auto result3_ct0 = result3.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result3_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result3[0]);
// CHECK-NEXT:}
__managed__ float result3;
int run_foo7 () {
  my_kernel<<<4, 8>>>(&result3);
  printf("%f ", result3);
}

// CHECK:dpct::shared_memory<float, 0> in;
// CHECK-NEXT:dpct::shared_memory<float, 0> out;
// CHECK-NEXT:void my_kernel2(float in, float *out, sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:  if (item_ct1.get_local_id(2) == 0) {
// CHECK-NEXT:    memcpy(out, &in, sizeof(float));
// CHECK-NEXT:  }
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo8() {
// CHECK-NEXT:  in[0] = 42;
// CHECK-NEXT:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      auto in_ct0 = in[0];
// CHECK-NEXT:      auto out_ct1 = out.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel2_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel2(in_ct0, out_ct1, item_ct1);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", out[0]);
// CHECK-NEXT:}
__managed__ float in;
__managed__ float out;
__global__ void my_kernel2(float in, float *out) {
  if (threadIdx.x == 0) {
    memcpy(out, &in, sizeof(float));
  }
}
int run_foo8() {
  in = 42;
  my_kernel2<<<4, 8>>>(in, &out);
  printf("%f ", out);
}
