// RUN: dpct --format-range=none -out-root %T %s --usm-level=restricted --cuda-include-path="%cuda-path/include" --sycl-named-lambda  -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/kernel-usm.dp.cpp

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>

// CHECK: void testDevice(const int *K) {
// CHECK-NEXT: int t = K[0];
// CHECK-NEXT: }
__device__ void testDevice(const int *K) {
  int t = K[0];
}

// CHECK: void testKernelPtr(const int *L, const int *M, int N,
// CHECK-NEXT: sycl::nd_item<3> item_ct1) {
// CHECK-NEXT: testDevice(L);
// CHECK-NEXT: int gtid = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  testDevice(L);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  int *karg1, *karg2;
  // CHECK: karg1 = (int *)sycl::malloc_device(32 * sizeof(int), dpct::get_current_device(), dpct::get_default_context());
  hipMalloc(&karg1, 32 * sizeof(int));
  // CHECK: karg2 = (int *)sycl::malloc_device(32 * sizeof(int), dpct::get_current_device(), dpct::get_default_context());
  hipMalloc(&karg2, 32 * sizeof(int));

  int karg3 = 80;
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernelPtr((const int *)karg1, karg2, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);
}

// CHECK:dpct::shared_memory<float, 1> result(32);
// CHECK-NEXT:void my_kernel(float* result, sycl::nd_item<3> item_ct1, float *resultInGroup) {
// CHECK-NEXT:  // __shared__ variable
// CHECK-NEXT:  resultInGroup[item_ct1.get_local_id(2)] = item_ct1.get_group(2);
// CHECK-NEXT:  memcpy(&result[item_ct1.get_group(2)*8], resultInGroup, sizeof(float)*8);
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo5 () {
// CHECK-NEXT:  {
// CHECK-NEXT:    auto result_ct0 = result.get_ptr();
// CHECK-NEXT:    dpct::get_default_queue().submit(
// CHECK-NEXT:      [&](sycl::handler &cgh) {
// CHECK-NEXT:        sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            my_kernel(result_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:          });
// CHECK-NEXT:      });
// CHECK-NEXT:  }
// CHECK-NEXT:  printf("%f ", result[10]);
// CHECK-NEXT:}
__managed__ __device__ float result[32];
__global__ void my_kernel(float* result) {
  __shared__ float resultInGroup[8]; // __shared__ variable
  resultInGroup[threadIdx.x] = blockIdx.x;
  memcpy(&result[blockIdx.x*8], resultInGroup, sizeof(float)*8);
}
int run_foo5 () {
  my_kernel<<<4, 8>>>(result);
  printf("%f ", result[10]);
}

// CHECK:dpct::shared_memory<float, 1> result2(32);
// CHECK-NEXT:int run_foo6 () {
// CHECK-NEXT:  {
// CHECK-NEXT:    auto result2_ct0 = result2.get_ptr();
// CHECK-NEXT:    dpct::get_default_queue().submit(
// CHECK-NEXT:      [&](sycl::handler &cgh) {
// CHECK-NEXT:        sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            my_kernel(result2_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:          });
// CHECK-NEXT:      });
// CHECK-NEXT:  }
// CHECK-NEXT:  printf("%f ", result2[10]);
// CHECK-NEXT:}
__managed__ float result2[32];
int run_foo6 () {
  my_kernel<<<4, 8>>>(result2);
  printf("%f ", result2[10]);
}

// CHECK:dpct::shared_memory<float, 0> result3;
// CHECK-NEXT:int run_foo7 () {
// CHECK-NEXT:  {
// CHECK-NEXT:    auto result3_ct0 = result3.get_ptr();
// CHECK-NEXT:    dpct::get_default_queue().submit(
// CHECK-NEXT:      [&](sycl::handler &cgh) {
// CHECK-NEXT:        sycl::accessor<float, 1, sycl::access::mode::read_write, sycl::access::target::local> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            my_kernel(result3_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:          });
// CHECK-NEXT:      });
// CHECK-NEXT:  }
// CHECK-NEXT:  printf("%f ", result3[0]);
// CHECK-NEXT:}
__managed__ float result3;
int run_foo7 () {
  my_kernel<<<4, 8>>>(&result3);
  printf("%f ", result3);
}