// RUN: cd %S/../build
// RUN: dpct -in-root ../src -out-root=%T -p ./  --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/kernel.dp.cpp

#include "hip/hip_runtime.h"
#include <stdio.h>

// CHECK: void kernel(){}
__global__ void kernel(){}
