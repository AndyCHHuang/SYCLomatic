// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture_object.dp.cpp --match-full-lines %s
// UNSUPPORTED: cdua-9.0, cuda-9.2, cuda-10.0, cuda-10.1
// UNSUPPORTED: v9.0, v9.2, v10.0, v10.1


#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheck(stmt) do {                         \
  hipError_t err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

void func(int i) {}

template <typename T>
void funcT(T t) {}

// CHECK: void device01(dpct::image_accessor<sycl::uint2, 1> tex21) {
// CHECK-NEXT: sycl::uint2 u21;
// CHECK-NEXT: dpct::read_image(&u21, tex21, 0.5f);
// CHECK-NEXT: dpct::read_image(&u21, tex21, 1);
__device__ void device01(hipTextureObject_t tex21) {
  uint2 u21;
  tex1D(&u21, tex21, 0.5f);
  tex1Dfetch(&u21, tex21, 1);
}

// CHECK: void kernel(dpct::image_accessor<sycl::uint2, 1> tex21, dpct::image_accessor<sycl::float4, 2> tex42) {
// CHECK-NEXT: device01(tex21);
// CHECK-NEXT: sycl::float4 f42;
// CHECK-NEXT: dpct::read_image(&f42, tex42, 0.5f, 0.5f);
/// Texture accessors should be passed down to __global__/__device__ function if used.
__global__ void kernel(hipTextureObject_t tex21, hipTextureObject_t tex42) {
  device01(tex21);
  float4 f42;
  tex2D(&f42, tex42, 0.5f, 0.5f);
}

int main() {

  // CHECK: sycl::float4 *d_data42;
  // CHECK-NEXT: dpct::image_matrix_p a42;
  // CHECK-NEXT: dpct::dpct_malloc(&d_data42, sizeof(sycl::float4) * 32 * 32);
  // CHECK-NEXT: dpct::image_channel desc42 = dpct::create_image_channel(32, 32, 32, 32, dpct::channel_float);
  // CHECK-NEXT: dpct::malloc_matrix(&a42, &desc42, 32, 32);
  // CHECK-NEXT: dpct::memcpy_to_matrix(a42, 0, 0, d_data42, 32 * 32 * sizeof(sycl::float4));
  // CHECK-NEXT: dpct::image_base_p tex42;
  // CHECK-NEXT: dpct::image_data res42;
  // CHECK-NEXT: dpct::image_info texDesc42;
  // CHECK-NEXT: res42.type = dpct::data_matrix;
  // CHECK-NEXT: res42.data.matrix = a42;
  // CHECK-NEXT: texDesc42.addr_mode() = sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: texDesc42.addr_mode() = sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: texDesc42.addr_mode() = sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: texDesc42.filter_mode() = sycl::filtering_mode::nearest;
  // CHECK-NEXT: dpct::create_image(&tex42, &res42, &texDesc42);
  float4 *d_data42;
  hipArray_t a42;
  hipMalloc(&d_data42, sizeof(float4) * 32 * 32);
  hipChannelFormatDesc desc42 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipMallocArray(&a42, &desc42, 32, 32);
  hipMemcpyToArray(a42, 0, 0, d_data42, 32 * 32 * sizeof(float4), hipMemcpyDeviceToDevice);
  hipTextureObject_t tex42;
  hipResourceDesc res42;
  hipTextureDesc texDesc42;
  res42.resType = hipResourceTypeArray;
  res42.res.array.array = a42;
  texDesc42.addressMode[0] = hipAddressModeClamp;
  texDesc42.addressMode[1] = hipAddressModeClamp;
  texDesc42.addressMode[2] = hipAddressModeClamp;
  texDesc42.filterMode = hipFilterModePoint;
  hipCreateTextureObject(&tex42, &res42, &texDesc42, NULL);

  // CHECK: sycl::uint2 *d_data21;
  // CHECK-NEXT: dpct::dpct_malloc(&d_data21, sizeof(sycl::uint2) * 32);
  // CHECK-NEXT: dpct::image_channel desc21 = dpct::create_image_channel(32, 32, 0, 0, dpct::channel_unsigned);
  // CHECK-NEXT: dpct::image_base_p tex21;
  // CHECK-NEXT: dpct::image_data res21;
  // CHECK-NEXT: dpct::image_info texDesc21;
  // CHECK-NEXT: res21.type = dpct::data_linear;
  // CHECK-NEXT: res21.data.linear.data = d_data21;
  // CHECK-NEXT: res21.data.linear.size = sizeof(sycl::uint2) * 32;
  // CHECK-NEXT: res21.data.linear.chn = desc21;
  // CHECK-NEXT: texDesc21.addr_mode() = sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: texDesc21.addr_mode() = sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: texDesc21.addr_mode() = sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: texDesc21.filter_mode() = sycl::filtering_mode::linear;
  // CHECK-NEXT: dpct::create_image(&tex21, &res21, &texDesc21);
  uint2 *d_data21;
  hipMalloc(&d_data21, sizeof(uint2) * 32);
  hipChannelFormatDesc desc21 = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindUnsigned);
  hipTextureObject_t tex21;
  hipResourceDesc res21;
  hipTextureDesc texDesc21;
  res21.resType = hipResourceTypeLinear;
  res21.res.linear.devPtr = d_data21;
  res21.res.linear.sizeInBytes = sizeof(uint2) * 32;
  res21.res.linear.desc = desc21;
  texDesc21.addressMode[0] = hipAddressModeClamp;
  texDesc21.addressMode[1] = hipAddressModeClamp;
  texDesc21.addressMode[2] = hipAddressModeClamp;
  texDesc21.filterMode = hipFilterModeLinear;
  hipCreateTextureObject(&tex21, &res21, &texDesc21, NULL);

  // CHECK: dpct::get_default_queue().submit(
  // CHECK-NEXT:   [&](sycl::handler &cgh) {
  // CHECK-NEXT:     auto tex21_acc = static_cast<dpct::image<sycl::uint2, 1> *>(tex21)->get_access(cgh);
  // CHECK-NEXT:     auto tex42_acc = static_cast<dpct::image<sycl::float4, 2> *>(tex42)->get_access(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         kernel(tex21_acc, tex42_acc);
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  kernel<<<1, 1>>>(tex21, tex42);

  // CHECK: dpct::dpct_free(tex42);
  // CHECK-NEXT: dpct::dpct_free(tex21);
  hipDestroyTextureObject(tex42);
  hipDestroyTextureObject(tex21);

  // CHECK: dpct::dpct_free(a42);
  hipFreeArray(a42);

  // CHECK: dpct::dpct_free(d_data42);
  // CHECK-NEXT: dpct::dpct_free(d_data21);
  hipFree(d_data42);
  hipFree(d_data21);

  // Test IsAssigned
  {
    int errorCode;
    // CHECK: errorCode = (dpct::create_image(&tex21, &res21, &texDesc21), 0);
    errorCode = hipCreateTextureObject(&tex21, &res21, &texDesc21, NULL);
    // CHECK: cudaCheck((dpct::create_image(&tex21, &res21, &texDesc21), 0));
    cudaCheck(hipCreateTextureObject(&tex21, &res21, &texDesc21, NULL));
    // CHECK: func((dpct::create_image(&tex21, &res21, &texDesc21), 0));
    func(hipCreateTextureObject(&tex21, &res21, &texDesc21, NULL));
    // CHECK: funcT((dpct::create_image(&tex21, &res21, &texDesc21), 0));
    funcT(hipCreateTextureObject(&tex21, &res21, &texDesc21, NULL));

    // CHECK: errorCode = (dpct::dpct_free(tex21), 0);
    errorCode = hipDestroyTextureObject(tex21);
    // CHECK: cudaCheck((dpct::dpct_free(tex21), 0));
    cudaCheck(hipDestroyTextureObject(tex21));
    // CHECK: func((dpct::dpct_free(tex21), 0));
    func(hipDestroyTextureObject(tex21));
    // CHECK: funcT((dpct::dpct_free(tex21), 0));
    funcT(hipDestroyTextureObject(tex21));
  }
}
