
#include <hip/hip_runtime.h>
// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda --always-use-async-handler -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/async-error-handler.dp.cpp --match-full-lines %s


int main() {
  // CHECK: sycl::queue *s0, *s1, *s2;
  hipStream_t s0, s1, s2;

  // CHECK: s0 = dpct::get_current_device().create_queue(true);
  hipStreamCreate(&s0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag/priority options.
  // CHECK-NEXT: */
  // CHECK-NEXT: s1 = dpct::get_current_device().create_queue(true);
  hipStreamCreateWithFlags(&s1, hipStreamDefault);

  // CHECK: /*
  // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag/priority options.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2 = dpct::get_current_device().create_queue(true);
  hipStreamCreateWithPriority(&s2, hipStreamDefault, 2);
}
