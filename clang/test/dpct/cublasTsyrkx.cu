// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasTsyrkx.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

//CHECK: #define macro_a (mkl::transpose)1
#define macro_a (hipblasOperation_t)1

//CHECK: #define macro_b (mkl::uplo)1
#define macro_b (hipblasFillMode_t)1

hipblasFillMode_t foo(){
  return HIPBLAS_FILL_MODE_LOWER;
}

hipblasOperation_t bar(){
  return HIPBLAS_OP_T;
}

int main() {
  int n = 275;
  int k = 275;
  int lda = 1;
  int ldb = 1;
  int ldc = 1;

  float alpha_s = 1;
  float beta_s = 1;

  double alpha_d = 1;
  double beta_d = 1;

  hipblasHandle_t handle;
  hipblasStatus_t status;

  float* A_s=0;
  float* B_s=0;
  float* C_s=0;

  double* A_d=0;
  double* B_d=0;
  double* C_d=0;

  int trans0 = 0;
  int trans1 = 1;
  int fill0 = 0;
  int fill1 = 1;


  //CHECK: {
  //CHECK-NEXT: auto A_s_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(A_s);
  //CHECK-NEXT: auto B_s_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(B_s);
  //CHECK-NEXT: auto C_s_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(C_s);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1003:0: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: status = (mkl::blas::gemmt(handle, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans0==0 ? mkl::transpose::trans : mkl::transpose::nontrans, n, k, alpha_s, A_s_buf_ct{{[0-9]+}}, lda, B_s_buf_ct{{[0-9]+}}, ldb, beta_s, C_s_buf_ct{{[0-9]+}}, ldc), 0);
  //CHECK-NEXT: }
  //CHECK-NEXT: {
  //CHECK-NEXT: auto A_s_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(A_s);
  //CHECK-NEXT: auto B_s_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(B_s);
  //CHECK-NEXT: auto C_s_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(C_s);
  //CHECK-NEXT: mkl::blas::gemmt(handle, fill1==0 ? mkl::uplo::lower : mkl::uplo::upper, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, trans1==0 ? mkl::transpose::trans : mkl::transpose::nontrans, n, k, alpha_s, A_s_buf_ct{{[0-9]+}}, lda, B_s_buf_ct{{[0-9]+}}, ldb, beta_s, C_s_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: }
  status = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);
  hipblasSsyrkx(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);

  //CHECK: {
  //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_d);
  //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_d);
  //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_d);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1003:1: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: status = (mkl::blas::gemmt(handle, mkl::uplo::lower, mkl::transpose::nontrans, mkl::transpose::trans, n, k, alpha_d, A_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, beta_d, C_d_buf_ct{{[0-9]+}}, ldc), 0);
  //CHECK-NEXT: }
  //CHECK-NEXT: {
  //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_d);
  //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_d);
  //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_d);
  //CHECK-NEXT: mkl::blas::gemmt(handle, mkl::uplo::upper, mkl::transpose::trans, mkl::transpose::nontrans, n, k, alpha_d, A_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, beta_d, C_d_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: }
  status = hipblasDsyrkx(handle, (hipblasFillMode_t)0, (hipblasOperation_t)0, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, (hipblasFillMode_t)1, (hipblasOperation_t)1, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  //CHECK: {
  //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_d);
  //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_d);
  //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_d);
  //CHECK-NEXT: mkl::blas::gemmt(handle, foo(), (int)macro_a==2 ? mkl::transpose::conjtrans : (mkl::transpose)macro_a, (int)macro_a==0 ? mkl::transpose::trans : mkl::transpose::nontrans, n, k, alpha_d, A_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, beta_d, C_d_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: }


  hipblasDsyrkx(handle, foo(), macro_a, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  //CHECK: {
  //CHECK-NEXT: auto bar_transpose_ct2 = bar();
  //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_d);
  //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_d);
  //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_d);
  //CHECK-NEXT: mkl::blas::gemmt(handle, (int)macro_b==0 ? mkl::uplo::lower : mkl::uplo::upper, bar_transpose_ct2, bar_transpose_ct2==mkl::transpose::nontrans ? mkl::transpose::trans : mkl::transpose::nontrans, n, k, alpha_d, A_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, beta_d, C_d_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: }
  hipblasDsyrkx(handle, macro_b, bar(), n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);


  //CHECK: {
  //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_d);
  //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_d);
  //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_d);
  //CHECK-NEXT: mkl::blas::gemmt(handle, mkl::uplo::lower, mkl::transpose::trans, mkl::transpose::nontrans, n, k, alpha_d, A_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, beta_d, C_d_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: }
  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  return 0;
}
