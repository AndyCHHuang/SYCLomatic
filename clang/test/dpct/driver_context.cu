// RUN: c2s --format-range=none -out-root %T/driver_context %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/driver_context/driver_context.dp.cpp
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NUM 1
#define MY_SAFE_CALL(CALL) do {    \
  int Error = CALL;                \
} while (0)

int main(){

  hipDevice_t device;

  // CHECK: int ctx;
  hipCtx_t ctx;

  // CHECK: int ctx2;
  hipCtx_t ctx2;

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipInit was removed because the function call is redundant in DPC++.
  // CHECK-NEXT: */
  hipInit(0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipInit was replaced with 0 because the function call is redundant in DPC++.
  // CHECK-NEXT: */
  // CHECK-NEXT: MY_SAFE_CALL(0);
  MY_SAFE_CALL(hipInit(0));

  // CHECK: ctx = device;
  hipCtxCreate(&ctx, hipDeviceLmemResizeToMax, device);

  // CHECK: MY_SAFE_CALL((ctx = device, 0));
  MY_SAFE_CALL(hipCtxCreate(&ctx, hipDeviceLmemResizeToMax, device));

  // CHECK: c2s::dev_mgr::instance().select_device(ctx);
  hipCtxSetCurrent(ctx);

  // CHECK: MY_SAFE_CALL((c2s::dev_mgr::instance().select_device(ctx), 0));
  MY_SAFE_CALL(hipCtxSetCurrent(ctx));

  // CHECK: ctx2 = c2s::dev_mgr::instance().current_device_id();
  hipCtxGetCurrent(&ctx2);

  // CHECK: MY_SAFE_CALL((ctx2 = c2s::dev_mgr::instance().current_device_id(), 0));
  MY_SAFE_CALL(hipCtxGetCurrent(&ctx2));

  // CHECK: c2s::get_current_device().queues_wait_and_throw();
  hipCtxSynchronize();

  // CHECK: MY_SAFE_CALL((c2s::get_current_device().queues_wait_and_throw(), 0));
  MY_SAFE_CALL(hipCtxSynchronize());

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipCtxDestroy was removed because the function call is redundant in DPC++.
  // CHECK-NEXT: */
  hipCtxDestroy(ctx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipCtxDestroy was replaced with 0 because the function call is redundant in DPC++.
  // CHECK-NEXT: */
  // CHECK-NEXT: MY_SAFE_CALL(0);
  MY_SAFE_CALL(hipCtxDestroy(ctx2));

  return 0;
}
