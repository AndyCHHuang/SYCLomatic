// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/template_initialization.dp.cpp

#include <hip/hip_runtime.h>

#include <cassert>

const int num_threads = 16;

template<typename T>
void run_test();

int main() {
  run_test<float>();
  return 0;
}

// CHECK: template<typename T>
// CHECK: void kernel(T* in, T* out, cl::sycl::nd_item<3> [[ITEM:item_ct1]]) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(0)] = in[{{.*}}[[ITEM]].get_local_id(0)];
// CHECK: }
template<typename T>
__global__ void kernel(T* in, T* out) {
  out[threadIdx.x] = in[threadIdx.x];
}

template<typename T>
void run_test() {
  const size_t mem_size = sizeof(T) * num_threads;

  T h_in[num_threads];
  for (int i = 0; i < num_threads; ++i) {
    h_in[i] = (T)i;
  }

  T h_out[num_threads] = { 0 };

  T* d_in;
  hipMalloc((void **)&d_in, mem_size);
  hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

  T* d_out;
  hipMalloc((void **)&d_out, mem_size);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_in);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(d_out);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(num_threads, 1, 1)), cl::sycl::range<3>(num_threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           T *arg_ct0 = (T *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           T *arg_ct1 = (T *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           kernel<T>(arg_ct0, arg_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  kernel<T><<<1, num_threads>>>(d_in, d_out);

  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

  for (int i = 0; i < num_threads; ++i) {
    assert(h_out[i] == h_in[i]);
  }
}
