// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/template_initialization.dp.cpp

#include <hip/hip_runtime.h>

#include <cassert>

const int num_threads = 16;

template<typename T>
void run_test();

int main() {
  run_test<float>();
  return 0;
}

// CHECK: template<typename T>
// CHECK: void kernel(T* in, T* out, cl::sycl::nd_item<3> [[ITEM:item_ct1]]) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(0)] = in[{{.*}}[[ITEM]].get_local_id(0)];
// CHECK: }
template<typename T>
__global__ void kernel(T* in, T* out) {
  out[threadIdx.x] = in[threadIdx.x];
}

template<typename T>
void run_test() {
  const size_t mem_size = sizeof(T) * num_threads;

  T h_in[num_threads];
  for (int i = 0; i < num_threads; ++i) {
    h_in[i] = (T)i;
  }

  T h_out[num_threads] = { 0 };

  T* d_in;
  hipMalloc((void **)&d_in, mem_size);
  hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

  T* d_out;
  hipMalloc((void **)&d_out, mem_size);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_in_buf_ct0 = dpct::get_buffer_and_offset(d_in);
  // CHECK-NEXT:   size_t d_in_offset_ct0 = d_in_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> d_out_buf_ct1 = dpct::get_buffer_and_offset(d_out);
  // CHECK-NEXT:   size_t d_out_offset_ct1 = d_out_buf_ct1.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto d_in_acc_ct0 = d_in_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto d_out_acc_ct1 = d_out_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(num_threads, 1, 1)), cl::sycl::range<3>(num_threads, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           T *d_in_ct0 = (T *)(&d_in_acc_ct0[0] + d_in_offset_ct0);
  // CHECK-NEXT:           T *d_out_ct1 = (T *)(&d_out_acc_ct1[0] + d_out_offset_ct1);
  // CHECK-NEXT:           kernel<T>(d_in_ct0, d_out_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  kernel<T><<<1, num_threads>>>(d_in, d_out);

  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

  for (int i = 0; i < num_threads; ++i) {
    assert(h_out[i] == h_in[i]);
  }
}
