
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/replace-align.dp.cpp

// CHECK:typedef struct __dpct_align__(4) dpct_type_{{[a-f0-9]+}}
typedef struct __align__(4)
{
    unsigned char r, g, b, a;
}
T0;

// CHECK:class __dpct_align__(8) T1 {
class __align__(8) T1 {
    unsigned int l, a;
};

// CHECK:struct __attribute__((aligned(16))) T2
struct __attribute__((aligned(16))) T2
{
    unsigned int r, g, b;
};

