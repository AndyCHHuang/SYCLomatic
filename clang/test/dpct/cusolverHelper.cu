// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverHelper.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

// CHECK: #define MACRO_A cusolverDnCreate
#define MACRO_A hipsolverDnCreate

// CHECK: void foo(int, int, int, int, int, int, int, int) {}
void foo(hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t) {}

// CHECK: void foo2(int){}
void foo2(hipsolverStatus_t){}

// CHECK: int foo3(int m, int n)
hipsolverStatus_t foo3(int m, int n)
{
    // CHECK: return 0;
    return HIPSOLVER_STATUS_SUCCESS;
}

// CHECK: extern cl::sycl::queue * cusolverH2 = NULL;
extern hipsolverHandle_t* cusolverH2 = NULL;

int main(int argc, char *argv[])
{
    // CHECK: cl::sycl::queue * cusolverH = NULL;
    // CHECK-NEXT: int status = 0;
    // CHECK-NEXT: status = 1;
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;

    // CHECK: foo(0, 1, 2, 3, 4, 6, 7, 8);
    // CHECK-NEXT: status = 0;
    foo(HIPSOLVER_STATUS_SUCCESS, HIPSOLVER_STATUS_NOT_INITIALIZED, HIPSOLVER_STATUS_ALLOC_FAILED, HIPSOLVER_STATUS_INVALID_VALUE, HIPSOLVER_STATUS_ARCH_MISMATCH, HIPSOLVER_STATUS_EXECUTION_FAILED, HIPSOLVER_STATUS_INTERNAL_ERROR, HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED);
    hipsolverDnCreate(cusolverH);
    status = hipsolverDnCreate(cusolverH);

    // CHECK: status = 0;
    status = MACRO_A(cusolverH);

    // CHECK: status = 0;
    status = hipsolverDnDestroy(*cusolverH);

    // CHECK: int a = sizeof(int);
    // CHECK-NEXT: int b = sizeof(cl::sycl::queue);
    int a = sizeof(hipblasStatus_t);
    int b = sizeof(hipsolverHandle_t);
}
