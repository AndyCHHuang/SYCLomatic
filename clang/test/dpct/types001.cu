#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types001.dp.cpp

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <hipfft/hipfft.h>

// CHECK: dpct::device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const dpct::device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile dpct::device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHDCK: sycl::event events[23];
hipEvent_t events[23];
// CHECK: const sycl::event *pevents[23];
const hipEvent_t *pevents[23];
// CHECK: const sycl::event **ppevents[23];
const hipEvent_t **ppevents[23];

// CHECK: int errors[23];
hipError_t errors[23];
// CHECK: const int *perrors[23];
const hipError_t *perrors[23];
// CHECK: const int **pperrors[23];
const hipError_t **pperrors[23];

// CHECK: int errors1[23];
hipError_t errors1[23];
// CHECK: const int *perrors1[23];
const hipError_t *perrors1[23];
// CHECK: const int **pperrors1[23];
const hipError_t **pperrors1[23];

// CHECK: sycl::range<3> dims[23];
dim3 dims[23];
// CHECK: const sycl::range<3> *pdims[23];
const dim3 *pdims[23];
// CHECK: const sycl::range<3> **ppdims[23];
const dim3 **ppdims[23];

struct s {
  // CHECK: sycl::event events[23];
  hipEvent_t events[23];
  // CHECK: const sycl::event *pevents[23];
  const hipEvent_t *pevents[23];
  // CHECK: const sycl::event **ppevents[23];
  const hipEvent_t **ppevents[23];

  // CHECK: int errors[23];
  hipError_t errors[23];
  // CHECK: const int *perrors[23];
  const hipError_t *perrors[23];
  // CHECK: const int **pperrors[23];
  const hipError_t **pperrors[23];

  // CHECK: int errors1[23];
  hipError_t errors1[23];
  // CHECK: const int *perrors1[23];
  const hipError_t *perrors1[23];
  // CHECK: const int **pperrors1[23];
  const hipError_t **pperrors1[23];

  // CHECK: sycl::range<3> dims[23];
  dim3 dims[23];
  // CHECK: const sycl::range<3> *pdims[23];
  const dim3 *pdims[23];
  // CHECK: const sycl::range<3> **ppdims[23];
  const dim3 **ppdims[23];
};

// CHECK:  void foo(dpct::device_info p) {
void foo(hipDeviceProp_t p) {
  return;
}

// CHECK: int e;
hipError_t e;

// CHECK: int ee;
hipError_t ee;

// CHECK: int foo_0(int);
hipError_t foo_0(hipError_t);

// CHECK: int foo_1(int);
hipError_t foo_1(hipError_t);

// CHECK: int apicall(int i) {
hipError_t apicall(int i) {
  return hipSuccess;
};

// CHECK: int err = apicall(0);
hipError_t err = apicall(0);

template <typename T>
// CHECK: void check(T result, char const *const func) {
void check(T result, char const *const func) {
}

#define checkCudaErrors(val) check((val), #val)

int main(int argc, char **argv) {
  //CHECK:sycl::range<3> d3(1, 1, 1);
  //CHECK-NEXT:int a = sizeof(sycl::range<3>);
  //CHECK-NEXT:a = sizeof(d3);
  //CHECK-NEXT:a = sizeof d3;
  dim3 d3;
  int a = sizeof(dim3);
  a = sizeof(d3);
  a = sizeof d3;

  //CHECK:int cudaErr_t;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(cudaErr_t);
  //CHECK-NEXT:a = sizeof cudaErr_t;
  hipError_t cudaErr_t;
  a = sizeof(hipError_t);
  a = sizeof(cudaErr_t);
  a = sizeof cudaErr_t;

  //CHECK:int res;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(res);
  //CHECK-NEXT:a = sizeof res;
  hipError_t res;
  a = sizeof(hipError_t);
  a = sizeof(res);
  a = sizeof res;

  //CHECK:void* context;
  //CHECK-NEXT:a = sizeof(void*);
  //CHECK-NEXT:a = sizeof(context);
  //CHECK-NEXT:a = sizeof context;
  hipCtx_t context;
  a = sizeof(hipCtx_t);
  a = sizeof(context);
  a = sizeof context;

  //CHECK:sycl::event event;
  //CHECK-NEXT:a = sizeof(sycl::event);
  //CHECK-NEXT:a = sizeof(event);
  //CHECK-NEXT:a = sizeof event;
  hipEvent_t event;
  a = sizeof(hipEvent_t);
  a = sizeof(event);
  a = sizeof event;

  //CHECK:queue_p stream;
  //CHECK-NEXT:a = sizeof(queue_p);
  //CHECK-NEXT:a = sizeof(stream);
  //CHECK-NEXT:a = sizeof stream;
  hipStream_t stream;
  a = sizeof(hipStream_t);
  a = sizeof(stream);
  a = sizeof stream;

  //CHECK:int cudaErr;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(cudaErr);
  //CHECK-NEXT:a = sizeof cudaErr;
  hipError_t cudaErr;
  a = sizeof(hipError_t);
  a = sizeof(cudaErr);
  a = sizeof cudaErr;

  //CHECK:sycl::half h;
  //CHECK-NEXT:a = sizeof(sycl::half);
  //CHECK-NEXT:a = sizeof(h);
  //CHECK-NEXT:a = sizeof h;
  half h;
  a = sizeof(half);
  a = sizeof(h);
  a = sizeof h;

  //CHECK:sycl::half2 h2;
  //CHECK-NEXT:a = sizeof(sycl::half2);
  //CHECK-NEXT:a = sizeof(h2);
  //CHECK-NEXT:a = sizeof h2;
  half2 h2;
  a = sizeof(half2);
  a = sizeof(h2);
  a = sizeof h2;

  //CHECK:int blasStatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(blasStatus);
  //CHECK-NEXT:a = sizeof blasStatus;
  hipblasStatus_t blasStatus;
  a = sizeof(hipblasStatus_t);
  a = sizeof(blasStatus);
  a = sizeof blasStatus;

  //CHECK:sycl::float2 complex;
  //CHECK-NEXT:a = sizeof(sycl::float2);
  //CHECK-NEXT:a = sizeof(complex);
  //CHECK-NEXT:a = sizeof complex;
  hipComplex complex;
  a = sizeof(hipComplex);
  a = sizeof(complex);
  a = sizeof complex;

  //CHECK:sycl::double2 doubleComplex;
  //CHECK-NEXT:a = sizeof(sycl::double2);
  //CHECK-NEXT:a = sizeof(doubleComplex);
  //CHECK-NEXT:a = sizeof doubleComplex;
  hipDoubleComplex doubleComplex;
  a = sizeof(hipDoubleComplex);
  a = sizeof(doubleComplex);
  a = sizeof doubleComplex;

  //CHECK:mkl::uplo fill;
  //CHECK-NEXT:a = sizeof(mkl::uplo);
  //CHECK-NEXT:a = sizeof(fill);
  //CHECK-NEXT:a = sizeof fill;
  hipblasFillMode_t fill;
  a = sizeof(hipblasFillMode_t);
  a = sizeof(fill);
  a = sizeof fill;

  //CHECK:mkl::diag diag;
  //CHECK-NEXT:a = sizeof(mkl::diag);
  //CHECK-NEXT:a = sizeof(diag);
  //CHECK-NEXT:a = sizeof diag;
  hipblasDiagType_t diag;
  a = sizeof(hipblasDiagType_t);
  a = sizeof(diag);
  a = sizeof diag;

  //CHECK:mkl::side side;
  //CHECK-NEXT:a = sizeof(mkl::side);
  //CHECK-NEXT:a = sizeof(side);
  //CHECK-NEXT:a = sizeof side;
  hipblasSideMode_t side;
  a = sizeof(hipblasSideMode_t);
  a = sizeof(side);
  a = sizeof side;

  //CHECK:mkl::transpose oper;
  //CHECK-NEXT:a = sizeof(mkl::transpose);
  //CHECK-NEXT:a = sizeof(oper);
  //CHECK-NEXT:a = sizeof oper;
  hipblasOperation_t oper;
  a = sizeof(hipblasOperation_t);
  a = sizeof(oper);
  a = sizeof oper;

  //CHECK:int blasStatus_legacy;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(blasStatus_legacy);
  //CHECK-NEXT:a = sizeof blasStatus_legacy;
  hipblasStatus_t blasStatus_legacy;
  a = sizeof(hipblasStatus_t);
  a = sizeof(blasStatus_legacy);
  a = sizeof blasStatus_legacy;

  //CHECK:int solverStatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(solverStatus);
  //CHECK-NEXT:a = sizeof solverStatus;
  hipsolverStatus_t solverStatus;
  a = sizeof(hipsolverStatus_t);
  a = sizeof(solverStatus);
  a = sizeof solverStatus;

  //CHECK:int64_t eigtype;
  //CHECK-NEXT:a = sizeof(int64_t);
  //CHECK-NEXT:a = sizeof(eigtype);
  //CHECK-NEXT:a = sizeof eigtype;
  hipsolverEigType_t eigtype;
  a = sizeof(hipsolverEigType_t);
  a = sizeof(eigtype);
  a = sizeof eigtype;

  //CHECK:mkl::job eigmode;
  //CHECK-NEXT:a = sizeof(mkl::job);
  //CHECK-NEXT:a = sizeof(eigmode);
  //CHECK-NEXT:a = sizeof eigmode;
  hipsolverEigMode_t eigmode;
  a = sizeof(hipsolverEigMode_t);
  a = sizeof(eigmode);
  a = sizeof eigmode;

  //CHECK:int randstatus_t;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(randstatus_t);
  //CHECK-NEXT:a = sizeof randstatus_t;
  hiprandStatus_t randstatus_t;
  a = sizeof(hiprandStatus_t);
  a = sizeof(randstatus_t);
  a = sizeof randstatus_t;

  //CHECK:int cudaerror;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(cudaerror);
  //CHECK-NEXT:a = sizeof cudaerror;
  hipError_t cudaerror;
  a = sizeof(hipError_t);
  a = sizeof(cudaerror);
  a = sizeof cudaerror;

  //CHECK:int fftresult;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(fftresult);
  //CHECK-NEXT:a = sizeof fftresult;
  hipfftResult_t fftresult;
  a = sizeof(hipfftResult_t);
  a = sizeof(fftresult);
  a = sizeof fftresult;

  //CHECK:hipError_t error_enum;
  //CHECK-NEXT:a = sizeof(hipError_t);
  //CHECK-NEXT:a = sizeof(error_enum);
  //CHECK-NEXT:a = sizeof error_enum;
  hipError_t error_enum;
  a = sizeof(hipError_t);
  a = sizeof(error_enum);
  a = sizeof error_enum;

  //CHECK:int randstatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(randstatus);
  //CHECK-NEXT:a = sizeof randstatus;
  hiprandStatus randstatus;
  a = sizeof(hiprandStatus);
  a = sizeof(randstatus);
  a = sizeof randstatus;

  //CHECK:dpct::device_info dp;
  //CHECK-NEXT:a = sizeof(dpct::device_info);
  //CHECK-NEXT:a = sizeof(dp);
  //CHECK-NEXT:a = sizeof dp;
  hipDeviceProp_t dp;
  a = sizeof(hipDeviceProp_t);
  a = sizeof(dp);
  a = sizeof dp;

  //CHECK:ihipStream_t *stream_st;
  //CHECK-NEXT:a = sizeof(ihipStream_t*);
  //CHECK-NEXT:a = sizeof(stream_st);
  //CHECK-NEXT:a = sizeof stream_st;
  ihipStream_t *stream_st;
  a = sizeof(ihipStream_t*);
  a = sizeof(stream_st);
  a = sizeof stream_st;

  //CHECK:ihipEvent_t *event_st;
  //CHECK-NEXT:a = sizeof(ihipEvent_t*);
  //CHECK-NEXT:a = sizeof(event_st);
  //CHECK-NEXT:a = sizeof event_st;
  ihipEvent_t *event_st;
  a = sizeof(ihipEvent_t*);
  a = sizeof(event_st);
  a = sizeof event_st;

  //CHECK:sycl::queue blashandle;
  //CHECK-NEXT:a = sizeof(sycl::queue);
  //CHECK-NEXT:a = sizeof(blashandle);
  //CHECK-NEXT:a = sizeof blashandle;
  hipblasHandle_t blashandle;
  a = sizeof(hipblasHandle_t);
  a = sizeof(blashandle);
  a = sizeof blashandle;

  //CHECK:sycl::queue solverdnhandle;
  //CHECK-NEXT:a = sizeof(sycl::queue);
  //CHECK-NEXT:a = sizeof(solverdnhandle);
  //CHECK-NEXT:a = sizeof solverdnhandle;
  hipsolverHandle_t solverdnhandle;
  a = sizeof(hipsolverHandle_t);
  a = sizeof(solverdnhandle);
  a = sizeof solverdnhandle;

  checkCudaErrors(apicall(0));
  return 0;
}

__global__ void foo() {
  void *p;
  // CHECK: (queue_p)p;
  // CHECK-NEXT: (queue_p *)p;
  // CHECK-NEXT: (queue_p **)p;
  // CHECK-NEXT: (queue_p ***)p;
  (hipStream_t)p;
  (hipStream_t *)p;
  (hipStream_t **)p;
  (hipStream_t ***)p;

  int i;
  // CHECK: (int)i;
  // CHECK-NEXT: (int *)p;
  // CHECK-NEXT: (int **)p;
  // CHECK-NEXT: (int ***)p;
  (hipError_t)i;
  (hipError_t *)p;
  (hipError_t **)p;
  (hipError_t ***)p;

  hipDeviceProp_t cdp;
  // CHECK: dpct::device_info cdp2 = (dpct::device_info)cdp;
  // CHECK-NEXT: (dpct::device_info *)p;
  // CHECK-NEXT: (dpct::device_info **)p;
  // CHECK-NEXT: (dpct::device_info ***)p;
  hipDeviceProp_t cdp2 = (hipDeviceProp_t)cdp;
  (hipDeviceProp_t *)p;
  (hipDeviceProp_t **)p;
  (hipDeviceProp_t ***)p;
}

