
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types001.dp.cpp

// CHECK: dpct::device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const dpct::device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile dpct::device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHDCK: sycl::event events[23];
hipEvent_t events[23];
// CHECK: const sycl::event *pevents[23];
const hipEvent_t *pevents[23];
// CHECK: const sycl::event **ppevents[23];
const hipEvent_t **ppevents[23];

// CHECK: int errors[23];
hipError_t errors[23];
// CHECK: const int *perrors[23];
const hipError_t *perrors[23];
// CHECK: const int **pperrors[23];
const hipError_t **pperrors[23];

// CHECK: int errors1[23];
hipError_t errors1[23];
// CHECK: const int *perrors1[23];
const hipError_t *perrors1[23];
// CHECK: const int **pperrors1[23];
const hipError_t **pperrors1[23];

// CHECK: sycl::range<3> dims[23];
dim3 dims[23];
// CHECK: const sycl::range<3> *pdims[23];
const dim3 *pdims[23];
// CHECK: const sycl::range<3> **ppdims[23];
const dim3 **ppdims[23];

struct s {
  // CHECK: sycl::event events[23];
  hipEvent_t events[23];
  // CHECK: const sycl::event *pevents[23];
  const hipEvent_t *pevents[23];
  // CHECK: const sycl::event **ppevents[23];
  const hipEvent_t **ppevents[23];

  // CHECK: int errors[23];
  hipError_t errors[23];
  // CHECK: const int *perrors[23];
  const hipError_t *perrors[23];
  // CHECK: const int **pperrors[23];
  const hipError_t **pperrors[23];

  // CHECK: int errors1[23];
  hipError_t errors1[23];
  // CHECK: const int *perrors1[23];
  const hipError_t *perrors1[23];
  // CHECK: const int **pperrors1[23];
  const hipError_t **pperrors1[23];

  // CHECK: sycl::range<3> dims[23];
  dim3 dims[23];
  // CHECK: const sycl::range<3> *pdims[23];
  const dim3 *pdims[23];
  // CHECK: const sycl::range<3> **ppdims[23];
  const dim3 **ppdims[23];
};

// CHECK:  void foo(dpct::device_info p) {
void foo(hipDeviceProp_t p) {
  return;
}

// CHECK: int e;
hipError_t e;

// CHECK: int ee;
hipError_t ee;

// CHECK: int foo_0(int);
hipError_t foo_0(hipError_t);

// CHECK: int foo_1(int);
hipError_t foo_1(hipError_t);

// CHECK: int apicall(int i) {
hipError_t apicall(int i) {
  return hipSuccess;
};

// CHECK: int err = apicall(0);
hipError_t err = apicall(0);

template <typename T>
// CHECK: void check(T result, char const *const func) {
void check(T result, char const *const func) {
}

#define checkCudaErrors(val) check((val), #val)

int main(int argc, char **argv) {

  checkCudaErrors(apicall(0));
  return 0;
}

__global__ void foo() {
  void *p;
  // CHECK: (queue_p)p;
  // CHECK-NEXT: (queue_p *)p;
  // CHECK-NEXT: (queue_p **)p;
  // CHECK-NEXT: (queue_p ***)p;
  (hipStream_t)p;
  (hipStream_t *)p;
  (hipStream_t **)p;
  (hipStream_t ***)p;

  int i;
  // CHECK: (int)i;
  // CHECK-NEXT: (int *)p;
  // CHECK-NEXT: (int **)p;
  // CHECK-NEXT: (int ***)p;
  (hipError_t)i;
  (hipError_t *)p;
  (hipError_t **)p;
  (hipError_t ***)p;

  hipDeviceProp_t cdp;
  // CHECK: dpct::device_info cdp2 = (dpct::device_info)cdp;
  // CHECK-NEXT: (dpct::device_info *)p;
  // CHECK-NEXT: (dpct::device_info **)p;
  // CHECK-NEXT: (dpct::device_info ***)p;
  hipDeviceProp_t cdp2 = (hipDeviceProp_t)cdp;
  (hipDeviceProp_t *)p;
  (hipDeviceProp_t **)p;
  (hipDeviceProp_t ***)p;
}
