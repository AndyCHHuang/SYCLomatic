// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <mkl_sycl_types.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern sycl::queue* handle2;
extern hipblasHandle_t handle2;

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: int status;
  // CHECK-NEXT: sycl::queue* handle;
  // CHECK-NEXT: handle = &q_ct1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = &q_ct1, 0);
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetPointerMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetPointerMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_HOST;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

  // CHECK: sycl::queue *stream1;
  // CHECK-NEXT: stream1 = dev_ct1.create_queue();
  // CHECK-NEXT: handle = stream1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = stream1, 0);
  // CHECK-NEXT: stream1 = handle;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (stream1 = handle, 0);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipblasSetStream(handle, stream1);
  status = hipblasSetStream(handle, stream1);
  hipblasGetStream(handle, &stream1);
  status = hipblasGetStream(handle, &stream1);


  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_S, *handle), d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_S, *handle), d_C_S_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(*handle, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, N, N, N, dpct::get_value(&alpha_S, *handle), d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_S, *handle), d_C_S_buf_ct{{[0-9]+}}, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_D, *handle), d_A_D_buf_ct{{[0-9]+}}, N, d_B_D_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_D, *handle), d_C_D_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(*handle, trans2==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans2, mkl::transpose::conjtrans, N, N, N, dpct::get_value(&alpha_D, *handle), d_A_D_buf_ct{{[0-9]+}}, N, d_B_D_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_D, *handle), d_C_D_buf_ct{{[0-9]+}}, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_H, *handle), d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_H, *handle), d_C_H_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(*handle, trans2==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans2, mkl::transpose::conjtrans, N, N, N, dpct::get_value(&alpha_H, *handle), d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_H, *handle), d_C_H_buf_ct{{[0-9]+}}, N);
  status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
  hipblasHgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (mkl::blas::gemm(*handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, dpct::get_value(&alpha_S, *handle), d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_S, *handle), d_C_S_buf_ct{{[0-9]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: mkl::blas::gemm(*handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, dpct::get_value(&alpha_S, *handle), d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_S, *handle), d_C_S_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: foo(bar((mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_S, *handle), d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, dpct::get_value(&beta_S, *handle), d_C_S_buf_ct{{[0-9]+}}, N), 0)));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = nullptr, 0);
  // CHECK-NEXT: handle = nullptr;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}
