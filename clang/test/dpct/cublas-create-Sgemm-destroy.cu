// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <mkl_sycl_types.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern cl::sycl::queue handle2;
extern hipblasHandle_t handle2;

int main() {
  // CHECK: int status;
  // CHECK-NEXT: cl::sycl::queue handle;
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }
  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_D), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_D), buffer_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, *(&alpha_D), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_D), buffer_ct{{[0-9]+}}, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  // CHECK: for (;;) {
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: status = (mkl::blas::gemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: foo(bar((mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N), 0)));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  // CHECK: status = 0;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}
