// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <mkl_sycl_types.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern sycl::queue handle2;
extern hipblasHandle_t handle2;

int main() {
  // CHECK: int status;
  // CHECK-NEXT: sycl::queue handle;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasCreate was removed, because Function call is redundant in DPC++.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasCreate was replaced with 0, because Function call is redundant in DPC++.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }
  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), d_A_S_buff_ct1, N, d_B_S_buff_ct1, N, *(&beta_S), d_C_S_buff_ct1, N), 0);
  // CHECK: mkl::blas::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, *(&alpha_S), d_A_S_buff_ct1, N, d_B_S_buff_ct1, N, *(&beta_S), d_C_S_buff_ct1, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_D), d_A_D_buff_ct1, N, d_B_D_buff_ct1, N, *(&beta_D), d_C_D_buff_ct1, N), 0);
  // CHECK: mkl::blas::gemm(handle, (((int)transpose_ct1)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct1)), (((int)transpose_ct2)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct2)), N, N, N, *(&alpha_D), d_A_D_buff_ct1, N, d_B_D_buff_ct1, N, *(&beta_D), d_C_D_buff_ct1, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  // CHECK: for (;;) {
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_C_S);
  // CHECK-NEXT: status = (mkl::blas::gemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_S), d_A_S_buff_ct1, N, d_B_S_buff_ct1, N, *(&beta_S), d_C_S_buff_ct1, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_C_S);
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, *(&alpha_S), d_A_S_buff_ct1, N, d_B_S_buff_ct1, N, *(&beta_S), d_C_S_buff_ct1, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buff_ct1 = dpct::mem_mgr::instance().get_buffer<float>(d_C_S);
  // CHECK-NEXT: foo(bar((mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), d_A_S_buff_ct1, N, d_B_S_buff_ct1, N, *(&beta_S), d_C_S_buff_ct1, N), 0)));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasDestroy was replaced with 0, because Function call is redundant in DPC++.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasDestroy was removed, because Function call is redundant in DPC++.
  // CHECK-NEXT: */
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}
