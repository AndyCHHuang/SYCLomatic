// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <mkl_sycl_types.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern sycl::queue* handle2;
extern hipblasHandle_t handle2;

int main() {
  // CHECK: int status;
  // CHECK-NEXT: sycl::queue* handle;
  // CHECK-NEXT: handle = &dpct::get_default_queue();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = &dpct::get_default_queue(), 0);
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  // CHECK: sycl::queue *stream1;
  // CHECK-NEXT: stream1 = dpct::get_current_device().create_queue();
  // CHECK-NEXT: handle = stream1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = stream1, 0);
  // CHECK-NEXT: stream1 = handle;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (stream1 = handle, 0);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipblasSetStream(handle, stream1);
  status = hipblasSetStream(handle, stream1);
  hipblasGetStream(handle, &stream1);
  status = hipblasGetStream(handle, &stream1);


  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(*handle, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, alpha_D, d_A_D_buf_ct{{[0-9]+}}, N, d_B_D_buf_ct{{[0-9]+}}, N, beta_D, d_C_D_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: mkl::blas::gemm(*handle, trans2==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans2, mkl::transpose::conjtrans, N, N, N, alpha_D, d_A_D_buf_ct{{[0-9]+}}, N, d_B_D_buf_ct{{[0-9]+}}, N, beta_D, d_C_D_buf_ct{{[0-9]+}}, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (mkl::blas::gemm(*handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: mkl::blas::gemm(*handle, mkl::transpose::trans, mkl::transpose::trans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: foo(bar((mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N), 0)));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = nullptr, 0);
  // CHECK-NEXT: handle = nullptr;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}
