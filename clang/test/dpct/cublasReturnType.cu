// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasReturnType.dp.cpp --match-full-lines %s
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: int foo(int m, int n) {
hipblasStatus_t foo(int m, int n) {
  // CHECK: return 0;
  return HIPBLAS_STATUS_SUCCESS;
}

// CHECK: sycl::queue* foo1(int m) {
hipblasHandle_t foo1(int m) {
  return 0;
}

// CHECK: sycl::float2 foo2(sycl::float2 m) {
hipComplex foo2(hipComplex m) {
  // CHECK: return sycl::float2(1, 0);
  return make_hipComplex(1, 0);
}

// CHECK: sycl::double2 foo3(sycl::double2 m) {
hipDoubleComplex foo3(hipDoubleComplex m) {
  // CHECK: return sycl::double2(1, 0);
  return make_hipDoubleComplex(1, 0);
}

// CHECK: mkl::transpose foo4(mkl::transpose m) {
hipblasOperation_t foo4(hipblasOperation_t m) {
  // CHECK: return mkl::transpose::conjtrans;
  return HIPBLAS_OP_C;
}

// CHECK: mkl::uplo foo5(mkl::uplo m) {
hipblasFillMode_t foo5(hipblasFillMode_t m) {
  // CHECK: return mkl::uplo::lower;
  return HIPBLAS_FILL_MODE_LOWER;
}

// CHECK: mkl::side foo6(mkl::side m) {
hipblasSideMode_t foo6(hipblasSideMode_t m) {
  // CHECK: return mkl::side::right;
  return HIPBLAS_SIDE_RIGHT;
}

// CHECK: mkl::diag foo7(mkl::diag m) {
hipblasDiagType_t foo7(hipblasDiagType_t m) {
  // CHECK: return mkl::diag::nonunit;
  return HIPBLAS_DIAG_NON_UNIT;
}
