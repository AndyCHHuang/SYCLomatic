// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasReturnType.dp.cpp --match-full-lines %s
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: int foo(int m, int n) {
hipblasStatus_t foo(int m, int n) {
  // CHECK: return 0;
  return HIPBLAS_STATUS_SUCCESS;
}

// CHECK: cl::sycl::queue foo1(int m) {
hipblasHandle_t foo1(int m) {
  return 0;
}

// CHECK: cl::sycl::float2 foo2(cl::sycl::float2 m) {
hipComplex foo2(hipComplex m) {
  // CHECK: return cl::sycl::float2(1, 0);
  return make_hipComplex(1, 0);
}

// CHECK: cl::sycl::double2 foo3(cl::sycl::double2 m) {
hipDoubleComplex foo3(hipDoubleComplex m) {
  // CHECK: return cl::sycl::double2(1, 0);
  return make_hipDoubleComplex(1, 0);
}

// CHECK: mkl::transpose foo4(mkl::transpose m) {
hipblasOperation_t foo4(hipblasOperation_t m) {
  // CHECK: return mkl::transpose::conjtrans;
  return HIPBLAS_OP_C;
}

// CHECK: mkl::uplo foo5(mkl::uplo m) {
hipblasFillMode_t foo5(hipblasFillMode_t m) {
  // CHECK: return mkl::uplo::lower;
  return HIPBLAS_FILL_MODE_LOWER;
}

// CHECK: mkl::side foo6(mkl::side m) {
hipblasSideMode_t foo6(hipblasSideMode_t m) {
  // CHECK: return mkl::side::right;
  return HIPBLAS_SIDE_RIGHT;
}

// CHECK: mkl::diag foo7(mkl::diag m) {
hipblasDiagType_t foo7(hipblasDiagType_t m) {
  // CHECK: return mkl::diag::nonunit;
  return HIPBLAS_DIAG_NON_UNIT;
}
