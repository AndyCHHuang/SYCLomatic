// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

hipblasHandle_t handle;
int N = 275;
float *h_a, *h_b, *h_c;
const float *d_A_S;
const float *d_B_S;
float *d_C_S;
float alpha_S = 1.0f;
float beta_S = 0.0f;
int trans0 = 0;
int trans1 = 1;
int trans2 = 2;
int fill0 = 0;
int side0 = 0;
int diag0 = 0;
int *result = 0;
const float *x_S = 0;
const float *y_S = 0;

const double *d_A_D;
const double  *d_B_D;
double  *d_C_D;
double alpha_D;
double beta_D;
const double *x_D;
const double *y_D;

const float2 *d_A_C;
const float2  *d_B_C;
float2  *d_C_C;
float2 alpha_C;
float2 beta_C;
const float2 *x_C;
const float2 *y_C;

const double2 *d_A_Z;
const double2  *d_B_Z;
double2  *d_C_Z;
double2 alpha_Z;
double2 beta_Z;
const double2 *x_Z;
const double2 *y_Z;

float* result_S;
double* result_D;
float2* result_C;
double2* result_Z;

int incx, incy, lda, ldb, ldc;

int main() {

  //CHECK:/*
  //CHECK-NEXT:DPCT1018:{{[0-9]+}}: The cublasSetVector was migrated, but due to parameter 11111 equals to parameter 11111 but greater than 1, the generated code performance may be sub-optimal.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int a = (dpct::matrix_mem_copy((void*)d_C_S, (void*)h_a, 11111, 11111, 1, 10, sizeof(float)), 0);
  //CHECK-NEXT:dpct::matrix_mem_copy((void*)d_C_S, (void*)h_b, 1, 1, 1, 10, sizeof(float));
  //CHECK-NEXT:dpct::matrix_mem_copy((void*)d_C_S, (void*)h_c, 1, 1, 1, 10, sizeof(float));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (dpct::matrix_mem_copy((void*)d_C_S, (void*)h_a, 100, 100, 100, 100, 10000), 0);
  int a = hipblasSetVector(10, sizeof(float), h_a, 11111, d_C_S, 11111);
  hipblasSetVector(10, sizeof(float), h_b, 1, d_C_S, 1);
  hipblasSetVector(10, sizeof(float), h_c, 1, d_C_S, 1);
  a = hipblasSetMatrix(100, 100, 10000, h_a, 100, d_C_S, 100);


  //level 1

  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (mkl::blas::iamax(*handle, N, x_S, N, res_temp_ptr_ct{{[0-9]+}}).wait(), 0);
  //CHECK-NEXT:*result = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  a = hipblasIsamax(handle, N, x_S, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:mkl::blas::iamax(*handle, N, x_D, N, res_temp_ptr_ct{{[0-9]+}}).wait();
  //CHECK-NEXT:*result = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  hipblasIdamax(handle, N, x_D, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (mkl::blas::iamax(*handle, N, (std::complex<float>*)x_C, N, res_temp_ptr_ct{{[0-9]+}}).wait(), 0);
  //CHECK-NEXT:*result = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  a = hipblasIcamax(handle, N, x_C, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:mkl::blas::iamax(*handle, N, (std::complex<double>*)x_Z, N, res_temp_ptr_ct{{[0-9]+}}).wait();
  //CHECK-NEXT:*result = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  hipblasIzamax(handle, N, x_Z, N, result);

  //CHECK:a = (mkl::blas::rotm(*handle, N, d_C_S, N, d_C_S, N, const_cast<float*>(x_S)).wait(), 0);
  a = hipblasSrotm(handle, N, d_C_S, N, d_C_S, N, x_S);
  //CHECK:mkl::blas::rotm(*handle, N, d_C_D, N, d_C_D, N, const_cast<double*>(x_D)).wait();
  hipblasDrotm(handle, N, d_C_D, N, d_C_D, N, x_D);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::copy(*handle, N, x_S, incx, d_C_S, incy).wait(), 0);
  a = hipblasScopy(handle, N, x_S, incx, d_C_S, incy);
  // CHECK:mkl::blas::copy(*handle, N, x_D, incx, d_C_D, incy).wait();
  hipblasDcopy(handle, N, x_D, incx, d_C_D, incy);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::copy(*handle, N, (std::complex<float>*)x_C, incx, (std::complex<float>*)d_C_C, incy).wait(), 0);
  a = hipblasCcopy(handle, N, x_C, incx, d_C_C, incy);
  // CHECK:mkl::blas::copy(*handle, N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)d_C_Z, incy).wait();
  hipblasZcopy(handle, N, x_Z, incx, d_C_Z, incy);


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::axpy(*handle, N, alpha_S, x_S, incx, result_S, incy).wait(), 0);
  a = hipblasSaxpy(handle, N, &alpha_S, x_S, incx, result_S, incy);
  // CHECK:mkl::blas::axpy(*handle, N, alpha_D, x_D, incx, result_D, incy).wait();
  hipblasDaxpy(handle, N, &alpha_D, x_D, incx, result_D, incy);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::axpy(*handle, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)result_C, incy).wait(), 0);
  a = hipblasCaxpy(handle, N, &alpha_C, x_C, incx, result_C, incy);
  // CHECK:mkl::blas::axpy(*handle, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)result_Z, incy).wait();
  hipblasZaxpy(handle, N, &alpha_Z, x_Z, incx, result_Z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::scal(*handle, N, alpha_S, result_S, incx).wait(), 0);
  a = hipblasSscal(handle, N, &alpha_S, result_S, incx);
  // CHECK:mkl::blas::scal(*handle, N, alpha_D, result_D, incx).wait();
  hipblasDscal(handle, N, &alpha_D, result_D, incx);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::scal(*handle, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)result_C, incx).wait(), 0);
  a = hipblasCscal(handle, N, &alpha_C, result_C, incx);
  // CHECK:mkl::blas::scal(*handle, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)result_Z, incx).wait();
  hipblasZscal(handle, N, &alpha_Z, result_Z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::nrm2(*handle, N, x_S, incx, result_S).wait(), 0);
  a = hipblasSnrm2(handle, N, x_S, incx, result_S);
  // CHECK:mkl::blas::nrm2(*handle, N, x_D, incx, result_D).wait();

  hipblasDnrm2(handle, N, x_D, incx, result_D);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::nrm2(*handle, N, (std::complex<float>*)x_C, incx, result_S).wait(), 0);
  a = hipblasScnrm2(handle, N, x_C, incx, result_S);
  // CHECK:mkl::blas::nrm2(*handle, N, (std::complex<double>*)x_Z, incx, result_D).wait();
  hipblasDznrm2(handle, N, x_Z, incx, result_D);


  //level 2

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::gemv(*handle, trans2==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans2, N, N, alpha_S, x_S, lda, y_S, incx, beta_S, result_S, incy).wait(), 0);
  a = hipblasSgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  // CHECK:mkl::blas::gemv(*handle, mkl::transpose::nontrans, N, N, alpha_D, x_D, lda, y_D, incx, beta_D, result_D, incy).wait();
  hipblasDgemv(handle, HIPBLAS_OP_N, N, N, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::gemv(*handle, trans2==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans2, N, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)x_C, lda, (std::complex<float>*)y_C, incx, std::complex<float>(beta_C.x(),beta_C.y()), (std::complex<float>*)result_C, incy).wait(), 0);
  a = hipblasCgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_C, x_C, lda, y_C, incx, &beta_C, result_C, incy);
  // CHECK:mkl::blas::gemv(*handle, mkl::transpose::nontrans, N, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)x_Z, lda, (std::complex<double>*)y_Z, incx, std::complex<double>(beta_Z.x(),beta_Z.y()), (std::complex<double>*)result_Z, incy).wait();
  hipblasZgemv(handle, HIPBLAS_OP_N, N, N, &alpha_Z, x_Z, lda, y_Z, incx, &beta_Z, result_Z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::ger(*handle, N, N, alpha_S, x_S, incx, y_S, incy, result_S, lda).wait(), 0);
  a = hipblasSger(handle, N, N, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  // CHECK:mkl::blas::ger(*handle, N, N, alpha_D, x_D, incx, y_D, incy, result_D, lda).wait();
  hipblasDger(handle, N, N, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::geru(*handle, N, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda).wait(), 0);
  a = hipblasCgeru(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  // CHECK:mkl::blas::gerc(*handle, N, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda).wait();
  hipblasCgerc(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (mkl::blas::geru(*handle, N, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda).wait(), 0);
  a = hipblasZgeru(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);
  // CHECK:mkl::blas::gerc(*handle, N, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda).wait();
  hipblasZgerc(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);








  //level 3

  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (mkl::blas::gemm(*handle, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N).wait(), 0);
  a = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  //CHECK:mkl::blas::gemm(*handle, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, N, N, N, alpha_D, d_A_D, N, d_B_D, N, beta_D, d_C_D, N).wait();
  hipblasDgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)d_A_C, N, (std::complex<float>*)d_B_C, N, std::complex<float>(beta_C.x(),beta_C.y()), (std::complex<float>*)d_C_C, N).wait(), 0);
  a = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  //CHECK:mkl::blas::gemm(*handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)d_A_Z, N, (std::complex<double>*)d_B_Z, N, std::complex<double>(beta_Z.x(),beta_Z.y()), (std::complex<double>*)d_C_Z, N).wait();
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);


  //CHECK:dpct::matrix_mem_copy(d_C_S, d_B_S, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (mkl::blas::trmm(*handle, (mkl::side)side0, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, (mkl::diag)diag0, N, N, alpha_S, d_A_S, N, d_C_S, N).wait(), 0);
  a = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
  //CHECK:dpct::matrix_mem_copy(d_C_D, d_B_D, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:mkl::blas::trmm(*handle, (mkl::side)side0, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, (mkl::diag)diag0, N, N, alpha_D, d_A_D, N, d_C_D, N).wait();
  hipblasDtrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_D, d_A_D, N, d_B_D, N, d_C_D, N);
  //CHECK:dpct::matrix_mem_copy(d_C_C, d_B_C, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (mkl::blas::trmm(*handle, mkl::side::left, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::unit, N, N, std::complex<float>(alpha_C.x(),alpha_C.y()), (std::complex<float>*)d_A_C, N, (std::complex<float>*)d_C_C, N).wait(), 0);
  a = hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N);
  //CHECK:dpct::matrix_mem_copy(d_C_Z, d_B_Z, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:mkl::blas::trmm(*handle, mkl::side::left, mkl::uplo::lower, mkl::transpose::nontrans, mkl::diag::unit, N, N, std::complex<double>(alpha_Z.x(),alpha_Z.y()), (std::complex<double>*)d_A_Z, N, (std::complex<double>*)d_C_Z, N).wait();
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, d_C_Z, N);


  //CHECK:a = (mkl::blas::gemmt(*handle, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, trans1==0 ? mkl::transpose::trans : mkl::transpose::nontrans, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N).wait(), 0);
  a = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  //CHECK:mkl::blas::gemmt(*handle, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, trans1==0 ? mkl::transpose::trans : mkl::transpose::nontrans, N, N, alpha_D, d_A_D, N, d_B_D, N, beta_D, d_C_D, N).wait();
  hipblasDsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  // CHECK: dpct::matrix_mem_copy(d_C_S, d_B_S, N, N, N, N, dpct::device_to_device, *handle);
  // CHECK-NEXT: mkl::blas::trmm(*handle, (mkl::side)side0, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, (mkl::diag)diag0, N, N, alpha_S, d_A_S, N, d_C_S, N).wait();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. 0 is used in if statement. You need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if(int stat = 0){}
  if(int stat = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N)){}

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if(int stat = (mkl::blas::gemm(*handle, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N).wait(), 0)){}
  if(int stat = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){}


}

// CHECK: int foo1() try {
// CHECK-NEXT:   dpct::matrix_mem_copy(d_C_S, d_B_S, N, N, N, N, dpct::device_to_device, *handle);
// CHECK-NEXT:   mkl::blas::trmm(*handle, (mkl::side)side0, fill0==0 ? mkl::uplo::lower : mkl::uplo::upper, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, (mkl::diag)diag0, N, N, alpha_S, d_A_S, N, d_C_S, N).wait();
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. 0 is used in return statement. You need to rewrite this code.
// CHECK-NEXT:   */
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
int foo1(){
  return hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
}

// CHECK:int foo2() try {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT:  return (mkl::blas::gemm(*handle, trans0==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans0, trans1==2 ? mkl::transpose::conjtrans : (mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N).wait(), 0);
// CHECK-NEXT:}
int foo2(){
  return hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
}