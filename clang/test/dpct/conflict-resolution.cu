
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none -out-root %T/conflict-resolution %s -passes "ErrorHandlingIfStmtRule,ErrorConstantsRule" --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/conflict-resolution/conflict-resolution.dp.cpp

int printf(const char *format, ...);

// CHECK: void test_00(cudaError_t err) {
// CHECK-NEXT:   {{ +}}
// CHECK-NEXT: }
void test_00(hipError_t err) {
  if (err != hipSuccess) {
    printf("Some error happenned\n");
    exit(1);
  }
}

