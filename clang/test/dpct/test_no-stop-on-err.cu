#include "hip/hip_runtime.h"
// RUN: dpct -no-stop-on-err  --format-range=none -usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
#include <hip/hip_runtime.h>
#include <stdio.h>
#define VECTOR_SIZE 256

__global__ void VectorAddKernel(float* A, float* B, float* C); 

int main()
{
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, VECTOR_SIZE*sizeof(float));
    hipMalloc(&d_B, VECTOR_SIZE*sizeof(float));
    hipMalloc(&d_C, VECTOR_SIZE*sizeof(float));

    VectorAddKernel<<<1, VECTOR_SIZE>>>(d_A, d_B, d_C);

    float Result[VECTOR_SIZE] = { };
    hipMemcpy(Result, d_C, VECTOR_SIZE*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < VECTOR_SIZE; i++) {
        if (i % 16 == 0) {
            printf("\n");
        }
        printf("%f ", Result[i]);
    }

    return 0;
}
__global__ void VectorAddKernel(float* A, float* B, float* C)
{
    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];

    A[blockDim.x] = threadIdx.x + 1.0f;
    B[gridDim.x] = threadIdx.x + 1.0f;
    C[gridDim.x] = A[threadIdx.x] + B[threadIdx.x];

    __shared__  svar[10];

    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

