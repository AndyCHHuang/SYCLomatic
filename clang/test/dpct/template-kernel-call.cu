
#include <hip/hip_runtime.h>
// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/template-kernel-call.dp.cpp --match-full-lines %s

void printf(const char *format, unsigned char data);

template <class TName, unsigned N, class TData>
// CHECK: void testKernelPtr(const TData *L, const TData *M, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const TData *L, const TData *M) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(2) * [[ITEMNAME]].get_local_range().get(2) + [[ITEMNAME]].get_local_id(2);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

template<class TData>
// CHECK: void testKernel(TData L, TData M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernel(TData L, TData M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(2) * [[ITEMNAME]].get_local_range().get(2) + [[ITEMNAME]].get_local_id(2);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  L = M;
}

// CHECK: struct __dpct_align__(8) LA {
struct __align__(8) LA {
  unsigned int l, a;
};

template<class T>
class TestTemplate {
public:
  T data;
};

const unsigned ktarg = 80;
dim3 griddim = 2;
dim3 threaddim = 32;

template<class T>
void runTest() {
  typedef TestTemplate<T> TT;
  const void *karg1 = 0;
  const T *karg2 = 0;
  T *karg3 = 0;
  const TestTemplate<T> *karg4 = 0;
  TT *karg5 = 0;

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg1_buf_ct0 = dpct::get_buffer_and_offset((const T *)karg1);
  // CHECK-NEXT:   size_t karg1_offset_ct0 = karg1_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg2_buf_ct1 = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t karg2_offset_ct1 = karg2_buf_ct1.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto karg1_acc_ct0 = karg1_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto karg2_acc_ct1 = karg2_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestName, dpct_kernel_scalar<ktarg>, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const T *karg1_ct0 = (const T *)(&karg1_acc_ct0[0] + karg1_offset_ct0);
  // CHECK-NEXT:           const T *karg2_ct1 = (const T *)(&karg2_acc_ct1[0] + karg2_offset_ct1);
  // CHECK-NEXT:           testKernelPtr<class TestName, ktarg, T>(karg1_ct0, karg2_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<class TestName, ktarg, T><<<griddim, threaddim>>>((const T *)karg1, karg2);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg1_buf_ct0 = dpct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:   size_t karg1_offset_ct0 = karg1_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg3_buf_ct1 = dpct::get_buffer_and_offset(karg3);
  // CHECK-NEXT:   size_t karg3_offset_ct1 = karg3_buf_ct1.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto karg1_acc_ct0 = karg1_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto karg3_acc_ct1 = karg3_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestTemplate<T>, dpct_kernel_scalar<ktarg>, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const void *karg1_ct0 = (const void *)(&karg1_acc_ct0[0] + karg1_offset_ct0);
  // CHECK-NEXT:           T *karg3_ct1 = (T *)(&karg3_acc_ct1[0] + karg3_offset_ct1);
  // CHECK-NEXT:           testKernelPtr<class TestTemplate<T>, ktarg, T>(karg1_ct0, karg3_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<class TestTemplate<T>, ktarg, T><<<griddim, threaddim>>>(karg1, karg3);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg4_buf_ct0 = dpct::get_buffer_and_offset(karg4);
  // CHECK-NEXT:   size_t karg4_offset_ct0 = karg4_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg5_buf_ct1 = dpct::get_buffer_and_offset(karg5);
  // CHECK-NEXT:   size_t karg5_offset_ct1 = karg5_buf_ct1.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto karg4_acc_ct0 = karg4_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto karg5_acc_ct1 = karg5_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, T, dpct_kernel_scalar<ktarg>, TestTemplate<T>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const TestTemplate<T> *karg4_ct0 = (const TestTemplate<T> *)(&karg4_acc_ct0[0] + karg4_offset_ct0);
  // CHECK-NEXT:           TT *karg5_ct1 = (TT *)(&karg5_acc_ct1[0] + karg5_offset_ct1);
  // CHECK-NEXT:           testKernelPtr<T, ktarg, TestTemplate<T>>(karg4_ct0, karg5_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<T, ktarg, TestTemplate<T> ><<<griddim, threaddim>>>(karg4, karg5);

  T karg1T, karg2T;
  // CHECK: {
  // CHECK-NEXT:   auto ktarg_ct2 = ktarg;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<T>(karg1T, karg2T, ktarg_ct2, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<T><<<griddim, threaddim>>>(karg1T, karg2T, ktarg);

  TestTemplate<T> karg3TT;
  TT karg4TT;

  // CHECK: {
  // CHECK-NEXT:   auto ktarg_ct2 = ktarg;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, TestTemplate<T>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<TestTemplate<T>>(karg3TT, karg4TT, ktarg_ct2, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<TestTemplate<T> ><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);

  // CHECK: {
  // CHECK-NEXT:   auto ktarg_ct2 = ktarg;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, TT>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<TT>(karg3TT, karg4TT, ktarg_ct2, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<TT><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);
}

int main() {
  void *karg1 = 0;
  LA *karg2 = 0;
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg1_buf_ct0 = dpct::get_buffer_and_offset((const LA *)karg1);
  // CHECK-NEXT:   size_t karg1_offset_ct0 = karg1_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg2_buf_ct1 = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t karg2_offset_ct1 = karg2_buf_ct1.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto karg1_acc_ct0 = karg1_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto karg2_acc_ct1 = karg2_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestName, dpct_kernel_scalar<ktarg>, LA>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const LA *karg1_ct0 = (const LA *)(&karg1_acc_ct0[0] + karg1_offset_ct0);
  // CHECK-NEXT:           const LA *karg2_ct1 = (const LA *)(&karg2_acc_ct1[0] + karg2_offset_ct1);
  // CHECK-NEXT:           testKernelPtr<class TestName, ktarg, LA>(karg1_ct0, karg2_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<class TestName, ktarg, LA><<<griddim, threaddim>>>((const LA *)karg1, karg2);

  LA karg1LA, karg2LA;
  int intvar = 20;
  // CHECK: {
  // CHECK-NEXT:   auto ktarg_ct2 = ktarg;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, LA>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 10) * cl::sycl::range<3>(1, 1, intvar), cl::sycl::range<3>(1, 1, intvar)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<LA>(karg1LA, karg2LA, ktarg_ct2, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<LA><<<10, intvar>>>(karg1LA, karg2LA, ktarg);
}
