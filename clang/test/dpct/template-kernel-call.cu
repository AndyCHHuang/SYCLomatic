
#include <hip/hip_runtime.h>
// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/template-kernel-call.dp.cpp --match-full-lines %s

void printf(const char *format, unsigned char data);

template <class TName, unsigned N, class TData>
// CHECK: void testKernelPtr(const TData *L, const TData *M, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const TData *L, const TData *M) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

template<class TData>
// CHECK: void testKernel(TData L, TData M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernel(TData L, TData M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  L = M;
}

// CHECK: struct __sycl_align__(8) LA {
struct __align__(8) LA {
  unsigned int l, a;
};

template<class T>
class TestTemplate {
public:
  T data;
};

const unsigned ktarg = 80;
dim3 griddim = 2;
dim3 threaddim = 32;

template<class T>
void runTest() {
  typedef TestTemplate<T> TT;
  const void *karg1 = 0;
  const T *karg2 = 0;
  T *karg3 = 0;
  const TestTemplate<T> *karg4 = 0;
  TT *karg5 = 0;

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset((const T *)karg1);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestName, dpct_kernel_scalar<ktarg>, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const T *arg_ct0 = (const T *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           const T *arg_ct1 = (const T *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr<class TestName, ktarg, T>(arg_ct0, arg_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<class TestName, ktarg, T><<<griddim, threaddim>>>((const T *)karg1, karg2);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(karg1);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(karg3);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestTemplate<T>, dpct_kernel_scalar<ktarg>, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const void *arg_ct0 = (const void *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           T *arg_ct1 = (T *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr<class TestTemplate<T>, ktarg, T>(arg_ct0, arg_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<class TestTemplate<T>, ktarg, T><<<griddim, threaddim>>>(karg1, karg3);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(karg4);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(karg5);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, T, dpct_kernel_scalar<ktarg>, TestTemplate<T>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const TestTemplate<T> *arg_ct0 = (const TestTemplate<T> *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           TT *arg_ct1 = (TT *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr<T, ktarg, TestTemplate<T>>(arg_ct0, arg_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<T, ktarg, TestTemplate<T> ><<<griddim, threaddim>>>(karg4, karg5);

  T karg1T, karg2T;
  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, T>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<T>(karg1T, karg2T, ktarg, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<T><<<griddim, threaddim>>>(karg1T, karg2T, ktarg);

  TestTemplate<T> karg3TT;
  TT karg4TT;

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, TestTemplate<T>>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<TestTemplate<T>>(karg3TT, karg4TT, ktarg, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<TestTemplate<T> ><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, TT>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<TT>(karg3TT, karg4TT, ktarg, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<TT><<<griddim, threaddim>>>(karg3TT, karg4TT, ktarg);
}

int main() {
  void *karg1 = 0;
  LA *karg2 = 0;
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset((const LA *)karg1);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}, class TestName, dpct_kernel_scalar<ktarg>, LA>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const LA *arg_ct0 = (const LA *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           const LA *arg_ct1 = (const LA *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr<class TestName, ktarg, LA>(arg_ct0, arg_ct1, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<class TestName, ktarg, LA><<<griddim, threaddim>>>((const LA *)karg1, karg2);

  LA karg1LA, karg2LA;
  int intvar = 20;
  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}, LA>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(10, 1, 1) * cl::sycl::range<3>(intvar, 1, 1)), cl::sycl::range<3>(intvar, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel<LA>(karg1LA, karg2LA, ktarg, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<LA><<<10, intvar>>>(karg1LA, karg2LA, ktarg);
}
