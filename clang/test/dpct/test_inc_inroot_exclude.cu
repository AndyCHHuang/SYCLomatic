// RUN: c2s --format-range=none --usm-level=none -out-root %T/test_inc_inroot_exclude %s --cuda-include-path="%cuda-path/include" -extra-arg="-I%S/inc"  --in-root-exclude %S/inc -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/test_inc_inroot_exclude/test_inc_inroot_exclude.dp.cpp

// CHECK:#include "foo.cuh"
#include "foo.cuh"

// CHECK:#include "no_cuda_syntax.cuh"
#include "no_cuda_syntax.cuh"

// CHECK:#include <no_cuda_syntax.cuh>
#include <no_cuda_syntax.cuh>

void test(){
 foo<<<1,1>>>();
}
