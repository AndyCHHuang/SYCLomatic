// FIXME:
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-stream-api.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <list>
// CHECK: using queue_p = cl::sycl::queue *;

template <typename T>
// CHECK: void check(T result, char const *const func) {
void check(T result, char const *const func) {
}

#define checkCudaErrors(val) check((val), #val)

__global__ void kernelFunc() {
}

// CHECK: void process(queue_p st, char *data, int status) {}
void process(hipStream_t st, char *data, hipError_t status) {}

template<typename T>
// CHECK: void callback(queue_p st, int status, void *vp) {
void callback(hipStream_t st, hipError_t status, void *vp) {
  T *data = static_cast<T *>( vp);
  process(st, data, status);
}

template<typename FloatN, typename Float>
static void func()
{
  // CHECK: std::list<queue_p> streams;
  std::list<hipStream_t> streams;
  for (auto Iter = streams.begin(); Iter != streams.end(); ++Iter)
    // CHECK: *Iter = new cl::sycl::queue{};
    hipStreamCreate(&*Iter);
  for (auto Iter = streams.begin(); Iter != streams.end(); ++Iter)
    // CHECK: delete *Iter;
    hipStreamDestroy(*Iter);

  // CHECK: queue_p s0, &s1 = s0;
  // CHECK-NEXT: queue_p s2, *s3 = &s2;
  // CHECK-NEXT: queue_p s4, s5;
  // CHECK-EMPTY:
  hipStream_t s0, &s1 = s0;
  hipStream_t s2, *s3 = &s2;
  hipStream_t s4, s5;

  // CHECK: if (1)
  // CHECK-NEXT: s0 = new cl::sycl::queue{};
  if (1)
    hipStreamCreate(&s0);

  // CHECK: while (0)
  // CHECK-NEXT: s0 = new cl::sycl::queue{};
  while (0)
    hipStreamCreate(&s0);

  // CHECK: do
  // CHECK-NEXT: s0 = new cl::sycl::queue{};
  // CHECK: while (0);
  do
    hipStreamCreate(&s0);
  while (0);

  // CHECK: for (; 0; )
  // CHECK-NEXT: s0 = new cl::sycl::queue{};
  for (; 0; )
    hipStreamCreate(&s0);

  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<16, 32, 0>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((s1 = new cl::sycl::queue{}, 0));
  checkCudaErrors(hipStreamCreate(&s1));

  // CHECK:   s0->submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<16, 32, 0, s0>>>();

  // CHECK:   s1->submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernelFunc<<<16, 32, 0, s1>>>();

  {
    // CHECK: /*
    // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag/priority options.
    // CHECK-NEXT: */
    // CHECK-NEXT: s2 = new cl::sycl::queue{};
    hipStreamCreateWithFlags(&s2, hipStreamDefault);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag/priority options.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((*(s3) = new cl::sycl::queue{}, 0));
    checkCudaErrors(hipStreamCreateWithFlags(s3, hipStreamNonBlocking));

    // CHECK:   s2->submit(
    // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
    // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
    // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
    // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
    // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
    // CHECK-NEXT:           kernelFunc();
    // CHECK-NEXT:         });
    // CHECK-NEXT:     });
    kernelFunc<<<16, 32, 0, s2>>>();

    // CHECK:   (*s3)->submit(
    // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
    // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
    // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
    // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
    // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
    // CHECK-NEXT:           kernelFunc();
    // CHECK-NEXT:         });
    // CHECK-NEXT:     });
    kernelFunc<<<16, 32, 0, *s3>>>();

    // CHECK: delete s2;
    hipStreamDestroy(s2);
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((delete *s3, 0));
    checkCudaErrors(hipStreamDestroy(*s3));
  }

  {
    {
      // CHECK: /*
      // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag/priority options.
      // CHECK-NEXT: */
      // CHECK-NEXT: s4 = new cl::sycl::queue{};
      hipStreamCreateWithPriority(&s4, hipStreamDefault, 2);

      // CHECK: /*
      // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
      // CHECK-NEXT: */
      // CHECK-NEXT: /*
      // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag/priority options.
      // CHECK-NEXT: */
      // CHECK-NEXT: checkCudaErrors((s5 = new cl::sycl::queue{}, 0));
      checkCudaErrors(hipStreamCreateWithPriority(&s5, hipStreamNonBlocking, 3));

      // CHECK:   s4->submit(
      // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
      // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
      // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
      // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
      // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
      // CHECK-NEXT:           kernelFunc();
      // CHECK-NEXT:         });
      // CHECK-NEXT:     });
      kernelFunc<<<16, 32, 0, s4>>>();
      // CHECK:   s5->submit(
      // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
      // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(16, 1, 1) * cl::sycl::range<3>(32, 1, 1);
      // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(32, 1, 1);
      // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
      // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
      // CHECK-NEXT:           kernelFunc();
      // CHECK-NEXT:         });
      // CHECK-NEXT:     });
      kernelFunc<<<16, 32, 0, s5>>>();

      // CHECK: delete s4;
      hipStreamDestroy(s4);
      // CHECK: /*
      // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
      // CHECK-NEXT: */
      // CHECK-NEXT: checkCudaErrors((delete s5, 0));
      checkCudaErrors(hipStreamDestroy(s5));
    }
  }

  int priority_low;
  int priority_hi;
  // CHECK: /*
  // CHECK-NEXT: DPCT1014:{{[0-9]+}}: The flag/priority options are not supported for SYCL queues; the output parameter(s) are set to 0.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(&priority_low) = 0, *(&priority_hi) = 0;
  hipDeviceGetStreamPriorityRange(&priority_low, &priority_hi);
  // CHECK: /*
  // CHECK-NEXT: DPCT1014:{{[0-9]+}}: The flag/priority options are not supported for SYCL queues; the output parameter(s) are set to 0.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((*(&priority_low) = 0, *(&priority_hi) = 0, 0));
  checkCudaErrors(hipDeviceGetStreamPriorityRange(&priority_low, &priority_hi));

  int priority;
  // CHECK: /*
  // CHECK-NEXT: DPCT1014:{{[0-9]+}}: The flag/priority options are not supported for SYCL queues; the output parameter(s) are set to 0.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((*(&priority) = 0, 0));
  checkCudaErrors(hipStreamGetPriority(s0, &priority));

  char str[256];

  unsigned int flags = 0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: int status = (std::async([&]() { s0->wait(); callback<char *>(s0, 0, str); }), 0);
  // CHECK-NEXT: std::async([&]() { s1->wait(); callback<char*>(s1, 0, str); });
  hipError_t status = hipStreamAddCallback(s0, callback<char *>, str, flags);
  hipStreamAddCallback(s1, callback<char*>, str, flags);

  // CHECK: /*
  // CHECK-NEXT: DPCT1014:{{[0-9]+}}: The flag/priority options are not supported for SYCL queues; the output parameter(s) are set to 0.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((*(&flags) = 0, 0));
  checkCudaErrors(hipStreamGetFlags(s0, &flags));

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cudaStreamAttachMemAsync was removed. DPC++ currently doesn't support associating USM with a specific queue.
  // CHECK-NEXT: */
  hipStreamAttachMemAsync(s0, nullptr);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cudaStreamAttachMemAsync was replaced with 0. DPC++ currently doesn't support associating USM with a specific queue.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors(0);
  checkCudaErrors(hipStreamAttachMemAsync(s0, nullptr));

  hipEvent_t e;
  // CHECK; e.wait();
  hipStreamWaitEvent(s0, e, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cudaStreamQuery was removed. DPC++ currently doesn't support query operations on queues.
  // CHECK-NEXT: */
  hipStreamQuery(s0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cudaStreamQuery was replaced with 0. DPC++ currently doesn't support query operations on queues.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors(0);
  checkCudaErrors(hipStreamQuery(s0));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((e.wait(), 0));
  checkCudaErrors(hipStreamWaitEvent(s0, e, 0));

  // CHECK: s0->wait();
  hipStreamSynchronize(s0);
  // CHECK: checkCudaErrors((s1->wait(), 0));
  // CHECK-EMPTY:
  checkCudaErrors(hipStreamSynchronize(s1));

  // CHECK: delete s0;
  hipStreamDestroy(s0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkCudaErrors((delete s1, 0));
  checkCudaErrors(hipStreamDestroy(s1));
}
