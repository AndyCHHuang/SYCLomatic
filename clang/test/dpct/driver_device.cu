// RUN: dpct --format-range=none -out-root %T/driver_device %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/driver_device/driver_device.dp.cpp
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NUM 1
#define CUDA_SAFE_CALL( call) do {\
  int err = call;                \
} while (0)
int main(){
  int result1, result2;

  int *presult1 = &result1, *presult2 = &result2;
  // CHECK: int device;
  hipDevice_t device;

  // CHECK: int *pdevice = &device;
  hipDevice_t *pdevice = &device;

  // CHECK: device = 0;
  hipDeviceGet(&device, 0);

  // CHECK: device = NUM;
  hipDeviceGet(&device, NUM);

  // CHECK: *pdevice = 0;
  hipDeviceGet(pdevice, 0);

  // CHECK: *((int *)pdevice) = 0;
  hipDeviceGet((hipDevice_t *)pdevice, 0);

  // CHECK: CUDA_SAFE_CALL((device = 0, 0));
  CUDA_SAFE_CALL(hipDeviceGet(&device, 0));

  // CHECK: /*
  // CHECK-NEXT: DPCT1076:{{[0-9]+}}: The device attribute was not recognized by the Intel(R) DPC++ Compatibility Tool. You may need to adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuDeviceGetAttribute(&result1, attr, device);
  hipDeviceAttribute_t attr = hipDeviceAttributeMaxThreadsPerBlock;
  hipDeviceGetAttribute(&result1, attr, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_major_version();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMajor, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_minor_version();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMinor, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_integrated();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeIntegrated, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_max_clock_frequency();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeClockRate, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_max_compute_units();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).is_native_atomic_supported();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeHostNativeAtomicSupported, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxThreadsPerBlock, device);

  // CHECK: CUDA_SAFE_CALL((result1 = dpct::dev_mgr::instance().get_device(device).get_max_compute_units(), 0));
  CUDA_SAFE_CALL(hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device));

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_major_version();
  // CHECK: result2 = dpct::dev_mgr::instance().get_device(device).get_minor_version();
  hipDeviceComputeCapability(&result1, &result2, device);

  // CHECK: CUDA_SAFE_CALL([&](){
  // CHECK-NEXT:   result1 = dpct::dev_mgr::instance().get_device(device).get_major_version();
  // CHECK-NEXT:   result2 = dpct::dev_mgr::instance().get_device(device).get_minor_version();
  // CHECK-NEXT:   return 0;
  // CHECK-NEXT: }());
  CUDA_SAFE_CALL(hipDeviceComputeCapability(&result1, &result2, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1043:{{[0-9]+}}: The version-related API is different in SYCL. An initial code was generated, but you need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: result1 = dpct::get_current_device().get_info<sycl::info::device::version>();
  hipDriverGetVersion(&result1);

  // CHECK: CUDA_SAFE_CALL((result1 = dpct::dev_mgr::instance().device_count(), 0));
  CUDA_SAFE_CALL(hipGetDeviceCount(&result1));

  // CHECK: result1 = dpct::dev_mgr::instance().device_count();
  hipGetDeviceCount(&result1);

  // CHECK: CUDA_SAFE_CALL((result1 = dpct::dev_mgr::instance().device_count(), 0));
  CUDA_SAFE_CALL(hipGetDeviceCount(&result1));

  return 0;
}
