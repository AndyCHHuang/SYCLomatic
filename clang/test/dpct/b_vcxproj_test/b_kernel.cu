// UNSUPPORTED: -linux-
// RUN: cat %S/SVMbenchmark.vcxproj > %T/SVMbenchmark.vcxproj
// RUN: dpct --format-range=none  --vcxprojfile=%T/SVMbenchmark.vcxproj  -in-root=%S -out-root=%T  %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only

// RUN: cat %S/check_compilation_ref.txt  >%T/check_compilation_db.txt
// RUN: cat %T/compile_commands.json >>%T/check_compilation_db.txt

// RUN: FileCheck --match-full-lines --input-file %T/check_compilation_db.txt %T/check_compilation_db.txt


#include "hip/hip_runtime.h"
#include <stdio.h>

// CHECK: void addKernel(int *c, const int *a, const int *b, cl::sycl::nd_item<3> item_ct1)
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // CHECK: int i = item_ct1.get_local_id(0);
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

