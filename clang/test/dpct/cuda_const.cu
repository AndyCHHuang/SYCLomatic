// RUN: dpct --usm-level=none -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda_const.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_ELEMENTS 16
const unsigned num_elements = 16;
// CHECK: dpct::constant_memory<float, 1> const_angle(360);
// CHECK: dpct::constant_memory<float, 2> const_float(NUM_ELEMENTS, num_elements * 2);
__constant__ float const_angle[360], const_float[NUM_ELEMENTS][num_elements * 2];
// CHECK: dpct::constant_memory<cl::sycl::double2, 0> vec_d;
__constant__ double2 vec_d;

// CHECK:void simple_kernel(float *d_array, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<float, dpct::constant, 1> const_angle) {
// CHECK-NEXT:  int index;
// CHECK-NEXT:  index = [[ITEM]].get_group(0) * [[ITEM]].get_local_range().get(0) + [[ITEM]].get_local_id(0);
// CHECK-NEXT:  if (index < 360) {
// CHECK-NEXT:    d_array[index] = const_angle[index];
// CHECK-NEXT:  }
// CHECK-NEXT:  return;
// CHECK-NEXT:}
__global__ void simple_kernel(float *d_array) {
  int index;
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < 360) {
    d_array[index] = const_angle[index];
  }
  return;
}

// CHECK: dpct::constant_memory<float, 0> const_one;
__constant__ float const_one;

// CHECK:void simple_kernel_one(float *d_array, cl::sycl::nd_item<3> [[ITEM:item_ct1]], dpct::dpct_accessor<float, dpct::constant, 2> const_float, dpct::dpct_accessor<float, dpct::constant, 0> const_one) {
// CHECK-NEXT:  int index;
// CHECK-NEXT:  index = [[ITEM]].get_group(0) * [[ITEM]].get_local_range().get(0) + [[ITEM]].get_local_id(0);
// CHECK-NEXT:  if (index < 33) {
// CHECK-NEXT:    d_array[index] = (float)const_one + const_float[index][index];
// CHECK-NEXT:  }
// CHECK-NEXT:  return;
// CHECK-NEXT:}
__global__ void simple_kernel_one(float *d_array) {
  int index;
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < 33) {
    d_array[index] = const_one + const_float[index][index];
  }
  return;
}

int main(int argc, char **argv) {
  int size = 3200;
  float *d_array;
  float h_array[360];

  // CHECK: dpct::dpct_malloc((void **)&d_array, sizeof(float) * size);
  hipMalloc((void **)&d_array, sizeof(float) * size);

  // CHECK: dpct::dpct_memset((void*)(d_array), 0, sizeof(float) * size);
  hipMemset(d_array, 0, sizeof(float) * size);

  for (int loop = 0; loop < 360; loop++)
    h_array[loop] = acos(-1.0f) * loop / 180.0f;

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   (dpct::dpct_memcpy(const_angle.get_ptr(), (void*)(&h_array[0]), sizeof(float) * 360), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[0]), &h_array[0], sizeof(float) * 360);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   (dpct::dpct_memcpy(const_angle.get_ptr() + sizeof(float) * (3), (void*)(&h_array[0]), sizeof(float) * 357), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[3]), &h_array[0], sizeof(float) * 357);

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  (dpct::dpct_memcpy((void*)(&h_array[0]), const_angle.get_ptr() + sizeof(float) * (3), sizeof(float) * 357), 0);
  hipMemcpyFromSymbol(&h_array[0], HIP_SYMBOL(&const_angle[3]), sizeof(float) * 357);

  #define NUM 3
  // CHECK:/*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: (dpct::dpct_memcpy(const_angle.get_ptr() + sizeof(float) * (3+NUM), (void*)(&h_array[0]), sizeof(float) * 354), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[3+NUM]), &h_array[0], sizeof(float) * 354);

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  (dpct::dpct_memcpy((void*)(&h_array[0]), const_angle.get_ptr() + sizeof(float) * (3+NUM), sizeof(float) * 354), 0);
  hipMemcpyFromSymbol(&h_array[0], HIP_SYMBOL(&const_angle[3+NUM]), sizeof(float) * 354);
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_array);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto const_angle_acc_ct1 = const_angle.get_access(cgh);
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class simple_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(size / 64, 1, 1) * cl::sycl::range<3>(64, 1, 1)), cl::sycl::range<3>(64, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           float *arg_ct0 = (float *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           simple_kernel(arg_ct0, item_ct1, dpct::dpct_accessor<float, dpct::constant, 1>(const_angle_acc_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  simple_kernel<<<size / 64, 64>>>(d_array);

  float hangle_h[360];
  // CHECK:  dpct::dpct_memcpy((void*)(hangle_h), (void*)(d_array), 360 * sizeof(float), dpct::device_to_host);
  hipMemcpy(hangle_h, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 360; i++) {
    if (fabs(h_array[i] - hangle_h[i]) > 1e-5) {
      exit(-1);
    }
  }

  h_array[0] = 10.0f; // Just to test
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT:  (dpct::dpct_memcpy(const_one.get_ptr(), (void*)(&h_array[0]), sizeof(float) * 1), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_one), &h_array[0], sizeof(float) * 1);

  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(d_array);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto const_float_acc_ct1 = const_float.get_access(cgh);
  // CHECK-NEXT:       auto const_one_acc_ct1 = const_one.get_access(cgh);
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class simple_kernel_one_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(size / 64, 1, 1) * cl::sycl::range<3>(64, 1, 1)), cl::sycl::range<3>(64, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           float *arg_ct0 = (float *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           simple_kernel_one(arg_ct0, item_ct1, dpct::dpct_accessor<float, dpct::constant, 2>(const_float_acc_ct1), dpct::dpct_accessor<float, dpct::constant, 0>(const_one_acc_ct1));
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  simple_kernel_one<<<size / 64, 64>>>(d_array);

  hangle_h[360];
  // CHECK:  dpct::dpct_memcpy((void*)(hangle_h), (void*)(d_array), 360 * sizeof(float), dpct::device_to_host);
  hipMemcpy(hangle_h, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 1; i < 360; i++) {
    if (fabs(h_array[i] + 30.0f - hangle_h[i]) > 1e-5) {
      exit(-1);
    }
  }

  hipFree(d_array);

  printf("Test Passed!\n");
  return 0;
}
