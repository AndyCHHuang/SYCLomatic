// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda_const.dp.cpp


#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_ELEMENTS 16
const unsigned num_elements = 16;

class TestStruct {
public:
  __device__ void test() {}
};

// CHECK: dpct::constant_memory<TestStruct, 0> t1;
__constant__ TestStruct t1;

// CHECK: void member_acc(TestStruct t1) {
// CHECK-NEXT:  t1.test();
// CHECK-NEXT:}
__global__ void member_acc() {
  t1.test();
}
// CHECK: dpct::constant_memory<float, 1> const_angle(360);
// CHECK: dpct::constant_memory<float, 2> const_float(NUM_ELEMENTS, num_elements * 2);
__constant__ float const_angle[360], const_float[NUM_ELEMENTS][num_elements * 2];
// CHECK: dpct::constant_memory<sycl::double2, 0> vec_d;
__constant__ double2 vec_d;

// CHECK: dpct::device_memory<int, 1> const_ptr;
__constant__ int *const_ptr;

// CHECK:void simple_kernel(float *d_array, sycl::nd_item<3> [[ITEM:item_ct1]],
// CHECK-NEXT:              float *const_angle, int *const_ptr) {
// CHECK-NEXT:  int index;
// CHECK-NEXT:  index = [[ITEM]].get_group(2) * [[ITEM]].get_local_range().get(2) + [[ITEM]].get_local_id(2);
// CHECK-NEXT:  const_ptr[index] = index;
// CHECK-NEXT:  if (index < 360) {
// CHECK-NEXT:    d_array[index] = const_angle[index];
// CHECK-NEXT:  }
// CHECK-NEXT:  return;
// CHECK-NEXT:}
__global__ void simple_kernel(float *d_array) {
  int index;
  index = blockIdx.x * blockDim.x + threadIdx.x;
  const_ptr[index] = index;
  if (index < 360) {
    d_array[index] = const_angle[index];
  }
  return;
}

// CHECK: dpct::constant_memory<float, 0> const_one;
__constant__ float const_one;

// CHECK:void simple_kernel_one(float *d_array, sycl::nd_item<3> [[ITEM:item_ct1]],
// CHECK-NEXT:                  dpct::accessor<float, dpct::constant, 2> const_float,
// CHECK-NEXT:                  float const_one) {
// CHECK-NEXT:  int index;
// CHECK-NEXT:  index = [[ITEM]].get_group(2) * [[ITEM]].get_local_range().get(2) + [[ITEM]].get_local_id(2);
// CHECK-NEXT:  if (index < 33) {
// CHECK-NEXT:    d_array[index] = const_one + const_float[index][index];
// CHECK-NEXT:  }
// CHECK-NEXT:  return;
// CHECK-NEXT:}
__global__ void simple_kernel_one(float *d_array) {
  int index;
  index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < 33) {
    d_array[index] = const_one + const_float[index][index];
  }
  return;
}

int main(int argc, char **argv) {
  int size = 3200;
  int *d_int;
  float *d_array;
  float h_array[360];

  // CHECK: dpct::dpct_malloc((void **)&d_array, sizeof(float) * size);
  hipMalloc((void **)&d_array, sizeof(float) * size);
  // CHECK: dpct::dpct_malloc(&d_int, sizeof(int) * size);
  hipMalloc(&d_int, sizeof(int) * size);

  // CHECK: dpct::dpct_memset(d_array, 0, sizeof(float) * size);
  hipMemset(d_array, 0, sizeof(float) * size);

  for (int loop = 0; loop < 360; loop++)
    h_array[loop] = acos(-1.0f) * loop / 180.0f;

  // CHECK:   const_ptr.assign(d_int, sizeof(int) * size);
  hipMemcpyToSymbol(HIP_SYMBOL(const_ptr), &d_int, sizeof(int *));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   (dpct::dpct_memcpy(const_angle.get_ptr(), &h_array[0], sizeof(float) * 360), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[0]), &h_array[0], sizeof(float) * 360);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   (dpct::dpct_memcpy(const_angle.get_ptr() + sizeof(float) * (3), &h_array[0], sizeof(float) * 357), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[3]), &h_array[0], sizeof(float) * 357);

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  (dpct::dpct_memcpy(&h_array[0], const_angle.get_ptr() + sizeof(float) * (3), sizeof(float) * 357), 0);
  hipMemcpyFromSymbol(&h_array[0], HIP_SYMBOL(&const_angle[3]), sizeof(float) * 357);

  #define NUM 3
  // CHECK:/*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: (dpct::dpct_memcpy(const_angle.get_ptr() + sizeof(float) * (3+NUM), &h_array[0], sizeof(float) * 354), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_angle[3+NUM]), &h_array[0], sizeof(float) * 354);

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  (dpct::dpct_memcpy(&h_array[0], const_angle.get_ptr() + sizeof(float) * (3+NUM), sizeof(float) * 354), 0);
  hipMemcpyFromSymbol(&h_array[0], HIP_SYMBOL(&const_angle[3+NUM]), sizeof(float) * 354);

  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto t1_acc_ct1 = t1.get_access(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class member_acc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           member_acc(t1_acc_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  member_acc<<<1, 1>>>();
  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_array_buf_ct0 = dpct::get_buffer(d_array);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto const_angle_acc_ct1 = const_angle.get_access(cgh);
  // CHECK-NEXT:       auto const_ptr_acc_ct1 = const_ptr.get_access(cgh);
  // CHECK-NEXT:       auto d_array_acc_ct0 = d_array_buf_ct0.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class simple_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, size / 64) * sycl::range<3>(1, 1, 64), sycl::range<3>(1, 1, 64)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           simple_kernel((float *)(&d_array_acc_ct0[0]), item_ct1, const_angle_acc_ct1.get_pointer(), const_ptr_acc_ct1.get_pointer());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  simple_kernel<<<size / 64, 64>>>(d_array);

  float hangle_h[360];
  // CHECK:  dpct::dpct_memcpy(hangle_h, d_array, 360 * sizeof(float), dpct::device_to_host);
  hipMemcpy(hangle_h, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 360; i++) {
    if (fabs(h_array[i] - hangle_h[i]) > 1e-5) {
      exit(-1);
    }
  }

  h_array[0] = 10.0f; // Just to test
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT:  (dpct::dpct_memcpy(const_one.get_ptr(), &h_array[0], sizeof(float) * 1), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(&const_one), &h_array[0], sizeof(float) * 1);

  // CHECK: {
  // CHECK-NEXT:   dpct::buffer_t d_array_buf_ct0 = dpct::get_buffer(d_array);
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto const_float_acc_ct1 = const_float.get_access(cgh);
  // CHECK-NEXT:       auto const_one_acc_ct1 = const_one.get_access(cgh);
  // CHECK-NEXT:       auto d_array_acc_ct0 = d_array_buf_ct0.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class simple_kernel_one_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, size / 64) * sycl::range<3>(1, 1, 64), sycl::range<3>(1, 1, 64)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           simple_kernel_one((float *)(&d_array_acc_ct0[0]), item_ct1, dpct::accessor<float, dpct::constant, 2>(const_float_acc_ct1), const_one_acc_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  simple_kernel_one<<<size / 64, 64>>>(d_array);

  // CHECK:  dpct::dpct_memcpy(hangle_h, d_array, 360 * sizeof(float), dpct::device_to_host);
  hipMemcpy(hangle_h, d_array, 360 * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 1; i < 360; i++) {
    if (fabs(h_array[i] + 30.0f - hangle_h[i]) > 1e-5) {
      exit(-1);
    }
  }

  hipFree(d_array);

  printf("Test Passed!\n");
  return 0;
}


// CHECK: dpct::constant_memory<float, 0> C;
__constant__ float C;

// CHECK: void foo(float d, float y, float C){
// CHECK-NEXT:   float temp;
// CHECK-NEXT:   float maxtemp = sycl::fmax(temp=(y*d)<(y==1?C:0) ? -(3*y) :-10, (float)(-10));
// CHECK-NEXT: }
__global__ void foo(float d, float y){
  float temp;
  float maxtemp = fmaxf(temp=(y*d)<(y==1?C:0) ? -(3*y) :-10, -10);
}
