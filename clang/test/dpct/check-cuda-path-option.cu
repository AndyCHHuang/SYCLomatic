// RUN: dpct --cuda-path="%cuda-path" -out-root %T %s  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/check-cuda-path-option.dp.cpp --match-full-lines %s

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CHECK: void foo (int s){
void foo (hipblasStatus_t s){
}