#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/sync_api %s --cuda-include-path="%cuda-path/include" --use-experimental-features=nd_range_barrier -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sync_api/sync_api.dp.cpp

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include "hip/hip_cooperative_groups.h"
namespace cg = cooperative_groups;
using namespace cooperative_groups;

// CHECK: #define TB(b) auto b = item_ct1.get_group();
#define TB(b) cg::thread_block b = cg::this_thread_block();

__device__ void foo(int i) {}

#define FOO(x) foo(x)

// CHECK: void k(sycl::nd_item<3> item_ct1) {
__global__ void k() {
  // CHECK: auto cta = item_ct1.get_group();
  cg::thread_block cta = cg::this_thread_block();
  // CHECK: item_ct1.barrier();
  cg::sync(cta);

  // CHECK: auto block = item_ct1.get_group();
  cg::thread_block block = cg::this_thread_block();
  // CHECK: item_ct1.barrier();
  __syncthreads();
  // CHECK: item_ct1.barrier();
  block.sync();
  // CHECK: item_ct1.barrier();
  cg::sync(block);
  // CHECK: item_ct1.barrier();
  cg::this_thread_block().sync();
  // CHECK: item_ct1.barrier();
  cg::sync(cg::this_thread_block());

  // CHECK: auto b0 = item_ct1.get_group(), b1 = item_ct1.get_group();
  cg::thread_block b0 = cg::this_thread_block(), b1 = cg::this_thread_block();

  TB(blk);

  int p;
  // CHECK: /*
  // CHECK-NEXT: DPCT1078:{{[0-9]+}}: Consider replacing memory_order::acq_rel with memory_order::seq_cst for correctness if strong memory order restrictions are needed.
  // CHECK-NEXT: */
  // CHECK-NEXT: sycl::ext::oneapi::atomic_fence(sycl::ext::oneapi::memory_order::acq_rel, sycl::ext::oneapi::memory_scope::work_group);
  __threadfence_block();
  // CHECK: /*
  // CHECK-NEXT: DPCT1078:{{[0-9]+}}: Consider replacing memory_order::acq_rel with memory_order::seq_cst for correctness if strong memory order restrictions are needed.
  // CHECK-NEXT: */
  // CHECK-NEXT: sycl::ext::oneapi::atomic_fence(sycl::ext::oneapi::memory_order::acq_rel, sycl::ext::oneapi::memory_scope::device);
  __threadfence();
  // CHECK: /*
  // CHECK-NEXT: DPCT1078:{{[0-9]+}}: Consider replacing memory_order::acq_rel with memory_order::seq_cst for correctness if strong memory order restrictions are needed.
  // CHECK-NEXT: */
  // CHECK-NEXT: sycl::ext::oneapi::atomic_fence(sycl::ext::oneapi::memory_order::acq_rel, sycl::ext::oneapi::memory_scope::system);
  __threadfence_system();
  // CHECK: item_ct1.barrier();
  // CHECK-NEXT: sycl::all_of_group(item_ct1.get_group(), p);
  __syncthreads_and(p);
  // CHECK: item_ct1.barrier();
  // CHECK-NEXT: sycl::any_of_group(item_ct1.get_group(), p);
  __syncthreads_or(p);
  // CHECK: item_ct1.barrier();
  // CHECK-NEXT: sycl::reduce_over_group(item_ct1.get_group(), p == 0 ? 0 : 1, sycl::ext::oneapi::plus<>());
  __syncthreads_count(p);
  // CHECK: item_ct1.barrier();
  __syncwarp(0xffffffff);

  // CHECK: int a = (item_ct1.barrier(), sycl::all_of_group(item_ct1.get_group(), p));
  int a = __syncthreads_and(p);
  // CHECK: int b = (item_ct1.barrier(), sycl::any_of_group(item_ct1.get_group(), p));
  int b = __syncthreads_or(p);
  // CHECK: int c = (item_ct1.barrier(), sycl::reduce_over_group(item_ct1.get_group(), p == 0 ? 0 : 1, sycl::ext::oneapi::plus<>()));
  int c = __syncthreads_count(p);

  // CHECK: foo((item_ct1.barrier(), sycl::all_of_group(item_ct1.get_group(), p)));
  foo(__syncthreads_and(p));
  // CHECK: foo((item_ct1.barrier(), sycl::any_of_group(item_ct1.get_group(), p)));
  foo(__syncthreads_or(p));
  // CHECK: foo((item_ct1.barrier(), sycl::reduce_over_group(item_ct1.get_group(), p == 0 ? 0 : 1, sycl::ext::oneapi::plus<>())));
  foo(__syncthreads_count(p));

  // CHECK: FOO((item_ct1.barrier(), sycl::all_of_group(item_ct1.get_group(), p)));
  FOO(__syncthreads_and(p));
  // CHECK: FOO((item_ct1.barrier(), sycl::any_of_group(item_ct1.get_group(), p)));
  FOO(__syncthreads_or(p));
  // CHECK: FOO((item_ct1.barrier(), sycl::reduce_over_group(item_ct1.get_group(), p == 0 ? 0 : 1, sycl::ext::oneapi::plus<>())));
  FOO(__syncthreads_count(p));
}

// CHECK: void kernel(sycl::nd_item<3> item_ct1,
// CHECK-NEXT:            sycl::ext::oneapi::atomic_ref<unsigned int,sycl::ext::oneapi::memory_order::seq_cst,sycl::ext::oneapi::memory_scope::device,sycl::access::address_space::global_space> &sync_ct1) {
// CHECK-NEXT:  dpct::experimental::nd_range_barrier(item_ct1, sync_ct1);
// CHECK-NEXT:}
__global__ void kernel() {
  cg::grid_group grid = cg::this_grid();
  grid.sync();
}

int main() {
// CHECK:  {
// CHECK-NEXT:    dpct::global_memory<unsigned int, 0> d_sync_ct1(0);
// CHECK-NEXT:    unsigned *sync_ct1 = d_sync_ct1.get_ptr(dpct::get_default_queue());
// CHECK-NEXT:    dpct::get_default_queue().memset(sync_ct1, 0, sizeof(int)).wait();
// CHECK-NEXT:    dpct::get_default_queue().parallel_for(
// CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 2) * sycl::range<3>(1, 1, 2), sycl::range<3>(1, 1, 2)), 
// CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1)  {
// CHECK-NEXT:        auto atm_sync_ct1 = sycl::ext::oneapi::atomic_ref<unsigned int,sycl::ext::oneapi::memory_order::seq_cst,sycl::ext::oneapi::memory_scope::device,sycl::access::address_space::global_space>(sync_ct1[0]);
// CHECK-NEXT:        kernel(item_ct1, atm_sync_ct1);
// CHECK-NEXT:      }).wait();
// CHECK-NEXT:  }
  kernel<<<2, 2>>>();
  hipDeviceSynchronize();
  return 0;
}