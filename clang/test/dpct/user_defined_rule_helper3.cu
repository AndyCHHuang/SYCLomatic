// RUN: dpct --out-root %T/user_defined_rule_helper3 %s --cuda-include-path="%cuda-path/include" --rule-file %S/xpu_3.yaml --format-range=none --use-syclcompat
// RUN: FileCheck --input-file %T/user_defined_rule_helper3/user_defined_rule_helper3.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DNO_BUILD_TEST  %T/user_defined_rule_helper3/user_defined_rule_helper3.dp.cpp -o %T/user_defined_rule_helper3/user_defined_rule_helper3.dp.o %}

#ifndef NO_BUILD_TEST

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <syclcompat/syclcompat.hpp>
// CHECK-NEXT: #include "xpu_helper3.h"
#include <hip/hip_runtime.h>

__global__ void foo1_kernel() {}
void foo1() {
  // CHECK: static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream3()).parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)), 
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:     foo1_kernel();
  // CHECK-NEXT:   });
  foo1_kernel<<<1, 1>>>();
}

__global__ void foo2_kernel(double *d) {}

void foo2() {
  double *d;
  // CHECK: d = sycl::malloc_device<double>(1, static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream3()));
  // CHECK-NEXT: {
  // CHECK-NEXT:   syclcompat::has_capability_or_fail(static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream3()).get_device(), {sycl::aspect::fp64});
  // CHECK-EMPTY:
  // CHECK-NEXT:   static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream3()).parallel_for(
  // CHECK-NEXT:     sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)), 
  // CHECK-NEXT:     [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:       foo2_kernel(d);
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  // CHECK-NEXT: syclcompat::wait_and_free(d, static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream3()));
  hipMalloc(&d, sizeof(double));
  foo2_kernel<<<1, 1>>>(d);
  hipFree(d);
}

#endif
