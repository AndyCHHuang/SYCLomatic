// RUN: dpct --format-range=none --no-cl-namespace-inline --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -fno-delayed-template-parsing

// RUN: FileCheck --input-file %T/kernel-call.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

// CHECK: void helloFromGPUDDefaultArgs(int i, int j, int k,
// CHECK-NEXT:   cl::sycl::nd_item<3> item_ct1,
// CHECK-NEXT: int l = 0,
// CHECK-NEXT: int m = 0, int n = 0) {
// CHECK-NEXT: int a = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2) + item_ct1.get_group(2) +
// CHECK-NEXT: item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void helloFromGPUDDefaultArgs(int i, int j, int k,
  int l = 0,
  int m = 0, int n = 0) {
int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
blockDim.x + threadIdx.x;
}

// CHECK: void testKernel(int L, int M, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]], int N);
__global__ void testKernel(int L, int M, int N);

// CHECK: void testKernel(int L, int M, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]], int N = 0);
__global__ void testKernel(int L, int M, int N = 0);

// CHECK: void testKernelPtr(const int *L, const int *M, int N,
// CHECK-NEXT: cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(2) * [[ITEMNAME]].get_local_range().get(2) + [[ITEMNAME]].get_local_id(2);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}


// CHECK: // Test Launch Bounds
// CHECK-NEXT: void testKernel(int L, int M, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]], int N) {
__launch_bounds__(256, 512) // Test Launch Bounds
__global__ void testKernel(int L, int M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(2) * [[ITEMNAME]].get_local_range().get(2) + [[ITEMNAME]].get_local_id(2);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU(int i, cl::sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:     int a = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2) + item_ct1.get_group(2) +
// CHECK-NEXT:     item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void helloFromGPU(int i) {
  int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
          blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU(cl::sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:     int a = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2) + item_ct1.get_group(2) +
// CHECK-NEXT:     item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void helloFromGPU(void) {
  int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
          blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU2(cl::sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:     int a = item_ct1.get_group(2) * item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2) + item_ct1.get_group(2) +
// CHECK-NEXT:     item_ct1.get_local_range().get(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void helloFromGPU2() {
  int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
          blockDim.x + threadIdx.x;
}

void testReference(const int &i) {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  // CHECK:  /*
  // CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
  // CHECK-NEXT:  */
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(i, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  helloFromGPU<<<griddim, threaddim>>>(i);

}

struct TestThis {
  struct TestMember {
    int arg1, arg2;
  } args;
  int arg3;
  dim3 griddim, threaddim;
  void test() {
    /// Kernel function is called in method declaration, and fields are used as arguments.
    /// Check the miggration of implicit "this" pointer.
    // CHECK:  /*
    // CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
    // CHECK-NEXT:  */
    // CHECK-NEXT:  dpct::get_default_queue().submit(
    // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
    // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
    // CHECK-EMPTY:
    // CHECK-NEXT:       auto args_arg1_ct0 = args.arg1;
    // CHECK-NEXT:       auto args_arg2_ct1 = args.arg2;
    // CHECK-NEXT:       auto arg3_ct2 = arg3;
    // CHECK-EMPTY:
    // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
    // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
    // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
    // CHECK-NEXT:           testKernel(args_arg1_ct0, args_arg2_ct1, item_ct1, arg3_ct2);
    // CHECK-NEXT:         });
    // CHECK-NEXT:     });
    testKernel<<<griddim, threaddim>>>(args.arg1, args.arg2, arg3);
  }
};

int arr[16];

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: cl::sycl::queue &q_ct1 = dev_ct1.default_queue();
  dim3 griddim = 2;
  dim3 threaddim = 32;
  void *karg1 = 0;
  const int *karg2 = 0;
  int karg3 = 80;

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
  // CHECK-NEXT:  */
  // CHECK-NEXT: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg1_buf_ct0 = dpct::get_buffer_and_offset((const int *)karg1);
  // CHECK-NEXT:   size_t karg1_offset_ct0 = karg1_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg2_buf_ct1 = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t karg2_offset_ct1 = karg2_buf_ct1.second;
  // CHECK-NEXT:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto karg1_acc_ct0 = karg1_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto karg2_acc_ct1 = karg2_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const int *karg1_ct0 = (const int *)(&karg1_acc_ct0[0] + karg1_offset_ct0);
  // CHECK-NEXT:           const int *karg2_ct1 = (const int *)(&karg2_acc_ct1[0] + karg2_offset_ct1);
  // CHECK-NEXT:           testKernelPtr(karg1_ct0, karg2_ct1, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);

  int karg1int = 1;
  int karg2int = 2;
  int karg3int = 3;
  int intvar = 20;
  TestThis *args_p;
  // CHECK:  /*
  // CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
  // CHECK-NEXT:  */
  // CHECK-NEXT:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto args_p_arg3_ct1 = args_p->arg3;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 10) * cl::sycl::range<3>(1, 1, intvar), cl::sycl::range<3>(1, 1, intvar)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, args_p_arg3_ct1, item_ct1, karg3int);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  testKernel<<<10, intvar>>>(karg1int, args_p->arg3, karg3int);

  struct KernelPointer {
    const int *arg1, *arg2;
  } args;
  // CHECK: {
  // CHECK-NEXT:  std::pair<dpct::buffer_t, size_t> args_arg1_buf_ct0 = dpct::get_buffer_and_offset(args.arg1);
  // CHECK-NEXT:  size_t args_arg1_offset_ct0 = args_arg1_buf_ct0.second;
  // CHECK-NEXT:  std::pair<dpct::buffer_t, size_t> args_arg2_buf_ct1 = dpct::get_buffer_and_offset(args.arg2);
  // CHECK-NEXT:  size_t args_arg2_offset_ct1 = args_arg2_buf_ct1.second;
  // CHECK-NEXT:  q_ct1.submit(
  // CHECK-NEXT:    [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:      auto args_arg1_acc_ct0 = args_arg1_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:      auto args_arg2_acc_ct1 = args_arg2_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 2, 1), cl::sycl::range<3>(1, 2, 1)),
  // CHECK-NEXT:        [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:          const int *args_arg1_ct0 = (const int *)(&args_arg1_acc_ct0[0] + args_arg1_offset_ct0);
  // CHECK-NEXT:          const int *args_arg2_ct1 = (const int *)(&args_arg2_acc_ct1[0] + args_arg2_offset_ct1);
  // CHECK-NEXT:          testKernelPtr(args_arg1_ct0, args_arg2_ct1, karg3int, item_ct1);
  // CHECK-NEXT:        });
  // CHECK-NEXT:    });
  // CHECK-NEXT:}
  testKernelPtr<<<dim3(1), dim3(1, 2)>>>(args.arg1, args.arg2, karg3int);

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 2, 1) * cl::sycl::range<3>(3, 2, 1), cl::sycl::range<3>(3, 2, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, item_ct1, karg3int);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  testKernel<<<dim3(1, 2), dim3(1, 2, 3)>>>(karg1int, karg2int, karg3int);

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
  // CHECK-NEXT:  */
  // CHECK-NEXT:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arr_karg3int_ct2 = arr[karg3int];
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, griddim[0]) * cl::sycl::range<3>(1, 1, griddim[1] + 2), cl::sycl::range<3>(1, 1, griddim[1] + 2)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, item_ct1, arr_karg3int_ct2);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  testKernel <<<griddim.x, griddim.y + 2 >>>(karg1int, karg2int, arr[karg3int]);

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 2) * cl::sycl::range<3>(1, 1, 4), cl::sycl::range<3>(1, 1, 4)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(23, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  helloFromGPU <<<2, 4>>>(23);

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 2) * cl::sycl::range<3>(1, 1, 4), cl::sycl::range<3>(1, 1, 4)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  helloFromGPU <<<2, 4>>>();

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU2_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 2) * cl::sycl::range<3>(1, 1, 3), cl::sycl::range<3>(1, 1, 3)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU2(item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  helloFromGPU2 <<<2, 3>>>();

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
  // CHECK-NEXT:  */
  // CHECK-NEXT:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 2) * cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0)), cl::sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  helloFromGPU<<<2, threaddim>>>();

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(griddim.get(2), griddim.get(1), griddim.get(0)) * cl::sycl::range<3>(1, 1, 4), cl::sycl::range<3>(1, 1, 4)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  helloFromGPU<<<griddim, 4>>>();

  // CHECK: q_ct1.submit(
  // CHECK-NEXT:   [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class helloFromGPUDDefaultArgs_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:       cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 2) * cl::sycl::range<3>(1, 1, 4), cl::sycl::range<3>(1, 1, 4)),
  // CHECK-NEXT:       [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         helloFromGPUDDefaultArgs(1, 2, 3, item_ct1, 4, 5, 6);
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  helloFromGPUDDefaultArgs <<<2, 4>>>(1,2,3,4,5,6);
}

struct config {
  int b;
  struct subconfig {
    int d;
  } c;
};

// CHECK: void foo_kernel(int a, int b, int c) {}
__global__ void foo_kernel(int a, int b, int c) {}

class foo_class {
public:
  foo_class(int n) : a(n) {}

  // CHECK:  int run_foo() {
  // CHECK-NEXT:    dpct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:         auto a_ct0 = a;
  // CHECK-NEXT:         auto aa_b_ct1 = aa.b;
  // CHECK-NEXT:         auto aa_c_d_ct2 = aa.c.d;
  // CHECK-EMPTY:
  // CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class foo_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:            foo_kernel(a_ct0, aa_b_ct1, aa_c_d_ct2);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  int run_foo() {
    foo_kernel<<<1, 1>>>(a, aa.b, aa.c.d);
  }

private:
  int a;
  struct config aa;
};

__global__ void foo_kernel3(int *d) {
}
//CHECK:void run_foo(cl::sycl::range<3> c, cl::sycl::range<3> d) {
//CHECK-NEXT:  if (1)
//CHECK-NEXT:    {
//CHECK-NEXT:      std::pair<dpct::buffer_t, size_t> buf_ct0 = dpct::get_buffer_and_offset(0);
//CHECK-NEXT:      size_t offset_ct0 = buf_ct0.second;
//CHECK-NEXT:      dpct::get_default_queue().submit(
//CHECK-NEXT:        [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:          auto acc_ct0 = buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:          cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:            cl::sycl::nd_range<3>(cl::sycl::range<3>(c.get(2), c.get(1), c.get(0)), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:            [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:              int *ct0 = (int *)(&acc_ct0[0] + offset_ct0);
//CHECK-NEXT:              foo_kernel3(ct0);
//CHECK-NEXT:            });
//CHECK-NEXT:        });
//CHECK-NEXT:    }
//CHECK-NEXT:}
void run_foo(dim3 c, dim3 d) {
  if (1)
    foo_kernel3<<<c, 1>>>(0);
}
//CHECK:void run_foo2(cl::sycl::range<3> c, cl::sycl::range<3> d) {
//CHECK-NEXT:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
//CHECK-NEXT:  cl::sycl::queue &q_ct1 = dev_ct1.default_queue();
//CHECK-NEXT:  if (1)
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
//CHECK-NEXT:  */
//CHECK-NEXT:    {
//CHECK-NEXT:      std::pair<dpct::buffer_t, size_t> buf_ct0 = dpct::get_buffer_and_offset(0);
//CHECK-NEXT:      size_t offset_ct0 = buf_ct0.second;
//CHECK-NEXT:      q_ct1.submit(
//CHECK-NEXT:        [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:          auto acc_ct0 = buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:          auto dpct_global_range = c * d;
//CHECK-EMPTY:
//CHECK-NEXT:          cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:            cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(d.get(2), d.get(1), d.get(0))),
//CHECK-NEXT:            [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:              int *ct0 = (int *)(&acc_ct0[0] + offset_ct0);
//CHECK-NEXT:              foo_kernel3(ct0);
//CHECK-NEXT:            });
//CHECK-NEXT:        });
//CHECK-NEXT:    }
//CHECK-NEXT:  else
//CHECK-NEXT:    {
//CHECK-NEXT:      std::pair<dpct::buffer_t, size_t> buf_ct0 = dpct::get_buffer_and_offset(0);
//CHECK-NEXT:      size_t offset_ct0 = buf_ct0.second;
//CHECK-NEXT:      q_ct1.submit(
//CHECK-NEXT:        [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:          auto acc_ct0 = buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:          cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:            cl::sycl::nd_range<3>(cl::sycl::range<3>(c.get(2), c.get(1), c.get(0)), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:            [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:              int *ct0 = (int *)(&acc_ct0[0] + offset_ct0);
//CHECK-NEXT:              foo_kernel3(ct0);
//CHECK-NEXT:            });
//CHECK-NEXT:        });
//CHECK-NEXT:    }
//CHECK-NEXT:}
void run_foo2(dim3 c, dim3 d) {
  if (1)
    foo_kernel3<<<c, d>>>(0);
  else
    foo_kernel3<<<c, 1>>>(0);
}
//CHECK:void run_foo3(cl::sycl::range<3> c, cl::sycl::range<3> d) {
//CHECK-NEXT:  for (;;)
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
//CHECK-NEXT:  */
//CHECK-NEXT:    {
//CHECK-NEXT:      std::pair<dpct::buffer_t, size_t> buf_ct0 = dpct::get_buffer_and_offset(0);
//CHECK-NEXT:      size_t offset_ct0 = buf_ct0.second;
//CHECK-NEXT:      dpct::get_default_queue().submit(
//CHECK-NEXT:        [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:          auto acc_ct0 = buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:          auto dpct_global_range = c * d;
//CHECK-EMPTY:
//CHECK-NEXT:          cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:            cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(d.get(2), d.get(1), d.get(0))),
//CHECK-NEXT:            [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:              int *ct0 = (int *)(&acc_ct0[0] + offset_ct0);
//CHECK-NEXT:              foo_kernel3(ct0);
//CHECK-NEXT:            });
//CHECK-NEXT:        });
//CHECK-NEXT:    }
//CHECK-NEXT:}
void run_foo3(dim3 c, dim3 d) {
  for (;;)
    foo_kernel3<<<c, d>>>(0);
}
//CHECK:void run_foo4(cl::sycl::range<3> c, cl::sycl::range<3> d) {
//CHECK-NEXT: while (1)
//CHECK-NEXT:   {
//CHECK-NEXT:     std::pair<dpct::buffer_t, size_t> buf_ct0 = dpct::get_buffer_and_offset(0);
//CHECK-NEXT:     size_t offset_ct0 = buf_ct0.second;
//CHECK-NEXT:     dpct::get_default_queue().submit(
//CHECK-NEXT:       [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:          auto acc_ct0 = buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:         cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:           cl::sycl::nd_range<3>(cl::sycl::range<3>(c.get(2), c.get(1), c.get(0)), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:           [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:             int *ct0 = (int *)(&acc_ct0[0] + offset_ct0);
//CHECK-NEXT:             foo_kernel3(ct0);
//CHECK-NEXT:           });
//CHECK-NEXT:       });
//CHECK-NEXT:   }
//CHECK-NEXT:}
void run_foo4(dim3 c, dim3 d) {
 while (1)
   foo_kernel3<<<c, 1>>>(0);
}

//CHECK:dpct::shared_memory<float, 1> result(32);
//CHECK-NEXT:void my_kernel(float* result, cl::sycl::nd_item<3> item_ct1,
//CHECK-NEXT:               float *resultInGroup) {
//CHECK-NEXT:  // __shared__ variable
//CHECK-NEXT:  resultInGroup[item_ct1.get_local_id(2)] = item_ct1.get_group(2);
//CHECK-NEXT:  memcpy(&result[item_ct1.get_group(2)*8], resultInGroup, sizeof(float)*8);
//CHECK-NEXT:}
//CHECK-NEXT:int run_foo5 () {
//CHECK-NEXT:  {
//CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> result_buf_ct0 = dpct::get_buffer_and_offset(result.get_ptr());
//CHECK-NEXT:    size_t result_offset_ct0 = result_buf_ct0.second;
//CHECK-NEXT:    dpct::get_default_queue().submit(
//CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:        cl::sycl::accessor<float, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> resultInGroup_acc_ct1(cl::sycl::range<1>(8), cgh);
//CHECK-NEXT:        auto result_acc_ct0 = result_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
//CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 4) * cl::sycl::range<3>(1, 1, 8), cl::sycl::range<3>(1, 1, 8)),
//CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:            float *result_ct0 = (float *)(&result_acc_ct0[0] + result_offset_ct0);
//CHECK-NEXT:            my_kernel(result_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
//CHECK-NEXT:          });
//CHECK-NEXT:      });
//CHECK-NEXT:  }
//CHECK-NEXT:  printf("%f ", result[10]);
//CHECK-NEXT:}
 __managed__ float result[32];
__global__ void my_kernel(float* result) {
  __shared__ float resultInGroup[8]; // __shared__ variable
  resultInGroup[threadIdx.x] = blockIdx.x;
  memcpy(&result[blockIdx.x*8], resultInGroup, sizeof(float)*8);
}
int run_foo5 () {
  my_kernel<<<4, 8>>>(result);
  printf("%f ", result[10]);
}

//CHECK:dpct::shared_memory<float, 1> result2(32);
//CHECK-NEXT:int run_foo6 () {
//CHECK-NEXT:  {
//CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> result2_buf_ct0 = dpct::get_buffer_and_offset(result2.get_ptr());
//CHECK-NEXT:    size_t result2_offset_ct0 = result2_buf_ct0.second;
//CHECK-NEXT:    dpct::get_default_queue().submit(
//CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:        cl::sycl::accessor<float, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> resultInGroup_acc_ct1(cl::sycl::range<1>(8), cgh);
//CHECK-NEXT:        auto result2_acc_ct0 = result2_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
//CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 4) * cl::sycl::range<3>(1, 1, 8), cl::sycl::range<3>(1, 1, 8)),
//CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:            float *result2_ct0 = (float *)(&result2_acc_ct0[0] + result2_offset_ct0);
//CHECK-NEXT:            my_kernel(result2_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
//CHECK-NEXT:          });
//CHECK-NEXT:      });
//CHECK-NEXT:  }
//CHECK-NEXT:  printf("%f ", result2[10]);
//CHECK-NEXT:}
 __managed__ float result2[32];
int run_foo6 () {
  my_kernel<<<4, 8>>>(result2);
  printf("%f ", result2[10]);
}

//CHECK:dpct::shared_memory<float, 0> result3;
//CHECK-NEXT:int run_foo7 () {
//CHECK-NEXT:  {
//CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> result3_buf_ct0 = dpct::get_buffer_and_offset(result3.get_ptr());
//CHECK-NEXT:    size_t result3_offset_ct0 = result3_buf_ct0.second;
//CHECK-NEXT:    dpct::get_default_queue().submit(
//CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:        cl::sycl::accessor<float, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> resultInGroup_acc_ct1(cl::sycl::range<1>(8), cgh);
//CHECK-NEXT:        auto result3_acc_ct0 = result3_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
//CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 4) * cl::sycl::range<3>(1, 1, 8), cl::sycl::range<3>(1, 1, 8)),
//CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:            float *result3_ct0 = (float *)(&result3_acc_ct0[0] + result3_offset_ct0);
//CHECK-NEXT:            my_kernel(result3_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
//CHECK-NEXT:          });
//CHECK-NEXT:      });
//CHECK-NEXT:  }
//CHECK-NEXT:  printf("%f ", result3[0]);
//CHECK-NEXT:}
__managed__ float result3;
int run_foo7 () {
  my_kernel<<<4, 8>>>(&result3);
  printf("%f ", result3);
}

//CHECK:dpct::shared_memory<float, 0> in;
//CHECK-NEXT:dpct::shared_memory<float, 0> out;
//CHECK-NEXT:void my_kernel2(float in, float *out, cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:  if (item_ct1.get_local_id(2) == 0) {
//CHECK-NEXT:    memcpy(out, &in, sizeof(float));
//CHECK-NEXT:  }
//CHECK-NEXT:}
//CHECK-NEXT:int run_foo8() {
//CHECK-NEXT:  in[0] = 42;
//CHECK-NEXT:  {
//CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> out_buf_ct1 = dpct::get_buffer_and_offset(out.get_ptr());
//CHECK-NEXT:    size_t out_offset_ct1 = out_buf_ct1.second;
//CHECK-NEXT:    dpct::get_default_queue().submit(
//CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:        auto out_acc_ct1 = out_buf_ct1.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:        auto in_ct0 = in[0];
//CHECK-EMPTY:
//CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class my_kernel2_{{[0-9a-z]+}}>>(
//CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 4) * cl::sycl::range<3>(1, 1, 8), cl::sycl::range<3>(1, 1, 8)),
//CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:            float *out_ct1 = (float *)(&out_acc_ct1[0] + out_offset_ct1);
//CHECK-NEXT:            my_kernel2(in_ct0, out_ct1, item_ct1);
//CHECK-NEXT:          });
//CHECK-NEXT:      });
//CHECK-NEXT:  }
//CHECK-NEXT:  printf("%f ", out[0]);
//CHECK-NEXT:}

__managed__ float in;
__managed__ float out;
__global__ void my_kernel2(float in, float *out) {
  if (threadIdx.x == 0) {
    memcpy(out, &in, sizeof(float));
  }
}
int run_foo8() {
  in = 42;
  my_kernel2<<<4, 8>>>(in, &out);
  printf("%f ", out);
}

//CHECK: void deviceFoo(int i, int j, int k, cl::sycl::nd_item<3> item_ct1,
//CHECK-NEXT: int l = 0,
//CHECK-NEXT: int m = 0, int n = 0){
//CHECK-NEXT: int a = item_ct1.get_group(2);
//CHECK-NEXT: }
__device__ void deviceFoo(int i, int j, int k,
  int l = 0,
  int m = 0, int n = 0){
  int a = blockIdx.x;
}


//CHECK: void deviceFoo2(cl::sycl::nd_item<3> item_ct1, int i = 0, int j = 0){
//CHECK-NEXT:   int a = item_ct1.get_group(2);
//CHECK-NEXT: }
__device__ void deviceFoo2(int i = 0, int j = 0){
  int a = blockIdx.x;
}

//CHECK: void callDeviceFoo(cl::sycl::nd_item<3> item_ct1){
//CHECK-NEXT:   deviceFoo(1,2,3, item_ct1,4,5,6);
//CHECK-NEXT:   deviceFoo2(item_ct1, 1,2);
//CHECK-NEXT: }
__global__ void callDeviceFoo(){
  deviceFoo(1,2,3,4,5,6);
  deviceFoo2(1,2);
}

struct A{
  int a;
  int* get_pointer(){
    return &a;
  }
};

__global__ void k(int *p){}

//CHECK:int run_foo9() {
//CHECK-NEXT:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
//CHECK-NEXT:  cl::sycl::queue &q_ct1 = dev_ct1.default_queue();
//CHECK-NEXT:  std::vector<A> vec(10);
//CHECK-NEXT:  A aa;
//CHECK-NEXT:  {
//CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> aa_get_pointer_buf_ct0 = dpct::get_buffer_and_offset(aa.get_pointer());
//CHECK-NEXT:    size_t aa_get_pointer_offset_ct0 = aa_get_pointer_buf_ct0.second;
//CHECK-NEXT:    q_ct1.submit(
//CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:        auto aa_get_pointer_acc_ct0 = aa_get_pointer_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class k_{{[0-9a-z]+}}>>(
//CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:            int *aa_get_pointer_ct0 = (int *)(&aa_get_pointer_acc_ct0[0] + aa_get_pointer_offset_ct0);
//CHECK-NEXT:            k(aa_get_pointer_ct0);
//CHECK-NEXT:          });
//CHECK-NEXT:      });
//CHECK-NEXT:  }
//CHECK-NEXT:  {
//CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> vec_get_pointer_buf_ct0 = dpct::get_buffer_and_offset(vec[2].get_pointer());
//CHECK-NEXT:    size_t vec_get_pointer_offset_ct0 = vec_get_pointer_buf_ct0.second;
//CHECK-NEXT:    q_ct1.submit(
//CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:        auto vec_get_pointer_acc_ct0 = vec_get_pointer_buf_ct0.first.get_access<cl::sycl::access::mode::read_write>(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class k_{{[0-9a-z]+}}>>(
//CHECK-NEXT:          cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:          [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:            int *vec_get_pointer_ct0 = (int *)(&vec_get_pointer_acc_ct0[0] + vec_get_pointer_offset_ct0);
//CHECK-NEXT:            k(vec_get_pointer_ct0);
//CHECK-NEXT:          });
//CHECK-NEXT:      });
//CHECK-NEXT:  }
//CHECK-NEXT:}
int run_foo9() {
  std::vector<A> vec(10);
  A aa;
  k<<<1,1>>>(aa.get_pointer());
  k<<<1,1>>>(vec[2].get_pointer());
}

//CHECK:void cuda_pme_forces_dev(float **afn_s) {
//CHECK-NEXT:  // __shared__ variable
//CHECK-NEXT:}
//CHECK-NEXT:int run_foo10() {
//CHECK-NEXT: dpct::get_default_queue().submit(
//CHECK-NEXT:   [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:     cl::sycl::accessor<float *, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> afn_s_acc_ct1(cl::sycl::range<1>(3), cgh);
//CHECK-EMPTY:
//CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class cuda_pme_forces_dev_{{[0-9a-z]+}}>>(
//CHECK-NEXT:       cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:       [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:         cuda_pme_forces_dev(afn_s_acc_ct1.get_pointer());
//CHECK-NEXT:       });
//CHECK-NEXT:   });
//CHECK-NEXT:}
__global__ void cuda_pme_forces_dev() {
  __shared__ float *afn_s[3]; // __shared__ variable
}
int run_foo10() {
  cuda_pme_forces_dev<<<1,1>>>();
}

struct test_class {
  __device__ test_class() = default;
  // CHECK: test_class(int *a, cl::sycl::nd_item<3> item_ct1, int *s1) {
  // CHECK-NEXT:  // __shared__ variable
  // CHECK-NEXT:   s1[0] = item_ct1.get_local_range().get(2);
  // CHECK-NEXT: }
  // CHECK-NEXT: test_class(int *a, int *b, cl::sycl::nd_item<3> item_ct1, float *s2) {
  // CHECK-NEXT:  // __shared__ variable
  // CHECK-NEXT:   int d = item_ct1.get_local_range().get(2);
  // CHECK-NEXT: }
  // CHECK-NEXT: template<class T>
  // CHECK-NEXT: test_class(T *a, T *b, cl::sycl::nd_item<3> item_ct1, T *s3) {
  // CHECK-NEXT:  // __shared__ variable
  // CHECK-NEXT:   int d = item_ct1.get_local_range().get(2);
  // CHECK-NEXT: }
  __device__ test_class(int *a) {
    __shared__ int s1[10]; // __shared__ variable
    s1[0] = blockDim.x;
  }
  __device__ test_class(int *a, int *b) {
    __shared__ float s2; // __shared__ variable
    int d = blockDim.x;
  }
  template<class T>
  __device__ test_class(T *a, T *b) {
    __shared__ T s3; // __shared__ variable
    int d = blockDim.x;
  }
};

// CHECK: void kernel_ctor(cl::sycl::nd_item<3> item_ct1, int *s1, float *s2, float *s3) {
// CHECK-NEXT:   float *fa, *fb;
// CHECK-NEXT:   int *la, *lb;
// CHECK-NEXT:   test_class tc(la, item_ct1, s1);
// CHECK-NEXT:   tc = test_class(la, lb, item_ct1, s2);
// CHECK-NEXT:   tc = test_class(fa, fb, item_ct1, s3);
// CHECK-NEXT: }
__global__ void kernel_ctor() {
  float *fa, *fb;
  int *la, *lb;
  test_class tc(la);
  tc = test_class(la, lb);
  tc = test_class(fa, fb);
}

void test_ctor() {
  // CHECK: dpct::get_default_queue().submit(
  // CHECK-NEXT:   [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:     cl::sycl::accessor<int, 1, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s1_acc_ct1(cl::sycl::range<1>(10), cgh);
  // CHECK-NEXT:     cl::sycl::accessor<float, 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s2_acc_ct1(cgh);
  // CHECK-NEXT:     cl::sycl::accessor<float, 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> s3_acc_ct1(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class kernel_ctor_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:       cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:       [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         kernel_ctor(item_ct1, s1_acc_ct1.get_pointer(), s2_acc_ct1.get_pointer(), (float *)s3_acc_ct1.get_pointer());
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });

  kernel_ctor<<<1,1>>>();
}

//CHECK:template <typename T>
//CHECK-NEXT:void k11(T a, uint8_t *temp_ct1, uint8_t *temp2_ct1){
//CHECK-NEXT:union  type_ct1{
//CHECK-NEXT:    T up;
//CHECK-NEXT:  };
//CHECK-NEXT:  type_ct1* temp = (type_ct1*)temp_ct1;
//CHECK-NEXT:  type_ct1* temp2 = (type_ct1*)temp2_ct1;
//CHECK-NEXT:  temp->up = a;
//CHECK-NEXT:  temp2->up = a;
//CHECK-NEXT:}
//CHECK-NEXT:template<typename TT>
//CHECK-NEXT:void foo11() {
//CHECK-NEXT:  TT a;
//CHECK-NEXT:  dpct::get_default_queue().submit(
//CHECK-NEXT:    [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:      /*
//CHECK-NEXT:      DPCT1054:{{[0-9]+}}: The type of variable temp is declared in device function with the name type_ct1. Adjust the code to make the type_ct1 declaration visible at the accessor declaration point.
//CHECK-NEXT:      */
//CHECK-NEXT:      cl::sycl::accessor<uint8_t[sizeof(type_ct1)], 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> temp_ct1_acc_ct1(cgh);
//CHECK-NEXT:      /*
//CHECK-NEXT:      DPCT1054:{{[0-9]+}}: The type of variable temp2 is declared in device function with the name type_ct1. Adjust the code to make the type_ct1 declaration visible at the accessor declaration point.
//CHECK-NEXT:      */
//CHECK-NEXT:      cl::sycl::accessor<uint8_t[sizeof(type_ct1)], 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> temp2_ct1_acc_ct1(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class k11_{{[0-9a-z]+}}, TT>>(
//CHECK-NEXT:        cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:        [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          k11<TT>(a, temp_ct1_acc_ct1.get_pointer(), temp2_ct1_acc_ct1.get_pointer());
//CHECK-NEXT:        });
//CHECK-NEXT:    });
//CHECK-NEXT:}
template <typename T>
__global__ void k11(T a){
__shared__ union {
    T up;
  } temp, temp2;
  temp.up = a;
  temp2.up = a;
}
template<typename TT>
void foo11() {
  TT a;
  k11<TT><<<1,1>>>(a);
}

//CHECK:template <typename T>
//CHECK-NEXT:void k12(T a, uint8_t *temp_ct1, uint8_t *temp2_ct1){
//CHECK-NEXT:  union UnionType {
//CHECK-NEXT:    T up;
//CHECK-NEXT:  };
//CHECK-NEXT:  UnionType* temp = (UnionType*)temp_ct1;
//CHECK-NEXT:  //shared variable
//CHECK-NEXT:  temp->up = a;
//CHECK-NEXT:  union  type_ct2{
//CHECK-NEXT:    T up;
//CHECK-NEXT:  };
//CHECK-NEXT:  type_ct2* temp2 = (type_ct2*)temp2_ct1;
//CHECK-NEXT:  temp2->up = a;
//CHECK-NEXT:}
//CHECK-NEXT:template<typename TT>
//CHECK-NEXT:void foo2() {
//CHECK-NEXT:  TT a;
//CHECK-NEXT:  dpct::get_default_queue().submit(
//CHECK-NEXT:    [&](cl::sycl::handler &cgh) {
//CHECK-NEXT:      /*
//CHECK-NEXT:      DPCT1054:{{[0-9]+}}: The type of variable temp is declared in device function with the name UnionType. Adjust the code to make the UnionType declaration visible at the accessor declaration point.
//CHECK-NEXT:      */
//CHECK-NEXT:      cl::sycl::accessor<uint8_t[sizeof(UnionType)], 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> temp_ct1_acc_ct1(cgh);
//CHECK-NEXT:      /*
//CHECK-NEXT:      DPCT1054:{{[0-9]+}}: The type of variable temp2 is declared in device function with the name type_ct2. Adjust the code to make the type_ct2 declaration visible at the accessor declaration point.
//CHECK-NEXT:      */
//CHECK-NEXT:      cl::sycl::accessor<uint8_t[sizeof(type_ct2)], 0, cl::sycl::access::mode::read_write, cl::sycl::access::target::local> temp2_ct1_acc_ct1(cgh);
//CHECK-EMPTY:
//CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class k12_{{[0-9a-z]+}}, TT>>(
//CHECK-NEXT:        cl::sycl::nd_range<3>(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:        [=](cl::sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          k12<TT>(a, temp_ct1_acc_ct1.get_pointer(), temp2_ct1_acc_ct1.get_pointer());
//CHECK-NEXT:        });
//CHECK-NEXT:    });
//CHECK-NEXT:}
template <typename T>
__global__ void k12(T a){
  union UnionType {
    T up;
  };
  __shared__ UnionType temp;//shared variable
  temp.up = a;
  __shared__ union {
    T up;
  } temp2;
  temp2.up = a;
}
template<typename TT>
void foo2() {
  TT a;
  k12<TT><<<1,1>>>(a);
}
