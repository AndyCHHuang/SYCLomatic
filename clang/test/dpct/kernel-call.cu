
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/kernel-call.dp.cpp --match-full-lines %s

// CHECK: void testKernel(int L, int M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]);
__global__ void testKernel(int L, int M, int N);

// CHECK: void testKernelPtr(const int *L, const int *M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

// CHECK: void testKernel(int L, int M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernel(int L, int M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU(int i, cl::sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:     int a = item_ct1.get_group(0) * item_ct1.get_local_range().get(0) + item_ct1.get_local_id(0) + item_ct1.get_group(0) +
// CHECK-NEXT:     item_ct1.get_local_range().get(0) + item_ct1.get_local_id(0);
// CHECK-NEXT: }
__global__ void helloFromGPU(int i) {
  int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
          blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU(cl::sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:     int a = item_ct1.get_group(0) * item_ct1.get_local_range().get(0) + item_ct1.get_local_id(0) + item_ct1.get_group(0) +
// CHECK-NEXT:     item_ct1.get_local_range().get(0) + item_ct1.get_local_id(0);
// CHECK-NEXT: }
__global__ void helloFromGPU(void) {
  int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
          blockDim.x + threadIdx.x;
}

// CHECK: void helloFromGPU2(cl::sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:     int a = item_ct1.get_group(0) * item_ct1.get_local_range().get(0) + item_ct1.get_local_id(0) + item_ct1.get_group(0) +
// CHECK-NEXT:     item_ct1.get_local_range().get(0) + item_ct1.get_local_id(0);
// CHECK-NEXT: }
__global__ void helloFromGPU2() {
  int a = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.x +
          blockDim.x + threadIdx.x;
}

void testReference(const int &i) {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(i, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  helloFromGPU<<<griddim, threaddim>>>(i);

}

struct TestThis {
  struct TestMember {
    int arg1, arg2;
  } args;
  int arg3;
  dim3 griddim, threaddim;
  void test() {
    /// Kernel function is called in method declaration, and fields are used as arguments.
    /// Check the miggration of implicit "this" pointer.
    // CHECK: {
    // CHECK-NEXT:   dpct::get_default_queue().submit(
    // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
    // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
    // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
    // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
    // CHECK-NEXT:           testKernel(args.arg1, args.arg2, arg3, item_ct1);
    // CHECK-NEXT:         });
    // CHECK-NEXT:     });
    // CHECK-NEXT: }
    testKernel<<<griddim, threaddim>>>(args.arg1, args.arg2, arg3);
  }
};

int main() {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  void *karg1 = 0;
  const int *karg2 = 0;
  int karg3 = 80;
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset((const int *)karg1);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const int *arg_ct0 = (const int *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           const int *arg_ct1 = (const int *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr(arg_ct0, arg_ct1, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);

  int karg1int = 1;
  int karg2int = 2;
  int karg3int = 3;
  int intvar = 20;
  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(10, 1, 1) * cl::sycl::range<3>(intvar, 1, 1)), cl::sycl::range<3>(intvar, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<<<10, intvar>>>(karg1int, karg2int, karg3int);

  struct KernelPointer {
    const int *arg1, *arg2;
  } args;
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct0_buf = dpct::get_buffer_and_offset(args.arg1);
  // CHECK-NEXT:   size_t arg_ct0_offset = arg_ct0_buf.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> arg_ct1_buf = dpct::get_buffer_and_offset(args.arg2);
  // CHECK-NEXT:   size_t arg_ct1_offset = arg_ct1_buf.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto arg_ct0_acc = arg_ct0_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto arg_ct1_acc = arg_ct1_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 2, 1)), cl::sycl::range<3>(1, 2, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const int *arg_ct0 = (const int *)(&arg_ct0_acc[0] + arg_ct0_offset);
  // CHECK-NEXT:           const int *arg_ct1 = (const int *)(&arg_ct1_acc[0] + arg_ct1_offset);
  // CHECK-NEXT:           testKernelPtr(arg_ct0, arg_ct1, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<<<dim3(1), dim3(1, 2)>>>(args.arg1, args.arg2, karg3int);

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 2, 1) * cl::sycl::range<3>(1, 2, 3)), cl::sycl::range<3>(1, 2, 3)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel<<<dim3(1, 2), dim3(1, 2, 3)>>>(karg1int, karg2int, karg3int);

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(griddim[0], 1, 1) * cl::sycl::range<3>(griddim[1] + 2, 1, 1)), cl::sycl::range<3>(griddim[1] + 2, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernel(karg1int, karg2int, karg3int, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernel <<<griddim.x, griddim.y + 2 >>>(karg1int, karg2int, karg3int);

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(2, 1, 1) * cl::sycl::range<3>(4, 1, 1)), cl::sycl::range<3>(4, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(23, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  helloFromGPU <<<2, 4>>>(23);

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(2, 1, 1) * cl::sycl::range<3>(4, 1, 1)), cl::sycl::range<3>(4, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU(item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  helloFromGPU <<<2, 4>>>();

  // CHECK: {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class helloFromGPU2_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>((cl::sycl::range<3>(2, 1, 1) * cl::sycl::range<3>(3, 1, 1)), cl::sycl::range<3>(3, 1, 1)),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           helloFromGPU2(item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  helloFromGPU2 <<<2, 3>>>();
}
