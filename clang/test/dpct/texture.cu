#include "hip/hip_runtime.h"
// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture.dp.cpp --match-full-lines %s

// CHECK: dpct::dpct_image<cl::sycl::float4, 2> tex42;
static texture<float4, 2> tex42;
// CHECK: dpct::dpct_image<cl::sycl::uint2, 1> tex21;
static texture<uint2, 1> tex21;
/// TODO: Expect to support 3D array in future.
// TODO-CHECK: dpct::dpct_image<int, 3> tex13;
// static texture<int, 3> tex13;

// CHECK: void device01(dpct::dpct_image_accessor<cl::sycl::uint2, 1> tex21) {
// CHECK-NEXT: cl::sycl::uint2 u21 = dpct::dpct_read_image(tex21, 1.0f);
// CHECK-NEXT: cl::sycl::uint2 u21_fetch = dpct::dpct_read_image(tex21, 1);
__device__ void device01() {
  uint2 u21 = tex1D(tex21, 1.0f);
  uint2 u21_fetch = tex1Dfetch(tex21, 1);
}

// CHECK: void kernel(dpct::dpct_image_accessor<cl::sycl::float4, 2> tex42, dpct::dpct_image_accessor<cl::sycl::uint2, 1> tex21) {
// CHECK-NEXT: device01(tex21);
// CHECK-NEXT: cl::sycl::float4 f42 = dpct::dpct_read_image(tex42, 1.0f, 1.0f);
/// Texture accessors should be passed down to __global__/__device__ function if used.
__global__ void kernel() {
  device01();
  float4 f42 = tex2D(tex42, 1.0f, 1.0f);
}

int main() {

  // CHECK: cl::sycl::float4 *d_data42;
  // CHECK-NEXT: dpct::dpct_matrix_p a42;
  // CHECK-NEXT: dpct::dpct_malloc(&d_data42, sizeof(cl::sycl::float4) * 32 * 32);
  // CHECK-NEXT: dpct::dpct_image_channel desc42 = dpct::create_image_channel(32, 32, 32, 32, dpct::channel_float);
  // CHECK-NEXT: dpct::dpct_malloc_matrix(&a42, &desc42, 32, 32);
  // CHECK-NEXT: dpct::dpct_memcpy_to_matrix(a42, 0, 0, d_data42, 32 * 32 * sizeof(cl::sycl::float4));
  // CHECK-NEXT: tex42.addr_mode() = cl::sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: tex42.addr_mode() = cl::sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: tex42.addr_mode() = cl::sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: tex42.filter_mode() = cl::sycl::filtering_mode::nearest;
  // CHECK-NEXT: dpct::dpct_attach_image(tex42, a42);
  float4 *d_data42;
  hipArray_t a42;
  hipMalloc(&d_data42, sizeof(float4) * 32 * 32);
  hipChannelFormatDesc desc42 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
  hipMallocArray(&a42, &desc42, 32, 32);
  hipMemcpyToArray(a42, 0, 0, d_data42, 32 * 32 * sizeof(float4), hipMemcpyDeviceToDevice);
  tex42.addressMode[0] = hipAddressModeClamp;
  tex42.addressMode[1] = hipAddressModeClamp;
  tex42.addressMode[2] = hipAddressModeClamp;
  tex42.filterMode = hipFilterModePoint;
  hipBindTextureToArray(tex42, a42, desc42);

  // CHECK: cl::sycl::uint2 *d_data21;
  // CHECK-NEXT: dpct::dpct_malloc(&d_data21, sizeof(cl::sycl::uint2) * 32);
  // CHECK-NEXT: dpct::dpct_image_channel desc21 = dpct::create_image_channel(32, 32, 0, 0, dpct::channel_unsigned);
  // CHECK-NEXT: tex21.addr_mode() = cl::sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: tex21.addr_mode() = cl::sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: tex21.addr_mode() = cl::sycl::addressing_mode::clamp_to_edge;
  // CHECK-NEXT: tex21.filter_mode() = cl::sycl::filtering_mode::linear;
  // CHECK-NEXT: dpct::dpct_attach_image(tex21, d_data21, desc21, 32 * sizeof(cl::sycl::uint2));
  uint2 *d_data21;
  hipMalloc(&d_data21, sizeof(uint2) * 32);
  hipChannelFormatDesc desc21 = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindUnsigned);
  tex21.addressMode[0] = hipAddressModeClamp;
  tex21.addressMode[1] = hipAddressModeClamp;
  tex21.addressMode[2] = hipAddressModeClamp;
  tex21.filterMode = hipFilterModeLinear;
  hipBindTexture(0, tex21, d_data21, desc21, 32 * sizeof(uint2));

  // CHECK:   {
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:       [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:         auto tex42_acc = tex42.get_access(cgh);
  // CHECK-NEXT:         auto tex21_acc = tex21.get_access(cgh);
  // CHECK-NEXT:         cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:             cl::sycl::nd_range<3>((cl::sycl::range<3>(1, 1, 1) * cl::sycl::range<3>(1, 1, 1)), cl::sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:             [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:               kernel(tex42_acc, tex21_acc);
  // CHECK-NEXT:             });
  // CHECK-NEXT:       });
  // CHECK-NEXT: }
  kernel<<<1, 1>>>();

  // CHECK: dpct::dpct_detach_image(tex42);
  // CHECK-NEXT: dpct::dpct_detach_image(tex21);
  hipUnbindTexture(tex42);
  hipUnbindTexture(tex21);

  // CHECK: dpct::dpct_free(a42);
  hipFreeArray(a42);
 
  // CHECK: dpct::dpct_free(d_data42);
  // CHECK-NEXT: dpct::dpct_free(d_data21);
  hipFree(d_data42);
  hipFree(d_data21);
}
