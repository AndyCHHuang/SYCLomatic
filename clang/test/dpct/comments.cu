#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" --comments -- -std=c++14  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/comments.dp.cpp

static texture<uint2, 1> tex21;

__constant__ int a = 1;
__device__ int b[36][36];

__device__ void test() {
  __shared__ int c[36];
  c[0] = b[0][0] + a;
}

__global__ void kernel() {
  test();
  __device__ uint2 a[16];
  __shared__ int b[12][12];
  a[0] = tex1D(tex21, 1);
  b[0][0] = 0;
  printf("test\n");
}

int main() {
    // CHECK: // These variables are defined for 3d matrix memory copy.
    // CHECK-NEXT: dpct::pitched_data p_from_data_ct1, p_to_data_ct1;
    // CHECK-NEXT: sycl::id<3> p_from_pos_ct1(0, 0, 0), p_to_pos_ct1(0, 0, 0);
    // CHECK-NEXT: sycl::range<3> p_size_ct1(0, 0, 0);
    // CHECK-NEXT: dpct::memcpy_direction p_direction_ct1;
    hipMemcpy3DParms p;
    dim3 griddim(1, 2, 3);
    dim3 threaddim(1, 2, 3);

// CHECK:    dpct::get_default_queue().submit(
// CHECK-NEXT:        [&](sycl::handler &cgh) {
// CHECK-NEXT:          sycl::stream stream_ct1(64 * 1024, 80, cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          dpct::device_memory<sycl::uint2, 1> a(16);
// CHECK-EMPTY:
// CHECK-NEXT:          // ranges used for accessors to device memory
// CHECK-NEXT:          sycl::range<2> b_range_ct1(12, 12);
// CHECK-EMPTY:  
// CHECK-NEXT:          // pointers to device memory
// CHECK-NEXT:          auto a_ptr_ct1 = a.get_ptr();
// CHECK-NEXT:          auto a_ptr_ct1 = a.get_ptr();
// CHECK-EMPTY:  
// CHECK-NEXT:          // accessors to device memory
// CHECK-NEXT:          sycl::accessor<int, 1, sycl::access::mode::read_write, sycl::access::target::local> c_acc_ct1(sycl::range<1>(36), cgh);
// CHECK-NEXT:          sycl::accessor<int, 2, sycl::access::mode::read_write, sycl::access::target::local> b_acc_ct1(b_range_ct1, cgh);
// CHECK-NEXT:          auto b_acc_ct1 = b.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          // accessors to image wrappers
// CHECK-NEXT:          auto tex21_acc = tex21.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          // ranges to define ND iteration space for the kernel
// CHECK-NEXT:          auto dpct_global_range = griddim * threaddim;
// CHECK-EMPTY:  
// CHECK-NEXT:          // run the kernel within defined ND range
// CHECK-NEXT:          cgh.parallel_for(
// CHECK-NEXT:            sycl::nd_range<3>(sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
// CHECK-NEXT:            [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:              kernel(stream_ct1, *a_ptr_ct1, b_acc_ct1, c_acc_ct1.get_pointer(), a_ptr_ct1, dpct::accessor<int, dpct::local, 2>(b_acc_ct1, b_range_ct1), tex21_acc);
// CHECK-NEXT:            });
// CHECK-NEXT:        });
    kernel<<<griddim, threaddim>>>();
}