#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" --comments -- -std=c++14  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/comments.dp.cpp

static texture<uint2, 1> tex21;

__constant__ int a = 1;
__device__ int b[36][36];

__device__ void test() {
  __shared__ int cl[36];
  cl[0] = b[0][0] + a;
}

__global__ void kernel() {
  test();
  __device__ uint2 al[16];
  __shared__ int bl[12][12];
  al[0] = tex1D(tex21, 1);
  bl[0][0] = 0;
  printf("test\n");
}

int main() {
    // CHECK: // These variables are defined for 3d matrix memory copy.
    // CHECK-NEXT: dpct::pitched_data p_from_data_ct1, p_to_data_ct1;
    // CHECK-NEXT: sycl::id<3> p_from_pos_ct1(0, 0, 0), p_to_pos_ct1(0, 0, 0);
    // CHECK-NEXT: sycl::range<3> p_size_ct1(0, 0, 0);
    // CHECK-NEXT: dpct::memcpy_direction p_direction_ct1;
    hipMemcpy3DParms p;
    dim3 griddim(1, 2, 3);
    dim3 threaddim(1, 2, 3);

// CHECK:    dpct::get_default_queue().submit(
// CHECK-NEXT:        [&](sycl::handler &cgh) {
// CHECK-NEXT:          sycl::stream stream_ct1(64 * 1024, 80, cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          dpct::device_memory<sycl::uint2, 1> al(16);
// CHECK-EMPTY:  
// CHECK-NEXT:          // init global memory
// CHECK-NEXT:          al.init();
// CHECK-NEXT:          a.init();
// CHECK-NEXT:          b.init();
// CHECK-EMPTY:
// CHECK-NEXT:          // ranges used for accessors to device memory
// CHECK-NEXT:          sycl::range<2> bl_range_ct1(12, 12);
// CHECK-EMPTY:  
// CHECK-NEXT:          // pointers to device memory
// CHECK-NEXT:          auto al_ptr_ct1 = al.get_ptr();
// CHECK-NEXT:          auto a_ptr_ct1 = a.get_ptr();
// CHECK-EMPTY:  
// CHECK-NEXT:          // accessors to device memory
// CHECK-NEXT:          sycl::accessor<int, 1, sycl::access::mode::read_write, sycl::access::target::local> cl_acc_ct1(sycl::range<1>(36), cgh);
// CHECK-NEXT:          sycl::accessor<int, 2, sycl::access::mode::read_write, sycl::access::target::local> bl_acc_ct1(bl_range_ct1, cgh);
// CHECK-NEXT:          auto b_acc_ct1 = b.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          // accessors to image objects
// CHECK-NEXT:          auto tex21_acc = tex21.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          // sampler of image objects
// CHECK-NEXT:          auto tex21_smpl = tex21.get_sampler();
// CHECK-EMPTY:  
// CHECK-NEXT:          // ranges to define ND iteration space for the kernel
// CHECK-NEXT:          auto dpct_global_range = griddim * threaddim;
// CHECK-EMPTY:  
// CHECK-NEXT:          // run the kernel within defined ND range
// CHECK-NEXT:          cgh.parallel_for(
// CHECK-NEXT:            sycl::nd_range<3>(sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), sycl::range<3>(threaddim.get(2), threaddim.get(1), threaddim.get(0))),
// CHECK-NEXT:            [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:              kernel(stream_ct1, *a_ptr_ct1, b_acc_ct1, cl_acc_ct1.get_pointer(), al_ptr_ct1, dpct::accessor<int, dpct::local, 2>(bl_acc_ct1, bl_range_ct1), dpct::image_accessor<sycl::uint2, 1>(tex21_smpl, tex21_acc));
// CHECK-NEXT:            });
// CHECK-NEXT:        });
    kernel<<<griddim, threaddim>>>();
}