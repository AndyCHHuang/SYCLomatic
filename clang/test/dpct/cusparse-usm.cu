// RUN: dpct --format-range=none --out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: mkl::transpose transA = mkl::transpose::nontrans;
//CHECK-NEXT: dpct::mat_prop descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: dpct::mat_prop descr1 , descr2 ;
  //CHECK-NEXT: dpct::mat_prop descr3 ;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  int zero = 0;
  //CHECK: mkl::diag diag0 = mkl::diag::nonunit;
  //CHECK-NEXT: mkl::uplo fill0 = mkl::uplo::lower;
  //CHECK-NEXT: mkl::index_base base0 = mkl::index_base::zero;
  //CHECK-NEXT: dpct::mat_type type0 = dpct::mat_type::ge;
  //CHECK-NEXT: descrA.type = (mkl::diag)zero;
  //CHECK-NEXT: descrA.uplo = dpct::get_uplo(zero);
  //CHECK-NEXT: descrA.index = (mkl::index_base)zero;
  //CHECK-NEXT: descrA.type = (dpct::mat_type)zero;
  //CHECK-NEXT: diag0 = descrA.type;
  //CHECK-NEXT: fill0 = descrA.uplo;
  //CHECK-NEXT: base0 = descrA.index;
  //CHECK-NEXT: type0 = descrA.type;
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)zero);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)zero);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)zero);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)zero);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseCreateMatDescr was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA.type = dpct::mat_type::ge;
  //CHECK-NEXT: descrA.index = mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support Hermitian sparse matrix for computing a sparse matrix-dense vector product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: dpct::sparse_csr_mv_wrapper(*handle, dpct::get_transpose(zero), dpct::get_value(&alpha, *handle), descrA, mat_handle_ct{{[0-9]+}}, x, dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDcsrmv(handle, (hipsparseOperation_t)zero, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support Hermitian sparse matrix for computing a sparse matrix-dense vector product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: dpct::sparse_csr_mv_wrapper(*handle, transA, dpct::get_value(&alpha_Z, *handle), descrA, mat_handle_ct{{[0-9]+}}, x_Z, dpct::get_value(&beta_Z, *handle), y_Z);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmv(handle, transA, m, n, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, &beta_Z, y_Z);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support symmetric/Hermitian/triangular sparse matrix for computing a sparse matrix-dense matrix product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemm(*handle, transA, dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), n, ldb, dpct::get_value(&beta, *handle), y, ldc);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseDcsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support symmetric/Hermitian/triangular sparse matrix for computing a sparse matrix-dense matrix product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: mkl::sparse::gemm(*handle, transA, dpct::get_value(&alpha_Z, *handle), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, n, ldb, dpct::get_value(&beta_Z, *handle), (std::complex<double>*)y_Z, ldc);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmm(handle, transA, m, n, k, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, ldb, &beta_Z, y_Z, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support Hermitian sparse matrix for computing a sparse matrix-dense vector product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: dpct::sparse_csr_mv_wrapper(*handle, transA, dpct::get_value(&alpha, *handle), descrA, mat_handle_ct{{[0-9]+}}, x, dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = 0){}
  if(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support Hermitian sparse matrix for computing a sparse matrix-dense vector product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: dpct::sparse_csr_mv_wrapper(*handle, transA, dpct::get_value(&alpha, *handle), descrA, mat_handle_ct{{[0-9]+}}, x, dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = 0;;){}
  for(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support Hermitian sparse matrix for computing a sparse matrix-dense vector product. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: dpct::sparse_csr_mv_wrapper(*handle, transA, dpct::get_value(&alpha, *handle), descrA, mat_handle_ct{{[0-9]+}}, x, dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = 0){}
  switch(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: int info;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCreateSolveAnalysisInfo was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDcsrsv_analysis was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support general/symmetric/Hermitian sparse matrix for solveing a system of linear equations for the sparse matrix. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1046:{{[0-9]+}}: The argument &alpha in original API call was removed, because mkl::sparse::trsv does not support specifing the scalar value. If the scalar value in the original code is not 1, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, m, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::optimize_trsv(*handle, descrA.uplo, transA, descrA.diag, mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::trsv(*handle, descrA.uplo, transA, descrA.diag, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroySolveAnalysisInfo was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseDcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDcsrsv_solve(handle, transA, m, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, info, x, y);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseZcsrsv_analysis was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support general/symmetric/Hermitian sparse matrix for solveing a system of linear equations for the sparse matrix. If the martix type is unsupported, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1046:{{[0-9]+}}: The argument &alpha_Z in original API call was removed, because mkl::sparse::trsv does not support specifing the scalar value. If the scalar value in the original code is not 1, you need adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, m, descrA.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: mkl::sparse::optimize_trsv(*handle, descrA.uplo, transA, descrA.diag, mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::trsv(*handle, descrA.uplo, transA, descrA.diag, mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, (std::complex<double>*)y_Z);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  cusparseZcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_Z, csrRowPtrA, csrColIndA, info);
  cusparseZcsrsv_solve(handle, transA, m, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, info, x_Z, y_Z);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(dpct::mat_prop descrB) try {
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1045:{{[0-9]+}}: The oneMKL library does not support Hermitian sparse matrix for computing a sparse matrix-dense vector product. If the martix type is unsupported, you need adjust the code.
//CHECK-NEXT: */
//CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB.index, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
//CHECK-NEXT: dpct::sparse_csr_mv_wrapper(*handle, transA, dpct::get_value(&alpha, *handle), descrB, mat_handle_ct{{[0-9]+}}, x, dpct::get_value(&beta, *handle), y);
//CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}