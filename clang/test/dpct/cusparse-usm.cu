// RUN: dpct --format-range=none --out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: mkl::transpose transA = mkl::transpose::nontrans;
//CHECK-NEXT: mkl::index_base descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int foo(int aaaaa){
  //CHECK: mkl::index_base descr1 , descr2 ;
  //CHECK-NEXT: mkl::index_base descr3 ;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  //CHECK: mkl::diag diag0 = mkl::diag::nonunit;
  //CHECK-NEXT: mkl::uplo fill0 = mkl::uplo::lower;
  //CHECK-NEXT: mkl::index_base base0 = mkl::index_base::zero;
  //CHECK-NEXT: int type0 = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatDiagType was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatFillMode was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = (mkl::index_base)aaaaa;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatDiagType was replaced with 0, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: diag0 = (mkl::diag)0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatFillMode was replaced with 0, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: fill0 = (mkl::uplo)0;
  //CHECK-NEXT: base0 = descrA;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatType was replaced with 0, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: type0 = 0;
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)aaaaa);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)aaaaa);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, dpct::get_transpose(aaaaa), dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDcsrmv(handle, (hipsparseOperation_t)aaaaa, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, dpct::get_value(&alpha_Z, *handle), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, dpct::get_value(&beta_Z, *handle), (std::complex<double>*)y_Z);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmv(handle, transA, m, n, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, &beta_Z, y_Z);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemm(*handle, transA, dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), n, ldb, dpct::get_value(&beta, *handle), y, ldc);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseDcsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: mkl::sparse::gemm(*handle, transA, dpct::get_value(&alpha_Z, *handle), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, n, ldb, dpct::get_value(&beta_Z, *handle), (std::complex<double>*)y_Z, ldc);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmm(handle, transA, m, n, k, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, ldb, &beta_Z, y_Z, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = 0){}
  if(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = 0;;){}
  for(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), dpct::get_value(&beta, *handle), y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = 0){}
  switch(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: int info;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCreateSolveAnalysisInfo was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDcsrsv_analysis was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroySolveAnalysisInfo was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseDcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseZcsrsv_analysis was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  cusparseZcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_Z, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(mkl::index_base descrB) try {
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1045:{{[0-9]+}}: The Intel(R) DPC++ Compatibility Tool supports migration of only general sparse matrix type for this API currently. You may need to adjust the code.
//CHECK-NEXT: */
//CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
//CHECK-NEXT: mkl::sparse::gemv(*handle, transA, dpct::get_value(&alpha, *handle), mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), dpct::get_value(&beta, *handle), y);
//CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}