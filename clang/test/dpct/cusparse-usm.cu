// RUN: dpct --format-range=none --out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: mkl::transpose transA = mkl::transpose::nontrans;
//CHECK-NEXT: mkl::index_base descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: mkl::index_base descr1 , descr2 ;
  //CHECK-NEXT: mkl::index_base descr3 ;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = mkl::index_base::zero;
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, csrRowPtrA, csrColIndA, csrValA);
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x, beta, y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(mkl::index_base descrB) try {
//CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB, csrRowPtrA, csrColIndA, csrValA);
//CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x, beta, y);
//CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. 0 is used in return statement. You need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}