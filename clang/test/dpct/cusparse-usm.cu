// RUN: dpct --format-range=none --out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: mkl::transpose transA = mkl::transpose::nontrans;
//CHECK-NEXT: mkl::index_base descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: mkl::index_base descr1 , descr2 ;
  //CHECK-NEXT: mkl::index_base descr3 ;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = mkl::index_base::zero;
  //CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  //CHECK: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, std::complex<double>(alpha_Z.x(),alpha_Z.y()), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, std::complex<double>(beta_Z.x(),beta_Z.y()), y_Z);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmv(handle, transA, m, n, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, &beta_Z, y_Z);

  //CHECK: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemm(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), n, ldb, beta, y, ldc);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseDcsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: mkl::sparse::gemm(*handle, transA, std::complex<double>(alpha_Z.x(),alpha_Z.y()), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, n, ldb, std::complex<double>(beta_Z.x(),beta_Z.y()), y_Z, ldc);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmm(handle, transA, m, n, k, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, ldb, &beta_Z, y_Z, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = 0){}
  if(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = 0;;){}
  for(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}
  
  //CHECK: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = 0){}
  switch(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed, because the function call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(mkl::index_base descrB) try {
//CHECK-NEXT: mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
//CHECK-NEXT: mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
//CHECK-NEXT: mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}