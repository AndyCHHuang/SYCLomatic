// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none -out-root %T/module_main %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/module_main/module_main.dp.cpp

//CHECK: #include <dlfcn.h>

#include <hip/hip_runtime.h>
#include <string>
int main(){
    //CHECK: void * M;
    hipModule_t M;
    //CHECK: dpct::kernel_functor F;
    hipFunction_t F;
    std::string Path, FunctionName, Data;
    //CHECK: /*
    //CHECK-NEXT: DPCT1079:{{[0-9]+}}: You need to replace the "PlaceHolder" with the file name of the dynamic library.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = dlopen(PlaceHolder/*Fix the module file name manually*/, RTLD_LAZY);
    hipModuleLoad(&M, Path.c_str());
    //CHECK: /*
    //CHECK-NEXT: DPCT1079:{{[0-9]+}}: You need to replace the "PlaceHolder" with the file name of the dynamic library.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = dlopen(PlaceHolder/*Fix the module file name manually*/, RTLD_LAZY);
    hipModuleLoadData(&M, Data.c_str());
    //CHECK: F = (dpct::kernel_functor)dlsym(M, (std::string(FunctionName.c_str()) + "_wrapper").c_str());
    hipModuleGetFunction(&F, M, FunctionName.c_str());

    int sharedSize;
    hipStream_t s;
    void **param, **extra;
    //CHECK:  F(*s, sycl::nd_range<3>(sycl::range<3>(32, 16, 1) * sycl::range<3>(64, 32, 4), sycl::range<3>(64, 32, 4)), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, s, param, extra);
    //CHECK:  F(q_ct1, sycl::nd_range<3>(sycl::range<3>(32, 16, 1) * sycl::range<3>(64, 32, 4), sycl::range<3>(64, 32, 4)), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, 0, param, extra);
    //CHECK:  F(q_ct1, sycl::nd_range<3>(sycl::range<3>(32, 16, 1) * sycl::range<3>(64, 32, 4), sycl::range<3>(64, 32, 4)), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, hipStreamLegacy, param, extra);

    //CHECK: dpct::image_wrapper_base_p tex;
    //CHECK: tex = (dpct::image_wrapper_base_p)dlsym(M, "tex");
    hipTexRef tex;
    hipModuleGetTexRef(&tex, M, "tex");

    //CHECK: dlclose(M);
    hipModuleUnload(M);
    return 0;
}