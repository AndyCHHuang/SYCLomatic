// RUN: dpct --format-range=none -out-root %T/cufft-placeholder %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-placeholder/cufft-placeholder.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
int rank;

//CHECK:/*
//CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK-NEXT:*/
//CHECK-NEXT:void foo1(std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan) {
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2* idata;
//CHECK-NEXT:  if ((void *)idata == (void *)odata) {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, idata);
//CHECK-NEXT:  } else {
//CHECK-NEXT:  plan->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
//CHECK-NEXT:  plan->commit(dpct::get_default_queue());
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, idata, odata);
//CHECK-NEXT:  }
//CHECK-NEXT:}
void foo1(hipfftHandle plan) {
  double* odata;
  double2* idata;
  hipfftExecZ2D(plan, idata, odata);
}

//CHECK:/*
//CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK-NEXT:*/
//CHECK-NEXT:void foo2(std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan) {
//CHECK-NEXT:  float* odata;
//CHECK-NEXT:  sycl::float2* idata;
//CHECK-NEXT:  if ((void *)idata == (void *)odata) {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, idata);
//CHECK-NEXT:  } else {
//CHECK-NEXT:  plan->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
//CHECK-NEXT:  plan->commit(dpct::get_default_queue());
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, idata, odata);
//CHECK-NEXT:  }
//CHECK-NEXT:}
void foo2(hipfftHandle plan) {
  float* odata;
  float2* idata;
  hipfftExecC2R(plan, idata, odata);
}

int main() {
  //CHECK:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan1;
  //CHECK-NEXT:int type1 = 108;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1068:{{[0-9]+}}: The argument of the dimensions and strides could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the cufftMakePlanMany could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1066:{{[0-9]+}}: Migration is supported only if the input distance is the same as the output distance. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan1 = std::make_shared<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>>(dpct_placeholder/*Fix the dimensions manually*/);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, idist);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 11);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, dpct_placeholder/*Fix the stride manually*/);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, dpct_placeholder/*Fix the stride manually*/);
  //CHECK-NEXT:plan1->commit(q_ct1);
  hipfftHandle plan1;
  hipfftType_t type1 = HIPFFT_Z2D;
  hipfftMakePlanMany(plan1, rank, n, inembed, istride, idist, onembed, ostride, odist, type1, 11, work_size);

  //CHECK:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan2;
  //CHECK-NEXT:int type2 = 44;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1068:{{[0-9]+}}: The argument of the dimensions and strides could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the cufftMakePlanMany could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1066:{{[0-9]+}}: Migration is supported only if the input distance is the same as the output distance. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan2 = std::make_shared<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>>(dpct_placeholder/*Fix the dimensions manually*/);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, idist);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, dpct_placeholder/*Fix the stride manually*/);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, dpct_placeholder/*Fix the stride manually*/);
  //CHECK-NEXT:plan2->commit(q_ct1);
  hipfftHandle plan2;
  hipfftType_t type2 = HIPFFT_C2R;
  hipfftMakePlanMany(plan2, rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12, work_size);

  foo1(plan1);
  foo2(plan2);

  return 0;
}

