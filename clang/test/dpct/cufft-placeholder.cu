// RUN: dpct --format-range=none -out-root %T/cufft-placeholder %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-placeholder/cufft-placeholder.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
int rank;

//CHECK:/*
//CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK-NEXT:*/
//CHECK-NEXT:void foo1(std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan) {
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2* idata;
//CHECK-NEXT:  if ((void *)idata == (void *)odata) {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (double*)idata);
//CHECK-NEXT:  } else {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (double*)idata, odata);
//CHECK-NEXT:  }
//CHECK-NEXT:}
void foo1(hipfftHandle plan) {
  double* odata;
  double2* idata;
  hipfftExecZ2D(plan, idata, odata);
}

//CHECK:/*
//CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK-NEXT:*/
//CHECK-NEXT:void foo2(std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan) {
//CHECK-NEXT:  float* odata;
//CHECK-NEXT:  sycl::float2* idata;
//CHECK-NEXT:  if ((void *)idata == (void *)odata) {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (float*)idata);
//CHECK-NEXT:  } else {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (float*)idata, odata);
//CHECK-NEXT:  }
//CHECK-NEXT:}
void foo2(hipfftHandle plan) {
  float* odata;
  float2* idata;
  hipfftExecC2R(plan, idata, odata);
}

int main() {
  //CHECK:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan1;
  //CHECK-NEXT:int type1 = 108;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1068:{{[0-9]+}}: The value of dimensions and strides could not be deduced. You need to update 'dpct_placeholder' manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The 'work_size' parameter could not be migrated. You may need to update the code manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function could not be deduced, so it is assumed out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1068:{{[0-9]+}}: The value of FFT type could not be deduced. You need to update 'FWD_DISTANCE' and 'BWD_DISTANCE' manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan1 = std::make_shared<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>>(dpct_placeholder/*Fix the dimensions manually*/);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 11);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[dpct_placeholder/*Fix the dimensions manually*/] = {dpct_placeholder/*Fix the stride manually*/};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[dpct_placeholder/*Fix the dimensions manually*/] = {dpct_placeholder/*Fix the stride manually*/};
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:plan1->commit(q_ct1);
  hipfftHandle plan1;
  hipfftType_t type1 = HIPFFT_Z2D;
  hipfftMakePlanMany(plan1, rank, n, inembed, istride, idist, onembed, ostride, odist, type1, 11, work_size);

  //CHECK:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> plan2;
  //CHECK-NEXT:int type2 = 44;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1068:{{[0-9]+}}: The value of dimensions and strides could not be deduced. You need to update 'dpct_placeholder' manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The 'work_size' parameter could not be migrated. You may need to update the code manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function could not be deduced, so it is assumed out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1068:{{[0-9]+}}: The value of FFT type could not be deduced. You need to update 'FWD_DISTANCE' and 'BWD_DISTANCE' manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan2 = std::make_shared<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>>(dpct_placeholder/*Fix the dimensions manually*/);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[dpct_placeholder/*Fix the dimensions manually*/] = {dpct_placeholder/*Fix the stride manually*/};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[dpct_placeholder/*Fix the dimensions manually*/] = {dpct_placeholder/*Fix the stride manually*/};
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:plan2->commit(q_ct1);
  hipfftHandle plan2;
  hipfftType_t type2 = HIPFFT_C2R;
  hipfftMakePlanMany(plan2, rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12, work_size);

  foo1(plan1);
  foo2(plan2);

  return 0;
}

