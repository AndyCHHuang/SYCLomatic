
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/unnamed-struct.dp.cpp --match-full-lines %s

//CHECK: struct __sycl_align__(4) dpct_type_{{[a-f0-9]+}}
struct __align__(4)
{
    unsigned i;
} A;

//CHECK: typedef class dpct_type_{{[a-f0-9]+}}{
typedef class{
    unsigned i;
} T1;


//CHECK: typedef struct dpct_type_{{[a-f0-9]+}}
typedef struct
	: public T1
{
    unsigned j;
} T2;

//CHECK: class dpct_type_{{[a-f0-9]+}}: public T2 {
class: public T2 {
    unsigned k;
} B;
