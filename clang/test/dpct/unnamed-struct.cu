
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/unnamed-struct.dp.cpp --match-full-lines %s

//CHECK: struct __dpct_align__(4) dpct_type_{{[a-f0-9]+}}
struct __align__(4)
{
    unsigned i;
} A;

//CHECK: typedef class dpct_type_{{[a-f0-9]+}}{
typedef class{
    unsigned i;
} T1;


//CHECK: typedef struct dpct_type_{{[a-f0-9]+}}
typedef struct
	: public T1
{
    unsigned j;
} T2;

//CHECK: class dpct_type_{{[a-f0-9]+}}: public T2 {
class: public T2 {
    unsigned k;
} B;
