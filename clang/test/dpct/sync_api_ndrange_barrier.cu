#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: c2s --format-range=none -out-root %T/sync_api_ndrange_barrier %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sync_api_ndrange_barrier/sync_api_ndrange_barrier.dp.cpp

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <c2s/c2s.hpp>
#include "hip/hip_cooperative_groups.h"
namespace cg = cooperative_groups;
using namespace cooperative_groups;

// CHECK: void kernel(const sycl::stream &stream_ct1) {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1087:{{[0-9]+}}: DPC++ currently does not support cross group synchronization, you can specify "--use-experimental-features=nd_range_barrier" to use the c2s::nd_range_barrier to migrate this_grid().
// CHECK-NEXT:  */
// CHECK-NEXT:  cg::grid_group grid = cg::this_grid();
// CHECK-NEXT:  stream_ct1 << "kernel run!\n";
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1087:{{[0-9]+}}: DPC++ currently does not support cross group synchronization, you can specify "--use-experimental-features=nd_range_barrier" to use the c2s::nd_range_barrier to migrate grid.sync().
// CHECK-NEXT:  */
// CHECK-NEXT:  grid.sync();
// CHECK-NEXT:}
__global__ void kernel() {
  cg::grid_group grid = cg::this_grid();
  printf("kernel run!\n");
  grid.sync();
}

int main() {
// CHECK:  c2s::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::stream stream_ct1(64 * 1024, 80, cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 2) * sycl::range<3>(1, 1, 2), sycl::range<3>(1, 1, 2)), 
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          kernel(stream_ct1);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
  kernel<<<2, 2>>>();

  hipDeviceSynchronize();
  return 0;
}
