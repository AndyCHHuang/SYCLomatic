// RUN: echo "[" > %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/main.cu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/main.cu\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/bar/util.gpu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/bar/util.gpu\"" >> %T/compile_commands.json
// RUN: echo "    }" >> %T/compile_commands.json
// RUN: echo "]\n" >> %T/compile_commands.json

// RUN: dpct --format-range=none --cuda-include-path="%cuda-path/include" -in-root=%S -out-root=%T -p=%T %s %S/bar/util.gpu  --sycl-named-lambda -extra-arg="-I%S/bar" 
// RUN: FileCheck %s --match-full-lines --input-file %T/main.dp.cpp
// RUN: FileCheck %S/bar/util.gpu --match-full-lines --input-file %T/bar/util.gpu.dp.cpp
// RUN: FileCheck %S/bar/util.gpuhead --match-full-lines --input-file %T/bar/util.gpuhead

// RUN: dpct --format-range=none --cuda-include-path="%cuda-path/include" -in-root=%S -out-root=%T  %S/main.gpu   --sycl-named-lambda 
// RUN: FileCheck %S/main.gpu --match-full-lines --input-file %T/main.gpu.dp.cpp

#include <stdio.h>
#include <hip/hip_runtime.h>

// CHECK:#include "util.gpuhead"
#include "util.gpuhead"

int main(){
 int *a, *b;
 hipMalloc((void **)&a, 4);
 hipMalloc((void **)&b, 4);
 kernel_util<<<1,1>>>(a,b);
}
