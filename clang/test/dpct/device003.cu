
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device003.dp.cpp

template <typename T>
void check(T result, char const *const func) {}

#define checkErrors(val) check((val), #val)

int main(int argc, char **argv)
{
int deviceCount = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors((deviceCount = dpct::get_device_manager().device_count(), 0));
checkErrors(hipGetDeviceCount(&deviceCount));

int dev_id;
// CHECK: checkErrors(dev_id = dpct::get_device_manager().current_device_id());
checkErrors(hipGetDevice(&dev_id));

hipDeviceProp_t deviceProp;
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkErrors((dpct::get_device_manager().get_device(0).get_device_info(deviceProp), 0));
checkErrors(hipGetDeviceProperties(&deviceProp, 0));

int atomicSupported;
// CHECK: checkErrors(atomicSupported = dpct::get_device_manager().get_device(dev_id).is_native_atomic_supported());
checkErrors(hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id));

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors(accessSupported = 0);
checkErrors(hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2));

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors(perfRank = 0);
checkErrors(hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2));

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors(atomicSupported = 0);
checkErrors(hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2));
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkErrors((dpct::get_device_manager().select_device(device2), 0));
checkErrors(hipSetDevice(device2));

return 0;
}
