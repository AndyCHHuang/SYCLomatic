
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device003.dp.cpp

template <typename T>
void check(T result, char const *const func) {}

#define checkErrors(val) check((val), #val)

int main(int argc, char **argv)
{
int deviceCount = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors((deviceCount = dpct::dev_mgr::instance().device_count(), 0));
checkErrors(hipGetDeviceCount(&deviceCount));

int dev_id;
// CHECK: checkErrors(dev_id = dpct::dev_mgr::instance().current_device_id());
checkErrors(hipGetDevice(&dev_id));

hipDeviceProp_t deviceProp;
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkErrors((dpct::dev_mgr::instance().get_device(0).get_device_info(deviceProp), 0));
checkErrors(hipGetDeviceProperties(&deviceProp, 0));

int atomicSupported;
// CHECK: checkErrors((atomicSupported = dpct::dev_mgr::instance().get_device(dev_id).is_native_atomic_supported(), 0));
checkErrors(hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id));

int device1 = 0;
int device2 = 1;
int perfRank = 0;
int accessSupported = 0;

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors(accessSupported = 0);
checkErrors(hipDeviceGetP2PAttribute(&accessSupported, hipDevP2PAttrAccessSupported, device1, device2));

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors(perfRank = 0);
checkErrors(hipDeviceGetP2PAttribute(&perfRank, hipDevP2PAttrPerformanceRank, device1, device2));

// CHECK:/*
// CHECK-NEXT:DPCT1004:{{[0-9]+}}: Could not generate replacement.
// CHECK-NEXT:*/
// CHECK-NEXT: checkErrors(atomicSupported = 0);
checkErrors(hipDeviceGetP2PAttribute(&atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1, device2));
// CHECK:/*
// CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:checkErrors((dpct::dev_mgr::instance().select_device(device2), 0));
checkErrors(hipSetDevice(device2));

return 0;
}
