// RUN: dpct --format-range=none --usm-level=none -out-root %T/cufft-different-locations %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-different-locations/cufft-different-locations.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
double* odata;
double2* idata;

#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line ) {
  if (err != HIPFFT_SUCCESS) {
    fprintf(stderr, "Cufft error in file '%s' in line %i : %s.\n",
            __FILE__, __LINE__, "error" );
  }
}

int main() {
  hipfftHandle plan1;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan1 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int res1 = (plan1->commit(q_ct1), 0);
  hipfftResult res1 = hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  //CHECK:int res2 = 0;
  //CHECK-NEXT:{
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan1, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:res2 = (oneapi::mkl::dft::compute_backward(*plan1, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}}), 0);
  //CHECK-NEXT:}
  //CHECK-NEXT:}
  hipfftResult res2 = hipfftExecZ2D(plan1, idata, odata);

  hipfftHandle plan2;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan2 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:res1 = (plan2->commit(q_ct1), 0);
  res1 = hipfftMakePlanMany(plan2, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  //CHECK:{
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan2, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:res2 = (oneapi::mkl::dft::compute_backward(*plan2, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}}), 0);
  //CHECK-NEXT:}
  //CHECK-NEXT:}
  res2 = hipfftExecZ2D(plan2, idata, odata);

  hipfftHandle plan3;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan3 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan3->commit(q_ct1);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in function-like macro statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR(0);
  HANDLE_CUFFT_ERROR(hipfftMakePlanMany(plan3, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  //CHECK:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR([&](){
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan3, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan3, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  HANDLE_CUFFT_ERROR(hipfftExecZ2D(plan3, idata, odata));

  hipfftHandle plan4;
  hipfftHandle plan5;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan4 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan4->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan4->commit(q_ct1);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(0) {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} else if ([&](){
  //CHECK-NEXT:plan5 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan5->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan5->commit(q_ct1);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}()) {
  //CHECK-NEXT:}
  if(hipfftMakePlanMany(plan4, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  } else if (hipfftMakePlanMany(plan5, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  //CHECK:{
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan4, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan4, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if (0) {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} else if([&](){
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan5, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan5, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}()) {
  //CHECK-NEXT:}
  if (hipfftExecZ2D(plan4, idata, odata)) {
  } else if(hipfftExecZ2D(plan5, idata, odata)) {
  }

  hipfftHandle plan6;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan6 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan6->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan6->commit(q_ct1);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(int res = 0) {
  //CHECK-NEXT:}
  if(hipfftResult res = hipfftMakePlanMany(plan6, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  //CHECK:{
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan6, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan6, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(int res = 0) {
  //CHECK-NEXT:}
  if(hipfftResult res = hipfftExecZ2D(plan6, idata, odata)) {
  }

  hipfftHandle plan7;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan7 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan7->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan7->commit(q_ct1);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (0;;) {
  //CHECK-NEXT:}
  for (hipfftMakePlanMany(plan7, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);;) {
  }
  //CHECK:{
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan7, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan7, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (0;;) {
  //CHECK-NEXT:}
  for (hipfftExecZ2D(plan7, idata, odata);;) {
  }

  hipfftHandle plan8;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (;[&](){
  //CHECK-NEXT:plan8 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan8->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan8->commit(q_ct1);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();) {
  //CHECK-NEXT:}
  for (;hipfftMakePlanMany(plan8, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);) {
  }
  //CHECK:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (;[&](){
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan8, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan8, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();) {
  //CHECK-NEXT:}
  for (;hipfftExecZ2D(plan8, idata, odata);) {
  }

  hipfftHandle plan9;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:while ([&](){
  //CHECK-NEXT:plan9 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan9->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan9->commit(q_ct1);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}() != 0) {
  //CHECK-NEXT:}
  while (hipfftMakePlanMany(plan9, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size) != 0) {
  }
  //CHECK:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:while ([&](){
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan9, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan9, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}() != 0) {
  //CHECK-NEXT:}
  while (hipfftExecZ2D(plan9, idata, odata) != 0) {
  }

  hipfftHandle plan10;
  //CHECK:do {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} while ([&](){
  //CHECK-NEXT:plan10 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan10->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan10->commit(q_ct1);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  do {
  } while (hipfftMakePlanMany(plan10, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  //CHECK:do {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1034:{{[0-9]+}}: Migrated API does not return error code. 0 is returned in the lambda. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} while ([&](){
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan10, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan10, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  do {
  } while (hipfftExecZ2D(plan10, idata, odata));

  hipfftHandle plan11;
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan11 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan11->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan11->commit(q_ct1);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:switch (int stat = 0){
  //CHECK-NEXT:}
  switch (int stat = hipfftMakePlanMany(plan11, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)){
  }
  //CHECK:{
  //CHECK-NEXT:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan11, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan11, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:switch (int stat = 0){
  //CHECK-NEXT:}
  switch (int stat = hipfftExecZ2D(plan11, idata, odata)){
  }
  return 0;
}

hipfftResult foo1(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan->commit(dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:return 0;
  return hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo2(hipfftHandle plan) {
  //CHECK:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:return 0;
  return hipfftExecZ2D(plan, idata, odata);
}

hipfftResult foo3(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan->commit(dpct::get_default_queue());
  hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo4(hipfftHandle plan) {
  //CHECK:auto idata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(idata);
  //CHECK-NEXT:if ((void *)idata == (void *)odata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan, idata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:  auto odata_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(odata);
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan, idata_buf_ct{{[0-9]+}}, odata_buf_ct{{[0-9]+}});
  //CHECK-NEXT:}
  hipfftExecZ2D(plan, idata, odata);
}

