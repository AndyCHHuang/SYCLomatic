// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-lambda.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <mkl_sycl_types.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>


int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;


  // CHECK: if ([&]() {
  // CHECK-NEXT: auto allocation_ct7 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct7.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct9 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct9.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct12 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct12 =
  // CHECK-NEXT:     allocation_ct12.buffer.reinterpret<float>(
  // CHECK-NEXT:         cl::sycl::range<1>(allocation_ct12.size / sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N,
  // CHECK-NEXT:                 N, N, *(&alpha_S), buffer_ct7, N, buffer_ct9, N, *(&beta_S),
  // CHECK-NEXT:                 buffer_ct12, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT:     }()) {
  // CHECK-NEXT: }
  if(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  // CHECK: if (int stat = [&]() {
  // CHECK-NEXT: auto allocation_ct7 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct7.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct9 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct9.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct12 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct12 =
  // CHECK-NEXT:     allocation_ct12.buffer.reinterpret<float>(
  // CHECK-NEXT:         cl::sycl::range<1>(allocation_ct12.size / sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N,
  // CHECK-NEXT:                 N, N, *(&alpha_S), buffer_ct7, N, buffer_ct9, N, *(&beta_S),
  // CHECK-NEXT:                 buffer_ct12, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT:     }()) {
  // CHECK-NEXT: }
  if(int stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  // CHECK: for ([&]() {
  // CHECK-NEXT: auto allocation_ct7 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct7.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct9 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct9.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct12 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct12 =
  // CHECK-NEXT:     allocation_ct12.buffer.reinterpret<float>(
  // CHECK-NEXT:         cl::sycl::range<1>(allocation_ct12.size / sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N,
  // CHECK-NEXT:                 N, N, *(&alpha_S), buffer_ct7, N, buffer_ct9, N, *(&beta_S),
  // CHECK-NEXT:                 buffer_ct12, N);
  // CHECK-NEXT:      }();
  // CHECK-NEXT:      ;) {
  // CHECK-NEXT: }
  for(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);;){
  }


  // CHECK: while ([&]() {
  // CHECK-NEXT: auto allocation_ct7 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct7.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct9 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct9.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct12 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct12 =
  // CHECK-NEXT:     allocation_ct12.buffer.reinterpret<float>(
  // CHECK-NEXT:         cl::sycl::range<1>(allocation_ct12.size / sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N,
  // CHECK-NEXT:                 N, N, *(&alpha_S), buffer_ct7, N, buffer_ct9, N, *(&beta_S),
  // CHECK-NEXT:                 buffer_ct12, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }() != 0) {
  // CHECK-NEXT: }
  while(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)!=0){
  }


  // CHECK: do{
  // CHECK-NEXT: } while ([&]() {
  // CHECK-NEXT: auto allocation_ct7 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct7.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct9 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct9.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct12 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct12 =
  // CHECK-NEXT:     allocation_ct12.buffer.reinterpret<float>(
  // CHECK-NEXT:         cl::sycl::range<1>(allocation_ct12.size / sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N,
  // CHECK-NEXT:                 N, N, *(&alpha_S), buffer_ct7, N, buffer_ct9, N, *(&beta_S),
  // CHECK-NEXT:                 buffer_ct12, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }());
  do{
  }while(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N));


  // CHECK: switch (int stat = [&]() {
  // CHECK-NEXT: auto allocation_ct7 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct7.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct9 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(
  // CHECK-NEXT:     cl::sycl::range<1>(allocation_ct9.size / sizeof(float)));
  // CHECK-NEXT: auto allocation_ct12 =
  // CHECK-NEXT:     dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct12 =
  // CHECK-NEXT:     allocation_ct12.buffer.reinterpret<float>(
  // CHECK-NEXT:         cl::sycl::range<1>(allocation_ct12.size / sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N,
  // CHECK-NEXT:                 N, N, *(&alpha_S), buffer_ct7, N, buffer_ct9, N, *(&beta_S),
  // CHECK-NEXT:                 buffer_ct12, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }()) {}
  switch (int stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  return 0;
}
