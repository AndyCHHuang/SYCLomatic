// RUN: dpct -out-root %T %s --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-lambda.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <mkl_blas_sycl.hpp>
// CHECK-NEXT: #include <mkl_lapack_sycl.hpp>
// CHECK-NEXT: #include <mkl_sycl_types.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>


int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;


  // CHECK: if([&](){
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }()){
  // CHECK-NEXT: }
  if(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  // CHECK: if(int stat = [&](){
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }()){
  // CHECK-NEXT: }
  if(int stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  // CHECK: for([&](){
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }();;){
  // CHECK-NEXT: }
  for(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);;){
  }


  // CHECK: while([&](){
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }()!=0){
  // CHECK-NEXT: }
  while(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)!=0){
  }


  // CHECK: do{
  // CHECK-NEXT: }while([&](){
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }());
  do{
  }while(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N));


  // CHECK: switch (int stat = [&](){
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_A_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_B_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(d_C_S);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: mkl::blas::gemm(handle, mkl::transpose::nontrans, mkl::transpose::nontrans, N, N, N, *(&alpha_S), buffer_ct{{[0-9]+}}, N, buffer_ct{{[0-9]+}}, N, *(&beta_S), buffer_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }()){
  // CHECK-NEXT: }
  switch (int stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){
  }


  return 0;
}
