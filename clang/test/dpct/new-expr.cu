// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/new-expr.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: #define NEW_STREAM new queue_p
// CHECK-NEXT: #define NEW_EVENT new sycl::event
// CHECK-EMPTY:
// CHECK-NEXT: #define NEW(T) new T
#define NEW_STREAM new hipStream_t
#define NEW_EVENT new hipEvent_t

#define NEW(T) new T

void foo() {
  int n = 16;

  // CHECK: queue_p *stream = new queue_p;
  // CHECK-NEXT: stream = new queue_p();
  // CHECK-NEXT: stream = NEW_STREAM;
  // CHECK-NEXT: stream = NEW(queue_p);
  // CHECK-NEXT: queue_p *streams = new queue_p[n];
  hipStream_t *stream = new hipStream_t;
  stream = new hipStream_t();
  stream = NEW_STREAM;
  stream = NEW(hipStream_t);
  hipStream_t *streams = new hipStream_t[n];

  // CHECK: sycl::event *event = new sycl::event;
  // CHECK-NEXT: event = new sycl::event();
  // CHECK-NEXT: event = NEW_EVENT;
  // CHECK-NEXT: event = NEW(sycl::event);
  // CHECK-NEXT: sycl::event *events = new sycl::event[n];
  hipEvent_t *event = new hipEvent_t;
  event = new hipEvent_t();
  event = NEW_EVENT;
  event = NEW(hipEvent_t);
  hipEvent_t *events = new hipEvent_t[n];
}
