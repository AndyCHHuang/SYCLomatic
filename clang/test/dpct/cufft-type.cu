#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/cufft-type %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-type/cufft-type.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t size;

int main() {
  //CHECK:float fftreal;
  //CHECK-NEXT:double fftdreal;
  //CHECK-NEXT:sycl::float2 fftcomplex;
  //CHECK-NEXT:sycl::double2 fftdcomplex;
  //CHECK-NEXT:sycl::float2 ccomplex;
  //CHECK-NEXT:sycl::double2 cdcomplex;
  //CHECK-NEXT:size = sizeof(float);
  //CHECK-NEXT:size = sizeof(double);
  //CHECK-NEXT:size = sizeof(sycl::float2);
  //CHECK-NEXT:size = sizeof(sycl::double2);
  //CHECK-NEXT:size = sizeof(sycl::float2);
  //CHECK-NEXT:size = sizeof(sycl::double2);
  hipfftReal fftreal;
  hipfftDoubleReal fftdreal;
  hipfftComplex fftcomplex;
  hipfftDoubleComplex fftdcomplex;
  hipComplex ccomplex;
  hipDoubleComplex cdcomplex;
  size = sizeof(hipfftReal);
  size = sizeof(hipfftDoubleReal);
  size = sizeof(hipfftComplex);
  size = sizeof(hipfftDoubleComplex);
  size = sizeof(hipComplex);
  size = sizeof(hipDoubleComplex);

  //CHECK:int forward = -1;
  //CHECK-NEXT:int inverse = 1;
  int forward = HIPFFT_FORWARD;
  int inverse = HIPFFT_BACKWARD;

  //CHECK:int fftt_t;
  //CHECK-NEXT:int fftt;
  //CHECK-NEXT:size = sizeof(int);
  //CHECK-NEXT:size = sizeof(int);
  //CHECK-NEXT:fftt = 42;
  //CHECK-NEXT:fftt = 44;
  //CHECK-NEXT:fftt = 41;
  //CHECK-NEXT:fftt = 106;
  //CHECK-NEXT:fftt = 108;
  //CHECK-NEXT:fftt = 105;
  hipfftType_t fftt_t;
  hipfftType fftt;
  size = sizeof(hipfftType_t);
  size = sizeof(hipfftType);
  fftt = HIPFFT_R2C;
  fftt = HIPFFT_C2R;
  fftt = HIPFFT_C2C;
  fftt = HIPFFT_D2Z;
  fftt = HIPFFT_Z2D;
  fftt = HIPFFT_Z2Z;

  //CHECK:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> ffth;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1050:{{[0-9]+}}: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:size = sizeof(std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>>);
  hipfftHandle ffth;
  size = sizeof(hipfftHandle);

  //CHECK:int fftr_t;
  //CHECK-NEXT:int fftr;
  //CHECK-NEXT:size = sizeof(int);
  //CHECK-NEXT:size = sizeof(int);
  //CHECK-NEXT:fftr = 0;
  //CHECK-NEXT:fftr = 1;
  //CHECK-NEXT:fftr = 2;
  //CHECK-NEXT:fftr = 3;
  //CHECK-NEXT:fftr = 4;
  //CHECK-NEXT:fftr = 5;
  //CHECK-NEXT:fftr = 6;
  //CHECK-NEXT:fftr = 7;
  //CHECK-NEXT:fftr = 8;
  //CHECK-NEXT:fftr = 9;
  //CHECK-NEXT:fftr = 10;
  //CHECK-NEXT:fftr = 11;
  //CHECK-NEXT:fftr = 12;
  //CHECK-NEXT:fftr = 13;
  //CHECK-NEXT:fftr = 14;
  //CHECK-NEXT:fftr = 15;
  //CHECK-NEXT:fftr = 16;
  hipfftResult_t fftr_t;
  hipfftResult fftr;
  size = sizeof(hipfftResult_t);
  size = sizeof(hipfftResult);
  fftr = HIPFFT_SUCCESS;
  fftr = HIPFFT_INVALID_PLAN;
  fftr = HIPFFT_ALLOC_FAILED;
  fftr = HIPFFT_INVALID_TYPE;
  fftr = HIPFFT_INVALID_VALUE;
  fftr = HIPFFT_INTERNAL_ERROR;
  fftr = HIPFFT_EXEC_FAILED;
  fftr = HIPFFT_SETUP_FAILED;
  fftr = HIPFFT_INVALID_SIZE;
  fftr = HIPFFT_UNALIGNED_DATA;
  fftr = HIPFFT_INCOMPLETE_PARAMETER_LIST;
  fftr = HIPFFT_INVALID_DEVICE;
  fftr = HIPFFT_PARSE_ERROR;
  fftr = HIPFFT_NO_WORKSPACE;
  fftr = HIPFFT_NOT_IMPLEMENTED;
  fftr = CUFFT_LICENSE_ERROR;
  fftr = HIPFFT_NOT_SUPPORTED;

  return 0;
}


//CHECK:template<
//CHECK-NEXT:typename A = float,
//CHECK-NEXT:typename B = double,
//CHECK-NEXT:typename C = sycl::float2,
//CHECK-NEXT:typename D = sycl::double2,
//CHECK-NEXT:typename E = sycl::float2,
//CHECK-NEXT:typename F = sycl::double2,
//CHECK-NEXT:typename G = int,
//CHECK-NEXT:typename H = int,
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1050:2: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK-NEXT:*/
//CHECK-NEXT:typename I = std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>>,
//CHECK-NEXT:typename J = int,
//CHECK-NEXT:typename K = int>
//CHECK-NEXT:void foo1(
//CHECK-NEXT:float a,
//CHECK-NEXT:double b,
//CHECK-NEXT:sycl::float2 c,
//CHECK-NEXT:sycl::double2 d,
//CHECK-NEXT:sycl::float2 e,
//CHECK-NEXT:sycl::double2 f,
//CHECK-NEXT:int g,
//CHECK-NEXT:int h,
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1050:3: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK-NEXT:*/
//CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> i,
//CHECK-NEXT:int j,
//CHECK-NEXT:int k
//CHECK-NEXT:){}
template<
typename A = hipfftReal,
typename B = hipfftDoubleReal,
typename C = hipfftComplex,
typename D = hipfftDoubleComplex,
typename E = hipComplex,
typename F = hipDoubleComplex,
typename G = hipfftType_t,
typename H = hipfftType,
typename I = hipfftHandle,
typename J = hipfftResult_t,
typename K = hipfftResult>
void foo1(
hipfftReal a,
hipfftDoubleReal b,
hipfftComplex c,
hipfftDoubleComplex d,
hipComplex e,
hipDoubleComplex f,
hipfftType_t g,
hipfftType h,
hipfftHandle i,
hipfftResult_t j,
hipfftResult k
){}


//CHECK:template<
//CHECK-NEXT:int A1 = 42,
//CHECK-NEXT:int A2 = 44,
//CHECK-NEXT:int A3 = 41,
//CHECK-NEXT:int A4 = 106,
//CHECK-NEXT:int A5 = 108,
//CHECK-NEXT:int A6 = 105,
//CHECK-NEXT:int B1 = 0,
//CHECK-NEXT:int B2 = 1,
//CHECK-NEXT:int B3 = 2,
//CHECK-NEXT:int B4 = 3,
//CHECK-NEXT:int B5 = 4,
//CHECK-NEXT:int B6 = 5,
//CHECK-NEXT:int B7 = 6,
//CHECK-NEXT:int B8 = 7,
//CHECK-NEXT:int B9 = 8,
//CHECK-NEXT:int B10 = 9,
//CHECK-NEXT:int B11 = 10,
//CHECK-NEXT:int B12 = 11,
//CHECK-NEXT:int B13 = 12,
//CHECK-NEXT:int B14 = 13,
//CHECK-NEXT:int B15 = 14,
//CHECK-NEXT:int B16 = 15,
//CHECK-NEXT:int B17 = 16>
//CHECK-NEXT:void foo2(
//CHECK-NEXT:int a1 = 42,
//CHECK-NEXT:int a2 = 44,
//CHECK-NEXT:int a3 = 41,
//CHECK-NEXT:int a4 = 106,
//CHECK-NEXT:int a5 = 108,
//CHECK-NEXT:int a6 = 105,
//CHECK-NEXT:int b1 = 0,
//CHECK-NEXT:int b2 = 1,
//CHECK-NEXT:int b3 = 2,
//CHECK-NEXT:int b4 = 3,
//CHECK-NEXT:int b5 = 4,
//CHECK-NEXT:int b6 = 5,
//CHECK-NEXT:int b7 = 6,
//CHECK-NEXT:int b8 = 7,
//CHECK-NEXT:int b9 = 8,
//CHECK-NEXT:int b10 = 9,
//CHECK-NEXT:int b11 = 10,
//CHECK-NEXT:int b12 = 11,
//CHECK-NEXT:int b13 = 12,
//CHECK-NEXT:int b14 = 13,
//CHECK-NEXT:int b15 = 14,
//CHECK-NEXT:int b16 = 15,
//CHECK-NEXT:int b17 = 16
//CHECK-NEXT:){}
template<
hipfftType A1 = HIPFFT_R2C,
hipfftType A2 = HIPFFT_C2R,
hipfftType A3 = HIPFFT_C2C,
hipfftType A4 = HIPFFT_D2Z,
hipfftType A5 = HIPFFT_Z2D,
hipfftType A6 = HIPFFT_Z2Z,
hipfftResult B1 = HIPFFT_SUCCESS,
hipfftResult B2 = HIPFFT_INVALID_PLAN,
hipfftResult B3 = HIPFFT_ALLOC_FAILED,
hipfftResult B4 = HIPFFT_INVALID_TYPE,
hipfftResult B5 = HIPFFT_INVALID_VALUE,
hipfftResult B6 = HIPFFT_INTERNAL_ERROR,
hipfftResult B7 = HIPFFT_EXEC_FAILED,
hipfftResult B8 = HIPFFT_SETUP_FAILED,
hipfftResult B9 = HIPFFT_INVALID_SIZE,
hipfftResult B10 = HIPFFT_UNALIGNED_DATA,
hipfftResult B11 = HIPFFT_INCOMPLETE_PARAMETER_LIST,
hipfftResult B12 = HIPFFT_INVALID_DEVICE,
hipfftResult B13 = HIPFFT_PARSE_ERROR,
hipfftResult B14 = HIPFFT_NO_WORKSPACE,
hipfftResult B15 = HIPFFT_NOT_IMPLEMENTED,
hipfftResult B16 = CUFFT_LICENSE_ERROR,
hipfftResult B17 = HIPFFT_NOT_SUPPORTED>
void foo2(
hipfftType a1 = HIPFFT_R2C,
hipfftType a2 = HIPFFT_C2R,
hipfftType a3 = HIPFFT_C2C,
hipfftType a4 = HIPFFT_D2Z,
hipfftType a5 = HIPFFT_Z2D,
hipfftType a6 = HIPFFT_Z2Z,
hipfftResult b1 = HIPFFT_SUCCESS,
hipfftResult b2 = HIPFFT_INVALID_PLAN,
hipfftResult b3 = HIPFFT_ALLOC_FAILED,
hipfftResult b4 = HIPFFT_INVALID_TYPE,
hipfftResult b5 = HIPFFT_INVALID_VALUE,
hipfftResult b6 = HIPFFT_INTERNAL_ERROR,
hipfftResult b7 = HIPFFT_EXEC_FAILED,
hipfftResult b8 = HIPFFT_SETUP_FAILED,
hipfftResult b9 = HIPFFT_INVALID_SIZE,
hipfftResult b10 = HIPFFT_UNALIGNED_DATA,
hipfftResult b11 = HIPFFT_INCOMPLETE_PARAMETER_LIST,
hipfftResult b12 = HIPFFT_INVALID_DEVICE,
hipfftResult b13 = HIPFFT_PARSE_ERROR,
hipfftResult b14 = HIPFFT_NO_WORKSPACE,
hipfftResult b15 = HIPFFT_NOT_IMPLEMENTED,
hipfftResult b16 = CUFFT_LICENSE_ERROR,
hipfftResult b17 = HIPFFT_NOT_SUPPORTED
){}


//CHECK:template<typename T>
//CHECK-NEXT:float foo3(){}
template<typename T>
hipfftReal foo3(){}

//CHECK:template<typename T>
//CHECK-NEXT:double foo4(){}
template<typename T>
hipfftDoubleReal foo4(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::float2 foo5(){}
template<typename T>
hipfftComplex foo5(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::double2 foo6(){}
template<typename T>
hipfftDoubleComplex foo6(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::float2 foo7(){}
template<typename T>
hipComplex foo7(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::double2 foo8(){}
template<typename T>
hipDoubleComplex foo8(){}

//CHECK:template<typename T>
//CHECK-NEXT:int foo9(){}
template<typename T>
hipfftType_t foo9(){}

//CHECK:template<typename T>
//CHECK-NEXT:int foo10(){}
template<typename T>
hipfftType foo10(){}

//CHECK:template<typename T>
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1050:4: The template argument of the FFT precision and domain type could not be deduced. You need to update this code.
//CHECK:*/
//CHECK-NEXT:std::shared_ptr<oneapi::mkl::dft::descriptor<dpct_placeholder/*Fix the precision and domain type manually*/>> foo11(){}
template<typename T>
hipfftHandle foo11(){}

//CHECK:template<typename T>
//CHECK-NEXT:int foo12(){}
template<typename T>
hipfftResult_t foo12(){}

//CHECK:template<typename T>
//CHECK-NEXT:int foo13(){}
template<typename T>
hipfftResult foo13(){}
