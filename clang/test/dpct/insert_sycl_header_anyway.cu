// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/insert_sycl_header_anyway.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
int main(){
  return 0;
}

