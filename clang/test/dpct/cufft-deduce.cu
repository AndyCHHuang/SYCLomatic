// RUN: dpct --format-range=none -out-root %T/cufft-deduce %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/cufft-deduce/cufft-deduce.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
constexpr int rank = 3;

//CHECK:void foo1(std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>> plan) {
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2* idata;
//CHECK-NEXT:  if ((void *)idata == (void *)odata) {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (double*)idata);
//CHECK-NEXT:  } else {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (double*)idata, odata);
//CHECK-NEXT:  }
//CHECK-NEXT:}
void foo1(hipfftHandle plan) {
  double* odata;
  double2* idata;
  hipfftExecZ2D(plan, idata, odata);
}

//CHECK:void foo2(std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>> plan) {
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2* idata;
//CHECK-NEXT:  if ((void *)idata == (void *)odata) {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (double*)idata);
//CHECK-NEXT:  } else {
//CHECK-NEXT:  oneapi::mkl::dft::compute_backward(*plan, (double*)idata, odata);
//CHECK-NEXT:  }
//CHECK-NEXT:}
void foo2(hipfftHandle plan) {
  double* odata;
  double2* idata;
  hipfftExecZ2D(plan, idata, odata);
}

int main() {
  //CHECK:constexpr int type = 108;
  constexpr hipfftType_t type = HIPFFT_Z2D;
  hipfftType_t type2 = type;

  //CHECK:std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>> plan1;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan1 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan1->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan1->commit(q_ct1);
  hipfftHandle plan1;
  hipfftMakePlanMany(plan1, rank, n, inembed, istride, idist, onembed, ostride, odist, type, 12, work_size);

  //CHECK:std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>> plan2;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan2 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan2->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan2->commit(q_ct1);
  hipfftHandle plan2;
  hipfftMakePlanMany(plan2, rank, n, inembed, istride, idist, onembed, ostride, odist, type, 12, work_size);


  //CHECK:std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>> plan3;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The argument work_size is not supported in the migrated API. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function cannot be deduced. It is migrated as out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan3 = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::REAL>>(std::vector<std::int64_t>{n[0], n[1], n[2]});
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed != nullptr && onembed != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, inembed[2] * inembed[1] * istride, inembed[2] * istride, istride};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[4] = {0, onembed[2] * onembed[1] * ostride, onembed[2] * ostride, ostride};
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:} else {
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[4] = {0, n[1]*(n[2]/2+1), (n[2]/2+1), 1};
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n[2]*n[1]*n[0]);
  //CHECK-NEXT:plan3->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n[2]*n[1]*(n[0]/2+1));
  //CHECK-NEXT:}
  //CHECK-NEXT:plan3->commit(q_ct1);
  hipfftHandle plan3;
  hipfftMakePlanMany(plan3, rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12, work_size);

  foo1(plan1);
  foo2(plan2);

  return 0;
}

