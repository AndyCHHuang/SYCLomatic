// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --usm-level=none -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --match-full-lines --input-file %T/memory_management.dp.cpp %s

#include <hip/hip_runtime.h>

__constant__ float constData[1234567 * 4];

void fooo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;
  // CHECK: dpct::dpct_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);
  // CHECK: dpct::dpct_memset((void*)(d_A), 0xf, size);
  hipMemset(d_A, 0xf, size);

  // CHECK: dpct::async_dpct_memset((void*)(d_A), 0xf, size);
  hipMemsetAsync(d_A, 0xf, size);
  // CHECK: dpct::async_dpct_memset((void*)(d_A), 0xf, size);
  hipMemsetAsync(d_A, 0xf, size, 0);
  // CHECK: dpct::async_dpct_memset((void*)(d_A), 0xf, size, stream);
  hipMemsetAsync(d_A, 0xf, size, stream);

  // CHECK: dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device, stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host, stream);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((void *)((char *)(constData.get_ptr()) + 2), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((void *)((char *)(constData.get_ptr()) + 3), (void*)(h_A), size, dpct::host_to_device, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // dpct::async_dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice, 0);
  // dpct::async_dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::host_to_device, stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 2), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 3), size, dpct::device_to_host, stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy((void*)(h_A), constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost, 0);
  // dpct::async_dpct_memcpy((void*)(h_A), constData.get_ptr(), size, dpct::device_to_host, stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::dpct_free(d_A);
  hipFree(d_A);
  free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  return (dpct::dpct_malloc(buffer, size), 0);
  return hipMalloc(buffer, size);
}

void checkError(hipError_t err) {
}

void testCommas() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  float *d_B = NULL;
  // CHECK:  dpct::dpct_malloc((void **)&d_A, size);
  hipMalloc((void **)&d_A, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  int err = (dpct::dpct_malloc((void **)&d_A, size), 0);
  hipError_t err = hipMalloc((void **)&d_A, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_malloc((void **)&d_A, size), 0));
  checkError(hipMalloc((void **)&d_A, size));
  // CHECK:  dpct::dpct_memset((void*)(d_A), 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memset((void*)(d_A), 0xf, size), 0);
  err = hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memset((void*)(d_A), 0xf, size), 0));
  checkError(hipMemset(d_A, 0xf, size));

  ///////// Host to host
  // CHECK:  dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_host);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_host), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_host), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(d_A), (void*)(h_A), size, dpct::host_to_device), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_host), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_device);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_device), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::device_to_device), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::automatic), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), (void*)(d_A), size, dpct::automatic), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice));

  // CHECK:  dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size, dpct::device_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(constData.get_ptr(), (void*)(h_A), size, dpct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyDeviceToDevice));

  // CHECK:  dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(d_B), size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(d_B), size, dpct::device_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(h_A), size, dpct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size, dpct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size, dpct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(d_B), size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(d_B), size, dpct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void *)((char *)(constData.get_ptr()) + 1), (void*)(d_B), size, dpct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(constData.get_ptr(), (void*)(d_B), size), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), constData.get_ptr(), size, dpct::device_to_host), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), constData.get_ptr(), size, dpct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost));

  // CHECK:  dpct::dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_host), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy((void*)(d_B), constData.get_ptr(), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(d_B), constData.get_ptr(), size, dpct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void*)(d_B), constData.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice));


  // CHECK:  dpct::dpct_memcpy((void*)(d_B), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(d_B), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void*)(d_B), (void *)((char *)(constData.get_ptr()) + 1), size, dpct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy((void*)(h_A), constData.get_ptr(), size), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void*)(h_A), constData.get_ptr(), size), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size));

  // CHECK: dpct::dpct_free(d_A);
  hipFree(d_A);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_free(d_A), 0);
  err = hipFree(d_A);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_free(d_A), 0));
  checkError(hipFree(d_A));
  // CHECK:  free(h_A);
  free(h_A);
}

// CHECK:  dpct::device_memory<float, 1> d_A(1234567);
// CHECK:  dpct::device_memory<float, 1> d_B(1234567);
static __device__ float d_A[1234567];
static __device__ float d_B[1234567];

void testCommas_in_device_memory() {
  size_t size = 1234567 * sizeof(float);
  hipError_t err;
  float *h_A = (float *)malloc(size);

  // CHECK:  dpct::dpct_memset(d_A.get_ptr(), 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memset(d_A.get_ptr(), 0xf, size), 0);
  err = hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memset(d_A.get_ptr(), 0xf, size), 0));
  checkError(hipMemset(d_A, 0xf, size));

  ///////// Host to host
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), (void*)(h_A), size, dpct::host_to_host);
  hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), (void*)(h_A), size, dpct::host_to_host), 0);
  err = hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), (void*)(h_A), size, dpct::host_to_host), 0));
  checkError(hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A.get_ptr(), (void*)(h_A), size, dpct::host_to_device), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A.get_ptr(), (void*)(h_A), size, dpct::host_to_device), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::device_to_host), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::device_to_host), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device);
  hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device), 0);
  err = hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::automatic), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::automatic), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), (void*)(h_A), size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A.get_ptr(), (void*)(h_A), size, dpct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A.get_ptr(), (void*)(h_A), size, dpct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size, dpct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size, dpct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::device_to_host), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(h_A), d_A.get_ptr(), size, dpct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((void*)(h_A), d_B.get_ptr(), size), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size));

  void *p_addr;
  // CHECK:  *(&p_addr) = d_A.get_ptr();
  hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A));

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  err = (*(&p_addr) = d_A.get_ptr(), 0);
  err = hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A));

  // CHECK: /*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:checkError((*(&p_addr) = d_A.get_ptr(), 0));
  checkError(hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A)));

  // CHECK:  free(h_A);
  free(h_A);
}

#define CUDA_CHECK(call)                                                           \
    if ((call) != hipSuccess) { \
        exit(-1); \
    }

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {}


template<typename T>
void uninstantiated_template_call(const T * d_data, size_t width, size_t height) {
  size_t datasize = width * height;
  T * data = new T[datasize];
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  assert_cuda((dpct::dpct_memcpy((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host), 0));
  assert_cuda(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: dpct::dpct_memcpy((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host), 0));
  checkError(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: int err = (dpct::dpct_memcpy((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host), 0);
  hipError_t err = hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: CUDA_CHECK((dpct::dpct_memcpy((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host), 0));
  CUDA_CHECK(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: checkCudaErrors((dpct::dpct_memcpy((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host), 0));
  checkCudaErrors(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: #define CUDAMEMCPY dpct::dpct_memcpy
  // CHECK-NEXT: CUDAMEMCPY((void*)(data), (void*)(d_data), datasize * sizeof(T), dpct::device_to_host);
  #define CUDAMEMCPY hipMemcpy
  CUDAMEMCPY(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  delete[] data;
}

#define N 1024
void test_segmentation_fault() {
  float *buffer;
  /*
  * Original code in getSizeString():
  * "SizeExpr->getBeginLoc()" cannot get the real SourceLocation of "N*sizeof(float)",
  * and results in boundary violation in "dpctGlobalInfo::getSourceManager().getCharacterData(SizeBegin)"
  * and fails with segmentation fault.
  * https://jira.devtools.intel.com/browse/CTST-527
  * https://jira.devtools.intel.com/browse/CTST-528
  */
  hipMalloc(&buffer, N*sizeof(float));
}

// CHECK: dpct::device_memory<uint32_t, 1> d_error(1);
static __device__ uint32_t d_error[1];

void test_foo(){
  // CHECK: dpct::dpct_memset(d_error.get_ptr(), 0, sizeof(uint32_t));
  hipMemset(d_error, 0, sizeof(uint32_t));
}
