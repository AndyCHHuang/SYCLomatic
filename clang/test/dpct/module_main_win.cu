// UNSUPPORTED: -linux-
// RUN: dpct --format-range=none -out-root %T/module_main_win %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/module_main_win/module_main_win.dp.cpp

//CHECK: #include <libloaderapi.h>
#include <string>
int main(){
    //CHECK: HMODULE M;
    hipModule_t M;
    //CHECK: dpct::kernel_functor F;
    hipFunction_t F;
    std::string Path, FunctionName;
    //CHECK: /*
    //CHECK-NEXT: DPCT1079:{{[0-9]+}}: Replace "Placeholder" with the file path of the dynamic library.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = LoadLibraryA(PlaceHolder/*Fix the module file name manually*/);
    hipModuleLoad(&M, Path.c_str());
    //CHECK: /*
    //CHECK-NEXT: DPCT1079:{{[0-9]+}}: Replace "Placeholder" with the file path of the dynamic library.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = LoadLibraryA(PlaceHolder/*Fix the module file name manually*/);
    hipModuleLoadData(&M, Data.c_str());
    //CHECK: F = (dpct::kernel_functor)GetProcAddress(M, (std::string(FunctionName.c_str()) + "_wrapper").c_str());
    hipModuleGetFunction(&F, M, FunctionName.c_str());

    int sharedSize;
    CUStream s;
    void **param, **extra;
    //CHECK:  F(*s, sycl::nd_range<3>(sycl::range<3>(32, 16, 1) * sycl::range<3>(64, 32, 4), sycl::range<3>(64, 32, 4)), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, s, param, extra);
    //CHECK:  F(q_ct1, sycl::nd_range<3>(sycl::range<3>(32, 16, 1) * sycl::range<3>(64, 32, 4), sycl::range<3>(64, 32, 4)), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, 0, param, extra);
    //CHECK:  F(q_ct1, sycl::nd_range<3>(sycl::range<3>(32, 16, 1) * sycl::range<3>(64, 32, 4), sycl::range<3>(64, 32, 4)), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, hipStreamLegacy, param, extra);

    //CHECK: dpct::image_wrapper_base_p tex;
    //CHECK: tex = (dpct::image_wrapper_base_p)GetProcAddress(M, "tex");
    hipTexRef tex;
    hipModuleGetTexRef(&tex, M, "tex");

    //CHECK: FreeLibrary(M);
    hipModuleUnload(M);
    return 0;
}