// RUN: dpct --format-range=none -out-root %T/api_recognization %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/api_recognization/api_recognization.dp.cpp
#include "hip/hip_runtime.h"


#include <stdio.h>

namespace at {
    namespace cub {
        void exclusive_scan(){}
    }
}

int main(int argc, char **argv) {
  // CHECK: at::cub::exclusive_scan();
  at::cub::exclusive_scan();

  return 0;

}