// RUN: dpct -out-root %T %s  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/cublasTtrmm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;

  int side0 = 0; int side1 = 1; int fill0 = 0; int fill1 = 1;
  int trans0 = 0; int trans1 = 1; int trans2 = 2; int diag0 = 0; int diag1 = 1;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:0: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans0;
  // CHECK-NEXT: auto ptr_ct8 = A_S;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto ptr_ct12 = C_S;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_S, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: status = (mkl::trmm(handle, (mkl::side)side0, (((int)fill0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag0, m_ct5, n_ct6, *(&alpha_S), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto ptr_ct8 = A_S;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto ptr_ct12 = C_S;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<float,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_S, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: mkl::trmm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m_ct5, n_ct6, *(&alpha_S), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13);
  // CHECK-NEXT: }
  status = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);
  hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:1: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans1;
  // CHECK-NEXT: auto ptr_ct8 = A_D;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto ptr_ct12 = C_D;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_D, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: status = (mkl::trmm(handle, (mkl::side)side1, (((int)fill1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)diag1, m_ct5, n_ct6, *(&alpha_D), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto ptr_ct8 = A_D;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto ptr_ct12 = C_D;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<double,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_D, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: mkl::trmm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m_ct5, n_ct6, *(&alpha_D), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13);
  // CHECK-NEXT: }
  status = hipblasDtrmm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, (hipblasDiagType_t)diag1, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);
  hipblasDtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);


  const hipComplex *A_C = 0;
  const hipComplex *B_C = 0;
  hipComplex *C_C = 0;
  hipComplex alpha_C = make_hipComplex(1.0f,0.0f);
  const hipDoubleComplex *A_Z = 0;
  const hipDoubleComplex *B_Z = 0;
  hipDoubleComplex *C_Z = 0;
  hipDoubleComplex alpha_Z = make_hipDoubleComplex(1.0,0.0);


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:2: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = trans2;
  // CHECK-NEXT: auto ptr_ct8 = A_C;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto ptr_ct12 = C_C;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_C, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: status = (mkl::trmm(handle, (mkl::side)0, (((int)0)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)0, m_ct5, n_ct6, std::complex<float>((&alpha_C)->x(),(&alpha_C)->y()), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto ptr_ct8 = A_C;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto ptr_ct12 = C_C;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<float>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_C, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: mkl::trmm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m_ct5, n_ct6, std::complex<float>((&alpha_C)->x(),(&alpha_C)->y()), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13);
  // CHECK-NEXT: }
  status = hipblasCtrmm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)0, (hipblasOperation_t)trans2, (hipblasDiagType_t)0, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);
  hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:3: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: {
  // CHECK-NEXT: auto transpose_ct3 = 2;
  // CHECK-NEXT: auto ptr_ct8 = A_Z;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto ptr_ct12 = C_Z;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_Z, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: status = (mkl::trmm(handle, (mkl::side)1, (((int)1)==0?(mkl::uplo::lower):(mkl::uplo::upper)), (((int)transpose_ct3)==2?(mkl::transpose::conjtrans):((mkl::transpose)transpose_ct3)), (mkl::diag)1, m_ct5, n_ct6, std::complex<double>((&alpha_Z)->x(),(&alpha_Z)->y()), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto ptr_ct8 = A_Z;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct8);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto ptr_ct12 = C_Z;
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::memory_manager::get_instance().translate_ptr(ptr_ct12);
  // CHECK-NEXT: cl::sycl::buffer<std::complex<double>,1> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>, 1>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT: auto ld_ct13 = ldc; auto m_ct5 = m; auto n_ct6 = n;
  // CHECK-NEXT: dpct::matrix_mem_copy(ptr_ct12, B_Z, ld_ct13, ldb, m_ct5, n_ct6, dpct::device_to_device);
  // CHECK-NEXT: mkl::trmm(handle, mkl::side::left, mkl::uplo::upper, mkl::transpose::nontrans, mkl::diag::nonunit, m_ct5, n_ct6, std::complex<double>((&alpha_Z)->x(),(&alpha_Z)->y()), buffer_ct{{[0-9]+}}, lda,  buffer_ct{{[0-9]+}}, ld_ct13);
  // CHECK-NEXT: }
  status = hipblasZtrmm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)1, (hipblasOperation_t)2, (hipblasDiagType_t)1, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);

}
