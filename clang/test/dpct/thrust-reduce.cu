#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: c2s --format-range=none -out-root %T/thrust-reduce %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -fno-delayed-template-parsing -std=c++17 -fsized-deallocation
// RUN: FileCheck --input-file %T/thrust-reduce/thrust-reduce.dp.cpp --match-full-lines %s
// CHECK: #include <oneapi/dpl/execution>
// CHECK-NEXT: #include <oneapi/dpl/algorithm>
// CHECK-NEXT: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <c2s/c2s.hpp>
// CHECK-NEXT: #include <c2s/dpl_utils.hpp>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

template <typename T>
struct square {
// CHECK:  T operator()(const T& x) const { return x * x; }
  __host__ __device__  T operator()(const T& x) const { return x * x; }
};

int main() {
  double sum;
  double *p;
// CHECK:  c2s::device_pointer<double> dp(p);
  thrust::device_ptr<double> dp(p);
// CHECK:  sum = std::reduce(oneapi::dpl::execution::make_device_policy(c2s::get_default_queue()), dp, dp + 10);
  sum = thrust::reduce(dp, dp + 10);
}

void check_transform_reduce() {
// CHECK:  c2s::device_ext &dev_ct1 = c2s::get_current_device();
// CHECK-NEXT:  sycl::queue &q_ct1 = dev_ct1.default_queue();
  float x[4] = {1.0, 2.0, 3.0, 4.0};
// CHECK:  c2s::device_vector<float> d_x(x, x + 4);
  thrust::device_vector<float> d_x(x, x + 4);
  square<float>        unary_op;
// CHECK:  std::plus<float> binary_op;
  thrust::plus<float> binary_op;
  float init = 0;

// CHECK:  float norm     = std::transform_reduce(oneapi::dpl::execution::make_device_policy(q_ct1), d_x.begin(), d_x.end(), init, binary_op, unary_op);
  float norm     = thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op);
// CHECK:  float normSqrt = std::sqrt(std::transform_reduce(oneapi::dpl::execution::make_device_policy(q_ct1), d_x.begin(), d_x.end(), init, binary_op, unary_op));
  float normSqrt = std::sqrt(thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op));
}

template <typename T>
class C {
  T *data;
public:
  C() {
    this->data = 0;
  }

  // CHECK:   inline T *raw() {
  // CHECK-NEXT:   return c2s::get_raw_pointer(this->data);
  // CHECK-NEXT: }
  // CHECK-NEXT: inline const T *raw() const {
  // CHECK-NEXT:   return c2s::get_raw_pointer(this->data + 2);
  // CHECK-NEXT: }
  inline T *raw() {
    return thrust::raw_pointer_cast(this->data);
  }
  inline const T *raw() const {
    return thrust::raw_pointer_cast(this->data + 2);
  }
};

