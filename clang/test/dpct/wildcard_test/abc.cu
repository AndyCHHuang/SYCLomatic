
#include <hip/hip_runtime.h>
// RUN: dpct -out-root=%T/abc -in-root=%S %S/*.cu  -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/abc/abc.dp.cpp --match-full-lines %S/abc.cu
// RUN: FileCheck --input-file %T/abc/abd.dp.cpp --match-full-lines %S/abd.cu

// CHECK: void testKernelPtr(const int *L, const int *M, int N, cl::sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(0) * [[ITEMNAME]].get_local_range().get(0) + [[ITEMNAME]].get_local_id(0);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  void *karg1 = 0;
  const int *karg2 = 0;
  int karg3 = 80;
  // CHECK:  {
  // CHECK-NEXT:    std::pair<dpct::buffer_t, size_t> karg2_buf = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:    size_t karg2_offset = karg2_buf.second;
  // CHECK-NEXT:    dpct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        auto karg2_acc = karg2_buf.first.get_access<cl::sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((griddim * threaddim), threaddim),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_ct1]]) {
  // CHECK-NEXT:            const int *karg2 = (const int*)(&karg2_acc[0] + karg2_offset);
  // CHECK-NEXT:            testKernelPtr((const int *)karg2, karg2, karg3, [[ITEM]]);
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
  // CHECK-NEXT:  }
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg2, karg2, karg3);

}
