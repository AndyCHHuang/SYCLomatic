
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --usm-level=none -out-root=%T/abc -in-root=%S %S/*.cu --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/abc/abc.dp.cpp --match-full-lines %S/abc.cu
// RUN: FileCheck --input-file %T/abc/abd.dp.cpp --match-full-lines %S/abd.cu

// CHECK: void testKernelPtr(const int *L, const int *M, int N, sycl::nd_item<3> [[ITEMNAME:item_ct1]]) {
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  // CHECK: int gtid = [[ITEMNAME]].get_group(2) * [[ITEMNAME]].get_local_range().get(2) + [[ITEMNAME]].get_local_id(2);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
  dim3 griddim = 2;
  dim3 threaddim = 32;
  void *karg1 = 0;
  const int *karg2 = 0;
  int karg3 = 80;
  // CHECK: {
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg2_buf_ct0 = dpct::get_buffer_and_offset((const int *)karg2);
  // CHECK-NEXT:   size_t karg2_offset_ct0 = karg2_buf_ct0.second;
  // CHECK-NEXT:   std::pair<dpct::buffer_t, size_t> karg2_buf_ct1 = dpct::get_buffer_and_offset(karg2);
  // CHECK-NEXT:   size_t karg2_offset_ct1 = karg2_buf_ct1.second;
  // CHECK-NEXT:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto karg2_acc_ct0 = karg2_buf_ct0.first.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-NEXT:       auto karg2_acc_ct1 = karg2_buf_ct1.first.get_access<sycl::access::mode::read_write>(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto dpct_global_range = griddim * threaddim;
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(dpct_global_range, threaddim),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           const int *karg2_ct0 = (const int *)(&karg2_acc_ct0[0] + karg2_offset_ct0);
  // CHECK-NEXT:           const int *karg2_ct1 = (const int *)(&karg2_acc_ct1[0] + karg2_offset_ct1);
  // CHECK-NEXT:           testKernelPtr(karg2_ct0, karg2_ct1, karg3, item_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg2, karg2, karg3);

}
