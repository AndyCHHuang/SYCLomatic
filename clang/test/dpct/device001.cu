#include "hip/hip_runtime.h"
// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck %s --match-full-lines --input-file %T/device001.dp.cpp

int main(int argc, char **argv) {

  // CHECK: dpct::dpct_device_info deviceProp;
  hipDeviceProp_t deviceProp;

  // CHECK: if (deviceProp.get_mode() == dpct::compute_mode::prohibited) {
  if (deviceProp.computeMode == hipComputeModeProhibited) {
    return 0;
  }

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:int major = deviceProp.get_major_version();
  int major = deviceProp.major;
// CHECK:/*
// CHECK-NEXT:DPCT1006:{{[0-9]+}}: SYCL doesn't provide standard API to differentiate between integrated/discrete GPU devices. Consider to re-implement the code which depends on this field
// CHECK-NEXT:*/
// CHECK-NEXT:int integrated = deviceProp.get_integrated();
  int integrated = deviceProp.integrated;

  // CHECK: int warpSize = deviceProp.get_max_sub_group_size();
  int warpSize = deviceProp.warpSize;

  // CHECK: int maxThreadsPerMultiProcessor = deviceProp.get_max_work_items_per_compute_unit();
  int maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:deviceProp.set_major_version(1);
  deviceProp.major=1;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:int minor = deviceProp.get_minor_version();
  int minor = deviceProp.minor;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:deviceProp.set_minor_version(120);
  deviceProp.minor=120;

  // CHECK:     char *name = deviceProp.get_name();
  char *name = deviceProp.name;

  // CHECK:     int clock = deviceProp.get_max_clock_frequency();
  int clock = deviceProp.clockRate;
  int xxxx = 10;
  int yyyy = 5;

  // CHECK:  deviceProp.set_max_clock_frequency ( xxxx * 100 + yyyy);
  deviceProp.clockRate = xxxx * 100 + yyyy;

  // CHECK: int count = deviceProp.get_max_compute_units();
  int count = deviceProp.multiProcessorCount;

  // CHECK: count = deviceProp.get_max_work_group_size();
  count = deviceProp.maxThreadsPerBlock;

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1022:{{[0-9]+}}: There is no exact match between maxGridSize and nd_range size. Please verify the correctness.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  int *maxGridSize = deviceProp.get_max_nd_range_size();
  int *maxGridSize = deviceProp.maxGridSize;

  // CHECK:/*
  // CHECK-NEXT:DPCT1019:{{[0-9]+}}: The sharedMemPerBlock is not necessarily the same as local_mem_size in DPC++
  // CHECK-NEXT:*/
  // CHECK-NEXT:size_t share_mem_size = deviceProp.get_local_mem_size();
  size_t share_mem_size = deviceProp.sharedMemPerBlock;

  // CHECK: cl::sycl::range<3> grid(deviceProp.get_max_compute_units() * (deviceProp.get_max_work_items_per_compute_unit() / deviceProp.get_max_sub_group_size()), 1, 1);
  dim3 grid(deviceProp.multiProcessorCount * (deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize));

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You may want to rewrite this code
// CHECK-NEXT:*/
// CHECK-NEXT:int n = deviceProp.get_minor_version() / deviceProp.get_major_version();
  int n = deviceProp.minor / deviceProp.major;

  // CHECK: size_t memsize = deviceProp.get_global_mem_size();
  size_t memsize = deviceProp.totalGlobalMem;

  return 0;
}

__global__ void foo_kernel(void)
{
}

void test()
{
  // CHECK: dpct::dpct_device_info deviceProp;
  hipDeviceProp_t deviceProp;
  // CHECK:    {
  // CHECK-NEXT:dpct::get_default_queue().submit(
  // CHECK-NEXT:      [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:        cgh.parallel_for<dpct_kernel_name<class foo_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:          cl::sycl::nd_range<3>((cl::sycl::range<3>(deviceProp.get_max_compute_units(), 1, 1) * cl::sycl::range<3>(deviceProp.get_max_work_group_size(), 1, 1)), cl::sycl::range<3>(deviceProp.get_max_work_group_size(), 1, 1)),
  // CHECK-NEXT:          [=](cl::sycl::nd_item<3> [[ITEM:item_ct1]]) {
  // CHECK-NEXT:            foo_kernel();
  // CHECK-NEXT:          });
  // CHECK-NEXT:      });
 // CHECK-NEXT:  }
  foo_kernel<<<deviceProp.multiProcessorCount, deviceProp.maxThreadsPerBlock,  deviceProp.maxThreadsPerBlock>>>();
}
