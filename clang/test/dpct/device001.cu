#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none --usm-level=none -out-root %T/device001 %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device001/device001.dp.cpp

int main(int argc, char **argv) {

  // CHECK: dpct::device_info deviceProp;
  hipDeviceProp_t deviceProp;

  // CHECK: /*
  // CHECK-NEXT: DPCT1035:{{[0-9]+}}: All DPC++ devices can be used by host to submit tasks. You may need to adjust this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (false) {
  if (deviceProp.computeMode == hipComputeModeProhibited) {
    return 0;
  }

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The SYCL device version is different from CUDA Compute Compatibility. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int major = deviceProp.get_major_version();
  int major = deviceProp.major;
// CHECK:/*
// CHECK-NEXT:DPCT1006:{{[0-9]+}}: DPC++ does not provide a standard API to differentiate between integrated/ discrete GPU devices.
// CHECK-NEXT:*/
// CHECK-NEXT:int integrated = deviceProp.get_integrated();
  int integrated = deviceProp.integrated;

  // CHECK: int warpSize = deviceProp.get_max_sub_group_size();
  int warpSize = deviceProp.warpSize;

  // CHECK: int maxThreadsPerMultiProcessor = deviceProp.get_max_work_items_per_compute_unit();
  int maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The SYCL device version is different from CUDA Compute Compatibility. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:deviceProp.set_major_version(1);
  deviceProp.major=1;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The SYCL device version is different from CUDA Compute Compatibility. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int minor = deviceProp.get_minor_version();
  int minor = deviceProp.minor;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The SYCL device version is different from CUDA Compute Compatibility. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:deviceProp.set_minor_version(120);
  deviceProp.minor=120;

  // CHECK:     char *name = deviceProp.get_name();
  char *name = deviceProp.name;

  // CHECK:     int clock = deviceProp.get_max_clock_frequency();
  int clock = deviceProp.clockRate;
  int xxxx = 10;
  int yyyy = 5;

  // CHECK:  deviceProp.set_max_clock_frequency ( xxxx * 100 + yyyy);
  deviceProp.clockRate = xxxx * 100 + yyyy;

  // CHECK: int count = deviceProp.get_max_compute_units();
  int count = deviceProp.multiProcessorCount;

  // CHECK: count = deviceProp.get_max_work_group_size();
  count = deviceProp.maxThreadsPerBlock;

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1022:{{[0-9]+}}: There is no exact match between the maxGridSize and the max_nd_range size. Verify the correctness of the code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  int *maxGridSize = deviceProp.get_max_nd_range_size();
  int *maxGridSize = deviceProp.maxGridSize;

  // CHECK:/*
  // CHECK-NEXT:DPCT1019:{{[0-9]+}}: local_mem_size in SYCL is not a complete equivalent of sharedMemPerBlock in CUDA. You may need to adjust the code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:size_t share_mem_size = deviceProp.get_local_mem_size();
  size_t share_mem_size = deviceProp.sharedMemPerBlock;

  // CHECK: sycl::range<3> grid(1, 1, deviceProp.get_max_compute_units() * (deviceProp.get_max_work_items_per_compute_unit() / deviceProp.get_max_sub_group_size()));
  dim3 grid(deviceProp.multiProcessorCount * (deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize));

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The SYCL device version is different from CUDA Compute Compatibility. You may need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int n = deviceProp.get_minor_version() / deviceProp.get_major_version();
  int n = deviceProp.minor / deviceProp.major;

  // CHECK: size_t memsize = deviceProp.get_global_mem_size();
  size_t memsize = deviceProp.totalGlobalMem;

  // CHECK: int i=true;
  int i=deviceProp.deviceOverlap;
  // CHECK: if(true){
  if(deviceProp.deviceOverlap){
  //dosomething.
  }
  return 0;
}

__global__ void foo_kernel(void)
{
}

void test()
{
  // CHECK: dpct::device_info deviceProp;
  hipDeviceProp_t deviceProp;
  // CHECK:   dpct::get_default_queue().parallel_for<dpct_kernel_name<class foo_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, deviceProp.get_max_compute_units()) * sycl::range<3>(1, 1, deviceProp.get_max_work_group_size()), sycl::range<3>(1, 1, deviceProp.get_max_work_group_size())),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           foo_kernel();
  // CHECK-NEXT:         });
  foo_kernel<<<deviceProp.multiProcessorCount, deviceProp.maxThreadsPerBlock,  deviceProp.maxThreadsPerBlock>>>();
}

void test2() {
  hipLimit_t limit;
  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceSetLimit was removed because DPC++ currently does not support setting resource limits on devices.
  // CHECK-NEXT: */
  hipDeviceSetLimit(limit, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipDeviceSetLimit was replaced with 0 because DPC++ currently does not support setting resource limits on devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: int i = 0;
  int i = hipDeviceSetLimit(limit, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cudaThreadSetLimit was removed because DPC++ currently does not support setting resource limits on devices.
  // CHECK-NEXT: */
  cudaThreadSetLimit(limit, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cudaThreadSetLimit was replaced with 0 because DPC++ currently does not support setting resource limits on devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: int j = 0;
  int j = cudaThreadSetLimit(limit, 0);
}

