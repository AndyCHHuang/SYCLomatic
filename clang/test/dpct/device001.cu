#include "hip/hip_runtime.h"
// RUN: dpct --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device001.dp.cpp

int main(int argc, char **argv) {

  // CHECK: dpct::device_info deviceProp;
  hipDeviceProp_t deviceProp;

  // CHECK: if (deviceProp.get_mode() == dpct::compute_mode::prohibited) {
  if (deviceProp.computeMode == hipComputeModeProhibited) {
    return 0;
  }

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int major = deviceProp.get_major_version();
  int major = deviceProp.major;
// CHECK:/*
// CHECK-NEXT:DPCT1006:{{[0-9]+}}: DPC++ does not provide a standard API to differentiate between integrated/ discrete GPU devices.
// CHECK-NEXT:*/
// CHECK-NEXT:int integrated = deviceProp.get_integrated();
  int integrated = deviceProp.integrated;

  // CHECK: int warpSize = deviceProp.get_max_sub_group_size();
  int warpSize = deviceProp.warpSize;

  // CHECK: int maxThreadsPerMultiProcessor = deviceProp.get_max_work_items_per_compute_unit();
  int maxThreadsPerMultiProcessor = deviceProp.maxThreadsPerMultiProcessor;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:deviceProp.set_major_version(1);
  deviceProp.major=1;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int minor = deviceProp.get_minor_version();
  int minor = deviceProp.minor;

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:deviceProp.set_minor_version(120);
  deviceProp.minor=120;

  // CHECK:     char *name = deviceProp.get_name();
  char *name = deviceProp.name;

  // CHECK:     int clock = deviceProp.get_max_clock_frequency();
  int clock = deviceProp.clockRate;
  int xxxx = 10;
  int yyyy = 5;

  // CHECK:  deviceProp.set_max_clock_frequency ( xxxx * 100 + yyyy);
  deviceProp.clockRate = xxxx * 100 + yyyy;

  // CHECK: int count = deviceProp.get_max_compute_units();
  int count = deviceProp.multiProcessorCount;

  // CHECK: count = deviceProp.get_max_work_group_size();
  count = deviceProp.maxThreadsPerBlock;

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1022:{{[0-9]+}}: There is no exact match between the maxGridSize and the max_nd_range size. Verify the correctness of the code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  int *maxGridSize = deviceProp.get_max_nd_range_size();
  int *maxGridSize = deviceProp.maxGridSize;

  // CHECK:/*
  // CHECK-NEXT:DPCT1019:{{[0-9]+}}: local_mem_size in SYCL is not a complete equivalent of sharedMemPerBlock in CUDA. You may need to adjust the code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:size_t share_mem_size = deviceProp.get_local_mem_size();
  size_t share_mem_size = deviceProp.sharedMemPerBlock;

  // CHECK: cl::sycl::range<3> grid(deviceProp.get_max_compute_units() * (deviceProp.get_max_work_items_per_compute_unit() / deviceProp.get_max_sub_group_size()), 1, 1);
  dim3 grid(deviceProp.multiProcessorCount * (deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize));

// CHECK:/*
// CHECK-NEXT:DPCT1005:{{[0-9]+}}: The device version is different. You need to rewrite this code.
// CHECK-NEXT:*/
// CHECK-NEXT:int n = deviceProp.get_minor_version() / deviceProp.get_major_version();
  int n = deviceProp.minor / deviceProp.major;

  // CHECK: size_t memsize = deviceProp.get_global_mem_size();
  size_t memsize = deviceProp.totalGlobalMem;

  // CHECK: int i=true;
  int i=deviceProp.deviceOverlap;
  // CHECK: if(true){
  if(deviceProp.deviceOverlap){
  //dosomething.
  }
  return 0;
}

__global__ void foo_kernel(void)
{
}

void test()
{
  // CHECK: dpct::device_info deviceProp;
  hipDeviceProp_t deviceProp;
  // CHECK:   dpct::get_default_queue().submit(
  // CHECK-NEXT:     [&](cl::sycl::handler &cgh) {
  // CHECK-NEXT:       auto dpct_global_range = cl::sycl::range<3>(deviceProp.get_max_compute_units(), 1, 1) * cl::sycl::range<3>(deviceProp.get_max_work_group_size(), 1, 1);
  // CHECK-NEXT:       auto dpct_local_range = cl::sycl::range<3>(deviceProp.get_max_work_group_size(), 1, 1);
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class foo_kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         cl::sycl::nd_range<3>(cl::sycl::range<3>(dpct_global_range.get(2), dpct_global_range.get(1), dpct_global_range.get(0)), cl::sycl::range<3>(dpct_local_range.get(2), dpct_local_range.get(1), dpct_local_range.get(0))),
  // CHECK-NEXT:         [=](cl::sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           foo_kernel();
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  foo_kernel<<<deviceProp.multiProcessorCount, deviceProp.maxThreadsPerBlock,  deviceProp.maxThreadsPerBlock>>>();
}

void test2() {
  hipLimit_t limit;
  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceSetLimit was removed. DPC++ currently doesn't support setting resource limits on devices.
  // CHECK-NEXT: */
  hipDeviceSetLimit(limit, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipDeviceSetLimit was replaced with 0. DPC++ currently doesn't support setting resource limits on devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: int i = 0;
  int i = hipDeviceSetLimit(limit, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cudaThreadSetLimit was removed. DPC++ currently doesn't support setting resource limits on devices.
  // CHECK-NEXT: */
  cudaThreadSetLimit(limit, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cudaThreadSetLimit was replaced with 0. DPC++ currently doesn't support setting resource limits on devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: int j = 0;
  int j = cudaThreadSetLimit(limit, 0);
}
