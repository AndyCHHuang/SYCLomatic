// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    //CHECK: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnSpotrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnDpotrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnCpotrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnZpotrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    status = hipsolverDnSpotrf_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotrf_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotrf_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotrf_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnSgetrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnDgetrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnCgetrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnZgetrf_bufferSize was replaced with 0, because Function call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    status = hipsolverDnSgetrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnDgetrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnCgetrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnZgetrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, buffer_ct3, lda,   result_temp_buffer7);
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct4 = allocation_ct4.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct4 = allocation_ct4.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);
    hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct4 = allocation_ct4.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct4 = allocation_ct4.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);
    hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);
    hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&C_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, buffer_ct4, lda, buffer_ct6, ldb, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);
    hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);
    hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);
    hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);
    hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct7 = allocation_ct7.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct7.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer7(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, buffer_ct3, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct7.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct4 = allocation_ct4.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct4 = allocation_ct4.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);
    hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct4 = allocation_ct4.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct7 = allocation_ct7.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct4 = allocation_ct4.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct7 = allocation_ct7.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);
    hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);
    hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct6 = allocation_ct6.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct6.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer6(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, buffer_ct4, lda, result_temp_buffer6, buffer_ct7, ldb, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct6.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);
    hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<float>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<float>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct5 = allocation_ct5.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct5 = allocation_ct5.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct5.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<double>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<double>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct5 = allocation_ct5.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct5 = allocation_ct5.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct5.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct5 = allocation_ct5.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct5.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, buffer_ct3, lda, buffer_ct5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    status = hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ormqr_get_lwork<float>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ormqr_get_lwork<float>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct8 = allocation_ct8.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct11 = allocation_ct11.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct11.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct8 = allocation_ct8.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct8.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct9 = allocation_ct9.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct9.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct11 = allocation_ct11.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct11.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    status = hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ormqr_get_lwork<double>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ormqr_get_lwork<double>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct8 = allocation_ct8.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct9 = allocation_ct9.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct11 = allocation_ct11.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct11.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct8 = allocation_ct8.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct8.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct9 = allocation_ct9.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct9.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct11 = allocation_ct11.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct11.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    status = hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::unmqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::unmqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    status = hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::unmqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::unmqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct8 = allocation_ct8.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct8.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&B_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct9 = allocation_ct9.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct9.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct11 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct11 = allocation_ct11.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct11.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct13 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct13 = allocation_ct13.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct13.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer13(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, buffer_ct6, lda, buffer_ct8, buffer_ct9, ldb, buffer_ct11, Lwork, result_temp_buffer13);
    // CHECK-NEXT: buffer_ct13.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    status = hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgqr_get_lwork<float>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgqr_get_lwork<float>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct4 = allocation_ct4.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct4 = allocation_ct4.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct4.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT: cl::sycl::buffer<float> buffer_ct7 = allocation_ct7.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct7.size/sizeof(float)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgqr_get_lwork<double>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgqr_get_lwork<double>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct4 = allocation_ct4.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct7 = allocation_ct7.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct4 = allocation_ct4.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct4.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct7 = allocation_ct7.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct7.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct4 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct4 = allocation_ct4.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct4.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&TAU_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct7 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct7 = allocation_ct7.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct7.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct9 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct9 = allocation_ct9.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct9.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer9(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungqr(*cusolverH, m, n, k, buffer_ct4, lda, buffer_ct6, buffer_ct7, Lwork, result_temp_buffer9);
    // CHECK-NEXT: buffer_ct9.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<float>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<float>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:{
    // CHECK-NEXT:auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT:cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT:cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT:auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT:cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT:cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT:status = (mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT:buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    // CHECK-NEXT:{
    // CHECK-NEXT:auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_f);
    // CHECK-NEXT:cl::sycl::buffer<float> buffer_ct3 = allocation_ct3.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct3.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT:cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT:auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_f);
    // CHECK-NEXT:cl::sycl::buffer<float> buffer_ct6 = allocation_ct6.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct6.size/sizeof(float)));
    // CHECK-NEXT:auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT:cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT:cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT:mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT:buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    status = hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    status = hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<double>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<double>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct3 = allocation_ct3.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct3.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_d);
    // CHECK-NEXT: cl::sycl::buffer<double> buffer_ct6 = allocation_ct6.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct6.size/sizeof(double)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    status = hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_c);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<float>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<float>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    status = hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);
    hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto allocation_ct3 = dpct::mem_mgr::instance().translate_ptr(&A_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct3 = allocation_ct3.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct3.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct5 = dpct::mem_mgr::instance().translate_ptr(&devIpiv);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct5 = allocation_ct5.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct5.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer5(cl::sycl::range<1>(1));
    // CHECK-NEXT: auto allocation_ct6 = dpct::mem_mgr::instance().translate_ptr(&workspace_z);
    // CHECK-NEXT: cl::sycl::buffer<std::complex<double>> buffer_ct6 = allocation_ct6.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct6.size/sizeof(std::complex<double>)));
    // CHECK-NEXT: auto allocation_ct8 = dpct::mem_mgr::instance().translate_ptr(&devInfo);
    // CHECK-NEXT: cl::sycl::buffer<int> buffer_ct8 = allocation_ct8.buffer.reinterpret<int>(cl::sycl::range<1>(allocation_ct8.size/sizeof(int)));
    // CHECK-NEXT: cl::sycl::buffer<int64_t> result_temp_buffer8(cl::sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrf(*cusolverH, uplo, n, buffer_ct3, lda, result_temp_buffer5, buffer_ct6, Lwork, result_temp_buffer8);
    // CHECK-NEXT: buffer_ct5.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: buffer_ct8.get_access<cl::sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<cl::sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    status = hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
    hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
}
