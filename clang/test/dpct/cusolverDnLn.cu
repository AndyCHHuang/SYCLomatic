// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    //CHECK: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnSpotrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnDpotrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnCpotrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnZpotrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = 0;
    status = hipsolverDnSpotrf_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotrf_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotrf_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotrf_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnSgetrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnDgetrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnCgetrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusolverDnZgetrf_bufferSize was replaced with 0, because this call is redundant in DPC++.
    // CHECK-NEXT: */
    // CHECK-NEXT: status = 0;
    status = hipsolverDnSgetrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnDgetrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnCgetrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnZgetrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, A_f_buf_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, A_f_buf_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, A_d_buf_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, A_d_buf_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, A_c_buf_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, A_c_buf_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrf(*cusolverH, uplo, n, A_z_buf_ct1, lda,   result_temp_buffer7), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrf(*cusolverH, uplo, n, A_z_buf_ct1, lda,   result_temp_buffer7);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_f_buf_ct1 = dpct::get_buffer<float>(&C_f);
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_f_buf_ct1, lda, B_f_buf_ct1, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_f_buf_ct1 = dpct::get_buffer<float>(&C_f);
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_f_buf_ct1, lda, B_f_buf_ct1, ldb, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);
    hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_d_buf_ct1 = dpct::get_buffer<double>(&C_d);
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_d_buf_ct1, lda, B_d_buf_ct1, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_d_buf_ct1 = dpct::get_buffer<double>(&C_d);
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_d_buf_ct1, lda, B_d_buf_ct1, ldb, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);
    hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&C_c);
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_c_buf_ct1, lda, B_c_buf_ct1, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&C_c);
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_c_buf_ct1, lda, B_c_buf_ct1, ldb, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);
    hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&C_z);
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_z_buf_ct1, lda, B_z_buf_ct1, ldb, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto C_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&C_z);
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::potrs(*cusolverH, uplo, n, nrhs, C_z_buf_ct1, lda, B_z_buf_ct1, ldb, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);
    hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, A_f_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, A_f_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);
    hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, A_d_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, A_d_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);
    hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, A_c_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, A_c_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);
    hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, A_z_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, A_z_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrf(*cusolverH, m, n, A_z_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer7(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrf(*cusolverH, m, n, A_z_buf_ct1, lda,  result_temp_buffer6, result_temp_buffer7);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer7.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_f_buf_ct1, lda, result_temp_buffer6, B_f_buf_ct1, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_f_buf_ct1, lda, result_temp_buffer6, B_f_buf_ct1, ldb, result_temp_buffer9);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);
    hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_d_buf_ct1, lda, result_temp_buffer6, B_d_buf_ct1, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_d_buf_ct1, lda, result_temp_buffer6, B_d_buf_ct1, ldb, result_temp_buffer9);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);
    hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_c_buf_ct1, lda, result_temp_buffer6, B_c_buf_ct1, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_c_buf_ct1, lda, result_temp_buffer6, B_c_buf_ct1, ldb, result_temp_buffer9);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);
    hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_z_buf_ct1, lda, result_temp_buffer6, B_z_buf_ct1, ldb, result_temp_buffer9), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::getrs(*cusolverH, trans, n, nrhs, A_z_buf_ct1, lda, result_temp_buffer6, B_z_buf_ct1, ldb, result_temp_buffer9);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);
    hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<float>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<float>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<double>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<double>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::geqrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::geqrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::geqrf(*cusolverH, m, n, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::geqrf(*cusolverH, m, n, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    status = hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ormqr_get_lwork<float>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ormqr_get_lwork<float>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, A_f_buf_ct1, lda, TAU_f_buf_ct1, B_f_buf_ct1, ldb, workspace_f_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto B_f_buf_ct1 = dpct::get_buffer<float>(&B_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, A_f_buf_ct1, lda, TAU_f_buf_ct1, B_f_buf_ct1, ldb, workspace_f_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    status = hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ormqr_get_lwork<double>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ormqr_get_lwork<double>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, A_d_buf_ct1, lda, TAU_d_buf_ct1, B_d_buf_ct1, ldb, workspace_d_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto B_d_buf_ct1 = dpct::get_buffer<double>(&B_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ormqr(*cusolverH, side, trans, m, n, k, A_d_buf_ct1, lda, TAU_d_buf_ct1, B_d_buf_ct1, ldb, workspace_d_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    status = hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);


    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::unmqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::unmqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, A_c_buf_ct1, lda, TAU_c_buf_ct1, B_c_buf_ct1, ldb, workspace_c_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto B_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&B_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, A_c_buf_ct1, lda, TAU_c_buf_ct1, B_c_buf_ct1, ldb, workspace_c_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    status = hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::unmqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::unmqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), side, trans, m, n, k,  lda,   ldc, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, A_z_buf_ct1, lda, TAU_z_buf_ct1, B_z_buf_ct1, ldb, workspace_z_buf_ct1, Lwork, result_temp_buffer13), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto B_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&B_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer13(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::unmqr(*cusolverH, side, trans, m, n, k, A_z_buf_ct1, lda, TAU_z_buf_ct1, B_z_buf_ct1, ldb, workspace_z_buf_ct1, Lwork, result_temp_buffer13);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer13.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    status = hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgqr_get_lwork<float>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgqr_get_lwork<float>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgqr(*cusolverH, m, n, k, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT: auto TAU_f_buf_ct1 = dpct::get_buffer<float>(&TAU_f);
    // CHECK-NEXT: auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgqr(*cusolverH, m, n, k, A_f_buf_ct1, lda, TAU_f_buf_ct1, workspace_f_buf_ct1, Lwork, result_temp_buffer9);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::orgqr_get_lwork<double>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::orgqr_get_lwork<double>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::orgqr(*cusolverH, m, n, k, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto TAU_d_buf_ct1 = dpct::get_buffer<double>(&TAU_d);
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::orgqr(*cusolverH, m, n, k, A_d_buf_ct1, lda, TAU_d_buf_ct1, workspace_d_buf_ct1, Lwork, result_temp_buffer9);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungqr_get_lwork<std::complex<float>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungqr(*cusolverH, m, n, k, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto TAU_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&TAU_c);
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungqr(*cusolverH, m, n, k, A_c_buf_ct1, lda, TAU_c_buf_ct1, workspace_c_buf_ct1, Lwork, result_temp_buffer9);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: status = (mkl::lapack::ungqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::lapack::ungqr_get_lwork<std::complex<double>>((*cusolverH).get_device(), m, n, k,  lda,  lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::ungqr(*cusolverH, m, n, k, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer9), 0);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto TAU_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&TAU_z);
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer9(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::ungqr(*cusolverH, m, n, k, A_z_buf_ct1, lda, TAU_z_buf_ct1, workspace_z_buf_ct1, Lwork, result_temp_buffer9);
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer9.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<float>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<float>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:{
    // CHECK-NEXT:auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT:auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT:sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT:auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT:auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT:sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT:status = (mkl::lapack::sytrf(*cusolverH, uplo, n, A_f_buf_ct1, lda, result_temp_buffer5, workspace_f_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT:devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT:devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    // CHECK-NEXT:{
    // CHECK-NEXT:auto A_f_buf_ct1 = dpct::get_buffer<float>(&A_f);
    // CHECK-NEXT:auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT:sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT:auto workspace_f_buf_ct1 = dpct::get_buffer<float>(&workspace_f);
    // CHECK-NEXT:auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT:sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT:mkl::lapack::sytrf(*cusolverH, uplo, n, A_f_buf_ct1, lda, result_temp_buffer5, workspace_f_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT:devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT:devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT:}
    status = hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    status = hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<double>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<double>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrf(*cusolverH, uplo, n, A_d_buf_ct1, lda, result_temp_buffer5, workspace_d_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_d_buf_ct1 = dpct::get_buffer<double>(&A_d);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT: auto workspace_d_buf_ct1 = dpct::get_buffer<double>(&workspace_d);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrf(*cusolverH, uplo, n, A_d_buf_ct1, lda, result_temp_buffer5, workspace_d_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    status = hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<std::complex<float>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrf(*cusolverH, uplo, n, A_c_buf_ct1, lda, result_temp_buffer5, workspace_c_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&A_c);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT: auto workspace_c_buf_ct1 = dpct::get_buffer<std::complex<float>>(&workspace_c);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrf(*cusolverH, uplo, n, A_c_buf_ct1, lda, result_temp_buffer5, workspace_c_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    status = hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);
    hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: status = (mkl::lapack::sytrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64), 0);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: int64_t lwork64 = *(&Lwork);
    // CHECK-NEXT: mkl::uplo uplo_ct_mkl_upper_lower;
    // CHECK-NEXT: mkl::lapack::sytrf_get_lwork<std::complex<double>>((*cusolverH).get_device(), uplo_ct_mkl_upper_lower, n,  lda, lwork64);
    // CHECK-NEXT: *(&Lwork) = lwork64;
    // CHECK-NEXT: }
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: status = (mkl::lapack::sytrf(*cusolverH, uplo, n, A_z_buf_ct1, lda, result_temp_buffer5, workspace_z_buf_ct1, Lwork, result_temp_buffer8), 0);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    // CHECK-NEXT: {
    // CHECK-NEXT: auto A_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&A_z);
    // CHECK-NEXT: auto devIpiv_buf_ct1 = dpct::get_buffer<int>(&devIpiv);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    // CHECK-NEXT: auto workspace_z_buf_ct1 = dpct::get_buffer<std::complex<double>>(&workspace_z);
    // CHECK-NEXT: auto devInfo_buf_ct1 = dpct::get_buffer<int>(&devInfo);
    // CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer8(sycl::range<1>(1));
    // CHECK-NEXT: mkl::lapack::sytrf(*cusolverH, uplo, n, A_z_buf_ct1, lda, result_temp_buffer5, workspace_z_buf_ct1, Lwork, result_temp_buffer8);
    // CHECK-NEXT: devIpiv_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: devInfo_buf_ct1.get_access<sycl::access::mode::write>()[0] = (int)result_temp_buffer8.get_access<sycl::access::mode::read>()[0];
    // CHECK-NEXT: }
    status = hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    status = hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
    hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
}
