
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_data_types.dp.cpp %s

void foo(int *data, int x, int y) {
  // CHECK: dpct::pitched_data p1 = dpct::pitched_data(data, x, x, y);
  hipPitchedPtr p1 = make_hipPitchedPtr(data, x, x, y);
  // CHECK: sycl::range<3> extent = sycl::range<3>(x, y, 1);
  hipExtent extent = make_hipExtent(x, y, 1);
  // CHECK: sycl::id<3> pos = sycl::id<3>(0, 0, 0);
  hipPos pos = make_hipPos(0, 0, 0);

  // CHECK: dpct::pitched_data p2_from_data_ct1, p2_to_data_ct1;
  // CHECK-NEXT: sycl::id<3> p2_from_pos_ct1(0, 0, 0), p2_to_pos_ct1(0, 0, 0);
  // CHECK-NEXT: sycl::range<3> p2_size_ct1(0, 0, 0);
  // CHECK-NEXT: dpct::memcpy_direction p2_direction_ct1;
  hipMemcpy3DParms p2;
  hipArray_t a1;

  // CHECK: p2_from_data_ct1 = a1->to_pitched_data();
  p2.srcArray = a1;
  // CHECK: p2_from_pos_ct1 = pos;
  p2.srcPos = pos;
  // CHECK: p2_to_data_ct1 = p1;
  p2.dstPtr = p1;
  // CHECK: p2_to_pos_ct1 = sycl::id<3>(0, 0, 0);
  p2.dstPos = make_hipPos(0, 0, 0);
  // CHECK: p2_size_ct1 = extent;
  p2.extent = extent;
  // CHECK: p2_direction_ct1 = dpct::device_to_host;
  p2.kind = hipMemcpyDeviceToHost;
  // CHECK: dpct::dpct_memcpy(p2_to_data_ct1, p2_to_pos_ct1, p2_from_data_ct1, p2_from_pos_ct1, p2_size_ct1, p2_direction_ct1);
  hipMemcpy3D(&p2);

  // CHECK: dpct::pitched_data p3;
  hipPitchedPtr p3;
  
  // CHECK: (dpct::dpct_malloc(&p3, sycl::range<3>(x, y, 1)), 0);
  hipMalloc3D(&p3, make_hipExtent(x, y, 1));

  // CHECK: p2_from_data_ct1 = dpct::pitched_data(data, x, x, y);
  p2.srcPtr = make_hipPitchedPtr(data, x, x, y);
  // CHECK: p2_to_data_ct1 = p3;
  p2.dstPtr = p3;
  // CHECK: p2_size_ct1[0] = x;
  p2.extent.width = x;
  // CHECK: p2_size_ct1[1] = y;
  p2.extent.height = y;
  // CHECK: p2_size_ct1[2] = 1;
  p2.extent.depth = 1;
  // CHECK: p2_direction_ct1 = dpct::host_to_device;
  p2.kind = hipMemcpyHostToDevice;
  // CHECK: dpct::dpct_memcpy(p2_to_data_ct1, p2_to_pos_ct1, p2_from_data_ct1, p2_from_pos_ct1, p2_size_ct1, p2_direction_ct1);
  hipMemcpy3D(&p2);
}