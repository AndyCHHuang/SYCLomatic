#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-math-intrinsics.dp.cpp --match-full-lines %s

#include <cmath>
#include <iomanip>
#include <iostream>
#include <limits>
#include <algorithm>

#include <stdio.h>

// CHECK: #include <algorithm>

#include "hip/hip_fp16.h"

using namespace std;

// CHECK: dpct::constant_memory<double, 0> d;
// CHECK-NEXT: dpct::constant_memory<double, 0> d2;
__constant__ double d;
__constant__ double d2;

// CHECK: double test(double d3, double d) {
// CHECK-NEXT:  return sycl::max(d, d3);
// CHECK-NEXT:}
__device__ double test(double d3) {
  return max(d, d3);
}

// CHECK:  double test2(double d, double d2) {
// CHECK-NEXT:   return sycl::max(d, d2);
// CHECK-NEXT: }
__device__ double test2() {
  return max(d, d2);
}

// CHECK:  double test3(double d4, double d5) {
// CHECK-NEXT:   return sycl::max(d4, d5);
// CHECK-NEXT: }
__device__ double test3(double d4, double d5) {
  return max(d4, d5);
}

// CHECK: dpct::constant_memory<float, 0> C;
// CHECK-NEXT:  int foo(int n, float C) {
// CHECK-NEXT:   return n == 1 ? C : 0;
// CHECK-NEXT: }
__constant__ float C;
__device__ int foo(int n) {
  return n == 1 ? C : 0;
}

__global__ void kernelFuncHalf(double *deviceArrayDouble) {
  __half h, h_1, h_2;
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half Arithmetic Functions

  // TODO:1CHECK: h2_2 = h2 / h2_1;
  //h2_2 = __h2div(h2, h2_1);
  // TODO:1CHECK: h_2 = h / h_1;
  //h_2 = __hdiv(h, h_1);
  // CHECK: h_2 = sycl::fma(h, h_1, h_2);
  h_2 = __hfma(h, h_1, h_2);
  // CHECK: h_2 = h * h_1;
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = -h;
  h_2 = __hneg(h);
  // CHECK: h_2 = h - h_1;
  h_2 = __hsub(h, h_1);

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::fma(h2, h2_1, h2_2);
  h2_2 = __hfma2(h2, h2_1, h2_2);
  // CHECK: h2_2 = h2 * h2_1;
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = -h2;
  h2_2 = __hneg2(h2);
  // CHECK: h2_2 = h2 - h2_1;
  h2_2 = __hsub2(h2, h2_1);

  // Half Comparison Functions

  // CHECK: b = h == h_1;
  b = __heq(h, h_1);
  // CHECK: b = h >= h_1;
  b = __hge(h, h_1);
  // CHECK: b = h > h_1;
  b = __hgt(h, h_1);
  // CHECK: b = sycl::isinf(h);
  b = __hisinf(h);
  // CHECK: b = sycl::isnan(h);
  b = __hisnan(h);
  // CHECK: b = h <= h_1;
  b = __hle(h, h_1);
  // CHECK: b = h < h_1;
  b = __hlt(h, h_1);
  // CHECK: b = h != h_1;
  b = __hne(h, h_1);

  // Half2 Comparison Functions

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __heq2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __hge2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __hisnan2(h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __hle2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = __hne2(h2, h2_1);

  // Half Math Functions

  // CHECK: h_2 = sycl::ceil(h);
  h_2 = hceil(h);
  // CHECK: h_2 = sycl::cos(h);
  h_2 = hcos(h);
  // CHECK: h_2 = sycl::exp(h);
  h_2 = hexp(h);
  // CHECK: h_2 = sycl::exp10(h);
  h_2 = hexp10(h);
  // CHECK: h_2 = sycl::exp2(h);
  h_2 = hexp2(h);
  // CHECK: h_2 = sycl::floor(h);
  h_2 = hfloor(h);
  // CHECK: h_2 = sycl::log(h);
  h_2 = hlog(h);
  // CHECK: h_2 = sycl::log10(h);
  h_2 = hlog10(h);
  // CHECK: h_2 = sycl::log2(h);
  h_2 = hlog2(h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h_2 = hrcp(h);
  // CHECK: h_2 = sycl::rint(h);
  h_2 = hrint(h);
  // CHECK: h_2 = sycl::rsqrt(h);
  h_2 = hrsqrt(h);
  // CHECK: h_2 = sycl::sin(h);
  h_2 = hsin(h);
  // CHECK: h_2 = sycl::sqrt(h);
  h_2 = hsqrt(h);
  // CHECK: h_2 = sycl::trunc(h);
  h_2 = htrunc(h);

  // Half2 Math Functions

  // CHECK: h2_2 = sycl::ceil(h2);
  h2_2 = h2ceil(h2);
  // CHECK: h2_2 = sycl::cos(h2);
  h2_2 = h2cos(h2);
  // CHECK: h2_2 = sycl::exp(h2);
  h2_2 = h2exp(h2);
  // CHECK: h2_2 = sycl::exp10(h2);
  h2_2 = h2exp10(h2);
  // CHECK: h2_2 = sycl::exp2(h2);
  h2_2 = h2exp2(h2);
  // CHECK: h2_2 = sycl::floor(h2);
  h2_2 = h2floor(h2);
  // CHECK: h2_2 = sycl::log(h2);
  h2_2 = h2log(h2);
  // CHECK: h2_2 = sycl::log10(h2);
  h2_2 = h2log10(h2);
  // CHECK: h2_2 = sycl::log2(h2);
  h2_2 = h2log2(h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  h2_2 = h2rcp(h2);
  // CHECK: h2_2 = sycl::rint(h2);
  h2_2 = h2rint(h2);
  // CHECK: h2_2 = sycl::rsqrt(h2);
  h2_2 = h2rsqrt(h2);
  // CHECK: h2_2 = sycl::sin(h2);
  h2_2 = h2sin(h2);
  // CHECK: h2_2 = sycl::sqrt(h2);
  h2_2 = h2sqrt(h2);
  // CHECK: h2_2 = sycl::trunc(h2);
  h2_2 = h2trunc(h2);
}

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::acos(d0);
  d2 = acos(d0);
  // CHECK: d2 = sycl::acos((double)i);
  d2 = acos(i);

  // CHECK: d2 = sycl::acosh(d0);
  d2 = acosh(d0);
  // CHECK: d2 = sycl::acosh((double)i);
  d2 = acosh(i);

  // CHECK: d2 = sycl::asin(d0);
  d2 = asin(d0);
  // CHECK: d2 = sycl::asin((double)i);
  d2 = asin(i);

  // CHECK: d2 = sycl::asinh(d0);
  d2 = asinh(d0);
  // CHECK: d2 = sycl::asinh((double)i);
  d2 = asinh(i);

  // CHECK: d2 = sycl::atan2(d0, d1);
  d2 = atan2(d0, d1);
  // CHECK: d2 = sycl::atan2((double)i, (double)i);
  d2 = atan2(i, i);
  // CHECK: d2 = sycl::atan2(d0, (double)i);
  d2 = atan2(d0, i);
  // CHECK: d2 = sycl::atan2((double)i, d1);
  d2 = atan2(i, d1);

  // CHECK: d2 = sycl::atan(d0);
  d2 = atan(d0);
  // CHECK: d2 = sycl::atan((double)i);
  d2 = atan(i);

  // CHECK: d2 = sycl::atanh(d0);
  d2 = atanh(d0);
  // CHECK: d2 = sycl::atanh((double)i);
  d2 = atanh(i);

  // CHECK: d2 = sycl::cbrt(d0);
  d2 = cbrt(d0);
  // CHECK: d2 = sycl::cbrt((double)i);
  d2 = cbrt(i);

  // CHECK: d2 = sycl::ceil(d0);
  d2 = ceil(d0);
  // CHECK: d2 = sycl::ceil((double)i);
  d2 = ceil(i);

  // CHECK: d2 = sycl::copysign(d0, d1);
  d2 = copysign(d0, d1);
  // CHECK: d2 = sycl::copysign((double)i, (double)i);
  d2 = copysign(i, i);
  // CHECK: d2 = sycl::copysign(d0, (double)i);
  d2 = copysign(d0, i);
  // CHECK: d2 = sycl::copysign((double)i, d1);
  d2 = copysign(i, d1);

  // CHECK: d2 = sycl::cos(d0);
  d2 = cos(d0);
  // CHECK: d2 = sycl::cos((double)i);
  d2 = cos(i);

  // CHECK: d2 = sycl::cosh(d0);
  d2 = cosh(d0);
  // CHECK: d2 = sycl::cosh((double)i);
  d2 = cosh(i);

  // CHECK: d2 = sycl::cospi(d0);
  d2 = cospi(d0);
  // CHECK: d2 = sycl::cospi((double)i);
  d2 = cospi((double)i);

  // CHECK: d2 = sycl::erfc(d0);
  d2 = erfc(d0);
  // CHECK: d2 = sycl::erfc((double)i);
  d2 = erfc(i);

  // CHECK: d2 = sycl::erf(d0);
  d2 = erf(d0);
  // CHECK: d2 = sycl::erf((double)i);
  d2 = erf(i);

  // CHECK: d2 = sycl::exp10(d0);
  d2 = exp10(d0);
  // CHECK: d2 = sycl::exp10((double)i);
  d2 = exp10((double)i);

  // CHECK: d2 = sycl::exp2(d0);
  d2 = exp2(d0);
  // CHECK: d2 = sycl::exp2((double)i);
  d2 = exp2(i);

  // CHECK: d2 = sycl::exp(d0);
  d2 = exp(d0);
  // CHECK: d2 = sycl::exp((double)i);
  d2 = exp(i);

  // CHECK: d2 = sycl::expm1(d0);
  d2 = expm1(d0);
  // CHECK: d2 = sycl::expm1((double)i);
  d2 = expm1(i);

  // CHECK: d2 = sycl::cos(d0);
  d2 = cos(d0);
  // CHECK: d2 = sycl::cos((double)i);
  d2 = cos(i);

  // CHECK: d2 = sycl::cosh(d0);
  d2 = cosh(d0);
  // CHECK: d2 = sycl::cosh((double)i);
  d2 = cosh(i);

  // CHECK: d2 = sycl::cospi(d0);
  d2 = cospi(d0);
  // CHECK: d2 = sycl::cospi((double)i);
  d2 = cospi((double)i);

  // CHECK: d2 = sycl::erfc(d0);
  d2 = erfc(d0);
  // CHECK: d2 = sycl::erfc((double)i);
  d2 = erfc(i);

  // CHECK: d2 = sycl::erf(d0);
  d2 = erf(d0);
  // CHECK: d2 = sycl::erf((double)i);
  d2 = erf(i);

  // CHECK: d2 = sycl::exp10(d0);
  d2 = exp10(d0);
  // CHECK: d2 = sycl::exp10((double)i);
  d2 = exp10((double)i);

  // CHECK: d2 = sycl::exp2(d0);
  d2 = exp2(d0);
  // CHECK: d2 = sycl::exp2((double)i);
  d2 = exp2(i);

  // CHECK: d2 = sycl::exp(d0);
  d2 = exp(d0);
  // CHECK: d2 = sycl::exp((double)i);
  d2 = exp(i);

  // CHECK: d2 = sycl::expm1(d0);
  d2 = expm1(d0);
  // CHECK: d2 = sycl::expm1((double)i);
  d2 = expm1(i);

  // CHECK: d2 = sycl::fabs(d0);
  d2 = fabs(d0);
  // CHECK: d2 = sycl::fabs((double)i);
  d2 = fabs(i);

  // CHECK: sycl::fabs(d0);
  abs(d0);
  // CHECK: sycl::fabs(d0 * d1);
  abs(d0 * d1);

  // CHECK: d2 = sycl::fdim(d0, d1);
  d2 = fdim(d0, d1);
  // CHECK: d2 = sycl::fdim((double)i, (double)i);
  d2 = fdim(i, i);
  // CHECK: d2 = sycl::fdim(d0, (double)i);
  d2 = fdim(d0, i);
  // CHECK: d2 = sycl::fdim((double)i, d1);
  d2 = fdim(i, d1);

  // CHECK: d2 = sycl::floor(d0);
  d2 = floor(d0);
  // CHECK: d2 = sycl::floor((double)i);
  d2 = floor(i);

  // CHECK: d2 = sycl::fma(d0, d1, d2);
  d2 = fma(d0, d1, d2);
  // CHECK: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = fma(i, i, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, (double)i);
  d2 = fma(d0, i, i);
  // CHECK: d2 = sycl::fma((double)i, d1, (double)i);
  d2 = fma(i, d1, i);
  // CHECK: d2 = sycl::fma((double)i, (double)i, d2);
  d2 = fma(i, i, d2);
  // CHECK: d2 = sycl::fma(d0, d1, (double)i);
  d2 = fma(d0, d1, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, d2);
  d2 = fma(d0, i, d2);
  // CHECK: d2 = sycl::fma((double)i, d1, d2);
  d2 = fma(i, d1, d2);

  // CHECK: d2 = sycl::fmax(d0, d1);
  d2 = fmax(d0, d1);
  // CHECK: d2 = sycl::fmax((double)i, (double)i);
  d2 = fmax(i, i);
  // CHECK: d2 = sycl::fmax(d0, (double)i);
  d2 = fmax(d0, i);
  // CHECK: d2 = sycl::fmax((double)i, d1);
  d2 = fmax(i, d1);

  // CHECK: d2 = sycl::fmin(d0, d1);
  d2 = fmin(d0, d1);
  // CHECK: d2 = sycl::fmin((double)i, (double)i);
  d2 = fmin(i, i);
  // CHECK: d2 = sycl::fmin(d0, (double)i);
  d2 = fmin(d0, i);
  // CHECK: d2 = sycl::fmin((double)i, d1);
  d2 = fmin(i, d1);

  // CHECK: d2 = sycl::fmod(d0, d1);
  d2 = fmod(d0, d1);
  // CHECK: d2 = sycl::fmod((double)i, (double)i);
  d2 = fmod(i, i);
  // CHECK: d2 = sycl::fmod(d0, (double)i);
  d2 = fmod(d0, i);
  // CHECK: d2 = sycl::fmod((double)i, d1);
  d2 = fmod(i, d1);

  // CHECK: d2 = sycl::frexp(d0, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  d2 = frexp(d0, &i);
  // CHECK: d2 = sycl::frexp((double)i, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  d2 = frexp(i, &i);

  // CHECK: d2 = sycl::hypot(d0, d1);
  d2 = hypot(d0, d1);
  // CHECK: d2 = sycl::hypot((double)i, (double)i);
  d2 = hypot(i, i);
  // CHECK: d2 = sycl::hypot(d0, (double)i);
  d2 = hypot(d0, i);
  // CHECK: d2 = sycl::hypot((double)i, d1);
  d2 = hypot(i, d1);

  // CHECK: d2 = sycl::ilogb(d0);
  d2 = ilogb(d0);
  // CHECK: d2 = sycl::ilogb((double)i);
  d2 = ilogb(i);

  // CHECK: d2 = sycl::ldexp(d0, i);
  d2 = ldexp(d0, i);
  // CHECK: d2 = sycl::ldexp((double)i, i);
  d2 = ldexp(i, i);

  // CHECK: d2 = sycl::lgamma(d0);
  d2 = lgamma(d0);
  // CHECK: d2 = sycl::lgamma((double)i);
  d2 = lgamma(i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = llrint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = llrint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = llround(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = llround(i);

  // CHECK: d2 = sycl::log10(d0);
  d2 = log10(d0);
  // CHECK: d2 = sycl::log10((double)i);
  d2 = log10(i);

  // CHECK: d2 = sycl::log1p(d0);
  d2 = log1p(d0);
  // CHECK: d2 = sycl::log1p((double)i);
  d2 = log1p(i);

  // CHECK: d2 = sycl::log2(d0);
  d2 = log2(d0);
  // CHECK: d2 = sycl::log2((double)i);
  d2 = log2(i);

  // CHECK: d2 = sycl::logb(d0);
  d2 = logb(d0);
  // CHECK: d2 = sycl::logb((double)i);
  d2 = logb(i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = lrint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = lrint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = lround(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = lround(i);

  // CHECK: d2 = sycl::modf(d0, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d1));
  d2 = modf(d0, &d1);
  // CHECK: d2 = sycl::modf((double)i, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d1));
  d2 = modf(i, &d1);

  // CHECK: d2 = sycl::nan(0u);
  d2 = nan("");

  // CHECK: d2 = sycl::pow(d0, d1);
  d2 = pow(d0, d1);
  // CHECK: d2 = sycl::pown((float)i, i);
  d2 = pow(i, i);
  // CHECK: d2 = sycl::pown(d0, i);
  d2 = pow(d0, i);
  // CHECK: d2 = sycl::pow((double)i, d1);
  d2 = pow(i, d1);

  // CHECK: sycl::pown(f, 1);
  float f;
  pow(f, 1);

  // CHECK: d2 = sycl::remainder(d0, d1);
  d2 = remainder(d0, d1);
  // CHECK: d2 = sycl::remainder((double)i, (double)i);
  d2 = remainder(i, i);
  // CHECK: d2 = sycl::remainder(d0, (double)i);
  d2 = remainder(d0, i);
  // CHECK: d2 = sycl::remainder((double)i, d1);
  d2 = remainder(i, d1);

  // CHECK: d2 = sycl::remquo(d0, d1, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  d2 = remquo(d0, d1, &i);
  // CHECK: d2 = sycl::remquo((double)i, (double)i, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  d2 = remquo(i, i, &i);
  // CHECK: d2 = sycl::remquo(d0, (double)i, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  d2 = remquo(d0, i, &i);
  // CHECK: d2 = sycl::remquo((double)i, d1, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  d2 = remquo(i, d1, &i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = rint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = rint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = round(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = round(i);

  // CHECK: d2 = sycl::rsqrt(d0);
  d2 = rsqrt(d0);
  // CHECK: d2 = sycl::rsqrt((double)i);
  d2 = rsqrt((double)i);

  // CHECK: d1 = sycl::sincos(d0, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d2));
  sincos(d0, &d1, &d2);
  // CHECK: d1 = sycl::sincos((double)i, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d2));
  sincos(i, &d1, &d2);

  // CHECK: d2 = sycl::sin(d0);
  d2 = sin(d0);
  // CHECK: d2 = sycl::sin((double)i);
  d2 = sin(i);

  // CHECK: d2 = sycl::sinh(d0);
  d2 = sinh(d0);
  // CHECK: d2 = sycl::sinh((double)i);
  d2 = sinh(i);

  // CHECK: d2 = sycl::sinpi(d0);
  d2 = sinpi(d0);
  // CHECK: d2 = sycl::sinpi((double)i);
  d2 = sinpi((double)i);

  // CHECK: d2 = sycl::sqrt(d0);
  d2 = sqrt(d0);
  // CHECK: d2 = sycl::sqrt((double)i);
  d2 = sqrt(i);

  // CHECK: d2 = sycl::tan(d0);
  d2 = tan(d0);
  // CHECK: d2 = sycl::tan((double)i);
  d2 = tan(i);

  // CHECK: d2 = sycl::tanh(d0);
  d2 = tanh(d0);
  // CHECK: d2 = sycl::tanh((double)i);
  d2 = tanh(i);

  // CHECK: d2 = sycl::tgamma(d0);
  d2 = tgamma(d0);
  // CHECK: d2 = sycl::tgamma((double)i);
  d2 = tgamma(i);

  // CHECK: d2 = sycl::trunc(d0);
  d2 = trunc(d0);
  // CHECK: d2 = sycl::trunc((double)i);
  d2 = trunc(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d1 = __drcp_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d1 = __drcp_rn(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d1 = __drcp_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d1 = __drcp_rz(d0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt(d0);
  d0 = __dsqrt_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt(d1);
  d1 = __dsqrt_rn(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt(d0);
  d0 = __dsqrt_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt(d1);
  d1 = __dsqrt_rz(d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt((double)i);
  d0 = __dsqrt_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt((double)i);
  d1 = __dsqrt_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt((double)i);
  d0 = __dsqrt_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt((double)i);
  d1 = __dsqrt_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rd(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rn(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_ru(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rz(d0, d1, d2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rd(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rn(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_ru(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rz(i, i, i);

  // CHECK: d0 = sycl::fmin(d0, d1);
  d0 = fmin(d0, d1);
  // CHECK: d0 = sycl::fmin((double)i, (double)i);
  d0 = fmin(i, i);
  // CHECK: d0 = sycl::fmin(d0, (double)i);
  d0 = fmin(d0, i);
  // CHECK: d0 = sycl::fmin((double)i, d1);
  d0 = fmin(i, d1);

  // CHECK: d0 = sycl::fmax(d0, d1);
  d0 = fmax(d0, d1);
  // CHECK: d0 = sycl::fmax((double)i, (double)i);
  d0 = fmax(i, i);
  // CHECK: d0 = sycl::fmax(d0, (double)i);
  d0 = fmax(d0, i);
  // CHECK: d0 = sycl::fmax((double)i, d1);
  d0 = fmax(i, d1);

  // CHECK: d1 = sycl::floor(d1);
  d1 = floor(d1);
  // CHECK: d1 = sycl::floor((double)i);
  d1 = floor(i);

  // CHECK: d2 = sycl::ceil(d2);
  d2 = ceil(d2);
  // CHECK: d2 = sycl::ceil((double)i);
  d2 = ceil(i);

  // CHECK: d2 = sycl::fma(d0, d1, d2);
  d2 = fma(d0, d1, d2);
  // CHECK: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = fma(i, i, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, (double)i);
  d2 = fma(d0, i, i);
  // CHECK: d2 = sycl::fma((double)i, d1, (double)i);
  d2 = fma(i, d1, i);
  // CHECK: d2 = sycl::fma((double)i, (double)i, d2);
  d2 = fma(i, i, d2);
  // CHECK: d2 = sycl::fma(d0, d1, (double)i);
  d2 = fma(d0, d1, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, d2);
  d2 = fma(d0, i, d2);
  // CHECK: d2 = sycl::fma((double)i, d1, d2);
  d2 = fma(i, d1, d2);

  // CHECK: d2 = sycl::nan(0u);
  d2 = nan("NaN");

  // CHECK: d0 = sycl::nextafter(d0, d0);
  d0 = nextafter(d0, d0);
  // CHECK: d0 = sycl::nextafter((double)i, (double)i);
  d0 = nextafter(i, i);
  // CHECK: d0 = sycl::nextafter(d0, (double)i);
  d0 = nextafter(d0, i);
  // CHECK: d0 = sycl::nextafter((double)i, d1);
  d0 = nextafter(i, d1);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::log(f0);
  f2 = logf(f0);
  // CHECK: f2 = sycl::log((float)i);
  f2 = logf(i);

  // CHECK: f2 = sycl::acos(f0);
  f2 = acosf(f0);
  // CHECK: f2 = sycl::acos((float)i);
  f2 = acosf(i);

  // CHECK: f2 = sycl::acosh(f0);
  f2 = acoshf(f0);
  // CHECK: f2 = sycl::acosh((float)i);
  f2 = acoshf(i);

  // CHECK: f2 = sycl::asin(f0);
  f2 = asinf(f0);
  // CHECK: f2 = sycl::asin((float)i);
  f2 = asinf(i);

  // CHECK: f2 = sycl::asinh(f0);
  f2 = asinhf(f0);
  // CHECK: f2 = sycl::asinh((float)i);
  f2 = asinhf(i);

  // CHECK: f2 = sycl::atan2(f0, f1);
  f2 = atan2f(f0, f1);
  // CHECK: f2 = sycl::atan2((float)i, (float)i);
  f2 = atan2f(i, i);
  // CHECK: f2 = sycl::atan2(f0, (float)i);
  f2 = atan2f(f0, i);
  // CHECK: f2 = sycl::atan2((float)i, f1);
  f2 = atan2f(i, f1);

  // CHECK: f2 = sycl::atan(f0);
  f2 = atanf(f0);
  // CHECK: f2 = sycl::atan((float)i);
  f2 = atanf(i);

  // CHECK: f2 = sycl::atanh(f0);
  f2 = atanhf(f0);
  // CHECK: f2 = sycl::atanh((float)i);
  f2 = atanhf(i);

  // CHECK: f2 = sycl::cbrt(f0);
  f2 = cbrtf(f0);
  // CHECK: f2 = sycl::cbrt((float)i);
  f2 = cbrtf(i);

  // CHECK: f2 = sycl::ceil(f0);
  f2 = ceilf(f0);
  // CHECK: f2 = sycl::ceil((float)i);
  f2 = ceilf(i);

  // CHECK: f2 = sycl::copysign(f0, f1);
  f2 = copysignf(f0, f1);
  // CHECK: f2 = sycl::copysign((float)i, (float)i);
  f2 = copysignf(i, i);
  // CHECK: f2 = sycl::copysign(f0, (float)i);
  f2 = copysignf(f0, i);
  // CHECK: f2 = sycl::copysign((float)i, f1);
  f2 = copysignf(i, f1);

  // CHECK: f2 = sycl::cos(f0);
  f2 = cosf(f0);
  // CHECK: f2 = sycl::cos((float)i);
  f2 = cosf(i);

  // CHECK: f2 = sycl::cosh(f0);
  f2 = coshf(f0);
  // CHECK: f2 = sycl::cosh((float)i);
  f2 = coshf(i);

  // CHECK: f2 = sycl::cospi(f0);
  f2 = cospif(f0);
  // CHECK: f2 = sycl::cospi((float)i);
  f2 = cospif(i);

  // CHECK: f2 = sycl::erfc(f0);
  f2 = erfcf(f0);
  // CHECK: f2 = sycl::erfc((float)i);
  f2 = erfcf(i);

  // CHECK: f2 = sycl::erf(f0);
  f2 = erff(f0);
  // CHECK: f2 = sycl::erf((float)i);
  f2 = erff(i);

  // CHECK: f2 = sycl::exp10(f0);
  f2 = exp10f(f0);
  // CHECK: f2 = sycl::exp10((float)i);
  f2 = exp10f(i);

  // CHECK: f2 = sycl::exp2(f0);
  f2 = exp2f(f0);
  // CHECK: f2 = sycl::exp2((float)i);
  f2 = exp2f(i);

  // CHECK: f2 = sycl::exp(f0);
  f2 = expf(f0);
  // CHECK: f2 = sycl::exp((float)i);
  f2 = expf(i);

  // CHECK: f2 = sycl::expm1(f0);
  f2 = expm1f(f0);
  // CHECK: f2 = sycl::expm1((float)i);
  f2 = expm1f(i);

  // CHECK: f2 = sycl::fabs(f0);
  f2 = fabsf(f0);
  // CHECK: f2 = sycl::fabs((float)i);
  f2 = fabsf(i);

  // CHECK: f2 = sycl::fdim(f0, f1);
  f2 = fdimf(f0, f1);
  // CHECK: f2 = sycl::fdim((float)i, (float)i);
  f2 = fdimf(i, i);
  // CHECK: f2 = sycl::fdim(f0, (float)i);
  f2 = fdimf(f0, i);
  // CHECK: f2 = sycl::fdim((float)i, f1);
  f2 = fdimf(i, f1);

  // CHECK: f2 = sycl::native::divide(f0, f1);
  f2 = fdividef(f0, f1);
  // CHECK: f2 = sycl::native::divide((float)i, (float)i);
  f2 = fdividef(i, i);
  // CHECK: f2 = sycl::native::divide(f0, (float)i);
  f2 = fdividef(f0, i);
  // CHECK: f2 = sycl::native::divide((float)i, f1);
  f2 = fdividef(i, f1);

  // CHECK: f2 = sycl::floor(f0);
  f2 = floorf(f0);
  // CHECK: f2 = sycl::floor((float)i);
  f2 = floorf(i);

  // CHECK: f2 = sycl::fma(f0, f1, f2);
  f2 = fmaf(f0, f1, f2);
  // CHECK: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = fmaf(i, i, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, (float)i);
  f2 = fmaf(f0, i, i);
  // CHECK: f2 = sycl::fma((float)i, f1, (float)i);
  f2 = fmaf(i, f1, i);
  // CHECK: f2 = sycl::fma((float)i, (float)i, f2);
  f2 = fmaf(i, i, f2);
  // CHECK: f2 = sycl::fma(f0, f1, (float)i);
  f2 = fmaf(f0, f1, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, f2);
  f2 = fmaf(f0, i, f2);
  // CHECK: f2 = sycl::fma((float)i, f1, f2);
  f2 = fmaf(i, f1, f2);

  // CHECK: f2 = sycl::fmax(f0, f1);
  f2 = fmaxf(f0, f1);
  // CHECK: f2 = sycl::fmax((float)i, (float)i);
  f2 = fmaxf(i, i);
  // CHECK: f2 = sycl::fmax(f0, (float)i);
  f2 = fmaxf(f0, i);
  // CHECK: f2 = sycl::fmax((float)i, f1);
  f2 = fmaxf(i, f1);

  // CHECK: f2 = sycl::fmin(f0, f1);
  f2 = fminf(f0, f1);
  // CHECK: f2 = sycl::fmin((float)i, (float)i);
  f2 = fminf(i, i);
  // CHECK: f2 = sycl::fmin(f0, (float)i);
  f2 = fminf(f0, i);
  // CHECK: f2 = sycl::fmin((float)i, f1);
  f2 = fminf(i, f1);

  // CHECK: f2 = sycl::fmod(f0, f1);
  f2 = fmodf(f0, f1);
  // CHECK: f2 = sycl::fmod((float)i, (float)i);
  f2 = fmodf(i, i);
  // CHECK: f2 = sycl::fmod(f0, (float)i);
  f2 = fmodf(f0, i);
  // CHECK: f2 = sycl::fmod((float)i, f1);
  f2 = fmodf(i, f1);

  // CHECK: f2 = sycl::frexp(f0, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  f2 = frexpf(f0, &i);
  // CHECK: f2 = sycl::frexp((float)i, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  f2 = frexpf(i, &i);

  // CHECK: f2 = sycl::hypot(f0, f1);
  f2 = hypotf(f0, f1);
  // CHECK: f2 = sycl::hypot((float)i, (float)i);
  f2 = hypotf(i, i);
  // CHECK: f2 = sycl::hypot(f0, (float)i);
  f2 = hypotf(f0, i);
  // CHECK: f2 = sycl::hypot((float)i, f1);
  f2 = hypotf(i, f1);

  // CHECK: f2 = sycl::ilogb(f0);
  f2 = ilogbf(f0);
  // CHECK: f2 = sycl::ilogb((float)i);
  f2 = ilogbf(i);

  // CHECK: i = sycl::isfinite(f0);
  i = isfinite(f0);
  // CHECK: i = sycl::isfinite((float)i);
  i = isfinite(i);

  // CHECK: i = sycl::isinf(f0);
  i = isinf(f0);
  // CHECK: i = sycl::isinf((float)i);
  i = isinf(i);

  // CHECK: i = sycl::isnan(f0);
  i = isnan(f0);
  // CHECK: i = sycl::isnan((float)i);
  i = isnan(i);

  // CHECK: f2 = sycl::ldexp(f0, i);
  f2 = ldexpf(f0, i);
  // CHECK: f2 = sycl::ldexp((float)i, i);
  f2 = ldexpf(i, i);

  // CHECK: f2 = sycl::lgamma(f0);
  f2 = lgammaf(f0);
  // CHECK: f2 = sycl::lgamma((float)i);
  f2 = lgammaf(i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = llrintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = llrintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = llroundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = llroundf(i);

  // CHECK: f2 = sycl::log10(f0);
  f2 = log10f(f0);
  // CHECK: f2 = sycl::log10((float)i);
  f2 = log10f(i);

  // CHECK: f2 = sycl::log1p(f0);
  f2 = log1pf(f0);
  // CHECK: f2 = sycl::log1p((float)i);
  f2 = log1pf(i);

  // CHECK: f2 = sycl::log2(f0);
  f2 = log2f(f0);
  // CHECK: f2 = sycl::log2((float)i);
  f2 = log2f(i);

  // CHECK: f2 = sycl::logb(f0);
  f2 = logbf(f0);
  // CHECK: f2 = sycl::logb((float)i);
  f2 = logbf(i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = lrintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = lrintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = lroundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = lroundf(i);

  // CHECK: f2 = sycl::modf(f0, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f1));
  f2 = modff(f0, &f1);
  // CHECK: f2 = sycl::modf((float)i, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f1));
  f2 = modff(i, &f1);

  // CHECK: f2 = sycl::nan(0u);
  f2 = nan("");

  // CHECK: f2 = sycl::pow(f0, f1);
  f2 = powf(f0, f1);
  // CHECK: f2 = sycl::pow((float)i, (float)i);
  f2 = powf(i, i);
  // CHECK: f2 = sycl::pow(f0, (float)i);
  f2 = powf(f0, i);
  // CHECK: f2 = sycl::pow((float)i, f1);
  f2 = powf(i, f1);

  // CHECK: f2 = sycl::remainder(f0, f1);
  f2 = remainderf(f0, f1);
  // CHECK: f2 = sycl::remainder((float)i, (float)i);
  f2 = remainderf(i, i);
  // CHECK: f2 = sycl::remainder(f0, (float)i);
  f2 = remainderf(f0, i);
  // CHECK: f2 = sycl::remainder((float)i, f1);
  f2 = remainderf(i, f1);

  // CHECK: f2 = sycl::remquo(f0, f1, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  f2 = remquof(f0, f1, &i);
  // CHECK: f2 = sycl::remquo((float)i, (float)i, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  f2 = remquof(i, i, &i);
  // CHECK: f2 = sycl::remquo(f0, (float)i, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  f2 = remquof(f0, i, &i);
  // CHECK: f2 = sycl::remquo((float)i, f1, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  f2 = remquof(i, f1, &i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = rintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = rintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = roundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = roundf(i);

  // CHECK: f2 = sycl::rsqrt(f0);
  f2 = rsqrtf(f0);
  // CHECK: f2 = sycl::rsqrt((float)i);
  f2 = rsqrtf(i);

  // CHECK: f2 = sycl::signbit(f0);
  f2 = signbit(f0);
  // CHECK: f2 = sycl::signbit((float)i);
  f2 = signbit(i);

  // CHECK: f1 = sycl::sincos(f0, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  sincosf(f0, &f1, &f2);
  // CHECK: f1 = sycl::sincos((float)i, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  sincosf(i, &f1, &f2);

  // CHECK: f2 = sycl::sin(f0);
  f2 = sinf(f0);
  // CHECK: f2 = sycl::sin((float)i);
  f2 = sinf(i);

  // CHECK: f2 = sycl::sinh(f0);
  f2 = sinhf(f0);
  // CHECK: f2 = sycl::sinh((float)i);
  f2 = sinhf(i);

  // CHECK: f2 = sycl::sinpi(f0);
  f2 = sinpif(f0);
  // CHECK: f2 = sycl::sinpi((float)i);
  f2 = sinpif(i);

  // CHECK: f2 = sycl::sqrt(f0);
  f2 = sqrtf(f0);
  // CHECK: f2 = sycl::sqrt((float)i);
  f2 = sqrtf(i);

  // CHECK: f2 = sycl::tan(f0);
  f2 = tanf(f0);
  // CHECK: f2 = sycl::tan((float)i);
  f2 = tanf(i);

  // CHECK: f2 = sycl::tanh(f0);
  f2 = tanhf(f0);
  // CHECK: f2 = sycl::tanh((float)i);
  f2 = tanhf(i);

  // CHECK: f2 = sycl::tgamma(f0);
  f2 = tgammaf(f0);
  // CHECK: f2 = sycl::tgamma((float)i);
  f2 = tgammaf(i);

  // CHECK: f2 = sycl::trunc(f0);
  f2 = truncf(f0);
  // CHECK: f2 = sycl::trunc((float)i);
  f2 = truncf(i);

  // CHECK: f0 = sycl::cos(f0);
  f0 = __cosf(f0);
  // CHECK: f0 = sycl::cos((float)i);
  f0 = __cosf(i);

  // CHECK: f0 = sycl::exp10(f0);
  f0 = __exp10f(f0);
  // CHECK: f0 = sycl::exp10((float)i);
  f0 = __exp10f(i);

  // CHECK: f0 = sycl::exp(f0);
  f0 = __expf(f0);
  // CHECK: f0 = sycl::exp((float)i);
  f0 = __expf(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rz(f0, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rz(f0, f1);

  // CHECK: f2 = sycl::native::divide(f0, f1);
  f2 = __fdividef(f0, f1);
  // CHECK: f2 = sycl::native::divide((float)i, (float)i);
  f2 = __fdividef(i, i);
  // CHECK: f2 = sycl::native::divide(f0, (float)i);
  f2 = __fdividef(f0, i);
  // CHECK: f2 = sycl::native::divide((float)i, f1);
  f2 = __fdividef(i, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rd(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rn(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_ru(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rz(f0, f1, f2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rd(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rn(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_ru(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rz(i, i, i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rz(f0, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rd(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rn(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_ru(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rz(f0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt(f0);
  f0 = __fsqrt_rd(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt(f1);
  f1 = __fsqrt_rn(f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt(f0);
  f0 = __fsqrt_ru(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt(f1);
  f1 = __fsqrt_rz(f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt((float)i);
  f0 = __fsqrt_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt((float)i);
  f1 = __fsqrt_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt((float)i);
  f0 = __fsqrt_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt((float)i);
  f1 = __fsqrt_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rz(f0, f1);

  // CHECK: f1 = sycl::log10(f1);
  f1 = __log10f(f1);
  // CHECK: f1 = sycl::log10((float)i);
  f1 = __log10f(i);

  // CHECK: f1 = sycl::log2(f1);
  f1 = __log2f(f1);
  // CHECK: f1 = sycl::log2((float)i);
  f1 = __log2f(i);

  // CHECK: f1 = sycl::log(f1);
  f1 = __logf(f1);
  // CHECK: f1 = sycl::log((float)i);
  f1 = __logf(i);

  // CHECK: f2 = sycl::pow(f0, f1);
  f2 = __powf(f0, f1);
  // CHECK: f2 = sycl::pow((float)i, (float)i);
  f2 = __powf(i, i);
  // CHECK: f2 = sycl::pow(f0, (float)i);
  f2 = __powf(f0, i);
  // CHECK: f2 = sycl::pow((float)i, f1);
  f2 = __powf(i, f1);

  // CHECK: f1 = sycl::sincos(f0, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  __sincosf(f0, &f1, &f2);
  // CHECK: f1 = sycl::sincos((float)i, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  __sincosf(i, &f1, &f2);

  // CHECK: f1 = sycl::sin(f1);
  f1 = __sinf(f1);
  // CHECK: f1 = sycl::sin((float)i);
  f1 = __sinf(i);

  // CHECK: f1 = sycl::tan(f1);
  f1 = __tanf(f1);
  // CHECK: f1 = sycl::tan((float)i);
  f1 = __tanf(i);

  // CHECK: f0 = sycl::fmin(f0, f1);
  f0 = fminf(f0, f1);
  // CHECK: f0 = sycl::fmin((float)i, (float)i);
  f0 = fminf(i, i);
  // CHECK: f0 = sycl::fmin(f0, (float)i);
  f0 = fminf(f0, i);
  // CHECK: f0 = sycl::fmin((float)i, f1);
  f0 = fminf(i, f1);

  // CHECK: f2 = sycl::fmax(f0, f1);
  f2 = fmaxf(f0, f1);
  // CHECK: f2 = sycl::fmax((float)i, (float)i);
  f2 = fmaxf(i, i);
  // CHECK: f2 = sycl::fmax(f0, (float)i);
  f2 = fmaxf(f0, i);
  // CHECK: f2 = sycl::fmax((float)i, f1);
  f2 = fmaxf(i, f1);

  // CHECK: f1 = sycl::floor(f1);
  f1 = floorf(f1);
  // CHECK: f1 = sycl::floor((float)i);
  f1 = floorf(i);

  // CHECK: f2 = sycl::ceil(f2);
  f2 = ceilf(f2);
  // CHECK: f2 = sycl::ceil((float)i);
  f2 = ceilf(i);

  // CHECK: f2 = sycl::fma(f0, f1, f2);
  f2 = fmaf(f0, f1, f2);
  // CHECK: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = fmaf(i, i, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, (float)i);
  f2 = fmaf(f0, i, i);
  // CHECK: f2 = sycl::fma((float)i, f1, (float)i);
  f2 = fmaf(i, f1, i);
  // CHECK: f2 = sycl::fma((float)i, (float)i, f2);
  f2 = fmaf(i, i, f2);
  // CHECK: f2 = sycl::fma(f0, f1, (float)i);
  f2 = fmaf(f0, f1, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, f2);
  f2 = fmaf(f0, i, f2);
  // CHECK: f2 = sycl::fma((float)i, f1, f2);
  f2 = fmaf(i, f1, f2);

  // CHECK: f2 = sycl::nan(0u);
  f2 = nanf("NaN");

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::rsqrt(f2);
  f2 = __frsqrt_rn(f2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::rsqrt((float)i);
  f2 = __frsqrt_rn(i);

  // CHECK: f0 = sycl::nextafter(f0, f0);
  f0 = nextafterf(f0, f0);
  // CHECK: f0 = sycl::nextafter((float)i, (float)i);
  f0 = nextafterf(i, i);
  // CHECK: f0 = sycl::nextafter(f0, (float)i);
  f0 = nextafterf(f0, i);
  // CHECK: f0 = sycl::nextafter((float)i, f1);
  f0 = nextafterf(i, f1);
}

__global__ void kernelFuncTypecasts() {
  short s, s_1;
  unsigned short us;
  int i, i_1;
  unsigned int ui, ui_1;
  long l;
  unsigned long ul;
  long long ll;
  unsigned long long ull;

  __half h;
  __half2 h2;
  float f;
  float2 f2;
  double d;
  double2 d2;

  // CHECK: h2 = f2.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __float22half2_rn(f2);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::automatic>().get_value(0);
  h = __float2half(f);

  // CHECK: h2 = sycl::float2{f,f}.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __float2half2_rn(f);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __float2half_rd(f);

  // sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  __float2half_rn(f);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __float2half_ru(f);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __float2half_rz(f);

  // CHECK: h2 = sycl::float2{f,f}.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __floats2half2_rn(f, f);

  // CHECK: f2 = h2.convert<float, sycl::rounding_mode::automatic>();
  f2 = __half22float2(h2);

  // CHECK: f = sycl::vec<sycl::half, 1>{h}.convert<float, sycl::rounding_mode::automatic>().get_value(0);
  f = __half2float(h);

  // CHECK: h2 = sycl::half2{h,h};
  h2 = __half2half2(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rtn>().get_value(0);
  i = __half2int_rd(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rte>().get_value(0);
  i = __half2int_rn(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rtp>().get_value(0);
  i = __half2int_ru(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rtz>().get_value(0);
  i = __half2int_rz(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rtn>().get_value(0);
  ll = __half2ll_rd(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rte>().get_value(0);
  ll = __half2ll_rn(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rtp>().get_value(0);
  ll = __half2ll_ru(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rtz>().get_value(0);
  ll = __half2ll_rz(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rtn>().get_value(0);
  s = __half2short_rd(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rte>().get_value(0);
  s = __half2short_rn(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rtp>().get_value(0);
  s = __half2short_ru(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rtz>().get_value(0);
  s = __half2short_rz(h);

  // CHECK: ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rtn>().get_value(0);
  ui = __half2uint_rd(h);

  // CHECK: ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rte>().get_value(0);
  ui = __half2uint_rn(h);

  // CHECK:ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rtp>().get_value(0);
  ui = __half2uint_ru(h);

  // CHECK: ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rtz>().get_value(0);
  ui = __half2uint_rz(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rtn>().get_value(0);
  ull = __half2ull_rd(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rte>().get_value(0);
  ull = __half2ull_rn(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rtp>().get_value(0);
  ull = __half2ull_ru(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rtz>().get_value(0);
  ull = __half2ull_rz(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rtn>().get_value(0);
  us = __half2ushort_rd(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rte>().get_value(0);
  us = __half2ushort_rn(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rtp>().get_value(0);
  us = __half2ushort_ru(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rtz>().get_value(0);
  us = __half2ushort_rz(h);

  // CHECK: s = dpct::bit_cast<sycl::half, short>(h);
  s = __half_as_short(h);

  // CHECK: us = dpct::bit_cast<sycl::half, unsigned short>(h);
  us = __half_as_ushort(h);

  // CHECK: h2 = sycl::half2{h,h};
  h2 = __halves2half2(h, h);

  // CHECK: f = h2.get_value(0);
  f = __high2float(h2);

  // CHECK: h = h2.get_value(0);
  h = __high2half(h2);

  // CHECK: h2 = sycl::half2{h2.get_value(0), h2.get_value(0)};
  h2 = __high2half2(h2);

  // CHECK: h2 = sycl::half2{h2.get_value(0), h2.get_value(0)};
  h2 = __highs2half2(h2, h2);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __int2half_rd(i);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  h = __int2half_rn(i);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __int2half_ru(i);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __int2half_rz(i);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __ll2half_rd(ll);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  h = __ll2half_rn(ll);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __ll2half_ru(ll);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __ll2half_rz(ll);

  // CHECK: f = h2.get_value(1);
  f = __low2float(h2);

  // CHECK: h = h2.get_value(1);
  h = __low2half(h2);

  // CHECK: h2 = sycl::half2{h2.get_value(1), h2.get_value(1)};
  h2 = __low2half2(h2);

  // CHECK: h2 = sycl::half2{h2.get_value(1), h2.get_value(0)};
  h2 = __lowhigh2highlow(h2);

  // CHECK: h2 = sycl::half2{h2.get_value(1), h2.get_value(1)};
  h2 = __lows2half2(h2, h2);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __short2half_rd(s);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  h = __short2half_rn(s);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __short2half_ru(s);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __short2half_rz(s);

  // CHECK: h = dpct::bit_cast<short, sycl::half>(s);
  h = __short_as_half(s);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __uint2half_rd(ui);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  h = __uint2half_rn(ui);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __uint2half_ru(ui);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __uint2half_rz(ui);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __ull2half_rd(ull);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  h = __ull2half_rn(ull);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __ull2half_ru(ull);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __ull2half_rz(ull);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rtn>().get_value(0);
  h = __ushort2half_rd(us);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rte>().get_value(0);
  h = __ushort2half_rn(us);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rtp>().get_value(0);
  h = __ushort2half_ru(us);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rtz>().get_value(0);
  h = __ushort2half_rz(us);

  // CHECK: h = dpct::bit_cast<unsigned short, sycl::half>(us);
  h = __ushort_as_half(us);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtn>().get_value(0);
  f = __double2float_rd(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rte>().get_value(0);
  f = __double2float_rn(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtp>().get_value(0);
  f = __double2float_ru(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtz>().get_value(0);
  f = __double2float_rz(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtn>().get_value(0);
  i = __double2int_rd(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rte>().get_value(0);
  i = __double2int_rn(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtp>().get_value(0);
  i = __double2int_ru(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtz>().get_value(0);
  i = __double2int_rz(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtn>().get_value(0);
  ll = __double2ll_rd(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rte>().get_value(0);
  ll = __double2ll_rn(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtp>().get_value(0);
  ll = __double2ll_ru(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtz>().get_value(0);
  ll = __double2ll_rz(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtn>().get_value(0);
  ui = __double2uint_rd(d);

  // CHECK:ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rte>().get_value(0);
  ui = __double2uint_rn(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtp>().get_value(0);
  ui = __double2uint_ru(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtz>().get_value(0);
  ui = __double2uint_rz(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtn>().get_value(0);
  ull = __double2ull_rd(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rte>().get_value(0);
  ull = __double2ull_rn(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtp>().get_value(0);
  ull = __double2ull_ru(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtz>().get_value(0);
  ull = __double2ull_rz(d);

  // CHECK: ll = dpct::bit_cast<double, long long>(d);
  ll = __double_as_longlong(d);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtn>().get_value(0);
  i = __float2int_rd(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rte>().get_value(0);
  i = __float2int_rn(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtp>().get_value(0);
  i = __float2int_ru(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtz>().get_value(0);
  i = __float2int_rz(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtn>().get_value(0);
  ll = __float2ll_rd(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rte>().get_value(0);
  ll = __float2ll_rn(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtp>().get_value(0);
  ll = __float2ll_ru(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtz>().get_value(0);
  ll = __float2ll_rz(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtn>().get_value(0);
  ui = __float2uint_rd(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rte>().get_value(0);
  ui = __float2uint_rn(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtp>().get_value(0);
  ui = __float2uint_ru(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtz>().get_value(0);
  ui = __float2uint_rz(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtn>().get_value(0);
  ull = __float2ull_rd(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rte>().get_value(0);
  ull = __float2ull_rn(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtp>().get_value(0);
  ull = __float2ull_ru(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtz>().get_value(0);
  ull = __float2ull_rz(f);

  // CHECK: i = dpct::bit_cast<float, int>(f);
  i = __float_as_int(f);

  // CHECK: ui = dpct::bit_cast<float, unsigned int>(f);
  ui = __float_as_uint(f);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<double, sycl::rounding_mode::rte>().get_value(0);
  d = __int2double_rn(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtn>().get_value(0);
  d = __int2float_rd(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rte>().get_value(0);
  d = __int2float_rn(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtp>().get_value(0);
  d = __int2float_ru(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtz>().get_value(0);
  d = __int2float_rz(i);

  // CHECK: f = dpct::bit_cast<int, float>(i);
  f = __int_as_float(i);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtn>().get_value(0);
  d = __ll2double_rd(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rte>().get_value(0);
  d = __ll2double_rn(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtp>().get_value(0);
  d = __ll2double_ru(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtz>().get_value(0);
  d = __ll2double_rz(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtn>().get_value(0);
  f = __ll2float_rd(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rte>().get_value(0);
  f = __ll2float_rn(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtp>().get_value(0);
  f = __ll2float_ru(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtz>().get_value(0);
  f = __ll2float_rz(ll);

  // CHECK: d = dpct::bit_cast<long long, double>(ll);
  d = __longlong_as_double(ll);

  // CHECK: d = sycl::vec<unsigned int, 1>{ui}.convert<double, sycl::rounding_mode::rte>().get_value(0);
  d = __uint2double_rn(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtn>().get_value(0);
  f = __uint2float_rd(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rte>().get_value(0);
  f = __uint2float_rn(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtp>().get_value(0);
  f = __uint2float_ru(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtz>().get_value(0);
  f = __uint2float_rz(ui);

  // CHECK: f = dpct::bit_cast<unsigned int, float>(ui);
  f = __uint_as_float(ui);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtn>().get_value(0);
  d = __ull2double_rd(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rte>().get_value(0);
  d = __ull2double_rn(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtp>().get_value(0);
  d = __ull2double_ru(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtz>().get_value(0);
  d = __ull2double_rz(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtn>().get_value(0);
  f = __ull2float_rd(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rte>().get_value(0);
  f = __ull2float_rn(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtp>().get_value(0);
  f = __ull2float_ru(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtz>().get_value(0);
  f = __ull2float_rz(ull);
}

void testDouble() {
  const unsigned int NUM = 3;
  const unsigned int bytes = NUM * sizeof(double);

  double *hostArrayDouble = (double *)malloc(bytes);
  memset(hostArrayDouble, 0, bytes);
  const long double pi = std::acos(-1.L);
  *hostArrayDouble = pi;
  *(hostArrayDouble + 1) = pi - 1;

  double *deviceArrayDouble;
  hipMalloc((double **)&deviceArrayDouble, bytes);

  hipMemcpy(deviceArrayDouble, hostArrayDouble, bytes, hipMemcpyHostToDevice);

  kernelFuncDouble<<<1, 1>>>(deviceArrayDouble);

  hipMemcpy(hostArrayDouble, deviceArrayDouble, bytes, hipMemcpyDeviceToHost);

  hipFree(deviceArrayDouble);

  cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1)
       << *(hostArrayDouble + 2) << endl;
}

void testFloat() {
  const unsigned int NUM = 3;
  const unsigned int bytes = NUM * sizeof(float);

  float *hostArrayFloat = (float *)malloc(bytes);
  memset(hostArrayFloat, 0, bytes);
  const long double pi = std::acos(-1.L);
  *hostArrayFloat = pi;
  *(hostArrayFloat + 1) = pi - 1;

  float *deviceArrayFloat;
  hipMalloc((float **)&deviceArrayFloat, bytes);

  hipMemcpy(deviceArrayFloat, hostArrayFloat, bytes, hipMemcpyHostToDevice);

  kernelFuncFloat<<<1, 1>>>(deviceArrayFloat);

  hipMemcpy(hostArrayFloat, deviceArrayFloat, bytes, hipMemcpyDeviceToHost);

  hipFree(deviceArrayFloat);

  cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1)
       << *(hostArrayFloat + 2) << endl;
}

__global__ void testUnsupported() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;
  half h;
  float f;
  double d;
  half2 h2;
  bool b;

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hadd_sat(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hfma_sat(h, h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hmul_sat(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hsub_sat(h, h);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hadd2_sat(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hfma2_sat(h2, h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hmul2_sat(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hsub2_sat(h2, h2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hequ(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hgeu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hgtu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hleu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hltu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hneu(h, h);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbeq2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbequ2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbge2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbgeu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbgt2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbgtu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hble2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbleu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hblt2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbltu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbne2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  b = __hbneu2(h2, h2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hequ2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hgeu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hgtu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hleu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hltu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  __hneu2(h2, h2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = cyl_bessel_i0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = cyl_bessel_i1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = erfcinvf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = erfcxf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = erfinvf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = j0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = j1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = jnf(i, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = norm3df(f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = norm4df(f, f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = normcdff(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = normcdfinvf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = normf(i, &f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = rcbrtf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = rnorm3df(f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = rnorm4df(f, f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = rnormf(i, &f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = scalblnf(f, l);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = scalbnf(f, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = y0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = y1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = ynf(i, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = cyl_bessel_i0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = cyl_bessel_i1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = erfcinv(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = erfcx(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = erfinv(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = j0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = j1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = jn(i, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = norm(i, &d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = norm3d(d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = norm4d(d, d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = normcdf(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = normcdfinv(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = rcbrt(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = rnorm3d(d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = rnorm4d(d, d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = rnorm(i, &d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = scalbln(d, l);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = scalbn(d, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = y0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = y1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = yn(i, d);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  f = __saturatef(f);

  // i = __shfl_down_sync(u, h, u, i);
  // i = __shfl_sync(u, h, u, i);
  // i = __shfl_up_sync(u, h, u, i);
  // i = __shfl_xor_sync(u, h, u, i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  i = __double2hiint(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  i = __double2loint(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  d = __hiloint2double(i, i);


  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __brev(u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  ull = __brevll(ull);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __byte_perm(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  i = __ffs(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  i = __ffsll(ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __funnelshift_l(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __funnelshift_lc(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __funnelshift_r(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __funnelshift_rc(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  ll = __mul64hi(ll, ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  i = __rhadd(i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __sad(i, i, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __uhadd(u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __umul24(u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  ull = __umul64hi(ull, ull);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __umulhi(u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __urhadd(u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1004:{{[0-9]+}}: Could not generate replacement.
  // CHECK-NEXT: */
  u = __usad(u, u, u);
}

__global__ void testSimulation() {
  float f;
  double d;

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::floor(f + 0.5);
  f = nearbyintf(f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = sycl::floor(d + 0.5);
  d = nearbyint(d);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::hypot call is used instead of the rhypotf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = 1 / sycl::hypot(f, f);
  f = rhypotf(f, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincospif call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::sincos(f * DPCT_PI_F, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f));
  sincospif(f, &f, &f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincospi call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = sycl::sincos(d * DPCT_PI, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d));
  sincospi(d, &d, &d);
}

__global__ void testIntegerFunctions() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;

  // CHECK: i = sycl::clz(i);
  // CHECK-NEXT: i = sycl::clz(ll);
  // CHECK-NEXT: i = sycl::hadd(i, i);
  // CHECK-NEXT: i = sycl::mul24(i, i);
  // CHECK-NEXT: i = sycl::mul_hi(i, i);
  // CHECK-NEXT: i = sycl::popcount(u);
  // CHECK-NEXT: i = sycl::popcount(ull);
  i = __clz(i);
  i = __clzll(ll);
  i = __hadd(i, i);
  i = __mul24(i, i);
  i = __mulhi(i, i);
  i = __popc(u);
  i = __popcll(ull);

  // CHECK: sycl::clz((int)u);
  // CHECK-NEXT: sycl::clz((long long)ull);
  // CHECK-NEXT: sycl::hadd((int)u, (int)u);
  // CHECK-NEXT: sycl::mul24((int)u, (int)u);
  // CHECK-NEXT: sycl::mul_hi((int)u, (int)u);
  __clz(u);
  __clzll(ull);
  __hadd(u, u);
  __mul24(u, u);
  __mulhi(u, u);
}

void testTypecasts() {

}

__global__ void testConditionalOperator(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1),
        &f2 = *(deviceArrayFloat + 2);
  // CHECK: f0 = sycl::fmax(f0 = (f1) > (f1 == 1 ? 0 : -f2) ? sycl::native::divide(sycl::pow(f1, 2.f), f1) : -f1, f1 + f1 < f2
  // CHECK-NEXT:         ? ((f1) > (f1 == 1 ? 0 : -f2) ? sycl::native::divide(sycl::pow(f2, 2.f), f1) : -f1)
  // CHECK-NEXT:         : -f1);
  // CHECK-NEXT: f0 = f1 > f2 ? sycl::native::divide(sycl::pow(f1, 2.f), f1) : f1;
  // CHECK-NEXT: f0 = sycl::fmax(0 ? sycl::native::divide(sycl::pow(f1, 2.f), f1) : f1, f2);
  f0 = fmaxf(
      f0 = (f1) > (f1 == 1 ? 0 : -f2) ? __fdividef(__powf(f1, 2.f), f1) : -f1,
      f1 + f1 < f2
          ? ((f1) > (f1 == 1 ? 0 : -f2) ? __fdividef(__powf(f2, 2.f), f1) : -f1)
          : -f1);
  f0 = f1 > f2 ? __fdividef(__powf(f1, 2.f), f1) : f1;
  f0 = fmaxf(0 ? __fdividef(__powf(f1, 2.f), f1) : f1, f2);
}

int main() {
  testDouble();
  testFloat();
  testTypecasts();
}

// Host max/min functions with integer parameters are in <algorithm> instead of <cmath>, so we need to
// migrate them to std versions and do necessary casts.
// The following migration is to make the resulted code compilable by dpcpp
// rule1: in pure __host__ functions, math functions are migrated to std alternatives
// rule2: math functions in __device__ or __global__ functions are always migrated to sycl alternatives
// rule3: functions in std namespace always remain untouched

// CHECK:  int foo(int i, int j) {
// CHECK-NEXT:   return std::max(i, j) + std::min(i, j);
// CHECK-NEXT: }
__host__ int foo(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float foo(float f, float g) {
// CHECK-NEXT:   return fmaxf(f, g) + fminf(f, g);
// CHECK-NEXT: }
__host__ float foo(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int foo2(int i, int j) {
// CHECK-NEXT:   return sycl::max(i, j) + sycl::min(i, j);
// CHECK-NEXT: }
__device__ int foo2(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float foo2(float f, float g) {
// CHECK-NEXT:   return sycl::max(f, g) + sycl::min(f, g);
// CHECK-NEXT: }
__device__ float foo2(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int  foo3(int i, int j) {
// CHECK-NEXT:   return sycl::max(i, j) + sycl::min(i, j);
// CHECK-NEXT: }
__device__ int __host__ foo3(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float  foo3(float f, float g) {
// CHECK-NEXT:   return sycl::max(f, g) + sycl::min(f, g);
// CHECK-NEXT: }
__device__ float __host__ foo3(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int bar(short i, long j) {
// CHECK-NEXT:   return std::max<long>(i, j) + std::min<long>(i, j);
// CHECK-NEXT: }
__host__ int bar(short i, long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(unsigned short i, unsigned long j) {
// CHECK-NEXT:   return std::max<unsigned long>(i, j) + std::min<unsigned long>(i, j);
// CHECK-NEXT: }
__host__ int bar(unsigned short i, unsigned long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(unsigned short i, long j) {
// CHECK-NEXT:   return max(i, j) + min(i, j);
// CHECK-NEXT: }
__host__ int bar(unsigned short i, long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(long i, unsigned short j) {
// CHECK-NEXT:   return max(i, j) + min(i, j);
// CHECK-NEXT: }
__host__ int bar(long i, unsigned short j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(short i, unsigned long j) {
// CHECK-NEXT:   return std::max<unsigned long>(i, j) + std::min<unsigned long>(i, j);
// CHECK-NEXT: }
__host__ int bar(short i, unsigned long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(unsigned long i, short j) {
// CHECK-NEXT:   return std::max<unsigned long>(i, j) + std::min<unsigned long>(i, j);
// CHECK-NEXT: }
__host__ int bar(unsigned long i, short j) {
  return max(i, j) + min(i, j);
}

typedef int INT;
typedef unsigned UINT;
using int_t = int;
using uint_t = unsigned;

// CHECK: int foo(UINT i, INT j) {
// CHECK-NEXT:   return std::max<UINT>(i, j) + std::min<UINT>(i, j);
// CHECK-NEXT: }
int foo(UINT i, INT j) {
  return max(i, j) + min(i, j);
}

// CHECK: int foo(INT i, UINT j) {
// CHECK-NEXT:   return std::max<UINT>(i, j) + std::min<UINT>(i, j);
// CHECK-NEXT: }
int foo(INT i, UINT j) {
  return max(i, j) + min(i, j);
}

// CHECK: int bar(uint_t i, int_t j) {
// CHECK-NEXT:   return std::max<uint_t>(i, j) + std::min<uint_t>(i, j);
// CHECK-NEXT: }
int bar(uint_t i, int_t j) {
  return max(i, j) + min(i, j);
}

// CHECK: int bar(int_t i, uint_t j) {
// CHECK-NEXT:   return std::max<uint_t>(i, j) + std::min<uint_t>(i, j);
// CHECK-NEXT: }
int bar(int_t i, uint_t j) {
  return max(i, j) + min(i, j);
}

__device__ void test_pow() {
  int i;
  float f;
  double d;

  // CHECK: sycl::pown((float)i, i);
  pow(i, i);
  // CHECK: sycl::pown(f, i);
  pow(f, i);
  // CHECK: sycl::pown(d, i);
  pow(d, i);

  // CHECK: sycl::pow((float)i, f);
  pow(i, f);
  // CHECK: sycl::pow(f, f);
  pow(f, f);
  // CHECK: sycl::pow(d, (double)f);
  pow(d, f);

  // CHECK: sycl::pow((double)i, d);
  pow(i, d);
  // CHECK: sycl::pow((double)f, d);
  pow(f, d);
  // CHECK: sycl::pow(d, d);
  pow(d, d);
}

__global__ void foobar(int i) {
  // CHECK: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_id(2));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_id(1));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_id(0));
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_id(2), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_id(1), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_id(0), (unsigned int)i);
  max(i, threadIdx.x);
  max(i, threadIdx.y);
  max(i, threadIdx.z);
  max(threadIdx.x, i);
  max(threadIdx.y, i);
  max(threadIdx.z, i);

  // CHECK: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_group(2));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_group(1));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_group(0));
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_group(2), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_group(1), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_group(0), (unsigned int)i);
  max(i, blockIdx.x);
  max(i, blockIdx.y);
  max(i, blockIdx.z);
  max(blockIdx.x, i);
  max(blockIdx.y, i);
  max(blockIdx.z, i);

  // CHECK: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_range(2));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_range(1));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_range(0));
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_range(2), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_range(1), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_range(0), (unsigned int)i);
  max(i, blockDim.x);
  max(i, blockDim.y);
  max(i, blockDim.z);
  max(blockDim.x, i);
  max(blockDim.y, i);
  max(blockDim.z, i);

  // CHECK: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_id(2));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_id(1));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_id(0));
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_id(2), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_id(1), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_id(0), (unsigned int)i);
  min(i, threadIdx.x);
  min(i, threadIdx.y);
  min(i, threadIdx.z);
  min(threadIdx.x, i);
  min(threadIdx.y, i);
  min(threadIdx.z, i);

  // CHECK: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_group(2));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_group(1));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_group(0));
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_group(2), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_group(1), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_group(0), (unsigned int)i);
  min(i, blockIdx.x);
  min(i, blockIdx.y);
  min(i, blockIdx.z);
  min(blockIdx.x, i);
  min(blockIdx.y, i);
  min(blockIdx.z, i);

  // CHECK: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_range(2));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_range(1));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_range(0));
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_range(2), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_range(1), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_range(0), (unsigned int)i);
  min(i, blockDim.x);
  min(i, blockDim.y);
  min(i, blockDim.z);
  min(blockDim.x, i);
  min(blockDim.y, i);
  min(blockDim.z, i);
}

void do_migration() {
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}
__global__ void do_migration2() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
__device__ void do_migration3() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
__host__ __device__ void do_migration4() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
int max(int i, int j) {
  return i > j ? i : j;
}
namespace t {
int max(int i, int j) {
  return i > j ? i : j;
}
}
void no_migration() {
  int i, j;
  // CHECK: max(i, j);
  max(i, j);
}
void no_migration2() {
  int i, j;
  // CHECK: t::max(i, j);
  t::max(i, j);
}
void no_migration3() {
  int i, j;
  // CHECK: std::max(i, j);
  std::max(i, j);
}
__host__ void no_migration4() {
  int i, j;
  // CHECK: max(i, j);
  max(i, j);
}

void ns() {
  using namespace std;
  int i, j;
  // CHECK: max(i, j);
  max(i, j);
}

void no_migration5() {
  float f;
  int i;

  //CHECK: std::max(i, i);
  //CHECK-NEXT: std::min(i, i);
  //CHECK-NEXT: std::fabs(f);
  //CHECK-NEXT: std::frexpf(f, &i);
  //CHECK-NEXT: std::modff(f, &f);
  //CHECK-NEXT: std::nearbyintf(f);
  //CHECK-NEXT: std::remquof(f, f, &i);
  //CHECK-NEXT: std::acosf(f);
  //CHECK-NEXT: std::acoshf(f);
  //CHECK-NEXT: std::asinf(f);
  //CHECK-NEXT: std::asinhf(f);
  //CHECK-NEXT: std::abs(f);
  //CHECK-NEXT: std::frexp(f, &i);
  //CHECK-NEXT: std::modf(f, &f);
  //CHECK-NEXT: std::nearbyint(f);
  //CHECK-NEXT: std::remquo(f, f, &i);
  //CHECK-NEXT: std::acos(f);
  //CHECK-NEXT: std::acosh(f);
  //CHECK-NEXT: std::asin(f);
  //CHECK-NEXT: std::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__device__ void do_migration5() {
  float f;
  int i;

  //CHECK: sycl::max(i, i);
  //CHECK-NEXT: sycl::min(i, i);
  //CHECK-NEXT:  sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexpf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modff call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<double, sycl::access::address_space::global_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__global__ void do_migration6() {
  float f;
  int i;

  //CHECK: sycl::max(i, i);
  //CHECK-NEXT: sycl::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexpf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modff call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<double, sycl::access::address_space::global_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__device__ __host__ void do_migration7() {
  float f;
  int i;

  //CHECK: sycl::max(i, i);
  //CHECK-NEXT: sycl::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexpf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modff call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<double, sycl::access::address_space::global_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::global_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh((double)f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh((double)f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

// crt wrapper API
__device__ void foo1() {
  int i;
  float f;
  unsigned int ui;
  long long ll;
  unsigned long long ull;

  // CHECK: i = sycl::mul24(i, i);
  i = mul24(i, i);
}

__device__ void test_recursive_unary() {
  int i, j, k;
  // CHECK: sycl::max(-sycl::max(-sycl::abs(i), j), k);
  max(-max(-abs(i), j), k);
}

__device__ void do_math(int i, int j) {
  // CHECK: sycl::sqrt((float)i);
  sqrtf(i);
  // CHECK: sycl::sqrt((double)i);
  sqrt(i);
  // CHECK: sycl::fmod((double)i, (double)j);
  fmod(i, j);
  // CHECK: sycl::sin((double)i);
  sin(i);
  // CHECK: sycl::cos((double)i);
  cos(i);
}

__device__ void do_math(float i, float j) {
  // CHECK: sycl::sqrt(i);
  sqrtf(i);
  // CHECK: sycl::sqrt(i);
  sqrt(i);
  // CHECK: sycl::fmod(i, j);
  fmod(i, j);
  // CHECK: sycl::sin(i);
  sin(i);
  // CHECK: sycl::cos(i);
  cos(i);
}

__device__ void do_math(double i, double j) {
  // CHECK: sycl::sqrt((float)i);
  sqrtf(i);
  // CHECK: sycl::sqrt(i);
  sqrt(i);
  // CHECK: sycl::fmod(i, j);
  fmod(i, j);
  // CHECK: sycl::sin(i);
  sin(i);
  // CHECK: sycl::cos(i);
  cos(i);
}
