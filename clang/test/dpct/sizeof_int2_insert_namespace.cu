
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T %s -- -x cuda --cuda-host-only --cuda-path="%cuda-path"
// RUN: FileCheck --input-file %T/sizeof_int2_insert_namespace.dp.cpp --match-full-lines %s

void fun() {
  // CHECK:  cl::sycl::int2 a, b, c, d[2], *e[2];
  int2 a, b, c, d[2], *e[2];
  // CHECK:  int i = sizeof(cl::sycl::int2);
  int i = sizeof(int2);
  // CHECK:  int j = sizeof(int);
  int j = sizeof(int);
  // CHECK:  cl::sycl::int2 k;
  int2 k;
  // CHECK:  int kk = sizeof(k);
  int kk = sizeof(k);
}
