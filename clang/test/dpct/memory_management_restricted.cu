// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T/memory_management_restricted %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++11
// RUN: FileCheck --match-full-lines --input-file %T/memory_management_restricted/memory_management_restricted.dp.cpp %s

#include <hip/hip_runtime.h>

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)
template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {}

#define DATAMACRO 32*32

//CHECK: template<typename ntype>
//CHECK-NEXT: void test(){
//CHECK-NEXT:   dpct::device_ext &dev_ct1 = dpct::get_current_device();
//CHECK-NEXT:   sycl::queue &q_ct1 = dev_ct1.default_queue();
//CHECK-NEXT:   int i = 0;
//CHECK-NEXT:   ntype** ptr;
//CHECK-NEXT:   ntype* array[10];
//CHECK-NEXT:   ptr[i] = (ntype *)sycl::malloc_device(10 * sizeof(ntype), q_ct1);
//CHECK-NEXT:   array[i] = (ntype *)sycl::malloc_device(10 * sizeof(ntype), q_ct1);
//CHECK-NEXT: }
template<typename ntype>
void test(){
  int i = 0;
  ntype** ptr;
  ntype* array[10];
  hipMalloc(&ptr[i], 10 * sizeof(ntype));
  hipMalloc(&array[i], 10 * sizeof(ntype));
}

int main(){
    //CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    //CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();

    float **data = NULL;
    float *d_A = NULL;
    float *h_A = NULL;
    int* a;
    hipStream_t stream;
    hipStream_t stream_array[10];
    int deviceID = 0;
    hipError_t err;

    //CHECK:  /*
    //CHECK-NEXT:  DPCT1003:0: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:  */
    //CHECK-NEXT:  MY_ERROR_CHECKER((*data = (float *)sycl::malloc_device(DATAMACRO, q_ct1), 0));
    MY_ERROR_CHECKER(hipMalloc((void **)data, DATAMACRO));

    //Currently, migration of using template version API only covers the simple case: the argument specifiy the size is sizeof(T)*Expr, Expr*sizeof(T) and sizeof(T)
    //CHECK:  *data = sycl::malloc_device<float>(10*10, q_ct1);
    hipMalloc(data, 10*10*sizeof(float));

    //CHECK:  *data = (float *)sycl::malloc_device(10*sizeof(float)*10, q_ct1);
    hipMalloc(data, 10*sizeof(float)*10);

    //CHECK:  *data = (float *)sycl::malloc_device(sizeof(float)*10*10, q_ct1);
    hipMalloc(data, sizeof(float)*10*10);

    size_t size2;
    // CHECK: size2 = d_A.get_size();
    hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:  */
    // CHECK-NEXT:  err = (size2 = d_A.get_size(), 0);
    err = hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:MY_ERROR_CHECKER((size2 = d_A.get_size(), 0));
    MY_ERROR_CHECKER(hipGetSymbolSize(&size2, HIP_SYMBOL(d_A)));

    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    MY_ERROR_CHECKER(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    MY_ERROR_CHECKER(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    MY_ERROR_CHECKER(hipMemPrefetchAsync(a, 100, deviceID, nullptr));

    //CHECK: stream_array[0]->memcpy(h_A, d_A, size2);
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, stream_array[0]);

    // CHECK: q_ct1.memcpy(h_A, d_A, size2);
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((q_ct1.memcpy(h_A, d_A, size2), 0));
    // CHECK-NEXT: q_ct1.memcpy(h_A, d_A, size2);
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((q_ct1.memcpy(h_A, d_A, size2), 0));
    // CHECK-NEXT: q_ct1.memcpy(h_A, d_A, size2);
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((q_ct1.memcpy(h_A, d_A, size2), 0));
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamDefault);
    MY_ERROR_CHECKER(hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamDefault));
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamPerThread);
    MY_ERROR_CHECKER(hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamPerThread));
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamLegacy);
    MY_ERROR_CHECKER(hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamLegacy));
}


template <typename T>
int foo() {
    T* a;
    hipStream_t stream;
    int deviceID = 0;
    hipError_t err;
    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    MY_ERROR_CHECKER(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    MY_ERROR_CHECKER(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    MY_ERROR_CHECKER(hipMemPrefetchAsync(a, 100, deviceID, nullptr));
    return 0;
}

template int foo<float>();
template int foo<int>();

void checkError(hipError_t err) {
}

void foobar() {
  int errorCode;

  hipChannelFormatDesc desc;
  hipExtent extent;
  unsigned int flags;
  hipArray_t array;

  // CHECK: desc = array->get_channel();
  // CHECK: extent = array->get_range();
  // CHECK: flags = 0;
  hipArrayGetInfo(&desc, &extent, &flags, array);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(([&](){
  // CHECK-NEXT:   desc = array->get_channel();
  // CHECK-NEXT:   extent = array->get_range();
  // CHECK-NEXT:   flags = 0;
  // CHECK-NEXT:   }(), 0));
  checkError(hipArrayGetInfo(&desc, &extent, &flags, array));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = ([&](){
  // CHECK-NEXT:   desc = array->get_channel();
  // CHECK-NEXT:   extent = array->get_range();
  // CHECK-NEXT:   flags = 0;
  // CHECK-NEXT:   }(), 0);
  errorCode = hipArrayGetInfo(&desc, &extent, &flags, array);

  int host;
  // CHECK: flags = 0;
  hipHostGetFlags(&flags, &host);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((flags = 0, 0));
  checkError(hipHostGetFlags(&flags, &host));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (flags = 0, 0);
  errorCode = hipHostGetFlags(&flags, &host);

  int *devPtr;
  size_t count;
  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: int advice = 0;
  hipMemoryAdvise advice = hipMemAdviseSetReadMostly;
  int device;
  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::get_device(device).default_queue().mem_advise(devPtr, count, advice);
  hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, advice), 0));
  checkError(hipMemAdvise(devPtr, count, advice, device));
  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, 0), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemoryAdvise(1), device));
  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, 0), 0));
  checkError(hipMemAdvise(devPtr, count, (hipMemoryAdvise)1, device));
  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, 0), 0));
  checkError(hipMemAdvise(devPtr, count, static_cast<hipMemoryAdvise>(1), device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (dpct::get_device(device).default_queue().mem_advise(devPtr, count, advice), 0);
  errorCode = hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::get_device(device).default_queue().mem_advise(devPtr, count, 0);
  hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, 0), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (dpct::get_device(device).default_queue().mem_advise(devPtr, count, 0), 0);
  errorCode = hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::cpu_device().default_queue().mem_advise(devPtr, count, 0), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, hipCpuDeviceId));


#define QRNG_DIMENSIONS 3
#define AAA(x)   (x + 2)
#define BBB(x)   x * 2
//CHECK: #define SIZEOF_FLOAT sizeof(float)
//CHECK-NEXT: #define SIZEOF(x) sizeof(x)
#define SIZEOF_FLOAT sizeof(float)
#define SIZEOF(x) sizeof(x)


  const int N = 1048576;
  float *d_Output;


  // a * sizeof
  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(AAA(N) * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, AAA(N) * N * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(N * AAA(N), q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, N * AAA(N) * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(N * QRNG_DIMENSIONS, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, N * QRNG_DIMENSIONS * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * N * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * N * SIZEOF_FLOAT));

  //CHECK: d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * N, q_ct1);
  hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * N * SIZEOF_FLOAT);

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * N * SIZEOF(float)));

  //CHECK: d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * N, q_ct1);
  hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * N * SIZEOF(float));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(N * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, N * N * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * QRNG_DIMENSIONS, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * QRNG_DIMENSIONS * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(AAA(N) * AAA(N), q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, AAA(N) * AAA(N) * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(AAA(N) * QRNG_DIMENSIONS, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, AAA(N) * QRNG_DIMENSIONS * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * AAA(N), q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * AAA(N) * sizeof(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(N * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, N * N * SIZEOF_FLOAT));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(N * N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, N * N * SIZEOF(float)));



  //CHECK: d_Output = sycl::malloc_device<float>(AAA(N) * N, q_ct1);
  hipMalloc((void **)&d_Output, AAA(N) * N * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(N * AAA(N), q_ct1);
  hipMalloc((void **)&d_Output, N * AAA(N) * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(N * QRNG_DIMENSIONS, q_ct1);
  hipMalloc((void **)&d_Output, N * QRNG_DIMENSIONS * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * N, q_ct1);
  hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * N * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(N * N, q_ct1);
  hipMalloc((void **)&d_Output, N * N * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * QRNG_DIMENSIONS, q_ct1);
  hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * QRNG_DIMENSIONS * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(AAA(N) * AAA(N), q_ct1);
  hipMalloc((void **)&d_Output, AAA(N) * AAA(N) * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(AAA(N) * QRNG_DIMENSIONS, q_ct1);
  hipMalloc((void **)&d_Output, AAA(N) * QRNG_DIMENSIONS * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS * AAA(N), q_ct1);
  hipMalloc((void **)&d_Output, QRNG_DIMENSIONS * AAA(N) * sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(N * N, q_ct1);
  hipMalloc((void **)&d_Output, N * N * SIZEOF_FLOAT);

  //CHECK: d_Output = sycl::malloc_device<float>(N * N, q_ct1);
  hipMalloc((void **)&d_Output, N * N * SIZEOF(float));



  // sizeof * a
  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, sizeof(float) * QRNG_DIMENSIONS));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(N, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, sizeof(float) * N));

  //CHECK: MY_ERROR_CHECKER((d_Output = (float *)sycl::malloc_device(sizeof(float) * BBB(N), q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, sizeof(float) * BBB(N)));

  //CHECK: d_Output = (float *)sycl::malloc_device(sizeof(float) * BBB(N), q_ct1);
  hipMalloc((void **)&d_Output, sizeof(float) * BBB(N));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(AAA(N), q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, sizeof(float) * AAA(N)));

  //CHECK: d_Output = sycl::malloc_device<float>(AAA(N), q_ct1);
  hipMalloc((void **)&d_Output, sizeof(float) * AAA(N));

  //CHECK: d_Output = sycl::malloc_device<float>(QRNG_DIMENSIONS, q_ct1);
  hipMalloc((void **)&d_Output, sizeof(float) * QRNG_DIMENSIONS);

  //CHECK: d_Output = sycl::malloc_device<float>(N, q_ct1);
  hipMalloc((void **)&d_Output, sizeof(float) * N);

  //CHECK: d_Output = sycl::malloc_device<float>(N, q_ct1);
  hipMalloc((void **)&d_Output, SIZEOF(float) * N);

  //CHECK: d_Output = sycl::malloc_device<float>(N, q_ct1);
  hipMalloc((void **)&d_Output, SIZEOF_FLOAT * N);



  // sizeof
  //CHECK: d_Output = sycl::malloc_device<float>(1, q_ct1);
  hipMalloc((void **)&d_Output, sizeof(float));

  //CHECK: d_Output = sycl::malloc_device<float>(1, q_ct1);
  hipMalloc((void **)&d_Output, SIZEOF_FLOAT);

  //CHECK: d_Output = sycl::malloc_device<float>(1, q_ct1);
  hipMalloc((void **)&d_Output, SIZEOF(float));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(1, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, SIZEOF_FLOAT));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(1, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, SIZEOF(float)));

  //CHECK: MY_ERROR_CHECKER((d_Output = sycl::malloc_device<float>(1, q_ct1), 0));
  MY_ERROR_CHECKER(hipMalloc((void **)&d_Output, sizeof(float)));
}

