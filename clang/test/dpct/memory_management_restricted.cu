// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++11
// RUN: FileCheck --match-full-lines --input-file %T/memory_management_restricted.dp.cpp %s

#include <hip/hip_runtime.h>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {}

#define DATAMACRO 32*32

int main(){
    //CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    //CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();

    float **data = NULL;
    float *d_A = NULL;
    float *h_A = NULL;
    int* a;
    hipStream_t stream;
    hipStream_t stream_array[10];
    int deviceID = 0;
    hipError_t err;

    //CHECK:  /*
    //CHECK-NEXT:  DPCT1003:0: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:  */
    //CHECK-NEXT:  checkCudaErrors((*data = (float *)sycl::malloc_device(DATAMACRO, q_ct1), 0));
    checkCudaErrors(hipMalloc((void **)data, DATAMACRO));

    //Currently, migration of using template version API only covers the simple case: the argument specifiy the size is sizeof(T)*Expr, Expr*sizeof(T) and sizeof(T)
    //CHECK:  *data = sycl::malloc_device<float>(10*10, q_ct1);
    hipMalloc(data, 10*10*sizeof(float));

    //CHECK:  *data = (float *)sycl::malloc_device(10*sizeof(float)*10, q_ct1);
    hipMalloc(data, 10*sizeof(float)*10);

    //CHECK:  *data = (float *)sycl::malloc_device(sizeof(float)*10*10, q_ct1);
    hipMalloc(data, sizeof(float)*10*10);

    size_t size2;
    // CHECK: size2 = d_A.get_size();
    hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:  */
    // CHECK-NEXT:  err = (size2 = d_A.get_size(), 0);
    err = hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:checkCudaErrors((size2 = d_A.get_size(), 0));
    checkCudaErrors(hipGetSymbolSize(&size2, HIP_SYMBOL(d_A)));

    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, nullptr));

    //CHECK: stream_array[0]->memcpy(h_A, d_A, size2);
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, stream_array[0]);

    // CHECK: q_ct1.memcpy(h_A, d_A, size2);
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((q_ct1.memcpy(h_A, d_A, size2), 0));
    // CHECK-NEXT: q_ct1.memcpy(h_A, d_A, size2);
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((q_ct1.memcpy(h_A, d_A, size2), 0));
    // CHECK-NEXT: q_ct1.memcpy(h_A, d_A, size2);
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((q_ct1.memcpy(h_A, d_A, size2), 0));
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamDefault);
    checkCudaErrors(hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamDefault));
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamPerThread);
    checkCudaErrors(hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamPerThread));
    hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamLegacy);
    checkCudaErrors(hipMemcpyAsync(h_A, d_A, size2, hipMemcpyDeviceToHost, hipStreamLegacy));
}


template <typename T>
int foo() {
    T* a;
    hipStream_t stream;
    int deviceID = 0;
    hipError_t err;
    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, nullptr));
    return 0;
}

template int foo<float>();
template int foo<int>();

void checkError(hipError_t err) {
}

void foobar() {
  int errorCode;

  hipChannelFormatDesc desc;
  hipExtent extent;
  unsigned int flags;
  hipArray_t array;

  // CHECK: desc = array->get_channel();
  // CHECK: extent = array->get_range();
  // CHECK: flags = 0;
  hipArrayGetInfo(&desc, &extent, &flags, array);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(([&](){
  // CHECK-NEXT:   desc = array->get_channel();
  // CHECK-NEXT:   extent = array->get_range();
  // CHECK-NEXT:   flags = 0;
  // CHECK-NEXT:   }(), 0));
  checkError(hipArrayGetInfo(&desc, &extent, &flags, array));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = ([&](){
  // CHECK-NEXT:   desc = array->get_channel();
  // CHECK-NEXT:   extent = array->get_range();
  // CHECK-NEXT:   flags = 0;
  // CHECK-NEXT:   }(), 0);
  errorCode = hipArrayGetInfo(&desc, &extent, &flags, array);

  int host;
  // CHECK: flags = 0;
  hipHostGetFlags(&flags, &host);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((flags = 0, 0));
  checkError(hipHostGetFlags(&flags, &host));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (flags = 0, 0);
  errorCode = hipHostGetFlags(&flags, &host);

  int *devPtr;
  size_t count;
  // CHECK: pi_mem_advice advice;
  hipMemoryAdvise advice;
  int device;
  // CHECK: dpct::get_device(device).default_queue().mem_advise(devPtr, count, pi_mem_advice(advice - 1));
  hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, pi_mem_advice(advice - 1)), 0));
  checkError(hipMemAdvise(devPtr, count, advice, device));
  // CHECK: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemoryAdvise(1), device));
  // CHECK: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, (hipMemoryAdvise)1, device));
  // CHECK: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, static_cast<hipMemoryAdvise>(1), device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (dpct::get_device(device).default_queue().mem_advise(devPtr, count, pi_mem_advice(advice - 1)), 0);
  errorCode = hipMemAdvise(devPtr, count, advice, device);

  // CHECK: dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY);
  hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0);
  errorCode = hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::cpu_device().default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, hipCpuDeviceId));
}
