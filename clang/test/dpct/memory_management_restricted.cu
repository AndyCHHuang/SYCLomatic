// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++11
// RUN: FileCheck --match-full-lines --input-file %T/memory_management_restricted.dp.cpp %s

#include <hip/hip_runtime.h>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {}

#define DATAMACRO 32*32

int main(){
    float **data = NULL;
    float *d_A = NULL;
    int* a;
    hipStream_t stream;
    int deviceID = 0;
    hipError_t err;

    //CHECK:  /*
    //CHECK-NEXT:  DPCT1003:0: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:  */
    //CHECK-NEXT:  checkCudaErrors((*data = (float *)sycl::malloc_device(DATAMACRO, dpct::get_current_device(), dpct::get_default_context()), 0));
    checkCudaErrors(hipMalloc((void **)data, DATAMACRO));

    size_t size2;
    // CHECK: size2 = d_A.get_size();
    hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:  */
    // CHECK-NEXT:  err = (size2 = d_A.get_size(), 0);
    err = hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:checkCudaErrors((size2 = d_A.get_size(), 0));
    checkCudaErrors(hipGetSymbolSize(&size2, HIP_SYMBOL(d_A)));

    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, nullptr));
}


template <typename T>
int foo() {
    T* a;
    hipStream_t stream;
    int deviceID = 0;
    hipError_t err;
    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, nullptr));
    return 0;
}

template int foo<float>();
template int foo<int>();

void checkError(hipError_t err) {
}

void foobar() {
  int errorCode;

  hipChannelFormatDesc desc;
  hipExtent extent;
  unsigned int flags;
  hipArray_t array;

  // CHECK: array->get_info(desc, extent, flags);
  hipArrayGetInfo(&desc, &extent, &flags, array);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((array->get_info(desc, extent, flags), 0));
  checkError(hipArrayGetInfo(&desc, &extent, &flags, array));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (array->get_info(desc, extent, flags), 0);
  errorCode = hipArrayGetInfo(&desc, &extent, &flags, array);

  int host;
  // CHECK: flags = 0;
  hipHostGetFlags(&flags, &host);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((flags = 0, 0));
  checkError(hipHostGetFlags(&flags, &host));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (flags = 0, 0);
  errorCode = hipHostGetFlags(&flags, &host);

  int *devPtr;
  size_t count;
  // CHECK: pi_mem_advice advice;
  hipMemoryAdvise advice;
  int device;
  // CHECK: dpct::get_device(device).default_queue().mem_advise(devPtr, count, pi_mem_advice(advice - 1));
  hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, pi_mem_advice(advice - 1)), 0));
  checkError(hipMemAdvise(devPtr, count, advice, device));
  // CHECK: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemoryAdvise(1), device));
  // CHECK: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, (hipMemoryAdvise)1, device));
  // CHECK: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, static_cast<hipMemoryAdvise>(1), device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (dpct::get_device(device).default_queue().mem_advise(devPtr, count, pi_mem_advice(advice - 1)), 0);
  errorCode = hipMemAdvise(devPtr, count, advice, device);

  // CHECK: dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY);
  hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (dpct::get_device(device).default_queue().mem_advise(devPtr, count, PI_MEM_ADVICE_SET_READ_MOSTLY), 0);
  errorCode = hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);
}
