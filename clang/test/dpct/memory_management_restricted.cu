// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++11
// RUN: FileCheck --match-full-lines --input-file %T/memory_management_restricted.dp.cpp %s

#include <hip/hip_runtime.h>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T result, char const *const func, const char *const file, int const line) {}

#define DATAMACRO 32*32

int main(){
    float **data = NULL;
    float *d_A = NULL;
    int* a;
    hipStream_t stream;
    int deviceID = 0;
    hipError_t err;

    //CHECK:  /*
    //CHECK-NEXT:  DPCT1003:0: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:  */
    //CHECK-NEXT:  checkCudaErrors((*data = (float *)sycl::malloc_device(DATAMACRO, dpct::get_current_device(), dpct::get_default_context()), 0));
    checkCudaErrors(hipMalloc((void **)data, DATAMACRO));

    size_t size2;
    // CHECK: size2 = d_A.get_size();
    hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:  */
    // CHECK-NEXT:  err = (size2 = d_A.get_size(), 0);
    err = hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

    // CHECK: /*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:checkCudaErrors((size2 = d_A.get_size(), 0));
    checkCudaErrors(hipGetSymbolSize(&size2, HIP_SYMBOL(d_A)));

    // CHECK: stream->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, stream);

    // CHECK: (*&stream)->prefetch(a,100);
    hipMemPrefetchAsync (a, 100, deviceID, *&stream);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: err = (dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0);
    err = hipMemPrefetchAsync(a, 100, deviceID);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, NULL));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, 0));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated api does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: checkCudaErrors((dpct::dev_mgr::instance().get_device(deviceID).default_queue().prefetch(a,100), 0));
    checkCudaErrors(hipMemPrefetchAsync(a, 100, deviceID, nullptr));
}