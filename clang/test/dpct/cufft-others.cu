// RUN: dpct --format-range=none -out-root %T/cufft-others %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/cufft-others/cufft-others.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


int main() {
  //CHECK:std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::SINGLE, oneapi::mkl::dft::domain::REAL>> plan;
  //CHECK-NEXT:sycl::float2* iodata;
  hipfftHandle plan;
  float2* iodata;

  //CHECK:plan = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::SINGLE, oneapi::mkl::dft::domain::REAL>>(10 + 2);
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[2] = {0, 1};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[2] = {0, 1};
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, ((10 + 2)/2+1)*2);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, (10 + 2)/2+1);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 3);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(dpct::get_default_queue());
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);

  return 0;
}

int foo2() {
  hipfftHandle plan_mmany64_Z2Z;
  size_t* work_size_mmany64_Z2Z;
  long long int odist_mmany64_Z2Z;
  long long int ostride_mmany64_Z2Z;
  long long int * onembed_mmany64_Z2Z;
  long long int idist_mmany64_Z2Z;
  long long int istride_mmany64_Z2Z;
  long long int* inembed_mmany64_Z2Z;
  long long int * n_mmany64_Z2Z;
  double2* odata_mmany64_Z2Z;
  double2* idata_mmany64_Z2Z;


  //CHECK:/*
  //CHECK-NEXT:DPCT1067:{{[0-9]+}}: The 'work_size_mmany64_Z2Z' parameter could not be migrated. You may need to update the code manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1071:{{[0-9]+}}: The placement of the FFT computational function could not be deduced, so it is assumed out-of-place. You may need to adjust the code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan_mmany64_Z2Z = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::DOUBLE, oneapi::mkl::dft::domain::COMPLEX>>(std::vector<std::int64_t>{n_mmany64_Z2Z[0], n_mmany64_Z2Z[1], n_mmany64_Z2Z[2]});
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::PLACEMENT, DFTI_CONFIG_VALUE::DFTI_NOT_INPLACE);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 12);
  //CHECK-NEXT:if (inembed_mmany64_Z2Z != nullptr && onembed_mmany64_Z2Z != nullptr) {
  //CHECK-NEXT:std::int64_t input_stride_ct2[4] = {0, inembed_mmany64_Z2Z[2] * inembed_mmany64_Z2Z[1] * istride_mmany64_Z2Z, inembed_mmany64_Z2Z[2] * istride_mmany64_Z2Z, istride_mmany64_Z2Z};
  //CHECK-NEXT:std::int64_t output_stride_ct3[4] = {0, onembed_mmany64_Z2Z[2] * onembed_mmany64_Z2Z[1] * ostride_mmany64_Z2Z, onembed_mmany64_Z2Z[2] * ostride_mmany64_Z2Z, ostride_mmany64_Z2Z};
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct2);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct3);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, n_mmany64_Z2Z[2]*n_mmany64_Z2Z[1]*n_mmany64_Z2Z[0]);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, n_mmany64_Z2Z[2]*n_mmany64_Z2Z[1]*n_mmany64_Z2Z[0]);
  //CHECK-NEXT:}
  hipfftMakePlanMany64(plan_mmany64_Z2Z, 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany64_Z2Z);

  //CHECK:/*
  //CHECK-NEXT:DPCT1066:{{[0-9]+}}: Input/Output distance are migrated to FWD/BWD_DISTANCE. You may need to update the code manually.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if (inembed_mmany64_Z2Z != nullptr && onembed_mmany64_Z2Z != nullptr) {
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, idist_mmany64_Z2Z);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, odist_mmany64_Z2Z);
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan_mmany64_Z2Z->commit(q_ct1);
  //CHECK-NEXT:if ((void *)idata_mmany64_Z2Z == (void *)odata_mmany64_Z2Z) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan_mmany64_Z2Z, (double*)idata_mmany64_Z2Z);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan_mmany64_Z2Z, (double*)idata_mmany64_Z2Z, (double*)odata_mmany64_Z2Z);
  //CHECK-NEXT:}
  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_FORWARD);

  //CHECK:/*
  //CHECK-NEXT:DPCT1066:{{[0-9]+}}: Input/Output distance are migrated to FWD/BWD_DISTANCE. You may need to update the code manually.
  //CHECK-NEXT:*/
  //CHECK:if (inembed_mmany64_Z2Z != nullptr && onembed_mmany64_Z2Z != nullptr) {
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, idist_mmany64_Z2Z);
  //CHECK-NEXT:plan_mmany64_Z2Z->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, odist_mmany64_Z2Z);
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan_mmany64_Z2Z->commit(q_ct1);
  //CHECK-NEXT:if ((void *)idata_mmany64_Z2Z == (void *)odata_mmany64_Z2Z) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan_mmany64_Z2Z, (double*)idata_mmany64_Z2Z);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_backward(*plan_mmany64_Z2Z, (double*)idata_mmany64_Z2Z, (double*)odata_mmany64_Z2Z);
  //CHECK-NEXT:}
  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_BACKWARD);

  return 0;
}

int foo3(hipStream_t stream) {
  //CHECK:std::shared_ptr<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::SINGLE, oneapi::mkl::dft::domain::REAL>> plan;
  //CHECK-NEXT:sycl::float2* iodata;
  hipfftHandle plan;
  float2* iodata;

  //CHECK:plan = std::make_shared<oneapi::mkl::dft::descriptor<oneapi::mkl::dft::precision::SINGLE, oneapi::mkl::dft::domain::REAL>>(10 + 2);
  //CHECK-NEXT:std::int64_t input_stride_ct{{[0-9]+}}[2] = {0, 1};
  //CHECK-NEXT:std::int64_t output_stride_ct{{[0-9]+}}[2] = {0, 1};
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::INPUT_STRIDES, input_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::OUTPUT_STRIDES, output_stride_ct{{[0-9]+}});
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::FWD_DISTANCE, ((10 + 2)/2+1)*2);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::BWD_DISTANCE, (10 + 2)/2+1);
  //CHECK-NEXT:plan->set_value(oneapi::mkl::dft::config_param::NUMBER_OF_TRANSFORMS, 3);
  hipfftSetStream(plan, stream);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  //CHECK:plan->commit(*stream);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);

  return 0;
}