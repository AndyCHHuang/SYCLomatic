// RUN: dpct --format-range=none --usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasTtrmmLegacy.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;
  int ldb = 275;
  const float *A_S = 0;
  float *B_S = 0;
  float alpha_S = 1.0f;
  const double *A_D = 0;
  double *B_D = 0;
  double alpha_D = 1.0;
  const hipComplex *A_C = 0;
  hipComplex *B_C = 0;
  hipComplex alpha_C = make_hipComplex(1.0f,0.0f);
  const hipDoubleComplex *A_Z = 0;
  hipDoubleComplex *B_Z = 0;
  hipDoubleComplex alpha_Z = make_hipDoubleComplex(1.0,0.0);


  //Legacy
  // CHECK: {
  // CHECK-NEXT: auto sidemode_ct0 = 'L';
  // CHECK-NEXT: auto fillmode_ct1 = 'U';
  // CHECK-NEXT: auto transpose_ct2 = 'N';
  // CHECK-NEXT: auto diagtype_ct3 = 'N';
  // CHECK-NEXT: auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(A_S);
  // CHECK-NEXT:cl::sycl::buffer<float> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(B_S);
  // CHECK-NEXT:cl::sycl::buffer<float> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<float>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(float)));
  // CHECK-NEXT:mkl::blas::trmm(dpct::get_default_queue(), (((sidemode_ct0)=='L'||(sidemode_ct0)=='l')?(mkl::side::left):(mkl::side::right)), (((fillmode_ct1)=='L'||(fillmode_ct1)=='l')?(mkl::uplo::lower):(mkl::uplo::upper)), (((transpose_ct2)=='N'||(transpose_ct2)=='n')?(mkl::transpose::nontrans):(((transpose_ct2)=='T'||(transpose_ct2)=='t')?(mkl::transpose::nontrans):(mkl::transpose::conjtrans))), (((diagtype_ct3)=='N'||(diagtype_ct3)=='n')?(mkl::diag::nonunit):(mkl::diag::unit)), m, n, alpha_S, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasStrmm('L', 'U', 'N', 'N', m, n, alpha_S, A_S, lda, B_S, ldb);

  // CHECK: {
  // CHECK-NEXT: auto sidemode_ct0 = 'L';
  // CHECK-NEXT: auto fillmode_ct1 = 'U';
  // CHECK-NEXT: auto transpose_ct2 = 'N';
  // CHECK-NEXT: auto diagtype_ct3 = 'N';
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(A_D);
  // CHECK-NEXT:cl::sycl::buffer<double> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(B_D);
  // CHECK-NEXT:cl::sycl::buffer<double> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<double>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(double)));
  // CHECK-NEXT:mkl::blas::trmm(dpct::get_default_queue(), (((sidemode_ct0)=='L'||(sidemode_ct0)=='l')?(mkl::side::left):(mkl::side::right)), (((fillmode_ct1)=='L'||(fillmode_ct1)=='l')?(mkl::uplo::lower):(mkl::uplo::upper)), (((transpose_ct2)=='N'||(transpose_ct2)=='n')?(mkl::transpose::nontrans):(((transpose_ct2)=='T'||(transpose_ct2)=='t')?(mkl::transpose::nontrans):(mkl::transpose::conjtrans))), (((diagtype_ct3)=='N'||(diagtype_ct3)=='n')?(mkl::diag::nonunit):(mkl::diag::unit)), m, n, alpha_D, buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasDtrmm('L', 'U', 'N', 'N', m, n, alpha_D, A_D, lda, B_D, ldb);

  // CHECK: {
  // CHECK-NEXT: auto sidemode_ct0 = 'L';
  // CHECK-NEXT: auto fillmode_ct1 = 'U';
  // CHECK-NEXT: auto transpose_ct2 = 'N';
  // CHECK-NEXT: auto diagtype_ct3 = 'N';
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(A_C);
  // CHECK-NEXT:cl::sycl::buffer<std::complex<float>> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(B_C);
  // CHECK-NEXT:cl::sycl::buffer<std::complex<float>> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<float>>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<float>)));
  // CHECK-NEXT:mkl::blas::trmm(dpct::get_default_queue(), (((sidemode_ct0)=='L'||(sidemode_ct0)=='l')?(mkl::side::left):(mkl::side::right)), (((fillmode_ct1)=='L'||(fillmode_ct1)=='l')?(mkl::uplo::lower):(mkl::uplo::upper)), (((transpose_ct2)=='N'||(transpose_ct2)=='n')?(mkl::transpose::nontrans):(((transpose_ct2)=='T'||(transpose_ct2)=='t')?(mkl::transpose::nontrans):(mkl::transpose::conjtrans))), (((diagtype_ct3)=='N'||(diagtype_ct3)=='n')?(mkl::diag::nonunit):(mkl::diag::unit)), m, n, std::complex<float>((alpha_C).x(),(alpha_C).y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasCtrmm('L', 'U', 'N', 'N', m, n, alpha_C, A_C, lda, B_C, ldb);

  // CHECK: {
  // CHECK-NEXT: auto sidemode_ct0 = 'L';
  // CHECK-NEXT: auto fillmode_ct1 = 'U';
  // CHECK-NEXT: auto transpose_ct2 = 'N';
  // CHECK-NEXT: auto diagtype_ct3 = 'N';
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(A_Z);
  // CHECK-NEXT:cl::sycl::buffer<std::complex<double>> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT:auto allocation_ct{{[0-9]+}} = dpct::mem_mgr::instance().translate_ptr(B_Z);
  // CHECK-NEXT:cl::sycl::buffer<std::complex<double>> buffer_ct{{[0-9]+}} = allocation_ct{{[0-9]+}}.buffer.reinterpret<std::complex<double>>(cl::sycl::range<1>(allocation_ct{{[0-9]+}}.size/sizeof(std::complex<double>)));
  // CHECK-NEXT:mkl::blas::trmm(dpct::get_default_queue(), (((sidemode_ct0)=='L'||(sidemode_ct0)=='l')?(mkl::side::left):(mkl::side::right)), (((fillmode_ct1)=='L'||(fillmode_ct1)=='l')?(mkl::uplo::lower):(mkl::uplo::upper)), (((transpose_ct2)=='N'||(transpose_ct2)=='n')?(mkl::transpose::nontrans):(((transpose_ct2)=='T'||(transpose_ct2)=='t')?(mkl::transpose::nontrans):(mkl::transpose::conjtrans))), (((diagtype_ct3)=='N'||(diagtype_ct3)=='n')?(mkl::diag::nonunit):(mkl::diag::unit)), m, n, std::complex<double>((alpha_Z).x(),(alpha_Z).y()), buffer_ct{{[0-9]+}}, lda, buffer_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasZtrmm('L', 'U', 'N', 'N', m, n, alpha_Z, A_Z, lda, B_Z, ldb);
}
