// RUN: dpct --format-range=none -out-root %T/time-measure-usm-restricted %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/time-measure-usm-restricted/time-measure-usm-restricted.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    hipStream_t stream;

    int ha[N], hb[N];
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
    hipEvent_t start, stop;
    hipError_t cudaStatus;

    int *da, *db;
    float elapsedTime;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }


    hipEventCreate(&start);
    hipEventCreate(&stop);

   // CHECK:    sycl::event stop_q_ct1_1;
   // CHECK-NEXT:    sycl::event stop_q_ct1_2;
   // CHECK-NEXT:    sycl::event stop_stream_1;
   // CHECK-MEXT:    start_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);

    // CHECK: stop_q_ct1_1 = q_ct1.memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    // CHECK: stop_q_ct1_2 = q_ct1.memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, 0);
    // CHECK: stop_stream_1 = stream->memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, stream);

    // CHECK: stop_stream_1.wait();
    // CHECK: stop_q_ct1_1.wait();
    // CHECK: stop_q_ct1_2.wait();
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    // CHECK: elapsedTime = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    add<<<N, 1>>>(da, db);

    // CHECK: q_ct1.memcpy(hb, db, N*sizeof(int));
    hipMemcpyAsync(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return 0;
}


