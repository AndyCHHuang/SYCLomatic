#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/devicemem_usm.dp.cpp

#include <hip/hip_runtime.h>

#include <cassert>

#define NUM_ELEMENTS (/* Threads per block */ 16)

class TestStruct {
public:
  __device__ void test() {}
};

// CHECK: dpct::device_memory<TestStruct, 0> t1;
__device__ TestStruct t1;

// CHECK: void member_acc(TestStruct *t1) {
// CHECK-NEXT:  t1->test();
// CHECK-NEXT:}
__global__ void member_acc() {
  t1.test();
}

// CHECK: dpct::device_memory<float, 1> in(NUM_ELEMENTS);
__device__ float in[NUM_ELEMENTS];
// CHECK: dpct::device_memory<int, 1> init(sycl::range<1>(4), {1, 2, 3, 4});
__device__ int init[4] = {1, 2, 3, 4};

// CHECK: void kernel1(float *out, sycl::nd_item<3> [[ITEM:item_ct1]], float *in) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(2)] = in[{{.*}}[[ITEM]].get_local_id(2)];
// CHECK: }
__global__ void kernel1(float *out) {
  out[threadIdx.x] = in[threadIdx.x];
}

// CHECK: dpct::device_memory<int, 0> al;
__device__ int al;
// CHECK: dpct::device_memory<int, 0> ainit(NUM_ELEMENTS);
__device__ int ainit = NUM_ELEMENTS;

const int num_elements = 16;
// CHECK: dpct::device_memory<float, 1> fx(2);
// CHECK: dpct::device_memory<float, 2> fy(num_elements, 4 * num_elements);
__device__ float fx[2], fy[num_elements][4 * num_elements];

// CHECK: void kernel2(float *out, sycl::nd_item<3> [[ITEM:item_ct1]], int *al, float *fx, dpct::accessor<float, dpct::device, 2> fy, float *tmp) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(2)] += *al;
// CHECK:   fx[{{.*}}[[ITEM]].get_local_id(2)] = fy[{{.*}}[[ITEM]].get_local_id(2)][{{.*}}[[ITEM]].get_local_id(2)];
// CHECK: }
__global__ void kernel2(float *out) {
  const int size = 64;
  __device__ float tmp[size];
  out[threadIdx.x] += al;
  fx[threadIdx.x] = fy[threadIdx.x][threadIdx.x];
}

int main() {
  float h_in[NUM_ELEMENTS] = {0};
  float h_out[NUM_ELEMENTS] = {0};

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    h_in[i] = i;
    h_out[i] = -i;
  }

  const size_t array_size = sizeof(float) * NUM_ELEMENTS;
  // CTST-50
  hipMemcpyToSymbol(HIP_SYMBOL(in), h_in, array_size);

  const int h_a = 3;
  // CTST-50
  hipMemcpyToSymbol(HIP_SYMBOL(al), &h_a, sizeof(int));

  float *d_out = NULL;
  hipMalloc((void **)&d_out, array_size);

  const int threads_per_block = NUM_ELEMENTS;
  // CHECK:   dpct::get_default_queue_wait().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto t1_ptr_ct1 = t1.get_ptr();
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class member_acc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, threads_per_block), sycl::range<3>(1, 1, threads_per_block)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           member_acc(t1_ptr_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  member_acc<<<1, threads_per_block>>>();
  // CHECK:   dpct::get_default_queue_wait().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto in_ptr_ct1 = in.get_ptr();
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel1_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, threads_per_block), sycl::range<3>(1, 1, threads_per_block)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel1(d_out, item_ct1, in_ptr_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel1<<<1, threads_per_block>>>(d_out);

  // CHECK:   dpct::get_default_queue_wait().submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       dpct::device_memory<float, 1> tmp(64/*size*/);
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto tmp_ptr_ct1 = tmp.get_ptr();
  // CHECK-NEXT:       auto al_ptr_ct1 = al.get_ptr();
  // CHECK-NEXT:       auto fx_ptr_ct1 = fx.get_ptr();
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto fy_acc_ct1 = fy.get_access(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel2_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1) * sycl::range<3>(1, 1, threads_per_block), sycl::range<3>(1, 1, threads_per_block)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel2(d_out, item_ct1, al_ptr_ct1, fx_ptr_ct1, fy_acc_ct1, tmp_ptr_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel2<<<1, threads_per_block>>>(d_out);

  hipMemcpy(h_out, d_out, array_size, hipMemcpyDeviceToHost);

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    assert(h_out[i] == i + h_a && "Value mis-calculated!");
  }

  return 0;
}
