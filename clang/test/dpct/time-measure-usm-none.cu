// RUN: dpct --format-range=none -usm-level=none -out-root %T %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/time-measure-usm-none.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
    hipStream_t stream;

    int ha[N], hb[N];
    // CHECK: std::chrono::time_point<std::chrono::high_resolution_clock> start_ct1;
    // CHECK: std::chrono::time_point<std::chrono::high_resolution_clock> stop_ct1;
    hipEvent_t start, stop;
    hipError_t cudaStatus;

    int *da, *db;
    float elapsedTime;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }


    hipEventCreate(&start);
    hipEventCreate(&stop);

    // CHECK: start_ct1 = std::chrono::high_resolution_clock::now();
    hipEventRecord(start, 0);

    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device);
    // CHECK: q_ct1.wait();
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device);
    // CHECK: q_ct1.wait();
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, 0);
    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device, *stream);
    // CHECK: stream->wait();
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, stream);

    // CHECK: stop_ct1 = std::chrono::high_resolution_clock::now();
    hipEventRecord(stop, 0);
    // CHECK: stop.wait_and_throw();
    hipEventSynchronize(stop);
    // CHECK: elapsedTime = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventElapsedTime(&elapsedTime, start, stop);

    add<<<N, 1>>>(da, db);

    // CHECK: dpct::async_dpct_memcpy(hb, db, N*sizeof(int), dpct::device_to_host);
    hipMemcpyAsync(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return 0;
}

