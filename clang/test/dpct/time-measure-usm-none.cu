#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -usm-level=none -out-root %T/time-measure-usm-none %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/time-measure-usm-none/time-measure-usm-none.dp.cpp --match-full-lines %s
#include <stdio.h>

#define N 1000

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}


__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
    hipStream_t stream;

    int ha[N], hb[N];
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
    hipEvent_t start, stop;
    hipError_t cudaStatus;

    int *da, *db;
    float elapsedTime;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }


    hipEventCreate(&start);
    hipEventCreate(&stop);

    // CHECK: start_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);

    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device);
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device);
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, 0);
    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device, *stream);
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, stream);

    // CHECK: stream->wait();
    // CHECK: q_ct1.wait();
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    // CHECK: elapsedTime = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    add<<<N, 1>>>(da, db);

    // CHECK: dpct::async_dpct_memcpy(hb, db, N*sizeof(int), dpct::device_to_host);
    hipMemcpyAsync(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return 0;
}


__global__ void kernel_foo(){}

void foo_test_1() {

    hipEvent_t     start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

// CHECK:    start_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:        for (int i=0; i<4; i++) {
// CHECK-NEXT:            dpct::get_default_queue().submit(
// CHECK-NEXT:              [&](sycl::handler &cgh) {
// CHECK-NEXT:                cgh.parallel_for<dpct_kernel_name<class kernel_foo_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                  sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:                  [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                    kernel_foo();
// CHECK-NEXT:                  });
// CHECK-NEXT:              });
// CHECK-NEXT:        }
// CHECK-NEXT:    dpct::get_current_device().queues_wait_and_throw();
    hipEventRecord( start, 0 );
        for (int i=0; i<4; i++) {
            kernel_foo<<<1, 1>>>();
        }
    hipDeviceSynchronize();

    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop ) ;
}

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + value;
}

void foo_test_2() {
    int num = 128;
    int nbytes = num * sizeof(int);
    float value = 10.0f;

    float *h_a = 0;
    float *d_a = 0;
    dim3 block = dim3(128);
    dim3 grid  = dim3((num + block.x - 1) / block.x);

    // create cuda event handles
    hipEvent_t stop;
    CHECK(hipEventCreate(&stop));

    // asynchronously issue work to the GPU (all to stream 0)
    CHECK(hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice));
    kernel<<<grid, block>>>(d_a, value);
    CHECK(hipMemcpyAsync(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // CHECK:    q_ct1.wait();
    // CHECK-NEXT:    stop_ct1 = std::chrono::steady_clock::now();
    // CHECK-NEXT:    CHECK(0);
    CHECK(hipEventRecord(stop));

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter = 0;
    while (hipEventQuery(stop) == hipErrorNotReady) {
        counter++;
    }
}

#define CHECK(call)                                                            \
  {                                                                            \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
    }                                                                          \
  }

#define NSTREAM 4
#define BDIM 128

__global__ void sumArrays(float *A, float *B, float *C, const int NN) {}

void foo_test_3() {
  int nElem = 1 << 18;
  size_t nBytes = nElem * sizeof(float);

  // malloc pinned host memory for async memcpy
  float *h_A, *h_B, *hostRef, *gpuRef;

  // malloc device global memory
  float *d_A, *d_B, *d_C;

  hipEvent_t start, stop;
  CHECK(hipEventCreate(&start));
  CHECK(hipEventCreate(&stop));

  // invoke kernel at host side
  dim3 block(BDIM);
  dim3 grid;

  // grid parallel operation
  int iElem = nElem / NSTREAM;
  size_t iBytes = iElem * sizeof(float);

  hipStream_t stream[NSTREAM];

  for (int i = 0; i < NSTREAM; ++i) {
    // CHECK:    CHECK((stream[i] = dpct::get_current_device().create_queue(), 0));
    CHECK(hipStreamCreate(&stream[i]));
  }

  CHECK(hipEventRecord(start, 0));

  // initiate all work on the device asynchronously in depth-first order
  for (int i = 0; i < NSTREAM; ++i) {
    int ioffset = i * iElem;
    CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes,
                          hipMemcpyHostToDevice, stream[i]));
    CHECK(hipMemcpyAsync(&d_B[ioffset], &h_B[ioffset], iBytes,
                          hipMemcpyHostToDevice, stream[i]));
    sumArrays<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], &d_B[ioffset],
                                             &d_C[ioffset], iElem);
    // CHECK:    CHECK((dpct::async_dpct_memcpy(&gpuRef[ioffset], &d_C[ioffset], iBytes,
    // CHECK-NEXT:                          dpct::device_to_host, *(stream[i])), 0));
    CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_C[ioffset], iBytes,
                          hipMemcpyDeviceToHost, stream[i]));
  }

  // CHECK: dpct::dev_mgr::instance().current_device().queues_wait_and_throw();
  // CHECK-NEXT: stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT: CHECK(0);
  // CHECK-NEXT: CHECK(0);
  CHECK(hipEventRecord(stop, 0));
  CHECK(hipEventSynchronize(stop));
  float execution_time;
  CHECK(hipEventElapsedTime(&execution_time, start, stop));
}

#define SAFE_CALL(call)                                                   \
  do {                                                                         \
    int err = call;                                                            \
  } while (0)

void foo_usm() {
  hipStream_t s1, s2;
  int *gpu_t, *host_t, n = 10;
  hipEvent_t start, stop;
  SAFE_CALL(hipEventRecord(start, 0));

  // CHECK:  DPCT1003:32: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  SAFE_CALL((dpct::async_dpct_memcpy(gpu_t, host_t, n * sizeof(int), dpct::host_to_device, *s1), 0));
  SAFE_CALL(hipMemcpyAsync(gpu_t, host_t, n * sizeof(int), hipMemcpyHostToDevice, s1));

  // CHECK:  s1->wait();
  // CHECK-NEXT:  stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT:  SAFE_CALL(0);
  // CHECK-NEXT:  SAFE_CALL(0);
  // CHECK-NEXT:  float Time = 0.0f;
  // CHECK-NEXT:  Time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
  SAFE_CALL(hipEventRecord(stop, 0));
  SAFE_CALL(hipEventSynchronize(stop));
  float Time = 0.0f;
  hipEventElapsedTime(&Time, start, stop);
}

__global__ void readTexels(int n, float *d_out, int width){}
__global__ void readTexelsFoo1(int n, float *d_out){}
__global__ void readTexelsFoo2(int n, float *d_out, int width, int height){}
texture<float4, 2, hipReadModeElementType> texA;

void foo()
{
    const unsigned int passes = 100;
    const unsigned int nsizes = 5;
    const unsigned int sizes[] = { 16, 64, 256, 1024, 4096 };
    const unsigned int kernelRepFoo[] = { 1024, 1024, 1024, 1024, 256 };
    const unsigned int iterations = 10;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int j = 0; j < nsizes; j++)
    {
        const unsigned int size      = 1024 * sizes[j];
        const unsigned int numFloat  = size / sizeof(float);
        const unsigned int numFloat4 = size / sizeof(float4);
        size_t width, height;
        const unsigned int kernelRepFactor = kernelRepFoo[j];

        // Image memory sizes should be power of 2.
        size_t sizeLog = lround(log2(double(numFloat4)));
        height = 1 << (sizeLog >> 1);  // height is the smaller size
        width = numFloat4 / height;

        const dim3 blockSize(16, 8);
        const dim3 gridSize(width/blockSize.x, height/blockSize.y);

        float *h_in = new float[numFloat];
        float *h_out = new float[numFloat4];
        float *d_out;
        hipMalloc((void**) &d_out, numFloat4 * sizeof(float));

        // Allocate a cuda array
        hipArray* cuArray;
        hipMallocArray(&cuArray, &texA.channelDesc, width, height);

        // Copy in source data
        hipMemcpyToArray(cuArray, 0, 0, h_in, size, hipMemcpyHostToDevice);

        // Bind texture to the array
        hipBindTextureToArray(texA, cuArray);

        for (int p = 0; p < passes; p++)
        {
            // Test 1: Repeated Linear Access
            float t = 0.0f;

            hipEventRecord(start, 0);
            // read texels from texture
            for (int iter = 0; iter < iterations; iter++)
            {
// CHECK:                DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                  dpct::buffer_t d_out_buf_ct1 = dpct::get_buffer(d_out);
// CHECK-NEXT:                  q_ct1.submit(
// CHECK-NEXT:                    [&](sycl::handler &cgh) {
// CHECK-NEXT:                      auto d_out_acc_ct1 = d_out_buf_ct1.get_access<sycl::access::mode::read_write>(cgh);
// CHECK-EMPTY:
// CHECK-NEXT:                      cgh.parallel_for<dpct_kernel_name<class readTexels_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                        sycl::nd_range<3>(gridSize * blockSize, blockSize), 
// CHECK-NEXT:                        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                          readTexels(kernelRepFactor, (float *)(&d_out_acc_ct1[0]), width);
// CHECK-NEXT:                        });
// CHECK-NEXT:                    });
                readTexels<<<gridSize, blockSize>>>(kernelRepFactor, d_out,
                                                    width);
            }

// CHECK:            DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:            */
// CHECK-NEXT:            dpct::dev_mgr::instance().current_device().queues_wait_and_throw();
// CHECK-NEXT:            stop_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:            t = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);
            t /= 1.e3;

            // Verify results
            hipMemcpy(h_out, d_out, numFloat4*sizeof(float),
                    hipMemcpyDeviceToHost);

            // Test 2 Repeated Cache Access
            hipEventRecord(start, 0);
            for (int iter = 0; iter < iterations; iter++)
            {
// CHECK:                DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                  dpct::buffer_t d_out_buf_ct1 = dpct::get_buffer(d_out);
// CHECK-NEXT:                  q_ct1.submit(
// CHECK-NEXT:                    [&](sycl::handler &cgh) {
// CHECK-NEXT:                      auto d_out_acc_ct1 = d_out_buf_ct1.get_access<sycl::access::mode::read_write>(cgh);
// CHECK-EMPTY:
// CHECK-NEXT:                      cgh.parallel_for<dpct_kernel_name<class readTexelsFoo1_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                        sycl::nd_range<3>(gridSize * blockSize, blockSize), 
// CHECK-NEXT:                        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                          readTexelsFoo1(kernelRepFactor, (float *)(&d_out_acc_ct1[0]));
// CHECK-NEXT:                        });
// CHECK-NEXT:                    });
                readTexelsFoo1<<<gridSize, blockSize>>>
                        (kernelRepFactor, d_out);
            }

// CHECK:            DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:             */
// CHECK-NEXT:             dpct::dev_mgr::instance().current_device().queues_wait_and_throw();
// CHECK-NEXT:             stop_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:             t = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

            // Verify results
            hipMemcpy(h_out, d_out, numFloat4*sizeof(float),
                    hipMemcpyDeviceToHost);

            // Test 3 Repeated "Random" Access
            hipEventRecord(start, 0);

            // read texels from texture
            for (int iter = 0; iter < iterations; iter++)
            {
// CHECK:                DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                  dpct::buffer_t d_out_buf_ct1 = dpct::get_buffer(d_out);
// CHECK-NEXT:                  q_ct1.submit(
// CHECK-NEXT:                    [&](sycl::handler &cgh) {
// CHECK-NEXT:                      auto d_out_acc_ct1 = d_out_buf_ct1.get_access<sycl::access::mode::read_write>(cgh);
// CHECK-EMPTY:
// CHECK-NEXT:                      cgh.parallel_for<dpct_kernel_name<class readTexelsFoo2_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                        sycl::nd_range<3>(gridSize * blockSize, blockSize), 
// CHECK-NEXT:                        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                          readTexelsFoo2(kernelRepFactor, (float *)(&d_out_acc_ct1[0]), width, height);
// CHECK-NEXT:                        });
// CHECK-NEXT:                    });
                readTexelsFoo2<<<gridSize, blockSize>>>
                                (kernelRepFactor, d_out, width, height);
            }

// CHECK:             DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:            */
// CHECK-NEXT:            dpct::dev_mgr::instance().current_device().queues_wait_and_throw();
// CHECK-NEXT:            stop_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:            t = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);
        }
        delete[] h_in;
        delete[] h_out;
        hipFree(d_out);
        hipFreeArray(cuArray);
        hipUnbindTexture(texA);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}



__global__ void foo_kernel_1(){}
__global__ void foo_kernel_2(){}
__global__ void foo_kernel_3(){}
__global__ void foo_kernel_4(){}

int foo_test_2()
{
    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    float elapsed_time;

    hipStream_t *streams = (hipStream_t *)malloc(n_streams*sizeof(hipStream_t));

    for (int i = 0 ; i < n_streams ; i++)
    {
        hipStreamCreate(&(streams[i]));
    }

    dim3 block (iblock);
    dim3 grid  (isize / iblock);

    // creat events
// CHECK:    sycl::event start, stop;
// CHECK-NEXT:    std::chrono::time_point<std::chrono::steady_clock> start_ct1;
// CHECK-NEXT:    std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
// CHECK-NEXT:    /*
// CHECK-NEXT:    DPCT1027:{{[0-9]+}}: The call to hipEventCreate was replaced with 0, because this call is redundant in DPC++.
// CHECK-NEXT:    */
// CHECK-NEXT:    CHECK(0);
// CHECK-NEXT:    /*
// CHECK-NEXT:    DPCT1027:{{[0-9]+}}: The call to hipEventCreate was replaced with 0, because this call is redundant in DPC++.
// CHECK-NEXT:    */
// CHECK-NEXT:    CHECK(0);
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    hipEvent_t *kernelEvent;
    kernelEvent = (hipEvent_t *) malloc(n_streams * sizeof(hipEvent_t));

    // record start event
// CHECK:    /*
// CHECK-NEXT:    DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:    */
// CHECK-NEXT:    /*
// CHECK-NEXT:    DPCT1024:{{[0-9]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT:    */
// CHECK-NEXT:    start_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:    CHECK(0);
    CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering
    for (int i = 0; i < n_streams; i++)
    {
// CHECK:        DPCT1049:{{[0-9]+}}: The workgroup size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the workgroup size if needed.
// CHECK-NEXT:        */
// CHECK-NEXT:        streams[i]->submit(
// CHECK-NEXT:          [&](sycl::handler &cgh) {
// CHECK-NEXT:            cgh.parallel_for<dpct_kernel_name<class foo_kernel_1_{{[a-z0-9]+}}>>(
// CHECK-NEXT:              sycl::nd_range<3>(grid * block, block), 
// CHECK-NEXT:              [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                foo_kernel_1();
// CHECK-NEXT:              });
// CHECK-NEXT:          });
        foo_kernel_1<<<grid, block, 0, streams[i]>>>();
        foo_kernel_2<<<grid, block, 0, streams[i]>>>();
        foo_kernel_3<<<grid, block, 0, streams[i]>>>();
        foo_kernel_4<<<grid, block, 0, streams[i]>>>();

// CHECK:        kernelEvent_ct1_i = std::chrono::steady_clock::now(); 
// CHECK-NEXT:        CHECK(0);
// CHECK-NEXT:        kernelEvent[i].wait();
        CHECK(hipEventRecord(kernelEvent[i], streams[i]));
        hipStreamWaitEvent(streams[n_streams - 1], kernelEvent[i], 0);
    }

// CHECK:    dpct::dev_mgr::instance().current_device().queues_wait_and_throw();
// CHECK-NEXT:    stop_ct1 = std::chrono::steady_clock::now(); 
// CHECK-NEXT:    CHECK(0);
// CHECK-NEXT:    CHECK(0);
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));

    return 0;
}
