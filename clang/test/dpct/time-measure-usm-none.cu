#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -usm-level=none -out-root %T/time-measure-usm-none %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/time-measure-usm-none/time-measure-usm-none.dp.cpp --match-full-lines %s
#include <stdio.h>

#define N 1000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
    hipStream_t stream;

    int ha[N], hb[N];
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
    hipEvent_t start, stop;
    hipError_t cudaStatus;

    int *da, *db;
    float elapsedTime;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }


    hipEventCreate(&start);
    hipEventCreate(&stop);

    // CHECK: start_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);

    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device);
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device);
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, 0);
    // CHECK: dpct::async_dpct_memcpy(da, ha, N*sizeof(int), dpct::host_to_device, *stream);
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, stream);

    // CHECK: q_ct1.wait();
    // CHECK: stream->wait();
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    // CHECK: elapsedTime = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    add<<<N, 1>>>(da, db);

    // CHECK: dpct::async_dpct_memcpy(hb, db, N*sizeof(int), dpct::device_to_host);
    hipMemcpyAsync(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return 0;
}


__global__ void kernel_foo(){}

void foo_test_1() {

    hipEvent_t     start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

// CHECK:    start_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:        for (int i=0; i<4; i++) {
// CHECK-NEXT:            dpct::get_default_queue().submit(
// CHECK-NEXT:              [&](sycl::handler &cgh) {
// CHECK-NEXT:                cgh.parallel_for<dpct_kernel_name<class kernel_foo_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                  sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:                  [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                    kernel_foo();
// CHECK-NEXT:                  });
// CHECK-NEXT:              });
// CHECK-NEXT:        }
// CHECK-NEXT:    dpct::get_current_device().queues_wait_and_throw();
    hipEventRecord( start, 0 );
        for (int i=0; i<4; i++) {
            kernel_foo<<<1, 1>>>();
        }
    hipDeviceSynchronize();

    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop ) ;
}
