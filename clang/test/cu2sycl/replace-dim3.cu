
#include <hip/hip_runtime.h>
// RUN: cu2sycl -out-root %T %s -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/replace-dim3.sycl.cpp --match-full-lines %s

// CHECK: void func(cl::sycl::range<3> a, cl::sycl::range<3> b, cl::sycl::range<3> c, cl::sycl::range<3> d) try {
void func(dim3 a, dim3 b, dim3 c, dim3 d) {
}

int main() {
  // range default constructor does the right thing.
  // CHECK: cl::sycl::range<3> deflt;
  dim3 deflt;

  // CHECK: cl::sycl::range<3> round1(1, 1, 1);
  dim3 round1(1);

  // CHECK: cl::sycl::range<3> round2(2, 1, 1);
  dim3 round2(2, 1);

  // CHECK: cl::sycl::range<3> assign = cl::sycl::range<3>(32, 1, 1);
  dim3 assign = 32;

  // CHECK: cl::sycl::range<3> castini = cl::sycl::range<3>(4, 1, 1);
  dim3 castini = (dim3)4;

  // CHECK: cl::sycl::range<3> castini2 = cl::sycl::range<3>(2, 2, 1);
  dim3 castini2 = dim3(2, 2);

  // CHECK: cl::sycl::range<3> castini3 = cl::sycl::range<3>(3, 1, 10);
  dim3 castini3 = dim3(3, 1, 10);

  // CHECK: deflt = cl::sycl::range<3>(3, 1, 1);
  deflt = dim3(3);

  // CHECK: cl::sycl::range<3> copyctor1 = cl::sycl::range<3>(cl::sycl::range<3>(33, 1, 1));
  dim3 copyctor1 = dim3((dim3)33);

  // CHECK: cl::sycl::range<3> copyctor2 = cl::sycl::range<3>(copyctor1);
  dim3 copyctor2 = dim3(copyctor1);

  // CHECK: cl::sycl::range<3> copyctor3(copyctor1);
  dim3 copyctor3(copyctor1);

  // CHECK: func(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(2, 1, 1), cl::sycl::range<3>(3, 2, 1));
  func((dim3)1, dim3(1), dim3(2, 1), dim3(3, 2, 1));
  // CHECK: func(cl::sycl::range<3>(1, 1, 1), cl::sycl::range<3>(2, 1, 1), cl::sycl::range<3>(3, 1, 1), cl::sycl::range<3>(4, 1, 1));
  func(1, 2, 3, 4);
  // CHECK: func(deflt, cl::sycl::range<3>(deflt), cl::sycl::range<3>(deflt), cl::sycl::range<3>(2+3*3, 1, 1));
  func(deflt, dim3(deflt), (dim3)deflt, 2+3*3);
}
