
#include <hip/hip_runtime.h>
// RUN: cu2sycl -out-root %T %s -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/iteration-space.sycl.cpp --match-full-lines %s

// Test that the replacement happens when it should to.
// CHECK: void test_00() {
__global__
void test_00() {
  // CHECK: size_t tix = item.get_local(0);
  // CHECK: size_t tiy = item.get_local(1);
  // CHECK: size_t tiz = item.get_local(2);
  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;

  // CHECK: size_t bix = item.get_group(0);
  // CHECK: size_t biy = item.get_group(1);
  // CHECK: size_t biz = item.get_group(2);

  size_t bix = blockIdx.x;
  size_t biy = blockIdx.y;
  size_t biz = blockIdx.z;

  // CHECK: size_t bdx = item.get_local_range().get(0);
  // CHECK: size_t bdy = item.get_local_range().get(1);
  // CHECK: size_t bdz = item.get_local_range().get(2);
  size_t bdx = blockDim.x;
  size_t bdy = blockDim.y;
  size_t bdz = blockDim.z;

  // CHECK: size_t gdx = item.get_num_groups(0);
  // CHECK: size_t gdy = item.get_num_groups(1);
  // CHECK: size_t gdz = item.get_num_groups(2);

  size_t gdx = gridDim.x;
  size_t gdy = gridDim.y;
  size_t gdz = gridDim.z;
}

// Test that the replacement doesn't happen in host functions.
// CHECK: void test_01() {
void test_01() {
  uint3 threadIdx, blockIdx, blockDim, gridDim;

  // CHECK: size_t tix = threadIdx.x;
  // CHECK: size_t tiy = threadIdx.y;
  // CHECK: size_t tiz = threadIdx.z;
  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;

  // CHECK: size_t bix = blockIdx.x;
  // CHECK: size_t biy = blockIdx.y;
  // CHECK: size_t biz = blockIdx.z;
  size_t bix = blockIdx.x;
  size_t biy = blockIdx.y;
  size_t biz = blockIdx.z;

  // CHECK: size_t bdx = blockDim.x;
  // CHECK: size_t bdy = blockDim.y;
  // CHECK: size_t bdz = blockDim.z;
  size_t bdx = blockDim.x;
  size_t bdy = blockDim.y;
  size_t bdz = blockDim.z;

  // CHECK: size_t gdx = gridDim.x;
  // CHECK: size_t gdy = gridDim.y;
  // CHECK: size_t gdz = gridDim.z;
  size_t gdx = gridDim.x;
  size_t gdy = gridDim.y;
  size_t gdz = gridDim.z;
}

// Test that the replacement doesn't happen if threadIdx is redefined.
// CHECK: void test_02() {
__global__ void test_02() {
  uint3 threadIdx, blockIdx, blockDim, gridDim;

  // CHECK: size_t tix = threadIdx.x;
  // CHECK: size_t tiy = threadIdx.y;
  // CHECK: size_t tiz = threadIdx.z;
  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;

  // CHECK: size_t bix = blockIdx.x;
  // CHECK: size_t biy = blockIdx.y;
  // CHECK: size_t biz = blockIdx.z;
  size_t bix = blockIdx.x;
  size_t biy = blockIdx.y;
  size_t biz = blockIdx.z;

  // CHECK: size_t bdx = blockDim.x;
  // CHECK: size_t bdy = blockDim.y;
  // CHECK: size_t bdz = blockDim.z;
  size_t bdx = blockDim.x;
  size_t bdy = blockDim.y;
  size_t bdz = blockDim.z;

  // CHECK: size_t gdx = gridDim.x;
  // CHECK: size_t gdy = gridDim.y;
  // CHECK: size_t gdz = gridDim.z;
  size_t gdx = gridDim.x;
  size_t gdy = gridDim.y;
  size_t gdz = gridDim.z;
}
