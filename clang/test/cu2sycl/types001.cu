
#include <hip/hip_runtime.h>
// RUN: cu2sycl -out-root %T %s -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types001.sycl.cpp

// CHECK: cu2sycl::sycl_device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const cu2sycl::sycl_device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile cu2sycl::sycl_device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHECK:  void foo(cu2sycl::sycl_device_info p) {
void foo(hipDeviceProp_t p) {
    return;
}
