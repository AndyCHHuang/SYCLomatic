
#include <hip/hip_runtime.h>
// RUN: cp %s %t
// RUN: cu2sycl %t -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %t

// CHECK: cu2sycl::sycl_device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const cu2sycl::sycl_device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile cu2sycl::sycl_device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHECK:  void foo(cu2sycl::sycl_device_info p) {
void foo(hipDeviceProp_t p) {
    return;
}
