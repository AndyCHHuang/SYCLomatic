// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: c2s --format-range=none -out-root %T/cublas-usm-11 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-usm-11/cublas-usm-11.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void foo1() {
  hipblasHandle_t handle;
  void *x, *y, *a, *b, *c, *alpha, *beta, *res, *cos, *sin;
  const void **a_array;
  const void **b_array;
  void **c_array;
  //CHECK:c2s::nrm2(*handle, 4, x, c2s::library_data_t::real_float, 1, res, c2s::library_data_t::real_float);
  //CHECK-NEXT:c2s::dot(*handle, 4, x, c2s::library_data_t::real_float, 1, y, c2s::library_data_t::real_float, 1, res, c2s::library_data_t::real_float);
  //CHECK-NEXT:c2s::dotc(*handle, 4, x, c2s::library_data_t::real_float, 1, y, c2s::library_data_t::real_float, 1, res, c2s::library_data_t::real_float);
  //CHECK-NEXT:c2s::scal(*handle, 4, alpha, c2s::library_data_t::real_float, x, c2s::library_data_t::real_float, 1);
  //CHECK-NEXT:c2s::axpy(*handle, 4, alpha, c2s::library_data_t::real_float, x, c2s::library_data_t::real_float, 1, y, c2s::library_data_t::real_float, 1);
  //CHECK-NEXT:c2s::rot(*handle, 4, x, c2s::library_data_t::real_float, 1, y, c2s::library_data_t::real_float, 1, cos, sin, c2s::library_data_t::real_float);
  //CHECK-NEXT:c2s::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a, c2s::library_data_t::real_half, 4, b, c2s::library_data_t::real_half, 4, beta, c, c2s::library_data_t::real_half, 4, c2s::library_data_t::real_half);
  //CHECK-NEXT:c2s::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a_array, c2s::library_data_t::real_half, 4, b_array, c2s::library_data_t::real_half, 4, beta, c_array, c2s::library_data_t::real_half, 4, 2, c2s::library_data_t::real_half);
  //CHECK-NEXT:c2s::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a, c2s::library_data_t::real_half, 4, 16, b, c2s::library_data_t::real_half, 4, 16, beta, c, c2s::library_data_t::real_half, 4, 16, 2, c2s::library_data_t::real_half);
  hipblasNrm2Ex(handle, 4, x, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasDotEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasDotcEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasScalEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, HIP_R_32F);
  hipblasAxpyEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, y, HIP_R_32F, 1, HIP_R_32F);
  hipblasRotEx(handle, 4, x, HIP_R_32F, 1,  y, HIP_R_32F, 1,  cos, sin, HIP_R_32F, HIP_R_32F);
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, b, HIP_R_16F, 4, beta, c, HIP_R_16F, 4, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a_array, HIP_R_16F, 4, b_array, HIP_R_16F, 4, beta, c_array, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, 16, b, HIP_R_16F, 4, 16, beta, c, HIP_R_16F, 4, 16, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
}
