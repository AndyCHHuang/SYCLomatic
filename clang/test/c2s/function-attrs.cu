
#include <hip/hip_runtime.h>
// RUN: c2s --format-range=none -out-root %T/function-attrs %s -passes "IterationSpaceBuiltinRule" --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/function-attrs/function-attrs.dp.cpp

// CHECK: void test_00();
__device__ void test_00();

// CHECK: void test_01();
__global__ void test_01();

// CHECK: void test_02();
__host__ void test_02();

// CHECK: void test_03();
__host__ __device__ void test_03();

// CHECK: void test_04();
void test_04();

// CHECK: void test_05();
__device__ void test_05();

// Test that the attribute is properly removed from all function declarations
// even if there are several of them.
// CHECK: void test_06();
// CHECK: void test_06();
// CHECK: void test_06() { }
__global__ void test_06();
__global__ void test_06();
__global__ void test_06() { }
// CHECK: void test_07();
__global__    void test_07();
// CHECK: void test_08();
__global__	    void test_08();


