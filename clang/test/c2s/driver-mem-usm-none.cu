// RUN: c2s --usm-level=none --format-range=none -out-root %T/driver-mem-usm-none %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem-usm-none/driver-mem-usm-none.dp.cpp %s

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
int main(){
    size_t result1, result2;
    int size = 32;
    float* f_A;
    // CHECK: f_A = (float *)malloc(size);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);


    // CHECK: void * f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: void * f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: f_D = c2s::c2s_malloc(size);
    hipMalloc(&f_D, size);

    // CHECK: sycl::queue * stream;
    hipStream_t stream;
    // CHECK: c2s::async_c2s_memcpy(f_D, f_A, size, c2s::automatic, *stream);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: c2s::async_c2s_memcpy(f_D, f_A, size, c2s::automatic);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: c2s::c2s_memcpy(f_D, f_A, size, c2s::automatic);
    hipMemcpyHtoD(f_D, f_A, size);

    // CHECK: c2s::async_c2s_memcpy(f_A, f_D, size, c2s::automatic, *stream);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: c2s::async_c2s_memcpy(f_A, f_D, size, c2s::automatic);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: c2s::c2s_memcpy(f_A, f_D, size, c2s::automatic);
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: c2s::async_c2s_memcpy(f_D, f_D2, size, c2s::automatic, *stream);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: c2s::async_c2s_memcpy(f_D, f_D2, size, c2s::automatic);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: c2s::c2s_memcpy(f_D, f_D2, size, c2s::automatic);
    hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: c2s::pitched_data cpy_from_data_ct1, cpy_to_data_ct1;
    // CHECK: sycl::id<3> cpy_from_pos_ct1(0, 0, 0), cpy_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy_size_ct1(1, 1, 1);
    hip_Memcpy2D cpy;
    //
    cpy.dstMemoryType = hipMemoryTypeHost;
    // CHECK: cpy_to_data_ct1.set_data_ptr(f_A);
    cpy.dstHost = f_A;
    // CHECK: cpy_to_data_ct1.set_pitch(20);
    cpy.dstPitch = 20;
    // CHECK: cpy_to_pos_ct1[1] = 10;
    cpy.dstY = 10;
    // CHECK: cpy_to_pos_ct1[0] = 15;
    cpy.dstXInBytes = 15;

    //
    cpy.srcMemoryType = hipMemoryTypeDevice;
    // CHECK: cpy_from_data_ct1.set_data_ptr(f_D);
    cpy.srcDevice = f_D;
    // CHECK: cpy_from_data_ct1.set_pitch(20);
    cpy.srcPitch = 20;
    // CHECK: cpy_from_pos_ct1[1] = 10;
    cpy.srcY = 10;
    // CHECK: cpy_from_pos_ct1[0] = 15;
    cpy.srcXInBytes = 15;

    // CHECK: cpy_size_ct1[0] = 4;
    cpy.WidthInBytes = 4;
    // CHECK: cpy_size_ct1[1] = 7;
    cpy.Height = 7;

    // CHECK: c2s::c2s_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1);
    hipMemcpyParam2D(&cpy);
    // CHECK: c2s::async_c2s_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1, c2s::automatic, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    // CHECK: c2s::pitched_data cpy2_from_data_ct1, cpy2_to_data_ct1;
    // CHECK: sycl::id<3> cpy2_from_pos_ct1(0, 0, 0), cpy2_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy2_size_ct1(1, 1, 1);
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    //
    cpy2.dstMemoryType = hipMemoryTypeArray;
    // CHECK: cpy2_to_data_ct1 = ca->to_pitched_data();
    cpy2.dstArray = ca;
    // CHECK: cpy2_to_data_ct1.set_pitch(5);
    cpy2.dstPitch = 5;
    // CHECK: cpy2_to_data_ct1.set_y(4);
    cpy2.dstHeight = 4;
    // CHECK: cpy2_to_pos_ct1[1] = 3;
    cpy2.dstY = 3;
    // CHECK: cpy2_to_pos_ct1[2] = 2;
    cpy2.dstZ = 2;
    // CHECK: cpy2_to_pos_ct1[0] = 1;
    cpy2.dstXInBytes = 1;
    //
    cpy2.dstLOD = 0;

    //
    cpy2.srcMemoryType = hipMemoryTypeHost;
    // CHECK: cpy2_from_data_ct1.set_data_ptr(f_A);
    cpy2.srcHost = f_A;
    // CHECK: cpy2_from_data_ct1.set_pitch(5);
    cpy2.srcPitch = 5;
    // CHECK: cpy2_from_data_ct1.set_y(4);
    cpy2.srcHeight = 4;
    // CHECK: cpy2_from_pos_ct1[1] = 3;
    cpy2.srcY = 3;
    // CHECK: cpy2_from_pos_ct1[2] = 2;
    cpy2.srcZ = 2;
    // CHECK: cpy2_from_pos_ct1[0] = 1;
    cpy2.srcXInBytes = 1;
    //
    cpy2.srcLOD = 0;

    // CHECK: cpy2_size_ct1[0] = 3;
    cpy2.WidthInBytes = 3;
    // CHECK: cpy2_size_ct1[1] = 2;
    cpy2.Height = 2;
    // CHECK: cpy2_size_ct1[2] = 1;
    cpy2.Depth = 1;

    // CHECK: c2s::c2s_memcpy(cpy2_to_data_ct1, cpy2_to_pos_ct1, cpy2_from_data_ct1, cpy2_from_pos_ct1, cpy2_size_ct1);
    hipDrvMemcpy3D(&cpy2);

    return 0;
}
